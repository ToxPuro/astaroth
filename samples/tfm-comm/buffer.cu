#include "buffer.h"

#if defined(__HIPCC__)
#include <hip/hip_runtime.h>
#elif defined(__HIP_PLATFORM_AMD__)
#include "hip.h"
#include <hip/hip_runtime.h>
#else
static_assert(false);
#endif

#include <iostream>

#include "errchk_gpu.h"

AcBuffer
acBufferCreate(const size_t count, const bool on_device)
{
    AcBuffer buffer = {
        .on_device = on_device,
        .count     = count,
        .data      = NULL,
    };
    const size_t bytes = sizeof(buffer.data[0]) * count;
    if (buffer.on_device) {
        ERRCHK_GPU_API(hipMalloc((void**)&buffer.data, bytes));
    }
    else {
        buffer.data = (double*)malloc(bytes);
    }
    ERRCHK(buffer.data != NULL);
    return buffer;
}

void
acBufferDestroy(AcBuffer* buffer)
{
    if (buffer->on_device)
        hipFree(buffer->data);
    else
        free(buffer->data);
    buffer->data  = NULL;
    buffer->count = 0;
}

void
acBufferMigrate(const AcBuffer in, AcBuffer* out)
{
    hipMemcpyKind kind;
    if (in.on_device) {
        if (out->on_device)
            kind = hipMemcpyDeviceToDevice;
        else
            kind = hipMemcpyDeviceToHost;
    }
    else {
        if (out->on_device)
            kind = hipMemcpyHostToDevice;
        else
            kind = hipMemcpyHostToHost;
    }

    ERRCHK(out->count >= in.count);
    const size_t bytes = sizeof(in.data[0]) * in.count;
    if (kind == hipMemcpyHostToHost)
        memmove(out->data, in.data, bytes);
    else
        ERRCHK_GPU_API(hipMemcpy(out->data, in.data, sizeof(in.data[0]) * in.count, kind));
}

void
acBufferPrint(const char* label, const AcBuffer buffer)
{
    std::cout << label << ": ";

    AcBuffer tmp;
    acBufferMigrate(buffer, &tmp);
    for (size_t i = 0; i < tmp.count; ++i)
        std::cout << tmp.data[i] << ((i + 1 < tmp.count) ? ", " : "\n");
    acBufferDestroy(&tmp);
}
