#include "hip/hip_runtime.h"
#include "pack.h"

static __device__ uint64_t
device_to_linear(const Index& coords, const Shape& shape)
{
    uint64_t result = 0;
    for (size_t j = 0; j < shape.count; ++j) {
        uint64_t factor = 1;
        for (size_t i = 0; i < j; ++i)
            factor *= shape[i];
        result += coords[j] * factor;
    }
    return result;
}

static __device__ Index
device_to_spatial(const uint64_t index, const Shape& shape)
{
    Index coords(shape.count);
    for (size_t j = 0; j < shape.count; ++j) {
        uint64_t divisor = 1;
        for (size_t i = 0; i < j; ++i)
            divisor *= shape[i];
        coords[j] = (index / divisor) % shape[j];
    }
    return coords;
}

template <typename T>
__global__ void
kernel_pack(const Shape mm, const Shape block_shape, const Index block_offset,
            const PackPtrArray<T*> inputs, T* output)
{
    const uint64_t i = static_cast<uint64_t>(threadIdx.x) + blockIdx.x * blockDim.x;
    const uint64_t block_nelems{prod(block_shape)};
    if (i < block_nelems) {
        for (size_t j = 0; j < inputs.count; ++j) {

            // Block coords
            const Index block_coords = device_to_spatial(i, block_shape);

            // Input coords
            const Index in_coords = block_offset + block_coords;
            const uint64_t in_idx = device_to_linear(in_coords, mm);

            output[i + j * block_nelems] = inputs[j][in_idx];
        }
    }
}

template <typename T>
__global__ void
kernel_unpack(const T* input, const Shape mm, const Shape block_shape, const Index block_offset,
              PackPtrArray<T*> outputs)
{
    const uint64_t i = static_cast<uint64_t>(threadIdx.x) + blockIdx.x * blockDim.x;
    const uint64_t block_nelems{prod(block_shape)};
    if (i < block_nelems) {
        for (size_t j = 0; j < outputs.count; ++j) {

            // Block coords
            const Index block_coords = device_to_spatial(i, block_shape);

            // Input coords
            const Index in_coords = block_offset + block_coords;
            const uint64_t in_idx = device_to_linear(in_coords, mm);

            outputs[j][in_idx] = input[i + j * block_nelems];
        }
    }
}

template <typename T>
void
pack(const Shape& mm, const Shape& block_shape, const Index& block_offset,
     const PackPtrArray<T*>& inputs, T* output)
{
    const uint64_t block_nelems{prod(block_shape)};
    const uint64_t tpb{256};
    const uint64_t bpg{(block_nelems + tpb - 1) / tpb};
    kernel_pack<<<bpg, tpb>>>(mm, block_shape, block_offset, inputs, output);
    ERRCHK_CUDA_KERNEL();
    hipDeviceSynchronize();
}

template <typename T>
void
unpack(const T* input, const Shape& mm, const Shape& block_shape, const Index& block_offset,
       PackPtrArray<T*>& outputs)
{
    const uint64_t block_nelems{prod(block_shape)};
    const uint64_t tpb{256};
    const uint64_t bpg{(block_nelems + tpb - 1) / tpb};
    kernel_unpack<<<bpg, tpb>>>(input, mm, block_shape, block_offset, outputs);
    ERRCHK_CUDA_KERNEL();
    hipDeviceSynchronize();
}

template void pack<AcReal>(const Shape&, const Shape&, const Index&, const PackPtrArray<AcReal*>&,
                           AcReal*);

template void unpack<AcReal>(const AcReal*, const Shape&, const Shape&, const Index&,
                             PackPtrArray<AcReal*>&);
