#include "hip/hip_runtime.h"
#include <cstdlib>

#if defined(__HIPCC__)
#include <hip/hip_runtime.h>
#elif defined(__HIP_PLATFORM_AMD__)
#include "hip.h"
#include <hip/hip_runtime.h>
#else
static_assert(false);
#endif

#include <iostream>
#include <vector>

#include "errchk.h"
#include "errchk_cuda.h"
#include "static_array.h"

// #include "device_buffer.h"
// #include "dbuffer.h"
#include "buffer.h"
#include "buffer_transfer.h"

__global__ void
kernel(const size_t count, const double* in, double* out)
{
    const size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < count)
        out[i] = 2 * in[i];
}

int
main()
{
    const size_t count = 10;
    Buffer<double> host_buffer(count);
    host_buffer.fill_arange(0, count);
    std::cout << "Before: " << host_buffer << std::endl;
    Buffer<double> device_buffer(count, BUFFER_DEVICE);
    // host_buffer.migrate(device_buffer);
    // device_buffer.migrate(host_buffer);
    HostToDeviceBufferExchangeTask<double> htod(count);
    htod.launch(host_buffer);
    htod.wait(device_buffer);
    HostToDeviceBufferExchangeTask<double> dtoh(count);
    dtoh.launch(device_buffer);
    dtoh.wait(host_buffer);
    std::cout << "After: " << host_buffer << std::endl;

    double* hin  = (double*)malloc(count * sizeof(hin[0]));
    double* hout = (double*)malloc(count * sizeof(hout[0]));
    ERRCHK(hin);
    ERRCHK(hout);

    double *din, *dout;
    ERRCHK_CUDA_API(hipMalloc(&din, count * sizeof(din[0])));
    ERRCHK_CUDA_API(hipMalloc(&dout, count * sizeof(dout[0])));

    for (size_t i = 0; i < count; ++i)
        hin[i] = static_cast<double>(i);

    ERRCHK_CUDA_API(hipMemcpy(din, hin, count * sizeof(hin[0]), hipMemcpyHostToDevice));
    const size_t tpb = 256;
    const size_t bpg = (count + tpb - 1) / tpb;
    kernel<<<bpg, tpb>>>(count, din, dout);
    ERRCHK_CUDA_KERNEL();
    ERRCHK_CUDA_API(hipMemcpy(hout, dout, count * sizeof(dout[0]), hipMemcpyDeviceToHost));

    for (size_t i = 0; i < count; ++i)
        std::cout << "i: " << hout[i] << std::endl;

    ERRCHK_CUDA_API(hipFree(dout));
    ERRCHK_CUDA_API(hipFree(din));
    free(hout);
    free(hin);
    return EXIT_SUCCESS;
}

// int
// main_draft(void)
// {
//     const size_t count = 10;
//     Buffer<double> hin(count);
//     Buffer<double> hout(count);
//     // Buffer<double> din(count, true); // count, on_device, pinned
//     // Buffer<double> dout(count, true);
//     // Buffer<double> din(count, true);
//     din.pin();
//     din.unpin();
//     Buffer::migrate_async();
//     Buffer::sync();

//     hin.fill_arange();
//     Buffer::migrate(hin, din);
//     // Kernel
//     Buffer::migrate(dout, hout);

//     return EXIT_SUCCESS;
// }
