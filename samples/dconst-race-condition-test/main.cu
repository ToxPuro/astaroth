/**
 * A program for checking whether it is possible to get a race condition
 * when writing to a device constant that is shared between two kernels.
 *
 *
 *    Building and running:
 *      nvcc ../samples/dconst-race-condition-test/main.cu && ./a.out
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__device__ __constant__ int dconst;

__global__ void
kernel(int* output, int* dummy_output)
{
    volatile int j = 0;
    for (int i = 0; i < 1000000000; ++i)
        j += i;

    *output       = dconst;
    *dummy_output = j; // For ensuring that the compiler does not optimize out the loop above
}

static void
timestamp(const char* msg)
{
    time_t ltime = time(NULL);
    printf("%s - %s", msg, asctime(localtime(&ltime)));
    fflush(stdout);
}

int
main(void)
{
    hipStream_t stream0, stream1;
    const int aa = 1;
    const int bb = 2;
    int *cc, *dd;
    hipMallocManaged((void**)&cc, 1);
    hipMallocManaged((void**)&dd, 1);

    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    timestamp("Calling hipMemcpyToSymbolAsync stream0");
    hipMemcpyToSymbolAsync(HIP_SYMBOL(dconst), &aa, 1, 0, hipMemcpyHostToDevice, stream0);
    timestamp("Calling kernel stream0");
    kernel<<<1, 1, 0, stream0>>>(cc, dd);
    timestamp("Calling hipMemcpyToSymbolAsync stream1");
    hipMemcpyToSymbolAsync(HIP_SYMBOL(dconst), &bb, 1, 0, hipMemcpyHostToDevice, stream1);

    timestamp("Calling hipDeviceSynchronize");
    hipDeviceSynchronize();
    timestamp("Synchronized");

    printf("-------------\n");
    if (*cc == aa)
        printf("OK! %d == %d\n", *cc, aa);
    else
        printf("FAILURE: %d != %d\n", *cc, aa);
    printf("-------------\n");

    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    hipFree(cc);
    return EXIT_SUCCESS;
}