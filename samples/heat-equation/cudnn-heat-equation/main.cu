#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hipDNN.h>

#include "timer_hires.h"

#define CONV_TYPE HIPDNN_CROSS_CORRELATION

#define USE_DOUBLE (0)
#if USE_DOUBLE
static hipdnnDataType_t dtype = HIPDNN_DATA_DOUBLE;
typedef double real;
#else
static hipdnnDataType_t dtype = HIPDNN_DATA_FLOAT;
typedef float real;
#endif

static inline void
cudnn_errchk(hipdnnStatus_t code, const char* file, int line, bool abort)
{
  if (code != HIPDNN_STATUS_SUCCESS) {
    time_t terr;
    time(&terr);
    fprintf(stderr, "%s", ctime(&terr));
    fprintf(stderr, "\tcuDNN error in file %s line %d: %d\n", file, line, code);
    fflush(stderr);
  }
}

#define ERRCHK(params)                                                         \
  {                                                                            \
    cudnn_errchk((params), __FILE__, __LINE__, true);                          \
  }

int
main(void)
{

  int ndevices;
  hipGetDeviceCount(&ndevices);
  printf("Devices: %d\n", ndevices);
  assert(ndevices == 1);

  const int device = 0;
  hipSetDevice(device);

  hipdnnHandle_t nn;
  hipdnnCreate(&nn);

  hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW;

  // Input
  const size_t fn = 1;
  const size_t fc = 1;
  const size_t fh = 4096;
  const size_t fw = 4096;
  hipdnnTensorDescriptor_t input_desc;
  ERRCHK(hipdnnCreateTensorDescriptor(&input_desc));
  ERRCHK(hipdnnSetTensor4dDescriptor(input_desc, format, dtype, fn, fc, fh, fw));

  real* input;
  const size_t input_count = fn * fc * fh * fw;
  const size_t input_bytes = input_count * sizeof(input[0]);
  hipMalloc((void**)&input, input_bytes);

  // Host
  real* host_input = (real*)malloc(input_bytes);
  for (size_t i = 0; i < input_count; ++i)
    host_input[i] = 1;
  hipMemcpy(input, host_input, input_bytes, hipMemcpyHostToDevice);

  // Kernel
  const size_t gk = 1;
  const size_t gc = 1;
  const size_t gh = 3;
  const size_t gw = 3;
  hipdnnFilterDescriptor_t filter_desc;
  ERRCHK(hipdnnCreateFilterDescriptor(&filter_desc));
  ERRCHK(
      hipdnnSetFilter4dDescriptor(filter_desc, dtype, format, gk, gc, gh, gw));

  real* filter;
  const size_t filter_count = gk * gc * gh * gw;
  const size_t filter_bytes = filter_count * sizeof(filter[0]);
  hipMalloc((void**)&filter, filter_bytes);

  // Host
  real* host_filter = (real*)malloc(filter_bytes);
  for (size_t i = 0; i < filter_count; ++i)
    host_filter[i] = 1;
  hipMemcpy(filter, host_filter, filter_bytes, hipMemcpyHostToDevice);

  // Convolution
  const size_t pad_h = 1;
  const size_t pad_w = 1;
  const size_t str_h = 1;
  const size_t str_w = 1;
  const size_t dil_h = 1;
  const size_t dil_w = 1;
  hipdnnConvolutionDescriptor_t convolution_desc;
  ERRCHK(hipdnnCreateConvolutionDescriptor(&convolution_desc));
  ERRCHK(hipdnnSetConvolution2dDescriptor(convolution_desc, pad_h, pad_w, str_h,
                                         str_w, dil_h, dil_w, CONV_TYPE,
                                         dtype));

  // Output
  int fn_out;
  int fc_out;
  int fh_out;
  int fw_out;
  ERRCHK(hipdnnGetConvolution2dForwardOutputDim(convolution_desc, input_desc,
                                               filter_desc, &fn_out, &fc_out,
                                               &fh_out, &fw_out));
  hipdnnTensorDescriptor_t output_desc;
  ERRCHK(hipdnnCreateTensorDescriptor(&output_desc));
  ERRCHK(hipdnnSetTensor4dDescriptor(output_desc, format, dtype, fn_out, fc_out,
                                    fh_out, fw_out));
  real* output;
  const size_t output_count = fn_out * fc_out * fh_out * fw_out;
  const size_t output_bytes = output_count * sizeof(output[0]);
  hipMalloc((void**)&output, output_bytes);

  // Algorithm
  //   const hipdnnConvolutionFwdAlgo_t
  //       algorithm = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
  // const hipdnnConvolutionFwdAlgo_t algorithm =
  // HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM; const
  // hipdnnConvolutionFwdAlgo_t algorithm = HIPDNN_CONVOLUTION_FWD_ALGO_FFT;
  // const hipdnnConvolutionFwdAlgo_t algorithm =
  // HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD;
  const int requested_algorithms = 1;
  hipdnnConvolutionFwdAlgoPerf_t algorithms[requested_algorithms];
  int returned_algorithms;
  ERRCHK(hipdnnFindConvolutionForwardAlgorithm(
      nn, input_desc, filter_desc, convolution_desc, output_desc,
      requested_algorithms, &returned_algorithms, algorithms));
  assert(returned_algorithms == requested_algorithms);
  const hipdnnConvolutionFwdAlgo_t algorithm = algorithms[0].algo;

  // Workspace
  size_t workspace_size;
  ERRCHK(hipdnnGetConvolutionForwardWorkspaceSize(nn, input_desc, filter_desc,
                                                 convolution_desc, output_desc,
                                                 algorithm, &workspace_size));

  real* workspace;
  hipMalloc((void**)&workspace, workspace_size);

  // Compute ---------------------------------------
  const real alpha = 1;
  const real beta  = 0;

  // Warmup
  for (size_t i = 0; i < 10; ++i)
    ERRCHK(hipdnnConvolutionForward(
        nn, &alpha, input_desc, input, filter_desc, filter, convolution_desc,
        algorithm, workspace, workspace_size, &beta, output_desc, output));

  // Benchmark
  Timer t;
  hipDeviceSynchronize();
  timer_reset(&t);
  for (size_t i = 0; i < 1; ++i)
    hipdnnConvolutionForward(nn, &alpha, input_desc, input, filter_desc, filter,
                            convolution_desc, algorithm, workspace,
                            workspace_size, &beta, output_desc, output);
  hipDeviceSynchronize();
  timer_diff_print(t);

  // Check results ------------------------------
  real* output_host = (real*)malloc(output_bytes);
  hipMemcpy(output_host, output, output_bytes, hipMemcpyDeviceToHost);
  for (size_t j = (gh - 1) / 2; j < fh_out - (gh - 1) / 2; ++j) {
    for (size_t i = (gw - 1) / 2; i < fw_out - (gw - 1) / 2; ++i) {
      assert(output_host[i + j * fw_out] == 9);
      // printf("%lu: %g\n", i, output_host[i + j * fw_out]);
    }
  }

  // Free ---------------------------------------
  // Workspace
  hipFree(workspace);

  // Output
  hipFree(output);
  ERRCHK(hipdnnDestroyTensorDescriptor(output_desc));

  // Convolution
  ERRCHK(hipdnnDestroyConvolutionDescriptor(convolution_desc));

  // Filter
  hipFree(filter);
  ERRCHK(hipdnnDestroyFilterDescriptor(filter_desc));

  // Input
  hipFree(input);
  ERRCHK(hipdnnDestroyTensorDescriptor(input_desc));

  // cuDNN
  hipdnnDestroy(nn);
  return EXIT_SUCCESS;
}