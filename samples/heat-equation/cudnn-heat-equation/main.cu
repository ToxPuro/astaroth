#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hipDNN.h>

#include "timer_hires.h"

int
main(void)
{

  int ndevices;
  hipGetDeviceCount(&ndevices);
  printf("Devices: %d\n", ndevices);
  assert(ndevices == 1);

  const int device = 0;
  hipSetDevice(device);

  hipdnnHandle_t nn;
  hipdnnCreate(&nn);

  hipdnnDataType_t dtype      = HIPDNN_DATA_FLOAT;
  hipdnnTensorFormat_t format = HIPDNN_TENSOR_NHWC;

  // Input
  const size_t fn = 1;
  const size_t fc = 1;
  const size_t fh = 4096;
  const size_t fw = 4096;
  hipdnnTensorDescriptor_t input_desc;
  hipdnnCreateTensorDescriptor(&input_desc);
  hipdnnSetTensor4dDescriptor(input_desc, format, dtype, fn, fc, fh, fw);

  float* input;
  hipMalloc((void**)&input, fn * fc * fh * fw * sizeof(input[0]));

  // Kernel
  const size_t gk = 1;
  const size_t gc = 1;
  const size_t gh = 3;
  const size_t gw = 3;
  hipdnnFilterDescriptor_t filter_desc;
  hipdnnCreateFilterDescriptor(&filter_desc);
  hipdnnSetFilter4dDescriptor(filter_desc, dtype, format, gk, gc, gh, gw);

  float* filter;
  hipMalloc((void**)&filter, gk * gc * gh * gw * sizeof(filter[0]));

  // Convolution
  const size_t pad_h = 1;
  const size_t pad_w = 1;
  const size_t str_h = 1;
  const size_t str_w = 1;
  const size_t dil_h = 1;
  const size_t dil_w = 1;
  hipdnnConvolutionDescriptor_t convolution_desc;
  hipdnnCreateConvolutionDescriptor(&convolution_desc);
  hipdnnSetConvolution2dDescriptor(convolution_desc, pad_h, pad_w, str_h, str_w,
                                  dil_h, dil_w, HIPDNN_CONVOLUTION, dtype);

  // Output
  int fn_out;
  int fc_out;
  int fh_out;
  int fw_out;
  hipdnnGetConvolution2dForwardOutputDim(convolution_desc, input_desc,
                                        filter_desc, &fn_out, &fc_out, &fh_out,
                                        &fw_out);

  hipdnnTensorDescriptor_t output_desc;
  hipdnnCreateTensorDescriptor(&output_desc);
  hipdnnSetTensor4dDescriptor(output_desc, format, dtype, fn_out, fc_out, fh_out,
                             fw_out);
  float* output;
  hipMalloc((void**)&output,
             fn_out * fc_out * fh_out * fw_out * sizeof(output[0]));

  // Algorithm
  const hipdnnConvolutionFwdAlgo_t
      algorithm = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
  // const hipdnnConvolutionFwdAlgo_t algorithm =
  // HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM; const
  // hipdnnConvolutionFwdAlgo_t algorithm = HIPDNN_CONVOLUTION_FWD_ALGO_FFT; const
  // hipdnnConvolutionFwdAlgo_t algorithm = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD;

  // Workspace
  size_t workspace_size;
  hipdnnGetConvolutionForwardWorkspaceSize(nn, input_desc, filter_desc,
                                          convolution_desc, output_desc,
                                          algorithm, &workspace_size);

  float* workspace;
  hipMalloc((void**)&workspace, workspace_size);

  // Compute ---------------------------------------
  const float alpha = 1.0f;
  const float beta  = 0.0f;

  // Warmup
  for (size_t i = 0; i < 10; ++i)
    hipdnnConvolutionForward(nn, &alpha, input_desc, input, filter_desc, filter,
                            convolution_desc, algorithm, workspace,
                            workspace_size, &beta, output_desc, output);

  // Benchmark
  Timer t;
  hipDeviceSynchronize();
  timer_reset(&t);
  for (size_t i = 0; i < 1; ++i)
    hipdnnConvolutionForward(nn, &alpha, input_desc, input, filter_desc, filter,
                            convolution_desc, algorithm, workspace,
                            workspace_size, &beta, output_desc, output);
  hipDeviceSynchronize();
  timer_diff_print(t);

  // Free ---------------------------------------
  // Workspace
  hipFree(workspace);

  // Output
  hipFree(output);
  hipdnnDestroyTensorDescriptor(output_desc);

  // Convolution
  hipdnnDestroyConvolutionDescriptor(convolution_desc);

  // Filter
  hipFree(filter);
  hipdnnDestroyFilterDescriptor(filter_desc);

  // Input
  hipFree(input);
  hipdnnDestroyTensorDescriptor(input_desc);

  // cuDNN
  hipdnnDestroy(nn);
  return EXIT_SUCCESS;
}