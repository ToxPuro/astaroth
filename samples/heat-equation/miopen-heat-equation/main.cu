#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <miopen/miopen.h>

#include "hip.h"

#define hipMallocManaged hipMallocManaged

#define hipdnnHandle_t miopenHandle_t
#define hipdnnCreate miopenCreate
#define hipdnnDataType_t miopenDataType_t

#define hipdnnTensorFormat_t miopenTensorFormat_t
#define hipdnnTensorDescriptor_t miopenTensorDescriptor_t
#define hipdnnCreateTensorDescriptor miopenCreateTensorDescriptor
#define hipdnnSetTensor4dDescriptor miopenSet4dTensorDescriptor

#define hipdnnFilterDescriptor_t miopenTensorDescriptor_t
#define hipdnnCreateFilterDescriptor miopenCreateTensorDescriptor
#define hipdnnSetFilter4dDescriptor miopenSet4dTensorDescriptor

#define hipdnnConvolutionDescriptor_t miopenConvolutionDescriptor_t
#define hipdnnCreateConvolutionDescriptor miopenCreateConvolutionDescriptor
#define HIPDNN_CONVOLUTION miopenConvolution
#define hipdnnSetConvolution2dDescriptor miopenInitConvolutionDescriptor
#define hipdnnGetConvolution2dForwardOutputDim                                  \
  miopenGetConvolutionForwardOutputDim

#define hipdnnGetConvolutionForwardWorkspaceSize                                \
  miopenConvolutionForwardGetWorkSpaceSize

#define hipdnnConvolutionForward miopenConvolutionForward

// #define HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM                               \
//   miopenConvolutionFwdAlgoImplicitGEMM
// miopenConvolutionFwdAlgoDirect

#define hipdnnDestroyConvolutionDescriptor miopenDestroyConvolutionDescriptor
#define hipdnnDestroy miopenDestroy
#define hipdnnDestroyTensorDescriptor miopenDestroyTensorDescriptor

#define hipdnnConvolutionFwdAlgo_t miopenConvFwdAlgorithm_t
#define hipdnnDestroyFilterDescriptor miopenDestroyTensorDescriptor

#define HIPDNN_DATA_FLOAT miopenFloat
#define HIPDNN_TENSOR_NHWC miopenTensor

#include "timer_hires.h"

int
main(void)
{

  int ndevices;
  hipGetDeviceCount(&ndevices);
  printf("Devices: %d\n", ndevices);
  assert(ndevices == 1);

  const int device = 0;
  hipSetDevice(device);

  hipdnnHandle_t nn;
  hipdnnCreate(&nn);

  hipdnnDataType_t dtype = HIPDNN_DATA_FLOAT;
  // hipdnnTensorFormat_t format = HIPDNN_TENSOR_NHWC; // NHWC not supported

  // Input
  const size_t fn = 1;
  const size_t fc = 1;
  const size_t fh = 4096;
  const size_t fw = 4096;
  hipdnnTensorDescriptor_t input_desc;
  hipdnnCreateTensorDescriptor(&input_desc);
  // hipdnnSetTensor4dDescriptor(input_desc, format, dtype, fn, fc, fh, fw);
  hipdnnSetTensor4dDescriptor(input_desc, dtype, fn, fc, fh, fw);

  float* input;
  hipMallocManaged((void**)&input, fn * fc * fh * fw * sizeof(input[0]));

  // // Kernel
  const size_t gk = 1;
  const size_t gc = 1;
  const size_t gh = 3;
  const size_t gw = 3;
  hipdnnFilterDescriptor_t filter_desc;
  hipdnnCreateFilterDescriptor(&filter_desc);
  hipdnnSetFilter4dDescriptor(filter_desc, dtype, gk, gc, gh, gw);

  float* filter;
  hipMallocManaged((void**)&filter, gk * gc * gh * gw * sizeof(filter[0]));

  // Convolution
  const size_t pad_h = 1;
  const size_t pad_w = 1;
  const size_t str_h = 1;
  const size_t str_w = 1;
  const size_t dil_h = 1;
  const size_t dil_w = 1;
  hipdnnConvolutionDescriptor_t convolution_desc;
  hipdnnCreateConvolutionDescriptor(&convolution_desc);
  // hipdnnSetConvolution2dDescriptor(convolution_desc, pad_h, pad_w, str_h,
  // str_w,
  //                                 dil_h, dil_w, HIPDNN_CONVOLUTION, dtype);
  hipdnnSetConvolution2dDescriptor(convolution_desc, HIPDNN_CONVOLUTION, pad_h,
                                  pad_w, str_h, str_w, dil_h, dil_w);

  // Output
  int fn_out;
  int fc_out;
  int fh_out;
  int fw_out;
  hipdnnGetConvolution2dForwardOutputDim(convolution_desc, input_desc,
                                        filter_desc, &fn_out, &fc_out, &fh_out,
                                        &fw_out);

  hipdnnTensorDescriptor_t output_desc;
  hipdnnCreateTensorDescriptor(&output_desc);
  hipdnnSetTensor4dDescriptor(output_desc, dtype, fn_out, fc_out, fh_out,
                             fw_out);
  float* output;
  hipMallocManaged((void**)&output,
                    fn_out * fc_out * fh_out * fw_out * sizeof(output[0]));

  // Algorithm
  // const hipdnnConvolutionFwdAlgo_t
  //     algorithm = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
  // const hipdnnConvolutionFwdAlgo_t algorithm =
  // HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM; const
  // hipdnnConvolutionFwdAlgo_t algorithm = HIPDNN_CONVOLUTION_FWD_ALGO_FFT;
  // const hipdnnConvolutionFwdAlgo_t algorithm =
  // HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD;

  // Workspace
  size_t workspace_size;
  // hipdnnGetConvolutionForwardWorkspaceSize(nn, input_desc, filter_desc,
  //                                         convolution_desc, output_desc,
  //                                         algorithm, &workspace_size);
  hipdnnGetConvolutionForwardWorkspaceSize(nn, input_desc, filter_desc,
                                          convolution_desc, output_desc,
                                          &workspace_size);

  float* workspace;
  hipMallocManaged((void**)&workspace, workspace_size);

  // // FindConvolution() is mandatory.
  // // Allocate workspace prior to running this API.
  // // A table with times and memory requirements
  // // for different algorithms is returned.
  // // Users can choose the top-most algorithm if
  // // they only care about the fastest algorithm.
  // miopenStatus_t
  // miopenFindConvolutionForwardAlgorithm(
  //     miopenHandle_t handle,
  //     const miopenTensorDescriptor_t xDesc,
  //     const void *x,
  //     const miopenTensorDescriptor_t wDesc,
  //     const void *w,
  //     const miopenConvolutionDescriptor_t convDesc,
  //     const miopenTensorDescriptor_t yDesc,
  //     void *y,
  //     const int requestAlgoCount,
  //     int *returnedAlgoCount,
  //     miopenConvAlgoPerf_t *perfResults,
  //     void *workSpace,
  //     size_t workSpaceSize,
  //     bool exhaustiveSearch)
  const int required_algorithms = 1;
  miopenConvAlgoPerf_t algorithms[required_algorithms];
  int returned_algorithms;
  miopenFindConvolutionForwardAlgorithm(
      nn, input_desc, input, filter_desc, filter, convolution_desc, output_desc,
      output, required_algorithms, &returned_algorithms, algorithms, workspace,
      workspace_size, true);
  assert(returned_algorithms == required_algorithms);

  printf("Convolution algorithm selected: ");
  switch (algorithms[0].fwd_algo) {
  case 0:
    printf("miOpenConvolutionAlgoGEMM\n");
    break;
  case 1:
    printf("miopenConvolutionAlgoDirect\n");
    break;
  case 2:
    printf("miopenConvolutionAlgoFFT\n");
    break;
  case 3:
    printf("miopenConvolutionAlgoWinograd\n");
    break;
  case 5: // Yes, skips 4
    printf("miopenConvolutionAlgoImplicitGEMM\n");
    break;
  }

  // Compute ---------------------------------------
  const float alpha = 1.0f;
  const float beta  = 0.0f;

  // Warmup
  for (size_t i = 0; i < 10; ++i)
    hipdnnConvolutionForward(nn, &alpha, input_desc, input, filter_desc, filter,
                            convolution_desc, algorithms[0].fwd_algo, &beta,
                            output_desc, output, workspace, workspace_size);

  // hipdnnConvolutionForward(nn, &alpha, input_desc, input, filter_desc, filter,
  //                         convolution_desc, workspace, workspace_size, &beta,
  //                         output_desc, output);

  /*
  miopenStatus_t
  miopenConvolutionForward(
      miopenHandle_t handle,
      const void *alpha,
      const miopenTensorDescriptor_t xDesc,
      const void *x,
      const miopenTensorDescriptor_t wDesc,
      const void *w,
      const miopenConvolutionDescriptor_t convDesc,
      miopenConvFwdAlgorithm_t algo,
      const void *beta,
      const miopenTensorDescriptor_t yDesc,´`
      void *y,
      void *workSpace,
      size_t workSpaceSize)
  */

  // Benchmark
  Timer t;
  hipDeviceSynchronize();
  timer_reset(&t);
  for (size_t i = 0; i < 1; ++i)
    hipdnnConvolutionForward(nn, &alpha, input_desc, input, filter_desc, filter,
                            convolution_desc, algorithms[0].fwd_algo, &beta,
                            output_desc, output, workspace, workspace_size);

  // hipdnnConvolutionForward(nn, &alpha, input_desc, input, filter_desc, filter,
  //                         convolution_desc, workspace, workspace_size, &beta,
  //                         output_desc, output);

  hipDeviceSynchronize();
  timer_diff_print(t);

  // Free ---------------------------------------
  // Workspace
  hipFree(workspace);

  // Output
  hipFree(output);
  hipdnnDestroyTensorDescriptor(output_desc);

  // Convolution
  hipdnnDestroyConvolutionDescriptor(convolution_desc);

  // Filter
  hipFree(filter);
  hipdnnDestroyFilterDescriptor(filter_desc);

  // Input
  hipFree(input);
  hipdnnDestroyTensorDescriptor(input_desc);

  // cuDNN
  hipdnnDestroy(nn);
  return EXIT_SUCCESS;
}