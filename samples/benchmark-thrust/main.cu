#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "acc_runtime.h"
#include "timer_hires.h"

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>

#include "astaroth_utils.h"

template <typename T> struct square {
    __host__ __device__ T operator()(const T& x) const { return x * x; }
};

static void
benchmark_thrust(const size_t mx, const size_t my, const size_t mz)
{
    std::vector<thrust::device_vector<double>> inputs(mz);
    for (size_t k = 0; k < mz; ++k)
        inputs[k] = thrust::device_vector<double>(mx * my, 1);

    thrust::device_vector<double> results(mz);

    for (size_t k = 0; k < mz; ++k)
        results[k] = thrust::reduce(inputs[k].begin(), inputs[k].end());

    Timer t;
    hipDeviceSynchronize();
    timer_reset(&t);
    for (size_t k = 0; k < mz; ++k)
        results[k] = thrust::reduce(inputs[k].begin(), inputs[k].end());
    hipDeviceSynchronize();
    timer_diff_print(t);
}

int
main(int argc, char** argv)
{
    hipProfilerStop();

    fprintf(stderr, "Usage: ./benchmark-thrust <nx> <ny> <nz>\n");
    const size_t nx     = (argc > 1) ? (size_t)atol(argv[1]) : 32;
    const size_t ny     = (argc > 2) ? (size_t)atol(argv[2]) : 32;
    const size_t nz     = (argc > 3) ? (size_t)atol(argv[3]) : 32;
    const size_t radius = STENCIL_ORDER / 2;
    const size_t mx     = nx + 2 * radius;
    const size_t my     = ny + 2 * radius;
    const size_t mz     = nz + 2 * radius;

    printf("Input parameters:\n");
    printf("\tnx: %zu\n", nx);
    printf("\tny: %zu\n", ny);
    printf("\tnz: %zu\n", nz);
    printf("\tradius: %zu\n", radius);
    printf("\tmx: %zu\n", mx);
    printf("\tmy: %zu\n", my);
    printf("\tmz: %zu\n", mz);

    // Device
    hipSetDevice(0);

#if 1
    hipProfilerStart();
    benchmark_thrust(mx, my, mz);
    hipProfilerStop();
#else
    // Mesh info
    AcMeshInfo info;
    acLoadConfig(AC_DEFAULT_CONFIG, &info);
    acSetMeshDims(nx, ny, nz, &info);
    acLoadMeshInfo(info, 0);

    // Random
    const size_t seed  = 12345;
    const size_t pid   = 0;
    const size_t count = acVertexBufferCompdomainSize(info);
    acRandInitAlt(seed, count, pid);

    AcMeshDims dims       = acGetMeshDims(info);
    VertexBufferArray vba = acVBACreate(mx, my, mz);
    acVBAReset(0, &vba);
    acLaunchKernel(randomize, 0, dims.n0, dims.n1, vba);
    acVBASwapBuffers(&vba);

    ProfileBufferArray pba = acPBACreate(mz);
    AcBufferArray ba0      = acBufferArrayCreate(12, nx * ny);
    AcBufferArray ba1      = acBufferArrayCreate(12, nx * ny);
    AcBufferArray ba2      = acBufferArrayCreate(12, nx * ny);
    // acMapCross(vba, 0, (int3){radius, radius, 0}, (int3){radius + nx, radius + ny, 1},
    // scratchpad);
    acMapCrossReduce(vba, 0, ba0, ba1, ba2, pba);

    Timer t;

    hipProfilerStart();
    hipDeviceSynchronize();
    timer_reset(&t);
    // for (size_t k = 0; k < mz; ++k)
    // acMapCross(vba, 0, (int3){radius, radius, k}, (int3){radius + nx, radius + ny, k +
    // 1},
    //            scratchpad);
    acMapCrossReduce(vba, 0, ba0, ba1, ba2, pba);

    hipDeviceSynchronize();
    timer_diff_print(t);
    hipProfilerStop();
    hipDeviceSynchronize();

    ERRCHK_CUDA_KERNEL_ALWAYS();
    acRandQuit();
    acBufferArrayDestroy(&ba2);
    acBufferArrayDestroy(&ba1);
    acBufferArrayDestroy(&ba0);
    acPBADestroy(&pba);
    acVBADestroy(&vba);
#endif
    return EXIT_SUCCESS;
}