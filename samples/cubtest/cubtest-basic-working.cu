#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include <hipcub/hipcub.hpp>

#include "timer_hires.h"

#define ARRAY_SIZE(x) (sizeof(x) / (sizeof(x[0])))
#define NUM_SAMPLES (10)

int
main(void)
{
    // Allocate
    const double in[1024]     = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    const size_t num_segments = 2;
    const size_t offsets[]    = {0, 5, ARRAY_SIZE(in)}; //{0, 3, 3, count};
    assert(num_segments + 1 == ARRAY_SIZE(offsets));
    double out[num_segments] = {0};

    double* d_in;
    hipMalloc(&d_in, sizeof(d_in[0]) * ARRAY_SIZE(in));
    hipMemcpy(d_in, in, sizeof(d_in[0]) * ARRAY_SIZE(in), hipMemcpyHostToDevice);

    size_t* d_offsets;
    hipMalloc(&d_offsets, sizeof(d_offsets[0]) * ARRAY_SIZE(offsets));
    hipMemcpy(d_offsets, offsets, sizeof(d_offsets[0]) * ARRAY_SIZE(offsets),
               hipMemcpyHostToDevice);

    double* d_out;
    hipMalloc(&d_out, sizeof(d_out[0]) * ARRAY_SIZE(out));

    void* d_temp_storage      = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_segments,
                                    d_offsets, d_offsets + 1);
    // hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, count);
    printf("Temp storage: %zu bytes\n", temp_storage_bytes);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Warmup
    hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_segments,
                                    d_offsets, d_offsets + 1);

    // Benchmark and compute
    double time_elapsed = 0;
    for (size_t i = 0; i < NUM_SAMPLES; ++i) {
        hipDeviceSynchronize();
        Timer t;
        timer_reset(&t);
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out,
                                        num_segments, d_offsets, d_offsets + 1);
        // hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, count);
        hipDeviceSynchronize();
        time_elapsed += timer_diff_nsec(t) / 1e6;
        // timer_diff_print(t);
    }
    time_elapsed /= NUM_SAMPLES;
    printf("Average time elapsed: %g ms\n", time_elapsed);

    // Get results
    hipMemcpy(out, d_out, sizeof(out[0]) * num_segments, hipMemcpyDeviceToHost);
    for (size_t i = 0; i < ARRAY_SIZE(out); ++i)
        printf("%zu: %g\n", i, out[i]);

    // Deallocate
    hipFree(d_temp_storage);
    hipFree(d_out);
    hipFree(d_offsets);
    hipFree(d_in);

    return EXIT_SUCCESS;
}