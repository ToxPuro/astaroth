#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#if USE_HIP
#include <hipcub/hipcub.hpp>

#define cub hipcub
#define hipMalloc hipMalloc
#define hipFree hipFree
#define hipMemcpy hipMemcpy
#define hipMemcpyHostToDevice hipMemcpyHostToDevice
#define hipMemcpyDeviceToHost hipMemcpyDeviceToHost
#define hipDeviceSynchronize hipDeviceSynchronize
#else
#include <hipcub/hipcub.hpp>
#endif

#include "timer_hires.h"

#define ARRAY_SIZE(x) (sizeof(x) / (sizeof(x[0])))
#define NUM_SAMPLES (10)

int
main(void)
{
    // Allocate
    const size_t nn           = 256;
    const size_t np           = 32;
    const size_t count        = nn * nn * nn * np;
    const size_t num_segments = nn * np;

    double* in = (double*)malloc(sizeof(in[0]) * count);
    assert(in);
    for (size_t i = 0; i < count; ++i)
        in[i] = !(i % (nn * nn)) ? i / (nn * nn) : 0;

    size_t* offsets = (size_t*)malloc(sizeof(offsets[0]) * (num_segments + 1));
    assert(offsets);
    for (size_t i = 0; i <= num_segments; ++i)
        offsets[i] = i * (count / num_segments);

    double* out = (double*)malloc(sizeof(out[0]) * num_segments);
    assert(out);

    double* d_in;
    hipMalloc(&d_in, sizeof(d_in[0]) * count);
    assert(d_in);
    hipMemcpy(d_in, in, sizeof(d_in[0]) * count, hipMemcpyHostToDevice);

    size_t* d_offsets;
    hipMalloc(&d_offsets, sizeof(d_offsets[0]) * (num_segments + 1));
    assert(d_offsets);
    hipMemcpy(d_offsets, offsets, sizeof(d_offsets[0]) * (num_segments + 1),
               hipMemcpyHostToDevice);

    double* d_out;
    hipMalloc(&d_out, sizeof(d_out[0]) * num_segments);
    assert(d_out);

    void* d_temp_storage      = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_segments,
                                    d_offsets, d_offsets + 1);
    // hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, count);
    printf("Temp storage: %zu bytes\n", temp_storage_bytes);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    assert(d_temp_storage);

    // Warmup
    hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_segments,
                                    d_offsets, d_offsets + 1);

    // Benchmark and compute
    double time_elapsed = 0;
    for (size_t i = 0; i < NUM_SAMPLES; ++i) {
        hipDeviceSynchronize();
        Timer t;
        timer_reset(&t);
        hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out,
                                        num_segments, d_offsets, d_offsets + 1);
        // hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, count);
        hipDeviceSynchronize();
        time_elapsed += timer_diff_nsec(t) / 1e6;
        // timer_diff_print(t);
    }
    time_elapsed /= NUM_SAMPLES;
    printf("Average time elapsed: %g ms\n", time_elapsed);

    // Get results
    hipMemcpy(out, d_out, sizeof(out[0]) * num_segments, hipMemcpyDeviceToHost);
    for (size_t i = 0; i < num_segments; ++i) {
        if (out[i] != i)
            printf("%zu: %g\n", i, out[i]);
        assert(out[i] == i);
    }

    // Deallocate
    hipFree(d_temp_storage);
    hipFree(d_out);
    hipFree(d_offsets);
    hipFree(d_in);
    free(out);
    free(offsets);
    free(in);

    return EXIT_SUCCESS;
}