#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2021, Johannes Pekkila, Miikka Vaisala.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "kernels.h"

#include "acc_runtime.cu"

static_assert(NUM_VTXBUF_HANDLES > 0, "ERROR: At least one uniform ScalarField must be declared.");

static __global__ void
dummy_kernel(void)
{
    DCONST((AcIntParam)0);
    DCONST((AcInt3Param)0);
    DCONST((AcRealParam)0);
    DCONST((AcReal3Param)0);
    acComplex a = exp(acComplex(1, 1) * AcReal(1));
    a* a;
}

AcResult
acKernelDummy(void)
{
    dummy_kernel<<<1, 1>>>();
    ERRCHK_CUDA_KERNEL_ALWAYS();
    return AC_SUCCESS;
}

static __global__ void
flush_kernel(AcReal* arr, const size_t n)
{
    const int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n)
        arr[idx] = (AcReal)NAN;
}

AcResult
acKernelFlush(AcReal* arr, const size_t n)
{
    const size_t tpb = 256;
    const size_t bpg = (size_t)(ceil((double)n / tpb));
    flush_kernel<<<bpg, tpb>>>(arr, n);
    ERRCHK_CUDA_KERNEL_ALWAYS();
    return AC_SUCCESS;
}

AcResult
acDeviceLoadScalarUniform(const Device device, const Stream stream, const AcRealParam param,
                          const AcReal value)
{
    hipSetDevice(device->id);
    return acLoadRealUniform(device->streams[stream], param, value);
}

AcResult
acDeviceLoadVectorUniform(const Device device, const Stream stream, const AcReal3Param param,
                          const AcReal3 value)
{
    hipSetDevice(device->id);
    return acLoadReal3Uniform(device->streams[stream], param, value);
}

AcResult
acDeviceLoadIntUniform(const Device device, const Stream stream, const AcIntParam param,
                       const int value)
{
    hipSetDevice(device->id);
    return acLoadIntUniform(device->streams[stream], param, value);
}

AcResult
acDeviceLoadInt3Uniform(const Device device, const Stream stream, const AcInt3Param param,
                        const int3 value)
{
    hipSetDevice(device->id);
    return acLoadInt3Uniform(device->streams[stream], param, value);
}

AcResult
acDeviceLoadMeshInfo(const Device device, const AcMeshInfo config)
{
    hipSetDevice(device->id);

    AcMeshInfo device_config = config;
    acHostUpdateBuiltinParams(&device_config);

    ERRCHK_ALWAYS(device_config.int_params[AC_nx] == device->local_config.int_params[AC_nx]);
    ERRCHK_ALWAYS(device_config.int_params[AC_ny] == device->local_config.int_params[AC_ny]);
    ERRCHK_ALWAYS(device_config.int_params[AC_nz] == device->local_config.int_params[AC_nz]);
    ERRCHK_ALWAYS(device_config.int_params[AC_multigpu_offset] ==
                  device->local_config.int_params[AC_multigpu_offset]);

    for (int i = 0; i < NUM_INT_PARAMS; ++i)
        acDeviceLoadIntUniform(device, STREAM_DEFAULT, (AcIntParam)i, device_config.int_params[i]);

    for (int i = 0; i < NUM_INT3_PARAMS; ++i)
        acDeviceLoadInt3Uniform(device, STREAM_DEFAULT, (AcInt3Param)i,
                                device_config.int3_params[i]);

    for (int i = 0; i < NUM_REAL_PARAMS; ++i)
        acDeviceLoadScalarUniform(device, STREAM_DEFAULT, (AcRealParam)i,
                                  device_config.real_params[i]);

    for (int i = 0; i < NUM_REAL3_PARAMS; ++i)
        acDeviceLoadVectorUniform(device, STREAM_DEFAULT, (AcReal3Param)i,
                                  device_config.real3_params[i]);

    return AC_SUCCESS;
}

// Built-in kernels
#include "boundconds.cuh"
#include "boundconds_miikka_GBC.cuh"
#include "packing.cuh"
#include "reductions.cuh"
#include "volume_copy.cuh"

AcResult
acKernel(const KernelParameters params, VertexBufferArray vba)
{
#if AC_INTEGRATION_ENABLED
    // TODO: Why is AC_step_number loaded here??
    acLoadIntUniform(params.stream, AC_step_number, params.step_number);
    acLaunchKernel(params.kernel, params.stream, params.start, params.end, vba);
    return AC_SUCCESS;
#else
    (void)params; // Unused
    (void)vba;    // Unused
    ERROR("acKernel() called but AC_step_number not defined!");
    return AC_FAILURE;
#endif
}
