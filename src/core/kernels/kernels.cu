#include "hip/hip_runtime.h"
#include "kernels.h"

#include "acc_runtime.cu"

static_assert(NUM_VTXBUF_HANDLES > 0, "ERROR: At least one uniform ScalarField must be declared.");

static __global__ void
dummy_kernel(void)
{
    DCONST((AcIntParam)0);
    DCONST((AcInt3Param)0);
    DCONST((AcRealParam)0);
    DCONST((AcReal3Param)0);
    acComplex a = exp(acComplex(1, 1) * AcReal(1));
    a* a;
}

AcResult
acKernelDummy(void)
{
    dummy_kernel<<<1, 1>>>();
    ERRCHK_CUDA_KERNEL_ALWAYS();
    return AC_SUCCESS;
}

AcResult
acDeviceLoadScalarUniform(const Device device, const Stream stream, const AcRealParam param,
                          const AcReal value)
{
    hipSetDevice(device->id);
    if (param < 0 || param >= NUM_REAL_PARAMS) {
        fprintf(stderr, "WARNING: invalid AcRealParam %d.\n", param);
        return AC_FAILURE;
    }

    if (!is_valid(value)) {
        fprintf(stderr,
                "WARNING: Passed an invalid value %g to device constant %s. "
                "Skipping.\n",
                (double)value, realparam_names[param]);
        return AC_FAILURE;
    }

    const size_t offset = (size_t)&d_mesh_info.real_params[param] - (size_t)&d_mesh_info;
    ERRCHK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset,
                                        hipMemcpyHostToDevice, device->streams[stream]));
    return AC_SUCCESS;
}

AcResult
acDeviceLoadVectorUniform(const Device device, const Stream stream, const AcReal3Param param,
                          const AcReal3 value)
{
    hipSetDevice(device->id);
    if (param < 0 || param >= NUM_REAL3_PARAMS) {
        fprintf(stderr, "WARNING: invalid AcReal3Param %d\n", param);
        return AC_FAILURE;
    }

    if (!is_valid(value)) {
        fprintf(stderr,
                "WARNING: Passed an invalid value (%g, %g, %g) to device constant "
                "%s. Skipping.\n",
                (double)value.x, (double)value.y, (double)value.z, real3param_names[param]);
        return AC_FAILURE;
    }

    const size_t offset = (size_t)&d_mesh_info.real3_params[param] - (size_t)&d_mesh_info;
    ERRCHK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset,
                                        hipMemcpyHostToDevice, device->streams[stream]));
    return AC_SUCCESS;
}

AcResult
acLoadIntUniform(const hipStream_t stream, const AcIntParam param, const int value)
{
    if (param < 0 || param >= NUM_INT_PARAMS) {
        fprintf(stderr, "WARNING: invalid AcIntParam %d\n", param);
        return AC_FAILURE;
    }

    if (!is_valid(value)) {
        fprintf(stderr,
                "WARNING: Passed an invalid value %d to device constant %s. "
                "Skipping.\n",
                value, intparam_names[param]);
        return AC_FAILURE;
    }

    const size_t offset = (size_t)&d_mesh_info.int_params[param] - (size_t)&d_mesh_info;
    ERRCHK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset,
                                        hipMemcpyHostToDevice, stream));
    return AC_SUCCESS;
}

AcResult
acDeviceLoadIntUniform(const Device device, const Stream stream, const AcIntParam param,
                       const int value)
{
    hipSetDevice(device->id);
    return acLoadIntUniform(device->streams[stream], param, value);
}

AcResult
acDeviceLoadInt3Uniform(const Device device, const Stream stream, const AcInt3Param param,
                        const int3 value)
{
    hipSetDevice(device->id);
    if (param < 0 || param >= NUM_INT3_PARAMS) {
        fprintf(stderr, "WARNING: invalid AcInt3Param %d\n", param);
        return AC_FAILURE;
    }

    if (!is_valid(value.x) || !is_valid(value.y) || !is_valid(value.z)) {
        fprintf(stderr,
                "WARNING: Passed an invalid value (%d, %d, %def) to device "
                "constant %s. "
                "Skipping.\n",
                value.x, value.y, value.z, int3param_names[param]);
        return AC_FAILURE;
    }

    const size_t offset = (size_t)&d_mesh_info.int3_params[param] - (size_t)&d_mesh_info;
    ERRCHK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset,
                                        hipMemcpyHostToDevice, device->streams[stream]));
    return AC_SUCCESS;
}

AcResult
acDeviceLoadMeshInfo(const Device device, const AcMeshInfo config)
{
    hipSetDevice(device->id);

    AcMeshInfo device_config = config;
    acHostUpdateBuiltinParams(&device_config);

    ERRCHK_ALWAYS(device_config.int_params[AC_nx] == device->local_config.int_params[AC_nx]);
    ERRCHK_ALWAYS(device_config.int_params[AC_ny] == device->local_config.int_params[AC_ny]);
    ERRCHK_ALWAYS(device_config.int_params[AC_nz] == device->local_config.int_params[AC_nz]);
    ERRCHK_ALWAYS(device_config.int_params[AC_multigpu_offset] ==
                  device->local_config.int_params[AC_multigpu_offset]);

    for (int i = 0; i < NUM_INT_PARAMS; ++i)
        acDeviceLoadIntUniform(device, STREAM_DEFAULT, (AcIntParam)i, device_config.int_params[i]);

    for (int i = 0; i < NUM_INT3_PARAMS; ++i)
        acDeviceLoadInt3Uniform(device, STREAM_DEFAULT, (AcInt3Param)i,
                                device_config.int3_params[i]);

    for (int i = 0; i < NUM_REAL_PARAMS; ++i)
        acDeviceLoadScalarUniform(device, STREAM_DEFAULT, (AcRealParam)i,
                                  device_config.real_params[i]);

    for (int i = 0; i < NUM_REAL3_PARAMS; ++i)
        acDeviceLoadVectorUniform(device, STREAM_DEFAULT, (AcReal3Param)i,
                                  device_config.real3_params[i]);

    return AC_SUCCESS;
}

AcResult
acDeviceLoadStencils(const Device device, const Stream stream,
                     AcReal stencil[NUM_STENCILS][STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
    hipSetDevice(device->id);

    const size_t bytes = sizeof(stencil[0][0][0][0]) * NUM_STENCILS * STENCIL_DEPTH *
                         STENCIL_HEIGHT * STENCIL_WIDTH;
    ERRCHK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(stencils), stencil, bytes, 0, hipMemcpyHostToDevice,
                                        device->streams[stream]));

    return AC_FAILURE;
}

// Built-in kernels
#include "boundconds.cuh"
#include "boundconds_miikka_GBC.cuh"
#include "packing.cuh"
#include "reductions.cuh"

AcResult
acKernel(const KernelParameters params, VertexBufferArray vba)
{
#ifndef AC_step_number
    (void)params; // Unused
    (void)vba;    // Unused
    ERROR("acKernel() called but AC_step_number not defined!");
    return AC_FAILURE;
#else

    // TODO: Why is AC_step_number loaded here??
    acLoadIntUniform(params.stream, AC_step_number, params.step_number);
    acLaunchKernel(params.kernel, params.stream, params.start, params.end, vba);
    return AC_SUCCESS;
#endif
}
