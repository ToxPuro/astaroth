#include "hip/hip_runtime.h"
#include "kernels.h"

#include <assert.h>
#include <hip/hip_complex.h>

#include "errchk.h"
#include "math_utils.h"

__device__ __constant__ AcMeshInfo d_mesh_info;

static int __device__ __forceinline__
DCONST(const AcIntParam param)
{
    return d_mesh_info.int_params[param];
}
static int3 __device__ __forceinline__
DCONST(const AcInt3Param param)
{
    return d_mesh_info.int3_params[param];
}
static AcReal __device__ __forceinline__
DCONST(const AcRealParam param)
{
    return d_mesh_info.real_params[param];
}
static AcReal3 __device__ __forceinline__
DCONST(const AcReal3Param param)
{
    return d_mesh_info.real3_params[param];
}
static __device__ constexpr VertexBufferHandle
DCONST(const VertexBufferHandle handle)
{
    return handle;
}
#define DEVICE_VTXBUF_IDX(i, j, k) ((i) + (j)*DCONST(AC_mx) + (k)*DCONST(AC_mxy))
#define DEVICE_1D_COMPDOMAIN_IDX(i, j, k) ((i) + (j)*DCONST(AC_nx) + (k)*DCONST(AC_nxy))
#define globalGridN (d_mesh_info.int3_params[AC_global_grid_n])
//#define globalMeshM // Placeholder
//#define localMeshN // Placeholder
//#define localMeshM // Placeholder
//#define localMeshN_min // Placeholder
//#define globalMeshN_min // Placeholder
#define d_multigpu_offset (d_mesh_info.int3_params[AC_multigpu_offset])
//#define d_multinode_offset (d_mesh_info.int3_params[AC_multinode_offset]) // Placeholder

static __device__ constexpr int
IDX(const int i)
{
    return i;
}

static __device__ __forceinline__ int
IDX(const int i, const int j, const int k)
{
    return DEVICE_VTXBUF_IDX(i, j, k);
}

static __device__ __forceinline__ int
IDX(const int3 idx)
{
    return DEVICE_VTXBUF_IDX(idx.x, idx.y, idx.z);
}

#if AC_DOUBLE_PRECISION == 1
typedef hipDoubleComplex acComplex;
#define acComplex(x, y) make_hipDoubleComplex(x, y)
#else
typedef hipFloatComplex acComplex;
#define acComplex(x, y) make_hipFloatComplex(x, y)
#endif

static __device__ inline acComplex
exp(const acComplex& val)
{
    return acComplex(exp(val.x) * cos(val.y), exp(val.x) * sin(val.y));
}
static __device__ inline acComplex operator*(const AcReal& a, const acComplex& b)
{
    return (acComplex){a * b.x, a * b.y};
}

static __device__ inline acComplex operator*(const acComplex& b, const AcReal& a)
{
    return (acComplex){a * b.x, a * b.y};
}

static __device__ inline acComplex operator*(const acComplex& a, const acComplex& b)
{
    return (acComplex){a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x};
}

// Kernels /////////////////////////////////////////////////////////////////////
#include "boundconds.cuh"
#include "integration.cuh"
#include "packing.cuh"
#include "reductions.cuh"

AcResult
acDeviceLoadMeshInfo(const Device device, const Stream stream, const AcMeshInfo device_config)
{
    hipSetDevice(device->id);

    ERRCHK_ALWAYS(device_config.int_params[AC_nx] == device->local_config.int_params[AC_nx]);
    ERRCHK_ALWAYS(device_config.int_params[AC_ny] == device->local_config.int_params[AC_ny]);
    ERRCHK_ALWAYS(device_config.int_params[AC_nz] == device->local_config.int_params[AC_nz]);
    ERRCHK_ALWAYS(device_config.int_params[AC_multigpu_offset] ==
                  device->local_config.int_params[AC_multigpu_offset]);

    ERRCHK_CUDA_ALWAYS(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mesh_info), &device_config, sizeof(device_config),
                                               0, hipMemcpyHostToDevice, device->streams[stream]));
    return AC_SUCCESS;
}

AcResult
acDeviceLoadScalarUniform(const Device device, const Stream stream, const AcRealParam param,
                          const AcReal value)
{
    hipSetDevice(device->id);

    if (param >= NUM_REAL_PARAMS)
        return AC_FAILURE;

    const size_t offset = (size_t)&d_mesh_info.real_params[param] - (size_t)&d_mesh_info;
    ERRCHK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset,
                                        hipMemcpyHostToDevice, device->streams[stream]));
    return AC_SUCCESS;
}

AcResult
acDeviceLoadVectorUniform(const Device device, const Stream stream, const AcReal3Param param,
                          const AcReal3 value)
{
    hipSetDevice(device->id);

    if (param >= NUM_REAL3_PARAMS || !NUM_REAL3_PARAMS)
        return AC_FAILURE;

    const size_t offset = (size_t)&d_mesh_info.real3_params[param] - (size_t)&d_mesh_info;
    ERRCHK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset,
                                        hipMemcpyHostToDevice, device->streams[stream]));
    return AC_SUCCESS;
}

AcResult
acDeviceLoadIntUniform(const Device device, const Stream stream, const AcIntParam param,
                       const int value)
{
    hipSetDevice(device->id);

    if (param >= NUM_INT_PARAMS)
        return AC_FAILURE;

    const size_t offset = (size_t)&d_mesh_info.int_params[param] - (size_t)&d_mesh_info;
    ERRCHK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset,
                                        hipMemcpyHostToDevice, device->streams[stream]));
    return AC_SUCCESS;
}

AcResult
acDeviceLoadInt3Uniform(const Device device, const Stream stream, const AcInt3Param param,
                        const int3 value)
{
    hipSetDevice(device->id);

    if (param >= NUM_INT3_PARAMS)
        return AC_FAILURE;

    const size_t offset = (size_t)&d_mesh_info.int3_params[param] - (size_t)&d_mesh_info;
    ERRCHK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset,
                                        hipMemcpyHostToDevice, device->streams[stream]));
    return AC_SUCCESS;
}
