#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2019, Johannes Pekkilae, Miikka Vaeisalae.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/

/**
 * @file
 * \brief Brief info.
 *
 * Detailed info.
 *
 */
#include "packing.cuh"

#include "common.cuh"
#include "src/core/errchk.h"

__global__ void
kernel_pack_data(const AcReal* unpacked, const int3 unpacked_start, const int3 packed_dimensions,
                 AcReal* packed)
{
    const int i_packed = threadIdx.x + blockIdx.x * blockDim.x;
    const int j_packed = threadIdx.y + blockIdx.y * blockDim.y;
    const int k_packed = threadIdx.z + blockIdx.z * blockDim.z;

    // If within the start-end range (this allows threadblock dims that are not
    // divisible by end - start)
    if (i_packed >= packed_dimensions.x || //
        j_packed >= packed_dimensions.y || //
        k_packed >= packed_dimensions.z) {
        return;
    }

    const int i_unpacked = i_packed + unpacked_start.x;
    const int j_unpacked = j_packed + unpacked_start.y;
    const int k_unpacked = k_packed + unpacked_start.z;

    const int unpacked_idx = DEVICE_VTXBUF_IDX(i_unpacked, j_unpacked, k_unpacked);
    const int packed_idx   = i_packed +                     //
                           j_packed * packed_dimensions.x + //
                           k_packed * packed_dimensions.x * packed_dimensions.y;

    packed[packed_idx] = unpacked[unpacked_idx];
}

__global__ void
kernel_unpack_data(const AcReal* packed, const int3 packed_dimensions, const int3 unpacked_start,
                   AcReal* unpacked)
{
    const int i_packed = threadIdx.x + blockIdx.x * blockDim.x;
    const int j_packed = threadIdx.y + blockIdx.y * blockDim.y;
    const int k_packed = threadIdx.z + blockIdx.z * blockDim.z;

    // If within the start-end range (this allows threadblock dims that are not
    // divisible by end - start)
    if (i_packed >= packed_dimensions.x || //
        j_packed >= packed_dimensions.y || //
        k_packed >= packed_dimensions.z) {
        return;
    }

    const int i_unpacked = i_packed + unpacked_start.x;
    const int j_unpacked = j_packed + unpacked_start.y;
    const int k_unpacked = k_packed + unpacked_start.z;

    const int unpacked_idx = DEVICE_VTXBUF_IDX(i_unpacked, j_unpacked, k_unpacked);
    const int packed_idx   = i_packed +                     //
                           j_packed * packed_dimensions.x + //
                           k_packed * packed_dimensions.x * packed_dimensions.y;

    unpacked[unpacked_idx] = packed[packed_idx];
}

AcResult
acKernelPackData(const hipStream_t stream, const AcReal* unpacked, const int3 unpacked_start,
                 const int3 packed_dimensions, AcReal* packed)
{
    const dim3 tpb(32, 8, 1);
    const dim3 bpg((unsigned int)ceil(packed_dimensions.x / (float)tpb.x),
                   (unsigned int)ceil(packed_dimensions.y / (float)tpb.y),
                   (unsigned int)ceil(packed_dimensions.z / (float)tpb.z));

    kernel_pack_data<<<bpg, tpb, 0, stream>>>(unpacked, unpacked_start, packed_dimensions, packed);
    ERRCHK_CUDA_KERNEL_ALWAYS(); // TODO SET W/ DEBUG ONLY

    return AC_SUCCESS;
}

AcResult
acKernelUnpackData(const hipStream_t stream, const AcReal* packed, const int3 packed_dimensions,
                   const int3 unpacked_start, AcReal* unpacked)
{
    const dim3 tpb(32, 8, 1);
    const dim3 bpg((unsigned int)ceil(packed_dimensions.x / (float)tpb.x),
                   (unsigned int)ceil(packed_dimensions.y / (float)tpb.y),
                   (unsigned int)ceil(packed_dimensions.z / (float)tpb.z));

    kernel_unpack_data<<<bpg, tpb, 0, stream>>>(packed, packed_dimensions, unpacked_start,
                                                unpacked);
    ERRCHK_CUDA_KERNEL_ALWAYS(); // TODO SET W/ DEBUG ONLY
    return AC_SUCCESS;
}

AcResult
acKernelPackCorner(void)
{
    return AC_FAILURE;
}
AcResult
acKernelUnpackCorner(void)
{
    return AC_FAILURE;
}

AcResult
acKernelPackEdge(void)
{
    return AC_FAILURE;
}
AcResult
acKernelUnpackEdge(void)
{
    return AC_FAILURE;
}

AcResult
acKernelPackSide(void)
{
    return AC_FAILURE;
}
AcResult
acKernelUnpackSide(void)
{
    return AC_FAILURE;
}
