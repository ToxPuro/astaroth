#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2019, Johannes Pekkilae, Miikka Vaeisalae.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/

/**
 * @file
 * \brief Multi-GPU implementation.
 *
 %JP: The old way for computing boundary conditions conflicts with the
 way we have to do things with multiple GPUs.

 The older approach relied on unified memory, which represented the whole
 memory area as one huge mesh instead of several smaller ones. However, unified memory
 in its current state is more meant for quick prototyping when performance is not an issue.
 Getting the CUDA driver to migrate data intelligently across GPUs is much more difficult
 than when managing the memory explicitly.

 In this new approach, I have simplified the multi- and single-GPU layers significantly.
 Quick rundown:
         New struct: Grid. There are two global variables, "grid" and "subgrid", which
         contain the extents of the whole simulation domain and the decomposed grids,
 respectively. To simplify thing, we require that each GPU is assigned the same amount of
 work, therefore each GPU in the node is assigned and "subgrid.m" -sized block of data to
 work with.

         The whole simulation domain is decomposed with respect to the z dimension.
         For example, if the grid contains (nx, ny, nz) vertices, then the subgrids
         contain (nx, ny, nz / num_devices) vertices.

         An local index (i, j, k) in some subgrid can be mapped to the global grid with
                 global idx = (i, j, k + device_id * subgrid.n.z)

 Terminology:
         - Single-GPU function: a function defined on the single-GPU layer (device.cu)

 Changes required to this commented code block:
         - The thread block dimensions (tpb) are no longer passed to the kernel here but in
 device.cu instead. Same holds for any complex index calculations. Instead, the local
 coordinates should be passed as an int3 type without having to consider how the data is
 actually laid out in device memory
         - The unified memory buffer no longer exists (d_buffer). Instead, we have an opaque
 handle of type "Device" which should be passed to single-GPU functions. In this file, all
 devices are stored in a global array "devices[num_devices]".
         - Every single-GPU function is executed asynchronously by default such that we
           can optimize Astaroth by executing memory transactions concurrently with
 computation. Therefore a StreamType should be passed as a parameter to single-GPU functions.
           Refresher: CUDA function calls are non-blocking when a stream is explicitly passed
           as a parameter and commands executing in different streams can be processed
           in parallel/concurrently.


 Note on periodic boundaries (might be helpful when implementing other boundary conditions):

         With multiple GPUs, periodic boundary conditions applied on indices ranging from

                 (0, 0, STENCIL_ORDER/2) to (subgrid.m.x, subgrid.m.y, subgrid.m.z -
 STENCIL_ORDER/2)

         on a single device are "local", in the sense that they can be computed without
 having to exchange data with neighboring GPUs. Special care is needed only for transferring
         the data to the fron and back plates outside this range. In the solution we use
 here, we solve the local boundaries first, and then just exchange the front and back plates
         in a "ring", like so
                                 device_id
                     (n) <-> 0 <-> 1 <-> ... <-> n <-> (0)

### Throughout this file we use the following notation and names for various index offsets

    Global coordinates: coordinates with respect to the global grid (static Grid grid)
    Local coordinates: coordinates with respect to the local subgrid (static Subgrid subgrid)

    s0, s1: source indices in global coordinates
    d0, d1: destination indices in global coordinates
    da = max(s0, d0);
    db = min(s1, d1);

    These are used in at least
    acLoad()
    acStore()
    acSynchronizeHalos()

     Here we decompose the host mesh and distribute it among the GPUs in
     the node.

     The host mesh is a huge contiguous block of data. Its dimensions are given by
     the global variable named "grid". A "grid" is decomposed into "subgrids",
     one for each GPU. Here we check which parts of the range s0...s1 maps
     to the memory space stored by some GPU, ranging d0...d1, and transfer
     the data if needed.

     The index mapping is inherently quite involved, but here's a picture which
     hopefully helps make sense out of all this.


     Grid
                                      |----num_vertices---|
     xxx|....................................................|xxx
              ^                   ^   ^                   ^
             d0                  d1  s0 (src)            s1

     Subgrid

              xxx|.............|xxx
              ^                   ^
             d0                  d1

                                  ^   ^
                                 db  da
 *
 */
#include "astaroth_node.h"

#include "astaroth_device.h"
#include "errchk.h"
#include "math_utils.h" // sum for reductions

static const int MAX_NUM_DEVICES = 32;

typedef struct {
    int3 m;
    int3 n;
} Grid;

struct node_s {
    int id;

    int num_devices;
    Device devices[MAX_NUM_DEVICES];

    Grid grid;
    Grid subgrid;

    AcMeshInfo config;
};

static int
gridIdx(const Grid grid, const int3 idx)
{
    return idx.x + idx.y * grid.m.x + idx.z * grid.m.x * grid.m.y;
}

static int3
gridIdx3d(const Grid grid, const int idx)
{
    return (int3){idx % grid.m.x, (idx % (grid.m.x * grid.m.y)) / grid.m.x,
                  idx / (grid.m.x * grid.m.y)};
}

static void
printInt3(const int3 vec)
{
    printf("(%d, %d, %d)", vec.x, vec.y, vec.z);
}

static inline void
print(const AcMeshInfo config)
{
    for (int i = 0; i < NUM_INT_PARAMS; ++i)
        printf("[%s]: %d\n", intparam_names[i], config.int_params[i]);
    for (int i = 0; i < NUM_REAL_PARAMS; ++i)
        printf("[%s]: %g\n", realparam_names[i], double(config.real_params[i]));
}

static void
update_builtin_params(AcMeshInfo* config)
{
    config->int_params[AC_mx] = config->int_params[AC_nx] + STENCIL_ORDER;
    ///////////// PAD TEST
    // config->int_params[AC_mx] = config->int_params[AC_nx] + STENCIL_ORDER + PAD_SIZE;
    ///////////// PAD TEST
    config->int_params[AC_my] = config->int_params[AC_ny] + STENCIL_ORDER;
    config->int_params[AC_mz] = config->int_params[AC_nz] + STENCIL_ORDER;

    // Bounds for the computational domain, i.e. nx_min <= i < nx_max
    config->int_params[AC_nx_min] = NGHOST;
    config->int_params[AC_nx_max] = config->int_params[AC_nx_min] + config->int_params[AC_nx];
    config->int_params[AC_ny_min] = NGHOST;
    config->int_params[AC_ny_max] = config->int_params[AC_ny] + NGHOST;
    config->int_params[AC_nz_min] = NGHOST;
    config->int_params[AC_nz_max] = config->int_params[AC_nz] + NGHOST;

    /* Additional helper params */
    // Int helpers
    config->int_params[AC_mxy]  = config->int_params[AC_mx] * config->int_params[AC_my];
    config->int_params[AC_nxy]  = config->int_params[AC_nx] * config->int_params[AC_ny];
    config->int_params[AC_nxyz] = config->int_params[AC_nxy] * config->int_params[AC_nz];
}

static Grid
createGrid(const AcMeshInfo config)
{
    Grid grid;

    grid.m = (int3){config.int_params[AC_mx], config.int_params[AC_my], config.int_params[AC_mz]};
    grid.n = (int3){config.int_params[AC_nx], config.int_params[AC_ny], config.int_params[AC_nz]};

    return grid;
}

AcResult
acNodeCreate(const int id, const AcMeshInfo node_config, Node* node_handle)
{
    struct node_s* node = (struct node_s*)malloc(sizeof(*node));
    node->id            = id;
    node->config        = node_config;

    // Get node->num_devices
    ERRCHK_CUDA_ALWAYS(hipGetDeviceCount(&node->num_devices));
    if (node->num_devices < 1) {
        ERROR("No CUDA devices found!");
        return AC_FAILURE;
    }
    if (node->num_devices > MAX_NUM_DEVICES) {
        WARNING("More devices found than MAX_NUM_DEVICES. Using only MAX_NUM_DEVICES");
        node->num_devices = MAX_NUM_DEVICES;
    }
    if (!AC_MULTIGPU_ENABLED) {
        WARNING("MULTIGPU_ENABLED was false. Using only one device");
        node->num_devices = 1; // Use only one device if multi-GPU is not enabled
    }
    // Check that node->num_devices is divisible with AC_nz. This makes decomposing the
    // problem domain to multiple GPUs much easier since we do not have to worry
    // about remainders
    ERRCHK_ALWAYS(node->config.int_params[AC_nz] % node->num_devices == 0);

    // Decompose the problem domain
    // The main grid
    node->grid = createGrid(node->config);

    // Subgrids
    AcMeshInfo subgrid_config = node->config;
    subgrid_config.int_params[AC_nz] /= node->num_devices;
    update_builtin_params(&subgrid_config);
#if VERBOSE_PRINTING // Defined in astaroth.h
    printf("###############################################################\n");
    printf("Config dimensions recalculated:\n");
    print(subgrid_config);
    printf("###############################################################\n");
#endif
    node->subgrid = createGrid(subgrid_config);

    // Periodic boundary conditions become weird if the system can "fold unto itself".
    ERRCHK_ALWAYS(node->subgrid.n.x >= STENCIL_ORDER);
    ERRCHK_ALWAYS(node->subgrid.n.y >= STENCIL_ORDER);
    ERRCHK_ALWAYS(node->subgrid.n.z >= STENCIL_ORDER);

#if VERBOSE_PRINTING
    // clang-format off
    printf("Grid m ");   printInt3(node->grid.m);    printf("\n");
    printf("Grid n ");   printInt3(node->grid.n);    printf("\n");
    printf("Subrid m "); printInt3(node->subgrid.m); printf("\n");
    printf("Subrid n "); printInt3(node->subgrid.n); printf("\n");
    // clang-format on
#endif

    // Initialize the devices
    // #pragma omp parallel for
    for (int i = 0; i < node->num_devices; ++i) {
        const int3 multinode_offset                    = (int3){0, 0, 0}; // Placeholder
        const int3 multigpu_offset                     = (int3){0, 0, i * node->subgrid.n.z};
        subgrid_config.int3_params[AC_global_grid_n]   = node->grid.n;
        subgrid_config.int3_params[AC_multigpu_offset] = multinode_offset + multigpu_offset;

        acDeviceCreate(i, subgrid_config, &node->devices[i]);
        acDevicePrintInfo(node->devices[i]);
    }

    // Enable peer access
    // #pragma omp parallel for
    for (int i = 0; i < node->num_devices; ++i) {
        const int front = (i + 1) % node->num_devices;
        const int back  = (i - 1 + node->num_devices) % node->num_devices;

        int can_access_front, can_access_back;
        hipDeviceCanAccessPeer(&can_access_front, i, front);
        hipDeviceCanAccessPeer(&can_access_back, i, back);
#if VERBOSE_PRINTING
        printf(
            "Trying to enable peer access from %d to %d (can access: %d) and %d (can access: %d)\n",
            i, front, can_access_front, back, can_access_back);
#endif

        hipSetDevice(i);
        if (can_access_front) {
            ERRCHK_CUDA_ALWAYS(hipDeviceEnablePeerAccess(front, 0));
        }
        if (can_access_back) {
            ERRCHK_CUDA_ALWAYS(hipDeviceEnablePeerAccess(back, 0));
        }
    }
    acNodeSynchronizeStream(node, STREAM_ALL);

    *node_handle = node;
    return AC_SUCCESS;
}

AcResult
acNodeDestroy(Node node)
{
    acNodeSynchronizeStream(node, STREAM_ALL);

    // #pragma omp parallel for
    for (int i = 0; i < node->num_devices; ++i) {
        acDeviceDestroy(node->devices[i]);
    }
    free(node);

    return AC_SUCCESS;
}

AcResult
acNodePrintInfo(const Node node)
{
    (void)node;
    WARNING("Not implemented");
    return AC_FAILURE;
}

AcResult
acNodeQueryDeviceConfiguration(const Node node, DeviceConfiguration* config)
{
    (void)node;
    (void)config;
    WARNING("Not implemented");
    return AC_FAILURE;
}

AcResult
acNodeAutoOptimize(const Node node)
{
    (void)node;
    WARNING("Not implemented");
    return AC_FAILURE;
}

AcResult
acNodeSynchronizeStream(const Node node, const Stream stream)
{
    // #pragma omp parallel for
    for (int i = 0; i < node->num_devices; ++i) {
        acDeviceSynchronizeStream(node->devices[i], stream);
    }

    return AC_SUCCESS;
}

AcResult
acNodeSynchronizeVertexBuffer(const Node node, const Stream stream,
                              const VertexBufferHandle vtxbuf_handle)
{
    acNodeSynchronizeStream(node, stream);
    // Exchanges the halos of subgrids
    // After this step, the data within the main grid ranging from
    // (0, 0, NGHOST) -> grid.m.x, grid.m.y, NGHOST + grid.n.z
    // has been synchronized and transferred to appropriate subgrids

    // We loop only to node->num_devices - 1 since the front and back plate of the grid is not
    // transferred because their contents depend on the boundary conditions.

    // IMPORTANT NOTE: the boundary conditions must be applied before
    // callingacNodeSynchronizeStream(node,  this function! I.e. the halos of subgrids must contain
    // up-to-date data!

    const size_t num_vertices = node->subgrid.m.x * node->subgrid.m.y * NGHOST;

    // #pragma omp parallel for
    for (int i = 0; i < node->num_devices - 1; ++i) {
        // ...|ooooxxx|... -> xxx|ooooooo|...
        const int3 src = (int3){0, 0, node->subgrid.n.z};
        const int3 dst = (int3){0, 0, 0};

        const Device src_device = node->devices[i];
        Device dst_device       = node->devices[i + 1];

        acDeviceTransferVertexBufferWithOffset(src_device, stream, vtxbuf_handle, src, dst,
                                               num_vertices, dst_device);
    }
    // #pragma omp parallel for
    for (int i = 1; i < node->num_devices; ++i) {
        // ...|ooooooo|xxx <- ...|xxxoooo|...
        const int3 src = (int3){0, 0, NGHOST};
        const int3 dst = (int3){0, 0, NGHOST + node->subgrid.n.z};

        const Device src_device = node->devices[i];
        Device dst_device       = node->devices[i - 1];

        acDeviceTransferVertexBufferWithOffset(src_device, stream, vtxbuf_handle, src, dst,
                                               num_vertices, dst_device);
    }
    return AC_SUCCESS;
}

AcResult
acNodeSynchronizeMesh(const Node node, const Stream stream)
{
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        acNodeSynchronizeVertexBuffer(node, stream, (VertexBufferHandle)i);
    }

    return AC_SUCCESS;
}

AcResult
acNodeSwapBuffers(const Node node)
{
    // #pragma omp parallel for
    for (int i = 0; i < node->num_devices; ++i) {
        acDeviceSwapBuffers(node->devices[i]);
    }
    return AC_SUCCESS;
}

AcResult
acNodeLoadConstant(const Node node, const Stream stream, const AcRealParam param,
                   const AcReal value)
{
    acNodeSynchronizeStream(node, stream);
    // #pragma omp parallel for
    for (int i = 0; i < node->num_devices; ++i) {
        acDeviceLoadConstant(node->devices[i], stream, param, value);
    }
    return AC_SUCCESS;
}

AcResult
acNodeLoadVertexBufferWithOffset(const Node node, const Stream stream, const AcMesh host_mesh,
                                 const VertexBufferHandle vtxbuf_handle, const int3 src,
                                 const int3 dst, const int num_vertices)
{
    acNodeSynchronizeStream(node, stream);
    // See the beginning of the file for an explanation of the index mapping
    // // #pragma omp parallel for
    // #pragma omp parallel for
    for (int i = 0; i < node->num_devices; ++i) {
        const int3 d0 = (int3){0, 0, i * node->subgrid.n.z}; // DECOMPOSITION OFFSET HERE
        const int3 d1 = (int3){node->subgrid.m.x, node->subgrid.m.y, d0.z + node->subgrid.m.z};

        const int3 s0 = src; // dst; // TODO fix
        (void)dst;           // TODO fix
        const int3 s1 = gridIdx3d(node->grid, gridIdx(node->grid, s0) + num_vertices);

        const int3 da = max(s0, d0);
        const int3 db = min(s1, d1);
        /*
        printf("Device %d\n", i);
        printf("\ts0: "); printInt3(s0); printf("\n");
        printf("\td0: "); printInt3(d0); printf("\n");
        printf("\tda: "); printInt3(da); printf("\n");
        printf("\tdb: "); printInt3(db); printf("\n");
        printf("\td1: "); printInt3(d1); printf("\n");
        printf("\ts1: "); printInt3(s1); printf("\n");
        printf("\t-> %s to device %d\n", db.z >= da.z ? "Copy" : "Do not copy", i);
        */
        if (db.z >= da.z) {
            const int copy_cells = gridIdx(node->subgrid, db) - gridIdx(node->subgrid, da);
            // DECOMPOSITION OFFSET HERE
            const int3 da_global = da; // src + da - dst; // TODO fix
            const int3 da_local = (int3){da.x, da.y, da.z - i * node->grid.n.z / node->num_devices};
            // printf("\t\tcopy %d cells to local index ", copy_cells); printInt3(da_local);
            // printf("\n");
            acDeviceLoadVertexBufferWithOffset(node->devices[i], stream, host_mesh, vtxbuf_handle,
                                               da_global, da_local, copy_cells);
        }
        // printf("\n");
    }
    return AC_SUCCESS;
}

AcResult
acNodeLoadMeshWithOffset(const Node node, const Stream stream, const AcMesh host_mesh,
                         const int3 src, const int3 dst, const int num_vertices)
{
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        acNodeLoadVertexBufferWithOffset(node, stream, host_mesh, (VertexBufferHandle)i, src, dst,
                                         num_vertices);
    }
    return AC_SUCCESS;
}

AcResult
acNodeLoadVertexBuffer(const Node node, const Stream stream, const AcMesh host_mesh,
                       const VertexBufferHandle vtxbuf_handle)
{
    const int3 src            = (int3){0, 0, 0};
    const int3 dst            = src;
    const size_t num_vertices = acVertexBufferSize(host_mesh.info);

    acNodeLoadVertexBufferWithOffset(node, stream, host_mesh, vtxbuf_handle, src, dst,
                                     num_vertices);
    return AC_SUCCESS;
}

AcResult
acNodeLoadMesh(const Node node, const Stream stream, const AcMesh host_mesh)
{
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        acNodeLoadVertexBuffer(node, stream, host_mesh, (VertexBufferHandle)i);
    }
    return AC_SUCCESS;
}

AcResult
acNodeStoreVertexBufferWithOffset(const Node node, const Stream stream,
                                  const VertexBufferHandle vtxbuf_handle, const int3 src,
                                  const int3 dst, const int num_vertices, AcMesh* host_mesh)
{
    acNodeSynchronizeStream(node, stream);
    // #pragma omp parallel for
    for (int i = 0; i < node->num_devices; ++i) {
        const int3 d0 = (int3){0, 0, i * node->subgrid.n.z}; // DECOMPOSITION OFFSET HERE
        const int3 d1 = (int3){node->subgrid.m.x, node->subgrid.m.y, d0.z + node->subgrid.m.z};

        const int3 s0 = src; // TODO fix
        (void)dst;           // TODO fix
        const int3 s1 = gridIdx3d(node->grid, gridIdx(node->grid, s0) + num_vertices);

        const int3 da = max(s0, d0);
        const int3 db = min(s1, d1);
        if (db.z >= da.z) {
            const int copy_cells = gridIdx(node->subgrid, db) - gridIdx(node->subgrid, da);
            // DECOMPOSITION OFFSET HERE
            const int3 da_local = (int3){da.x, da.y, da.z - i * node->grid.n.z / node->num_devices};
            const int3 da_global = da; // dst + da - src; // TODO fix
            acDeviceStoreVertexBufferWithOffset(node->devices[i], stream, vtxbuf_handle, da_local,
                                                da_global, copy_cells, host_mesh);
        }
    }
    return AC_SUCCESS;
}

AcResult
acNodeStoreMeshWithOffset(const Node node, const Stream stream, const int3 src, const int3 dst,
                          const int num_vertices, AcMesh* host_mesh)
{
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        acNodeStoreVertexBufferWithOffset(node, stream, (VertexBufferHandle)i, src, dst,
                                          num_vertices, host_mesh);
    }
    return AC_SUCCESS;
}

AcResult
acNodeStoreVertexBuffer(const Node node, const Stream stream,
                        const VertexBufferHandle vtxbuf_handle, AcMesh* host_mesh)
{
    const int3 src            = (int3){0, 0, 0};
    const int3 dst            = src;
    const size_t num_vertices = acVertexBufferSize(host_mesh->info);

    acNodeStoreVertexBufferWithOffset(node, stream, vtxbuf_handle, src, dst, num_vertices,
                                      host_mesh);

    return AC_SUCCESS;
}

AcResult
acNodeStoreMesh(const Node node, const Stream stream, AcMesh* host_mesh)
{
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        acNodeStoreVertexBuffer(node, stream, (VertexBufferHandle)i, host_mesh);
    }
    return AC_SUCCESS;
}

AcResult
acNodeIntegrateSubstep(const Node node, const Stream stream, const int isubstep, const int3 start,
                       const int3 end, const AcReal dt)
{
    acNodeSynchronizeStream(node, stream);

    // #pragma omp parallel for
    for (int i = 0; i < node->num_devices; ++i) {
        // DECOMPOSITION OFFSET HERE
        const int3 d0 = (int3){NGHOST, NGHOST, NGHOST + i * node->subgrid.n.z};
        const int3 d1 = d0 + (int3){node->subgrid.n.x, node->subgrid.n.y, node->subgrid.n.z};

        const int3 da = max(start, d0);
        const int3 db = min(end, d1);

        if (db.z >= da.z) {
            const int3 da_local = da - (int3){0, 0, i * node->subgrid.n.z};
            const int3 db_local = db - (int3){0, 0, i * node->subgrid.n.z};
            acDeviceIntegrateSubstep(node->devices[i], stream, isubstep, da_local, db_local, dt);
        }
    }
    return AC_SUCCESS;
}

static AcResult
local_boundcondstep(const Node node, const Stream stream, const VertexBufferHandle vtxbuf)
{
    acNodeSynchronizeStream(node, stream);

    if (node->num_devices > 1) {
        // Local boundary conditions
        // #pragma omp parallel for
        for (int i = 0; i < node->num_devices; ++i) {
            const int3 d0 = (int3){0, 0, NGHOST}; // DECOMPOSITION OFFSET HERE
            const int3 d1 = (int3){node->subgrid.m.x, node->subgrid.m.y, d0.z + node->subgrid.n.z};
            acDevicePeriodicBoundcondStep(node->devices[i], stream, vtxbuf, d0, d1);
        }
    }
    else {
        acDevicePeriodicBoundcondStep(node->devices[0], stream, vtxbuf, (int3){0, 0, 0},
                                      node->subgrid.m);
    }
    return AC_SUCCESS;
}

static AcResult
global_boundcondstep(const Node node, const Stream stream, const VertexBufferHandle vtxbuf_handle)
{
    acNodeSynchronizeStream(node, stream);

    if (node->num_devices > 1) {
        const size_t num_vertices = node->subgrid.m.x * node->subgrid.m.y * NGHOST;
        {
            // ...|ooooxxx|... -> xxx|ooooooo|...
            const int3 src = (int3){0, 0, node->subgrid.n.z};
            const int3 dst = (int3){0, 0, 0};

            const Device src_device = node->devices[node->num_devices - 1];
            Device dst_device       = node->devices[0];

            acDeviceTransferVertexBufferWithOffset(src_device, stream, vtxbuf_handle, src, dst,
                                                   num_vertices, dst_device);
        }
        {
            // ...|ooooooo|xxx <- ...|xxxoooo|...
            const int3 src = (int3){0, 0, NGHOST};
            const int3 dst = (int3){0, 0, NGHOST + node->subgrid.n.z};

            const Device src_device = node->devices[0];
            Device dst_device       = node->devices[node->num_devices - 1];

            acDeviceTransferVertexBufferWithOffset(src_device, stream, vtxbuf_handle, src, dst,
                                                   num_vertices, dst_device);
        }
    }
    return AC_SUCCESS;
}

AcResult
acNodeIntegrate(const Node node, const AcReal dt)
{
    acNodeSynchronizeStream(node, STREAM_ALL);
    // xxx|OOO OOOOOOOOO OOO|xxx
    //    ^    ^         ^  ^
    //   n0   n1        n2  n3
    // const int3 n0 = (int3){NGHOST, NGHOST, NGHOST};
    // const int3 n1 = (int3){2 * NGHOST, 2 * NGHOST, 2 * NGHOST};
    // const int3 n2 = node->grid.n;
    // const int3 n3 = n0 + node->grid.n;

    for (int isubstep = 0; isubstep < 3; ++isubstep) {
        acNodeSynchronizeStream(node, STREAM_ALL);
        for (int vtxbuf = 0; vtxbuf < NUM_VTXBUF_HANDLES; ++vtxbuf) {
            local_boundcondstep(node, (Stream)vtxbuf, (VertexBufferHandle)vtxbuf);
        }
        acNodeSynchronizeStream(node, STREAM_ALL);

        // Inner inner
        // #pragma omp parallel for
        for (int i = 0; i < node->num_devices; ++i) {
            const int3 m1 = (int3){2 * NGHOST, 2 * NGHOST, 2 * NGHOST};
            const int3 m2 = node->subgrid.n;
            acDeviceIntegrateSubstep(node->devices[i], STREAM_16, isubstep, m1, m2, dt);
        }

        for (int vtxbuf = 0; vtxbuf < NUM_VTXBUF_HANDLES; ++vtxbuf) {
            acNodeSynchronizeVertexBuffer(node, (Stream)vtxbuf, (VertexBufferHandle)vtxbuf);
            global_boundcondstep(node, (Stream)vtxbuf, (VertexBufferHandle)vtxbuf);
        }
        for (int vtxbuf = 0; vtxbuf < NUM_VTXBUF_HANDLES; ++vtxbuf) {
            acNodeSynchronizeStream(node, (Stream)vtxbuf);
        }

        // #pragma omp parallel for
        for (int i = 0; i < node->num_devices; ++i) { // Front
            const int3 m1 = (int3){NGHOST, NGHOST, NGHOST};
            const int3 m2 = m1 + (int3){node->subgrid.n.x, node->subgrid.n.y, NGHOST};
            acDeviceIntegrateSubstep(node->devices[i], STREAM_0, isubstep, m1, m2, dt);
        }
        // #pragma omp parallel for
        for (int i = 0; i < node->num_devices; ++i) { // Back
            const int3 m1 = (int3){NGHOST, NGHOST, node->subgrid.n.z};
            const int3 m2 = m1 + (int3){node->subgrid.n.x, node->subgrid.n.y, NGHOST};
            acDeviceIntegrateSubstep(node->devices[i], STREAM_1, isubstep, m1, m2, dt);
        }
        // #pragma omp parallel for
        for (int i = 0; i < node->num_devices; ++i) { // Bottom
            const int3 m1 = (int3){NGHOST, NGHOST, 2 * NGHOST};
            const int3 m2 = m1 + (int3){node->subgrid.n.x, NGHOST, node->subgrid.n.z - 2 * NGHOST};
            acDeviceIntegrateSubstep(node->devices[i], STREAM_2, isubstep, m1, m2, dt);
        }
        // #pragma omp parallel for
        for (int i = 0; i < node->num_devices; ++i) { // Top
            const int3 m1 = (int3){NGHOST, node->subgrid.n.y, 2 * NGHOST};
            const int3 m2 = m1 + (int3){node->subgrid.n.x, NGHOST, node->subgrid.n.z - 2 * NGHOST};
            acDeviceIntegrateSubstep(node->devices[i], STREAM_3, isubstep, m1, m2, dt);
        }
        // #pragma omp parallel for
        for (int i = 0; i < node->num_devices; ++i) { // Left
            const int3 m1 = (int3){NGHOST, 2 * NGHOST, 2 * NGHOST};
            const int3 m2 = m1 + (int3){NGHOST, node->subgrid.n.y - 2 * NGHOST,
                                        node->subgrid.n.z - 2 * NGHOST};
            acDeviceIntegrateSubstep(node->devices[i], STREAM_4, isubstep, m1, m2, dt);
        }
        // #pragma omp parallel for
        for (int i = 0; i < node->num_devices; ++i) { // Right
            const int3 m1 = (int3){node->subgrid.n.x, 2 * NGHOST, 2 * NGHOST};
            const int3 m2 = m1 + (int3){NGHOST, node->subgrid.n.y - 2 * NGHOST,
                                        node->subgrid.n.z - 2 * NGHOST};
            acDeviceIntegrateSubstep(node->devices[i], STREAM_5, isubstep, m1, m2, dt);
        }
        acNodeSwapBuffers(node);
    }
    acNodeSynchronizeStream(node, STREAM_ALL);
    return AC_SUCCESS;
}

AcResult
acNodePeriodicBoundcondStep(const Node node, const Stream stream,
                            const VertexBufferHandle vtxbuf_handle)
{
    local_boundcondstep(node, stream, vtxbuf_handle);
    acNodeSynchronizeVertexBuffer(node, stream, vtxbuf_handle);

    // TODO NOTE GLOBAL BOUNDCONDS NOT DONE HERE IF MORE THAN 1 NODE
    global_boundcondstep(node, stream, vtxbuf_handle);
    // WARNING("Global boundconds should not be done here with multinode");

    return AC_SUCCESS;
}

AcResult
acNodePeriodicBoundconds(const Node node, const Stream stream)
{
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        acNodePeriodicBoundcondStep(node, stream, (VertexBufferHandle)i);
    }
    return AC_SUCCESS;
}

static AcReal
simple_final_reduce_scal(const Node node, const ReductionType& rtype, const AcReal* results,
                         const int& n)
{
    AcReal res = results[0];
    for (int i = 1; i < n; ++i) {
        if (rtype == RTYPE_MAX) {
            res = max(res, results[i]);
        }
        else if (rtype == RTYPE_MIN) {
            res = min(res, results[i]);
        }
        else if (rtype == RTYPE_RMS || rtype == RTYPE_RMS_EXP || rtype == RTYPE_SUM) {
            res = sum(res, results[i]);
        }
        else {
            ERROR("Invalid rtype");
        }
    }

    if (rtype == RTYPE_RMS || rtype == RTYPE_RMS_EXP) {
        const AcReal inv_n = AcReal(1.) / (node->grid.n.x * node->grid.n.y * node->grid.n.z);
        res                = sqrt(inv_n * res);
    }
    return res;
}

AcResult
acNodeReduceScal(const Node node, const Stream stream, const ReductionType rtype,
                 const VertexBufferHandle vtxbuf_handle, AcReal* result)
{
    acNodeSynchronizeStream(node, STREAM_ALL);

    AcReal results[node->num_devices];
    // #pragma omp parallel for
    for (int i = 0; i < node->num_devices; ++i) {
        acDeviceReduceScal(node->devices[i], stream, rtype, vtxbuf_handle, &results[i]);
    }

    *result = simple_final_reduce_scal(node, rtype, results, node->num_devices);
    return AC_SUCCESS;
}

AcResult
acNodeReduceVec(const Node node, const Stream stream, const ReductionType rtype,
                const VertexBufferHandle a, const VertexBufferHandle b, const VertexBufferHandle c,
                AcReal* result)
{
    acNodeSynchronizeStream(node, STREAM_ALL);

    AcReal results[node->num_devices];
    // #pragma omp parallel for
    for (int i = 0; i < node->num_devices; ++i) {
        acDeviceReduceVec(node->devices[i], stream, rtype, a, b, c, &results[i]);
    }

    *result = simple_final_reduce_scal(node, rtype, results, node->num_devices);
    return AC_SUCCESS;
}

AcResult
acNodeLoadYZPlate(const Node node, const int3 start, const int3 end, AcMesh* host_mesh, AcReal* yzPlateBuffer)
{
    int kmin, kmax, nzloc=node->subgrid.n.z;
    size_t src_idx;

    int i,j,k,ind,iv;
    for (int id = 0; id <= node->num_devices; ++id) {

        kmin=max( NGHOST,       start.z-id*nzloc );
        kmax=min( NGHOST+nzloc, end.z  -id*nzloc );

        ind=0;
        for (k=kmin; k<=kmax; k++) {
            for (j=start.y; j<=end.y; j++) {
               for (i=start.x; i<end.x; i++) {
                   src_idx = acVertexBufferIdx(i,j,k,host_mesh->info);
                   for (iv = 0; iv < NUM_VTXBUF_HANDLES; ++iv) {
                       yzPlateBuffer[ind] = host_mesh->vertex_buffer[iv][src_idx];
                   }
                   ind++;
               }
            }
        }
        //copyMeshToDevice(devices[id], STREAM_PRIMARY, yzPlateBuffer, da, da_local, copy_cells);
    }

    return AC_SUCCESS;
}

