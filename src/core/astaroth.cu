#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2019, Johannes Pekkilae, Miikka Vaeisalae.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "astaroth.h"

#include "errchk.h"
#include "math_utils.h" // int3 + int3

#define AC_GEN_STR(X) #X
const char* intparam_names[]    = {AC_FOR_BUILTIN_INT_PARAM_TYPES(AC_GEN_STR) //
                                AC_FOR_USER_INT_PARAM_TYPES(AC_GEN_STR)};
const char* int3param_names[]   = {AC_FOR_BUILTIN_INT3_PARAM_TYPES(AC_GEN_STR) //
                                 AC_FOR_USER_INT3_PARAM_TYPES(AC_GEN_STR)};
const char* realparam_names[]   = {AC_FOR_BUILTIN_REAL_PARAM_TYPES(AC_GEN_STR) //
                                 AC_FOR_USER_REAL_PARAM_TYPES(AC_GEN_STR)};
const char* real3param_names[]  = {AC_FOR_BUILTIN_REAL3_PARAM_TYPES(AC_GEN_STR) //
                                  AC_FOR_USER_REAL3_PARAM_TYPES(AC_GEN_STR)};
const char* scalararray_names[] = {AC_FOR_SCALARARRAY_HANDLES(AC_GEN_STR)};
const char* vtxbuf_names[]      = {AC_FOR_VTXBUF_HANDLES(AC_GEN_STR)};
#undef AC_GEN_STR

static const int num_nodes = 1;
static Node nodes[num_nodes];

void
acPrintMeshInfo(const AcMeshInfo config)
{
    for (int i = 0; i < NUM_INT_PARAMS; ++i)
        printf("[%s]: %d\n", intparam_names[i], config.int_params[i]);
    for (int i = 0; i < NUM_INT3_PARAMS; ++i)
        printf("[%s]: (%d, %d, %d)\n", int3param_names[i], config.int3_params[i].x,
               config.int3_params[i].y, config.int3_params[i].z);
    for (int i = 0; i < NUM_REAL_PARAMS; ++i)
        printf("[%s]: %g\n", realparam_names[i], double(config.real_params[i]));
    for (int i = 0; i < NUM_REAL3_PARAMS; ++i)
        printf("[%s]: (%g, %g, %g)\n", real3param_names[i], double(config.real3_params[i].x),
               double(config.real3_params[i].y), double(config.real3_params[i].z));
}

AcResult
acInit(const AcMeshInfo mesh_info)
{
    return acNodeCreate(0, mesh_info, &nodes[0]);
}

AcResult
acQuit(void)
{
    return acNodeDestroy(nodes[0]);
}

AcResult
acLoadPlate(const int3& start, const int3& end, AcMesh* host_mesh, AcReal* plateBuffer, PlateType plate)
{
    return acNodeLoadPlate(nodes[0], STREAM_DEFAULT, start, end, host_mesh, plateBuffer, plate);
}

AcResult
acCheckDeviceAvailability(void)
{
    int device_count; // Separate from num_devices to avoid side effects
    ERRCHK_CUDA_ALWAYS(hipGetDeviceCount(&device_count));
    if (device_count > 0)
        return AC_SUCCESS;
    else
        return AC_FAILURE;
}

AcResult
acSynchronize(void)
{
    return acNodeSynchronizeStream(nodes[0], STREAM_ALL);
}

AcResult
acSynchronizeStream(const Stream stream)
{
    return acNodeSynchronizeStream(nodes[0], stream);
}

AcResult
acLoadDeviceConstant(const AcRealParam param, const AcReal value)
{
    return acNodeLoadConstant(nodes[0], STREAM_DEFAULT, param, value);
}

AcResult
acLoadVectorConstant(const AcReal3Param param, const AcReal3 value)
{
    return acNodeLoadVectorConstant(nodes[0], STREAM_DEFAULT, param, value);
}

AcResult
acLoad(const AcMesh host_mesh)
{
    return acNodeLoadMesh(nodes[0], STREAM_DEFAULT, host_mesh);
}

AcResult
acStore(AcMesh* host_mesh)
{
    return acNodeStoreMesh(nodes[0], STREAM_DEFAULT, host_mesh);
}

AcResult
acIntegrate(const AcReal dt)
{
    return acNodeIntegrate(nodes[0], dt);
}

AcResult
acIntegrateStep(const int isubstep, const AcReal dt)
{
    DeviceConfiguration config;
    acNodeQueryDeviceConfiguration(nodes[0], &config);

    const int3 start = (int3){NGHOST, NGHOST, NGHOST};
    const int3 end   = start + config.grid.n;
    return acNodeIntegrateSubstep(nodes[0], STREAM_DEFAULT, isubstep, start, end, dt);
}

AcResult
acIntegrateStepWithOffset(const int isubstep, const AcReal dt, const int3 start, const int3 end)
{
    return acNodeIntegrateSubstep(nodes[0], STREAM_DEFAULT, isubstep, start, end, dt);
}

AcResult
acBoundcondStep(void)
{
    return acNodePeriodicBoundconds(nodes[0], STREAM_DEFAULT);
}

AcReal
acReduceScal(const ReductionType rtype, const VertexBufferHandle vtxbuf_handle)
{
    AcReal result;
    acNodeReduceScal(nodes[0], STREAM_DEFAULT, rtype, vtxbuf_handle, &result);
    return result;
}

AcReal
acReduceVec(const ReductionType rtype, const VertexBufferHandle a, const VertexBufferHandle b,
            const VertexBufferHandle c)
{
    AcReal result;
    acNodeReduceVec(nodes[0], STREAM_DEFAULT, rtype, a, b, c, &result);
    return result;
}

AcResult
acStoreWithOffset(const int3 dst, const size_t num_vertices, AcMesh* host_mesh)
{
    return acNodeStoreMeshWithOffset(nodes[0], STREAM_DEFAULT, dst, dst, num_vertices, host_mesh);
}

AcResult
acLoadWithOffset(const AcMesh host_mesh, const int3 src, const int num_vertices)
{
    return acNodeLoadMeshWithOffset(nodes[0], STREAM_DEFAULT, host_mesh, src, src, num_vertices);
}

AcResult
acSynchronizeMesh(void)
{
    return acNodeSynchronizeMesh(nodes[0], STREAM_DEFAULT);
}

AcResult
acGetNode(Node * node)
{
    *node = nodes[0];
    return (AcResult) AC_SUCCESS;
}

int
acGetNumDevicesPerNode(void)
{
    int num_devices;
    ERRCHK_CUDA_ALWAYS(hipGetDeviceCount(&num_devices));
    return num_devices;
}
