#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2018, Johannes Pekkilae, Miikka Vaeisalae.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/

/**
 * @file
 * \brief Multi-GPU implementation.
 *
 %JP: The old way for computing boundary conditions conflicts with the
 way we have to do things with multiple GPUs.

 The older approach relied on unified memory, which represented the whole
 memory area as one huge mesh instead of several smaller ones. However, unified memory
 in its current state is more meant for quick prototyping when performance is not an issue.
 Getting the CUDA driver to migrate data intelligently across GPUs is much more difficult
 than when managing the memory explicitly.

 In this new approach, I have simplified the multi- and single-GPU layers significantly.
 Quick rundown:
         New struct: Grid. There are two global variables, "grid" and "subgrid", which
         contain the extents of the whole simulation domain and the decomposed grids,
 respectively. To simplify thing, we require that each GPU is assigned the same amount of
 work, therefore each GPU in the node is assigned and "subgrid.m" -sized block of data to
 work with.

         The whole simulation domain is decomposed with respect to the z dimension.
         For example, if the grid contains (nx, ny, nz) vertices, then the subgrids
         contain (nx, ny, nz / num_devices) vertices.

         An local index (i, j, k) in some subgrid can be mapped to the global grid with
                 global idx = (i, j, k + device_id * subgrid.n.z)

 Terminology:
         - Single-GPU function: a function defined on the single-GPU layer (device.cu)

 Changes required to this commented code block:
         - The thread block dimensions (tpb) are no longer passed to the kernel here but in
 device.cu instead. Same holds for any complex index calculations. Instead, the local
 coordinates should be passed as an int3 type without having to consider how the data is
 actually laid out in device memory
         - The unified memory buffer no longer exists (d_buffer). Instead, we have an opaque
 handle of type "Device" which should be passed to single-GPU functions. In this file, all
 devices are stored in a global array "devices[num_devices]".
         - Every single-GPU function is executed asynchronously by default such that we
           can optimize Astaroth by executing memory transactions concurrently with
 computation. Therefore a StreamType should be passed as a parameter to single-GPU functions.
           Refresher: CUDA function calls are non-blocking when a stream is explicitly passed
           as a parameter and commands executing in different streams can be processed
           in parallel/concurrently.


 Note on periodic boundaries (might be helpful when implementing other boundary conditions):

         With multiple GPUs, periodic boundary conditions applied on indices ranging from

                 (0, 0, STENCIL_ORDER/2) to (subgrid.m.x, subgrid.m.y, subgrid.m.z -
 STENCIL_ORDER/2)

         on a single device are "local", in the sense that they can be computed without
 having to exchange data with neighboring GPUs. Special care is needed only for transferring
         the data to the fron and back plates outside this range. In the solution we use
 here, we solve the local boundaries first, and then just exchange the front and back plates
         in a "ring", like so
                                 device_id
                     (n) <-> 0 <-> 1 <-> ... <-> n <-> (0)

### Throughout this file we use the following notation and names for various index offsets

    Global coordinates: coordinates with respect to the global grid (static Grid grid)
    Local coordinates: coordinates with respect to the local subgrid (static Subgrid subgrid)

    s0, s1: source indices in global coordinates
    d0, d1: destination indices in global coordinates
    da = max(s0, d0);
    db = min(s1, d1);

    These are used in at least
    acLoad()
    acStore()
    acSynchronizeHalos()

     Here we decompose the host mesh and distribute it among the GPUs in
     the node.

     The host mesh is a huge contiguous block of data. Its dimensions are given by
     the global variable named "grid". A "grid" is decomposed into "subgrids",
     one for each GPU. Here we check which parts of the range s0...s1 maps
     to the memory space stored by some GPU, ranging d0...d1, and transfer
     the data if needed.

     The index mapping is inherently quite involved, but here's a picture which
     hopefully helps make sense out of all this.


     Grid
                                      |----num_vertices---|
     xxx|....................................................|xxx
              ^                   ^   ^                   ^
             d0                  d1  s0 (src)            s1

     Subgrid

              xxx|.............|xxx
              ^                   ^
             d0                  d1

                                  ^   ^
                                 db  da
 *
 */
#include "astaroth.h"
#include "errchk.h"

#include "device.cuh"
#include "math_utils.h"               // sum for reductions
#include "standalone/config_loader.h" // update_config

const char* intparam_names[]  = {AC_FOR_BUILTIN_INT_PARAM_TYPES(AC_GEN_STR)
                                    AC_FOR_USER_INT_PARAM_TYPES(AC_GEN_STR)};
const char* realparam_names[] = {AC_FOR_REAL_PARAM_TYPES(AC_GEN_STR)};
const char* vtxbuf_names[]    = {AC_FOR_VTXBUF_HANDLES(AC_GEN_STR)};

static const int MAX_NUM_DEVICES       = 32;
static int num_devices                 = 0;
static Device devices[MAX_NUM_DEVICES] = {};

static Grid grid; // A grid consists of num_devices subgrids
static Grid subgrid;

static int
gridIdx(const Grid grid, const int3 idx)
{
    return idx.x + idx.y * grid.m.x + idx.z * grid.m.x * grid.m.y;
}

static int3
gridIdx3d(const Grid& grid, const int idx)
{
    return (int3){idx % grid.m.x, (idx % (grid.m.x * grid.m.y)) / grid.m.x,
                  idx / (grid.m.x * grid.m.y)};
}

static void
printInt3(const int3 vec)
{
    printf("(%d, %d, %d)", vec.x, vec.y, vec.z);
}

static Grid
createGrid(const AcMeshInfo& config)
{
    Grid grid;

    grid.m = (int3){config.int_params[AC_mx], config.int_params[AC_my], config.int_params[AC_mz]};
    grid.n = (int3){config.int_params[AC_nx], config.int_params[AC_ny], config.int_params[AC_nz]};

    return grid;
}

AcResult
acCheckDeviceAvailability(void)
{
    int device_count; // Separate from num_devices to avoid side effects
    ERRCHK_CUDA_ALWAYS(hipGetDeviceCount(&device_count));
    if (device_count > 0)
        return AC_SUCCESS;
    else
        return AC_FAILURE;
}

AcResult
acSynchronizeStream(const StreamType stream)
{
    // #pragma omp parallel for
    for (int i = 0; i < num_devices; ++i) {
        synchronize(devices[i], stream);
    }

    return AC_SUCCESS;
}

static AcResult
synchronize_halos(const StreamType stream)
{
    // Exchanges the halos of subgrids
    // After this step, the data within the main grid ranging from
    // (0, 0, NGHOST) -> grid.m.x, grid.m.y, NGHOST + grid.n.z
    // has been synchronized and transferred to appropriate subgrids

    // We loop only to num_devices - 1 since the front and back plate of the grid is not
    // transferred because their contents depend on the boundary conditions.

    // IMPORTANT NOTE: the boundary conditions must be applied before calling this function!
    // I.e. the halos of subgrids must contain up-to-date data!
    // #pragma omp parallel for
    for (int i = 0; i < num_devices - 1; ++i) {
        const int num_vertices = subgrid.m.x * subgrid.m.y * NGHOST;
        // ...|ooooxxx|... -> xxx|ooooooo|...
        {
            const int3 src = (int3){0, 0, subgrid.n.z};
            const int3 dst = (int3){0, 0, 0};
            copyMeshDeviceToDevice(devices[i], stream, src, devices[(i + 1) % num_devices], dst,
                                   num_vertices);
        }
        // ...|ooooooo|xxx <- ...|xxxoooo|...
        {
            const int3 src = (int3){0, 0, NGHOST};
            const int3 dst = (int3){0, 0, NGHOST + subgrid.n.z};
            copyMeshDeviceToDevice(devices[(i + 1) % num_devices], stream, src, devices[i], dst,
                                   num_vertices);
        }
    }
    return AC_SUCCESS;
}

AcResult
acSynchronizeMesh(void)
{
    acSynchronizeStream(STREAM_ALL);
    synchronize_halos(STREAM_DEFAULT);
    acSynchronizeStream(STREAM_ALL);

    return AC_SUCCESS;
}

AcResult
acInit(const AcMeshInfo& config)
{
    // Get num_devices
    ERRCHK_CUDA_ALWAYS(hipGetDeviceCount(&num_devices));
    if (num_devices < 1) {
        ERROR("No CUDA devices found!");
        return AC_FAILURE;
    }
    if (num_devices > MAX_NUM_DEVICES) {
        WARNING("More devices found than MAX_NUM_DEVICES. Using only MAX_NUM_DEVICES");
        num_devices = MAX_NUM_DEVICES;
    }
    if (!AC_MULTIGPU_ENABLED) {
        WARNING("MULTIGPU_ENABLED was false. Using only one device");
        num_devices = 1; // Use only one device if multi-GPU is not enabled
    }
    // Check that num_devices is divisible with AC_nz. This makes decomposing the
    // problem domain to multiple GPUs much easier since we do not have to worry
    // about remainders
    ERRCHK_ALWAYS(config.int_params[AC_nz] % num_devices == 0);

    // Decompose the problem domain
    // The main grid
    grid = createGrid(config);

    // Subgrids
    AcMeshInfo subgrid_config = config;
    subgrid_config.int_params[AC_nz] /= num_devices;
    update_config(&subgrid_config);
    subgrid = createGrid(subgrid_config);

    // Periodic boundary conditions become weird if the system can "fold unto itself".
    ERRCHK_ALWAYS(subgrid.n.x >= STENCIL_ORDER);
    ERRCHK_ALWAYS(subgrid.n.y >= STENCIL_ORDER);
    ERRCHK_ALWAYS(subgrid.n.z >= STENCIL_ORDER);

#if VERBOSE_PRINTING
    // clang-format off
    printf("Grid m ");   printInt3(grid.m);    printf("\n");
    printf("Grid n ");   printInt3(grid.n);    printf("\n");
    printf("Subrid m "); printInt3(subgrid.m); printf("\n");
    printf("Subrid n "); printInt3(subgrid.n); printf("\n");
    // clang-format on
#endif

    // Initialize the devices
    for (int i = 0; i < num_devices; ++i) {
        createDevice(i, subgrid_config, &devices[i]);
        loadGlobalGrid(devices[i], grid);
        printDeviceInfo(devices[i]);
    }

    acSynchronizeStream(STREAM_ALL);
    return AC_SUCCESS;
}

AcResult
acQuit(void)
{
    acSynchronizeStream(STREAM_ALL);

    for (int i = 0; i < num_devices; ++i) {
        destroyDevice(devices[i]);
    }
    return AC_SUCCESS;
}

AcResult
acIntegrateStepWithOffsetAsync(const int& isubstep, const AcReal& dt, const int3& start,
                               const int3& end, const StreamType stream)
{
    // See the beginning of the file for an explanation of the index mapping
    // #pragma omp parallel for
    for (int i = 0; i < num_devices; ++i) {
        // DECOMPOSITION OFFSET HERE
        const int3 d0 = (int3){NGHOST, NGHOST, NGHOST + i * subgrid.n.z};
        const int3 d1 = d0 + (int3){subgrid.n.x, subgrid.n.y, subgrid.n.z};

        const int3 da = max(start, d0);
        const int3 db = min(end, d1);

        if (db.z >= da.z) {
            const int3 da_local = da - (int3){0, 0, i * subgrid.n.z};
            const int3 db_local = db - (int3){0, 0, i * subgrid.n.z};
            rkStep(devices[i], stream, isubstep, da_local, db_local, dt);
        }
    }
    return AC_SUCCESS;
}

AcResult
acIntegrateStepWithOffset(const int& isubstep, const AcReal& dt, const int3& start, const int3& end)
{
    return acIntegrateStepWithOffsetAsync(isubstep, dt, start, end, STREAM_DEFAULT);
}

AcResult
acIntegrateStepAsync(const int& isubstep, const AcReal& dt, const StreamType stream)
{
    const int3 start = (int3){NGHOST, NGHOST, NGHOST};
    const int3 end   = start + grid.n;
    return acIntegrateStepWithOffsetAsync(isubstep, dt, start, end, stream);
}

AcResult
acIntegrateStep(const int& isubstep, const AcReal& dt)
{
    return acIntegrateStepAsync(isubstep, dt, STREAM_DEFAULT);
}

static AcResult
local_boundcondstep(const StreamType stream)
{
    if (num_devices == 1) {
        boundcondStep(devices[0], stream, (int3){0, 0, 0}, subgrid.m);
    }
    else {
        // Local boundary conditions
        // #pragma omp parallel for
        for (int i = 0; i < num_devices; ++i) {
            const int3 d0 = (int3){0, 0, NGHOST}; // DECOMPOSITION OFFSET HERE
            const int3 d1 = (int3){subgrid.m.x, subgrid.m.y, d0.z + subgrid.n.z};
            boundcondStep(devices[i], stream, d0, d1);
        }
    }
    return AC_SUCCESS;
}

static AcResult
global_boundcondstep(const StreamType stream)
{
    if (num_devices > 1) {
        // With periodic boundary conditions we exchange the front and back plates of the
        // grid. The exchange is done between the first and last device (0 and num_devices - 1).
        const int num_vertices = subgrid.m.x * subgrid.m.y * NGHOST;
        // ...|ooooxxx|... -> xxx|ooooooo|...
        {
            const int3 src = (int3){0, 0, subgrid.n.z};
            const int3 dst = (int3){0, 0, 0};
            copyMeshDeviceToDevice(devices[num_devices - 1], stream, src, devices[0], dst,
                                   num_vertices);
        }
        // ...|ooooooo|xxx <- ...|xxxoooo|...
        {
            const int3 src = (int3){0, 0, NGHOST};
            const int3 dst = (int3){0, 0, NGHOST + subgrid.n.z};
            copyMeshDeviceToDevice(devices[0], stream, src, devices[num_devices - 1], dst,
                                   num_vertices);
        }
    }
    return AC_SUCCESS;
}

AcResult
acBoundcondStepAsync(const StreamType stream)
{
    ERRCHK_ALWAYS(stream < NUM_STREAM_TYPES);

    local_boundcondstep(stream);
    acSynchronizeStream(stream);
    global_boundcondstep(stream);
    synchronize_halos(stream);
    acSynchronizeStream(stream);
    return AC_SUCCESS;
}

AcResult
acBoundcondStep(void)
{
    return acBoundcondStepAsync(STREAM_DEFAULT);
}

static AcResult
swap_buffers(void)
{
    // #pragma omp parallel for
    for (int i = 0; i < num_devices; ++i) {
        swapBuffers(devices[i]);
    }
    return AC_SUCCESS;
}

AcResult
acIntegrate(const AcReal& dt)
{
    acSynchronizeStream(STREAM_ALL);
    for (int isubstep = 0; isubstep < 3; ++isubstep) {
        acIntegrateStep(isubstep, dt); // Note: boundaries must be initialized.
        swap_buffers();
        acBoundcondStep();
    }
    return AC_SUCCESS;
}

static AcReal
simple_final_reduce_scal(const ReductionType& rtype, const AcReal* results, const int& n)
{
    AcReal res = results[0];
    for (int i = 1; i < n; ++i) {
        if (rtype == RTYPE_MAX) {
            res = max(res, results[i]);
        }
        else if (rtype == RTYPE_MIN) {
            res = min(res, results[i]);
        }
        else if (rtype == RTYPE_RMS || rtype == RTYPE_RMS_EXP) {
            res = sum(res, results[i]);
        }
        else {
            ERROR("Invalid rtype");
        }
    }

    if (rtype == RTYPE_RMS || rtype == RTYPE_RMS_EXP) {
        const AcReal inv_n = AcReal(1.) / (grid.n.x * grid.n.y * grid.n.z);
        res                = sqrt(inv_n * res);
    }

    return res;
}

AcReal
acReduceScal(const ReductionType& rtype, const VertexBufferHandle& vtxbuffer_handle)
{
    acSynchronizeStream(STREAM_ALL);

    AcReal results[num_devices];
    // #pragma omp parallel for
    for (int i = 0; i < num_devices; ++i) {
        reduceScal(devices[i], STREAM_DEFAULT, rtype, vtxbuffer_handle, &results[i]);
    }

    return simple_final_reduce_scal(rtype, results, num_devices);
}

AcReal
acReduceVec(const ReductionType& rtype, const VertexBufferHandle& a, const VertexBufferHandle& b,
            const VertexBufferHandle& c)
{
    acSynchronizeStream(STREAM_ALL);

    AcReal results[num_devices];
    // #pragma omp parallel for
    for (int i = 0; i < num_devices; ++i) {
        reduceVec(devices[i], STREAM_DEFAULT, rtype, a, b, c, &results[i]);
    }

    return simple_final_reduce_scal(rtype, results, num_devices);
}

AcResult
acLoadWithOffsetAsync(const AcMesh& host_mesh, const int3& src, const int num_vertices,
                      const StreamType stream)
{
    // See the beginning of the file for an explanation of the index mapping
    // #pragma omp parallel for
    for (int i = 0; i < num_devices; ++i) {
        const int3 d0 = (int3){0, 0, i * subgrid.n.z}; // DECOMPOSITION OFFSET HERE
        const int3 d1 = (int3){subgrid.m.x, subgrid.m.y, d0.z + subgrid.m.z};

        const int3 s0 = src;
        const int3 s1 = gridIdx3d(grid, gridIdx(grid, s0) + num_vertices);

        const int3 da = max(s0, d0);
        const int3 db = min(s1, d1);
        /*
        printf("Device %d\n", i);
        printf("\ts0: "); printInt3(s0); printf("\n");
        printf("\td0: "); printInt3(d0); printf("\n");
        printf("\tda: "); printInt3(da); printf("\n");
        printf("\tdb: "); printInt3(db); printf("\n");
        printf("\td1: "); printInt3(d1); printf("\n");
        printf("\ts1: "); printInt3(s1); printf("\n");
        printf("\t-> %s to device %d\n", db.z >= da.z ? "Copy" : "Do not copy", i);
        */
        if (db.z >= da.z) {
            const int copy_cells = gridIdx(subgrid, db) - gridIdx(subgrid, da);
            // DECOMPOSITION OFFSET HERE
            const int3 da_local = (int3){da.x, da.y, da.z - i * grid.n.z / num_devices};
            // printf("\t\tcopy %d cells to local index ", copy_cells); printInt3(da_local);
            // printf("\n");
            copyMeshToDevice(devices[i], stream, host_mesh, da, da_local, copy_cells);
        }
        // printf("\n");
    }
    return AC_SUCCESS;
}

AcResult
acLoadWithOffset(const AcMesh& host_mesh, const int3& src, const int num_vertices)
{
    return acLoadWithOffsetAsync(host_mesh, src, num_vertices, STREAM_DEFAULT);
}

AcResult
acLoad(const AcMesh& host_mesh)
{
    acLoadWithOffset(host_mesh, (int3){0, 0, 0}, AC_VTXBUF_SIZE(host_mesh.info));
    acSynchronizeStream(STREAM_ALL);
    return AC_SUCCESS;
}

AcResult
acStoreWithOffsetAsync(const int3& src, const int num_vertices, AcMesh* host_mesh,
                       const StreamType stream)
{
    // See the beginning of the file for an explanation of the index mapping
    // #pragma omp parallel for
    for (int i = 0; i < num_devices; ++i) {
        const int3 d0 = (int3){0, 0, i * subgrid.n.z}; // DECOMPOSITION OFFSET HERE
        const int3 d1 = (int3){subgrid.m.x, subgrid.m.y, d0.z + subgrid.m.z};

        const int3 s0 = src;
        const int3 s1 = gridIdx3d(grid, gridIdx(grid, s0) + num_vertices);

        const int3 da = max(s0, d0);
        const int3 db = min(s1, d1);
        if (db.z >= da.z) {
            const int copy_cells = gridIdx(subgrid, db) - gridIdx(subgrid, da);
            // DECOMPOSITION OFFSET HERE
            const int3 da_local = (int3){da.x, da.y, da.z - i * grid.n.z / num_devices};
            copyMeshToHost(devices[i], stream, da_local, da, copy_cells, host_mesh);
        }
    }
    return AC_SUCCESS;
}

AcResult
acStoreWithOffset(const int3& src, const int num_vertices, AcMesh* host_mesh)
{
    return acStoreWithOffsetAsync(src, num_vertices, host_mesh, STREAM_DEFAULT);
}

AcResult
acStore(AcMesh* host_mesh)
{
    acStoreWithOffset((int3){0, 0, 0}, AC_VTXBUF_SIZE(host_mesh->info), host_mesh);
    acSynchronizeStream(STREAM_ALL);
    return AC_SUCCESS;
}

AcResult
acLoadDeviceConstantAsync(const AcRealParam param, const AcReal value, const StreamType stream)
{
    // #pragma omp parallel for
    for (int i = 0; i < num_devices; ++i) {
        loadDeviceConstant(devices[i], stream, param, value);
    }
    return AC_SUCCESS;
}

AcResult
acLoadDeviceConstant(const AcRealParam param, const AcReal value)
{
    return acLoadDeviceConstantAsync(param, value, STREAM_DEFAULT);
}
