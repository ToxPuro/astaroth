#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2019, Johannes Pekkilae, Miikka Vaeisalae.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/
// #include "astaroth_defines.h"
#include "astaroth.h"

#define AC_GEN_STR(X) #X
const char* intparam_names[]   = {AC_FOR_BUILTIN_INT_PARAM_TYPES(AC_GEN_STR) //
                                AC_FOR_USER_INT_PARAM_TYPES(AC_GEN_STR)};
const char* int3param_names[]  = {AC_FOR_BUILTIN_INT3_PARAM_TYPES(AC_GEN_STR) //
                                 AC_FOR_USER_INT3_PARAM_TYPES(AC_GEN_STR)};
const char* realparam_names[]  = {AC_FOR_BUILTIN_REAL_PARAM_TYPES(AC_GEN_STR) //
                                 AC_FOR_USER_REAL_PARAM_TYPES(AC_GEN_STR)};
const char* real3param_names[] = {AC_FOR_BUILTIN_REAL3_PARAM_TYPES(AC_GEN_STR) //
                                  AC_FOR_USER_REAL3_PARAM_TYPES(AC_GEN_STR)};
const char* vtxbuf_names[]     = {AC_FOR_VTXBUF_HANDLES(AC_GEN_STR)};
#undef AC_GEN_STR

static const int num_nodes = 1;
static Node nodes[num_nodes];

AcResult
acInit(const AcMeshInfo mesh_info)
{
    return acNodeCreate(0, mesh_info, &nodes[0]);
}

AcResult
acQuit(void)
{
    return acNodeDestroy(nodes[0]);
}

AcResult
acSynchronizeStream(const Stream stream)
{
    return acNodeSynchronizeStream(nodes[0], stream);
}

AcResult
acLoadDeviceConstant(const AcRealParam param, const AcReal value)
{
    return acNodeLoadConstant(nodes[0], STREAM_DEFAULT, param, value);
}

AcResult
acLoad(const AcMesh host_mesh)
{
    return acNodeLoadMesh(nodes[0], STREAM_DEFAULT, host_mesh);
}

AcResult
acStore(AcMesh* host_mesh)
{
    return acNodeStoreMesh(nodes[0], STREAM_DEFAULT, host_mesh);
}

AcResult
acIntegrate(const AcReal dt)
{
    /*
    acNodeIntegrate(nodes[0], dt);
    return acBoundcondStep();
    */
    return acNodeIntegrate(nodes[0], dt);
}

AcResult
acBoundcondStep(void)
{
    return acNodePeriodicBoundconds(nodes[0], STREAM_DEFAULT);
}

AcReal
acReduceScal(const ReductionType rtype, const VertexBufferHandle vtxbuf_handle)
{
    AcReal result;
    acNodeReduceScal(nodes[0], STREAM_DEFAULT, rtype, vtxbuf_handle, &result);
    return result;
}

AcReal
acReduceVec(const ReductionType rtype, const VertexBufferHandle a, const VertexBufferHandle b,
            const VertexBufferHandle c)
{
    AcReal result;
    acNodeReduceVec(nodes[0], STREAM_DEFAULT, rtype, a, b, c, &result);
    return result;
}

AcResult
acStoreWithOffset(const int3 dst, const size_t num_vertices, AcMesh* host_mesh)
{
    return acNodeStoreMeshWithOffset(nodes[0], STREAM_DEFAULT, dst, dst, num_vertices, host_mesh);
}
