#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2018, Johannes Pekkilae, Miikka Vaeisalae.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/

/**
 * @file
 * \brief Brief info.
 *
 * Detailed info.
 *
 */
#include "device.cuh"

#include "errchk.h"

typedef struct {
    AcReal* in[NUM_VTXBUF_HANDLES];
    AcReal* out[NUM_VTXBUF_HANDLES];
} VertexBufferArray;

__constant__ AcMeshInfo d_mesh_info;
__constant__ int3 d_multigpu_offset;
__constant__ Grid globalGrid;
#define DCONST_INT(X) (d_mesh_info.int_params[X])
#define DCONST_REAL(X) (d_mesh_info.real_params[X])
#define DEVICE_VTXBUF_IDX(i, j, k) ((i) + (j)*DCONST_INT(AC_mx) + (k)*DCONST_INT(AC_mxy))
#define DEVICE_1D_COMPDOMAIN_IDX(i, j, k) ((i) + (j)*DCONST_INT(AC_nx) + (k)*DCONST_INT(AC_nxy))
#include "kernels/kernels.cuh"

#if PACKED_DATA_TRANSFERS // Defined in device.cuh
// #include "kernels/pack_unpack.cuh"
#endif

struct device_s {
    int id;
    AcMeshInfo local_config;

    // Concurrency
    hipStream_t streams[NUM_STREAM_TYPES];

    // Memory
    VertexBufferArray vba;
    AcReal* reduce_scratchpad;
    AcReal* reduce_result;

#if PACKED_DATA_TRANSFERS
// Declare memory for buffers needed for packed data transfers here
// AcReal* data_packing_buffer;
#endif
};

AcResult
printDeviceInfo(const Device device)
{
    const int device_id = device->id;

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device_id);
    printf("--------------------------------------------------\n");
    printf("Device Number: %d\n", device_id);
    const size_t bus_id_max_len = 128;
    char bus_id[bus_id_max_len];
    hipDeviceGetPCIBusId(bus_id, bus_id_max_len, device_id);
    printf("  PCI bus ID: %s\n", bus_id);
    printf("    Device name: %s\n", props.name);
    printf("    Compute capability: %d.%d\n", props.major, props.minor);

    // Compute
    printf("  Compute\n");
    printf("    Clock rate (GHz): %g\n", props.clockRate / 1e6); // KHz -> GHz
    printf("    Stream processors: %d\n", props.multiProcessorCount);
    printf("    SP to DP flops performance ratio: %d:1\n", props.singleToDoublePrecisionPerfRatio);
    printf(
        "    Compute mode: %d\n",
        (int)props
            .computeMode); // https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1g7eb25f5413a962faad0956d92bae10d0
    // Memory
    printf("  Global memory\n");
    printf("    Memory Clock Rate (MHz): %d\n", props.memoryClockRate / (1000));
    printf("    Memory Bus Width (bits): %d\n", props.memoryBusWidth);
    printf("    Peak Memory Bandwidth (GiB/s): %f\n",
           2 * (props.memoryClockRate * 1e3) * props.memoryBusWidth / (8. * 1024. * 1024. * 1024.));
    printf("    ECC enabled: %d\n", props.ECCEnabled);
    // Memory usage
    size_t free_bytes, total_bytes;
    hipMemGetInfo(&free_bytes, &total_bytes);
    const size_t used_bytes = total_bytes - free_bytes;
    printf("    Total global mem: %.2f GiB\n", props.totalGlobalMem / (1024.0 * 1024 * 1024));
    printf("    Gmem used (GiB): %.2f\n", used_bytes / (1024.0 * 1024 * 1024));
    printf("    Gmem memory free (GiB): %.2f\n", free_bytes / (1024.0 * 1024 * 1024));
    printf("    Gmem memory total (GiB): %.2f\n", total_bytes / (1024.0 * 1024 * 1024));
    printf("  Caches\n");
    printf("    Local L1 cache supported: %d\n", props.localL1CacheSupported);
    printf("    Global L1 cache supported: %d\n", props.globalL1CacheSupported);
    printf("    L2 size: %d KiB\n", props.l2CacheSize / (1024));
    printf("    Total const mem: %ld KiB\n", props.totalConstMem / (1024));
    printf("    Shared mem per block: %ld KiB\n", props.sharedMemPerBlock / (1024));
    printf("  Other\n");
    printf("    Warp size: %d\n", props.warpSize);
    // printf("    Single to double perf. ratio: %dx\n",
    // props.singleToDoublePrecisionPerfRatio); //Not supported with older CUDA
    // versions
    printf("    Stream priorities supported: %d\n", props.streamPrioritiesSupported);
    printf("--------------------------------------------------\n");

    return AC_SUCCESS;
}

static __global__ void
dummy_kernel(void)
{
}

AcResult
createDevice(const int id, const AcMeshInfo device_config, Device* device_handle)
{
    hipSetDevice(id);
    hipDeviceReset();

    // Create Device
    struct device_s* device = (struct device_s*)malloc(sizeof(*device));
    ERRCHK_ALWAYS(device);

    device->id           = id;
    device->local_config = device_config;

    // Check that the code was compiled for the proper GPU architecture
    printf("Trying to run a dummy kernel. If this fails, make sure that your\n"
           "device supports the CUDA architecture you are compiling for.\n"
           "Running dummy kernel... ");
    fflush(stdout);
    dummy_kernel<<<1, 1>>>();
    ERRCHK_CUDA_KERNEL_ALWAYS();
    printf("Success!\n");

    // Concurrency
    for (int i = 0; i < NUM_STREAM_TYPES; ++i) {
        hipStreamCreate(&device->streams[i]);
    }

    // Memory
    const size_t vba_size_bytes = AC_VTXBUF_SIZE_BYTES(device_config);
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        ERRCHK_CUDA_ALWAYS(hipMalloc(&device->vba.in[i], vba_size_bytes));
        ERRCHK_CUDA_ALWAYS(hipMalloc(&device->vba.out[i], vba_size_bytes));
    }
    ERRCHK_CUDA_ALWAYS(
        hipMalloc(&device->reduce_scratchpad, AC_VTXBUF_COMPDOMAIN_SIZE_BYTES(device_config)));
    ERRCHK_CUDA_ALWAYS(hipMalloc(&device->reduce_result, sizeof(AcReal)));

#if PACKED_DATA_TRANSFERS
// Allocate data required for packed transfers here (hipMalloc)
#endif

    // Device constants
    ERRCHK_CUDA_ALWAYS(hipMemcpyToSymbol(HIP_SYMBOL(d_mesh_info), &device_config, sizeof(device_config), 0,
                                          hipMemcpyHostToDevice));

    // Multi-GPU offset. This is used to compute globalVertexIdx.
    // Might be better to calculate this in astaroth.cu instead of here, s.t.
    // everything related to the decomposition is limited to the multi-GPU layer
    const int3 multigpu_offset = (int3){0, 0, device->id * device->local_config.int_params[AC_nz]};
    ERRCHK_CUDA_ALWAYS(hipMemcpyToSymbol(HIP_SYMBOL(d_multigpu_offset), &multigpu_offset,
                                          sizeof(multigpu_offset), 0, hipMemcpyHostToDevice));

    printf("Created device %d (%p)\n", device->id, device);
    *device_handle = device;
    return AC_SUCCESS;
}

AcResult
destroyDevice(Device device)
{
    hipSetDevice(device->id);
    printf("Destroying device %d (%p)\n", device->id, device);

    // Memory
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        hipFree(device->vba.in[i]);
        hipFree(device->vba.out[i]);
    }
    hipFree(device->reduce_scratchpad);
    hipFree(device->reduce_result);

#if PACKED_DATA_TRANSFERS
// Free data required for packed tranfers here (hipFree)
#endif

    // Concurrency
    for (int i = 0; i < NUM_STREAM_TYPES; ++i)
        hipStreamDestroy(device->streams[i]);

    // Destroy Device
    free(device);
    return AC_SUCCESS;
}

AcResult
boundcondStep(const Device device, const StreamType stream_type, const int3& start, const int3& end)
{
    hipSetDevice(device->id);
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        periodic_boundconds(device->streams[stream_type], start, end, device->vba.in[i]);
    }
    return AC_SUCCESS;
}

AcResult
reduceScal(const Device device, const StreamType stream_type, const ReductionType rtype,
           const VertexBufferHandle vtxbuf_handle, AcReal* result)
{
    hipSetDevice(device->id);

    const int3 start = (int3){device->local_config.int_params[AC_nx_min],
                              device->local_config.int_params[AC_ny_min],
                              device->local_config.int_params[AC_nz_min]};

    const int3 end = (int3){device->local_config.int_params[AC_nx_max],
                            device->local_config.int_params[AC_ny_max],
                            device->local_config.int_params[AC_nz_max]};

    *result = reduce_scal(device->streams[stream_type], rtype, start, end,
                          device->vba.in[vtxbuf_handle], device->reduce_scratchpad,
                          device->reduce_result);
    return AC_SUCCESS;
}

AcResult
reduceVec(const Device device, const StreamType stream_type, const ReductionType rtype,
          const VertexBufferHandle vtxbuf0, const VertexBufferHandle vtxbuf1,
          const VertexBufferHandle vtxbuf2, AcReal* result)
{
    hipSetDevice(device->id);

    const int3 start = (int3){device->local_config.int_params[AC_nx_min],
                              device->local_config.int_params[AC_ny_min],
                              device->local_config.int_params[AC_nz_min]};

    const int3 end = (int3){device->local_config.int_params[AC_nx_max],
                            device->local_config.int_params[AC_ny_max],
                            device->local_config.int_params[AC_nz_max]};

    *result = reduce_vec(device->streams[stream_type], rtype, start, end, device->vba.in[vtxbuf0],
                         device->vba.in[vtxbuf1], device->vba.in[vtxbuf2],
                         device->reduce_scratchpad, device->reduce_result);
    return AC_SUCCESS;
}

AcResult
rkStep(const Device device, const StreamType stream_type, const int step_number, const int3& start,
       const int3& end, const AcReal dt)
{
    hipSetDevice(device->id);
    rk3_step_async(device->streams[stream_type], step_number, start, end, dt, &device->vba);
    return AC_SUCCESS;
}

AcResult
synchronize(const Device device, const StreamType stream_type)
{
    hipSetDevice(device->id);
    if (stream_type == STREAM_ALL) {
        hipDeviceSynchronize();
    }
    else {
        hipStreamSynchronize(device->streams[stream_type]);
    }
    return AC_SUCCESS;
}

static AcResult
loadWithOffset(const Device device, const StreamType stream_type, const AcReal* src,
               const size_t bytes, AcReal* dst)
{
    hipSetDevice(device->id);
    ERRCHK_CUDA(
        hipMemcpyAsync(dst, src, bytes, hipMemcpyHostToDevice, device->streams[stream_type]));
    return AC_SUCCESS;
}

static AcResult
storeWithOffset(const Device device, const StreamType stream_type, const AcReal* src,
                const size_t bytes, AcReal* dst)
{
    hipSetDevice(device->id);
    ERRCHK_CUDA(
        hipMemcpyAsync(dst, src, bytes, hipMemcpyDeviceToHost, device->streams[stream_type]));
    return AC_SUCCESS;
}

AcResult
copyMeshToDevice(const Device device, const StreamType stream_type, const AcMesh& host_mesh,
                 const int3& src, const int3& dst, const int num_vertices)
{
    const size_t src_idx = AC_VTXBUF_IDX(src.x, src.y, src.z, host_mesh.info);
    const size_t dst_idx = AC_VTXBUF_IDX(dst.x, dst.y, dst.z, device->local_config);
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        loadWithOffset(device, stream_type, &host_mesh.vertex_buffer[i][src_idx],
                       num_vertices * sizeof(AcReal), &device->vba.in[i][dst_idx]);
    }
    return AC_SUCCESS;
}

AcResult
copyMeshToHost(const Device device, const StreamType stream_type, const int3& src, const int3& dst,
               const int num_vertices, AcMesh* host_mesh)
{
    const size_t src_idx = AC_VTXBUF_IDX(src.x, src.y, src.z, device->local_config);
    const size_t dst_idx = AC_VTXBUF_IDX(dst.x, dst.y, dst.z, host_mesh->info);
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        storeWithOffset(device, stream_type, &device->vba.in[i][src_idx],
                        num_vertices * sizeof(AcReal), &host_mesh->vertex_buffer[i][dst_idx]);
    }
    return AC_SUCCESS;
}

AcResult
copyMeshDeviceToDevice(const Device src_device, const StreamType stream_type, const int3& src,
                       Device dst_device, const int3& dst, const int num_vertices)
{
    hipSetDevice(src_device->id);
    const size_t src_idx = AC_VTXBUF_IDX(src.x, src.y, src.z, src_device->local_config);
    const size_t dst_idx = AC_VTXBUF_IDX(dst.x, dst.y, dst.z, dst_device->local_config);

    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        ERRCHK_CUDA(hipMemcpyPeerAsync(&dst_device->vba.in[i][dst_idx], dst_device->id,
                                        &src_device->vba.in[i][src_idx], src_device->id,
                                        sizeof(src_device->vba.in[i][0]) * num_vertices,
                                        src_device->streams[stream_type]));
    }
    return AC_SUCCESS;
}

AcResult
swapBuffers(const Device device)
{
    hipSetDevice(device->id);
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        AcReal* tmp        = device->vba.in[i];
        device->vba.in[i]  = device->vba.out[i];
        device->vba.out[i] = tmp;
    }
    return AC_SUCCESS;
}

AcResult
loadDeviceConstant(const Device device, const AcIntParam param, const int value)
{
    hipSetDevice(device->id);
    // CUDA 10 apparently creates only a single name for a device constant (d_mesh_info here)
    // and something like d_mesh_info.real_params[] cannot be directly accessed.
    // Therefore we have to obfuscate the code a bit and compute the offset address before
    // invoking hipMemcpyToSymbol.
    const size_t offset = (size_t)&d_mesh_info.int_params[param] - (size_t)&d_mesh_info;
    ERRCHK_CUDA_ALWAYS(
        hipMemcpyToSymbol(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset, hipMemcpyHostToDevice));
    return AC_SUCCESS;
}

AcResult
loadDeviceConstant(const Device device, const AcRealParam param, const AcReal value)
{
    hipSetDevice(device->id);
    const size_t offset = (size_t)&d_mesh_info.real_params[param] - (size_t)&d_mesh_info;
    ERRCHK_CUDA_ALWAYS(
        hipMemcpyToSymbol(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset, hipMemcpyHostToDevice));
    return AC_SUCCESS;
}

AcResult
loadGlobalGrid(const Device device, const Grid grid)
{
    hipSetDevice(device->id);
    ERRCHK_CUDA_ALWAYS(
        hipMemcpyToSymbol(HIP_SYMBOL(globalGrid), &grid, sizeof(grid), 0, hipMemcpyHostToDevice));
    return AC_SUCCESS;
}

#if PACKED_DATA_TRANSFERS
// Functions for calling packed data transfers
#endif
