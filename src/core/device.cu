#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2019, Johannes Pekkilae, Miikka Vaeisalae.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/

/**
 * @file
 * \brief Brief info.
 *
 * Detailed info.
 *
 */
#include "device.cuh"

#include "errchk.h"

// Device info
#define REGISTERS_PER_THREAD (255)
#define MAX_REGISTERS_PER_BLOCK (65536)
#define MAX_THREADS_PER_BLOCK (1024)
#define WARP_SIZE (32)

typedef struct {
    AcReal* in[NUM_VTXBUF_HANDLES];
    AcReal* out[NUM_VTXBUF_HANDLES];
} VertexBufferArray;

__constant__ AcMeshInfo d_mesh_info;
__constant__ int3 d_multigpu_offset;
__constant__ Grid globalGrid;
#define DCONST_INT(X) (d_mesh_info.int_params[X])
#define DCONST_INT3(X) (d_mesh_info.int3_params[X])
#define DCONST_REAL(X) (d_mesh_info.real_params[X])
#define DCONST_REAL3(X) (d_mesh_info.real3_params[X])
#define DEVICE_VTXBUF_IDX(i, j, k) ((i) + (j)*DCONST_INT(AC_mx) + (k)*DCONST_INT(AC_mxy))
#define DEVICE_1D_COMPDOMAIN_IDX(i, j, k) ((i) + (j)*DCONST_INT(AC_nx) + (k)*DCONST_INT(AC_nxy))
#include "kernels/kernels.cuh"

static dim3 rk3_tpb = (dim3){32, 1, 4};

#if PACKED_DATA_TRANSFERS // Defined in device.cuh
// #include "kernels/pack_unpack.cuh"
#endif

struct device_s {
    int id;
    AcMeshInfo local_config;

    // Concurrency
    hipStream_t streams[NUM_STREAM_TYPES];

    // Memory
    VertexBufferArray vba;
    AcReal* reduce_scratchpad;
    AcReal* reduce_result;

#if PACKED_DATA_TRANSFERS
// Declare memory for buffers needed for packed data transfers here
// AcReal* data_packing_buffer;
#endif
};

AcResult
printDeviceInfo(const Device device)
{
    const int device_id = device->id;

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device_id);
    printf("--------------------------------------------------\n");
    printf("Device Number: %d\n", device_id);
    const size_t bus_id_max_len = 128;
    char bus_id[bus_id_max_len];
    hipDeviceGetPCIBusId(bus_id, bus_id_max_len, device_id);
    printf("  PCI bus ID: %s\n", bus_id);
    printf("    Device name: %s\n", props.name);
    printf("    Compute capability: %d.%d\n", props.major, props.minor);

    // Compute
    printf("  Compute\n");
    printf("    Clock rate (GHz): %g\n", props.clockRate / 1e6); // KHz -> GHz
    printf("    Stream processors: %d\n", props.multiProcessorCount);
    printf("    SP to DP flops performance ratio: %d:1\n", props.singleToDoublePrecisionPerfRatio);
    printf(
        "    Compute mode: %d\n",
        (int)props
            .computeMode); // https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1g7eb25f5413a962faad0956d92bae10d0
    // Memory
    printf("  Global memory\n");
    printf("    Memory Clock Rate (MHz): %d\n", props.memoryClockRate / (1000));
    printf("    Memory Bus Width (bits): %d\n", props.memoryBusWidth);
    printf("    Peak Memory Bandwidth (GiB/s): %f\n",
           2 * (props.memoryClockRate * 1e3) * props.memoryBusWidth / (8. * 1024. * 1024. * 1024.));
    printf("    ECC enabled: %d\n", props.ECCEnabled);

    // Memory usage
    size_t free_bytes, total_bytes;
    hipMemGetInfo(&free_bytes, &total_bytes);
    const size_t used_bytes = total_bytes - free_bytes;
    printf("    Total global mem: %.2f GiB\n", props.totalGlobalMem / (1024.0 * 1024 * 1024));
    printf("    Gmem used (GiB): %.2f\n", used_bytes / (1024.0 * 1024 * 1024));
    printf("    Gmem memory free (GiB): %.2f\n", free_bytes / (1024.0 * 1024 * 1024));
    printf("    Gmem memory total (GiB): %.2f\n", total_bytes / (1024.0 * 1024 * 1024));
    printf("  Caches\n");
    printf("    Local L1 cache supported: %d\n", props.localL1CacheSupported);
    printf("    Global L1 cache supported: %d\n", props.globalL1CacheSupported);
    printf("    L2 size: %d KiB\n", props.l2CacheSize / (1024));
    // MV: props.totalConstMem and props.sharedMemPerBlock cause assembler error
    // MV: while compiling in TIARA gp cluster. Therefore commeted out.
    //!!    printf("    Total const mem: %ld KiB\n", props.totalConstMem / (1024));
    //!!    printf("    Shared mem per block: %ld KiB\n", props.sharedMemPerBlock / (1024));
    printf("  Other\n");
    printf("    Warp size: %d\n", props.warpSize);
    // printf("    Single to double perf. ratio: %dx\n",
    // props.singleToDoublePrecisionPerfRatio); //Not supported with older CUDA
    // versions
    printf("    Stream priorities supported: %d\n", props.streamPrioritiesSupported);
    printf("--------------------------------------------------\n");

    return AC_SUCCESS;
}

static __global__ void
dummy_kernel(void)
{
}

AcResult
createDevice(const int id, const AcMeshInfo device_config, Device* device_handle)
{
    hipSetDevice(id);
    hipDeviceReset();

    // Create Device
    struct device_s* device = (struct device_s*)malloc(sizeof(*device));
    ERRCHK_ALWAYS(device);

    device->id           = id;
    device->local_config = device_config;

    // Check that the code was compiled for the proper GPU architecture
    printf("Trying to run a dummy kernel. If this fails, make sure that your\n"
           "device supports the CUDA architecture you are compiling for.\n"
           "Running dummy kernel... ");
    fflush(stdout);
    dummy_kernel<<<1, 1>>>();
    ERRCHK_CUDA_KERNEL_ALWAYS();
    printf("Success!\n");

    // Concurrency
    for (int i = 0; i < NUM_STREAM_TYPES; ++i) {
        hipStreamCreateWithPriority(&device->streams[i], hipStreamNonBlocking, 0);
    }

    // Memory
    const size_t vba_size_bytes = acVertexBufferSizeBytes(device_config);
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        ERRCHK_CUDA_ALWAYS(hipMalloc(&device->vba.in[i], vba_size_bytes));
        ERRCHK_CUDA_ALWAYS(hipMalloc(&device->vba.out[i], vba_size_bytes));
    }
    ERRCHK_CUDA_ALWAYS(
        hipMalloc(&device->reduce_scratchpad, acVertexBufferCompdomainSizeBytes(device_config)));
    ERRCHK_CUDA_ALWAYS(hipMalloc(&device->reduce_result, sizeof(AcReal)));

#if PACKED_DATA_TRANSFERS
// Allocate data required for packed transfers here (hipMalloc)
#endif

    // Device constants
    ERRCHK_CUDA_ALWAYS(hipMemcpyToSymbol(HIP_SYMBOL(d_mesh_info), &device_config, sizeof(device_config), 0,
                                          hipMemcpyHostToDevice));

    // Multi-GPU offset. This is used to compute globalVertexIdx.
    // Might be better to calculate this in astaroth.cu instead of here, s.t.
    // everything related to the decomposition is limited to the multi-GPU layer
    const int3 multigpu_offset = (int3){0, 0, device->id * device->local_config.int_params[AC_nz]};
    ERRCHK_CUDA_ALWAYS(hipMemcpyToSymbol(HIP_SYMBOL(d_multigpu_offset), &multigpu_offset,
                                          sizeof(multigpu_offset), 0, hipMemcpyHostToDevice));

    printf("Created device %d (%p)\n", device->id, device);
    *device_handle = device;

    // Autoptimize
    if (id == 0)
        autoOptimize(device);

    return AC_SUCCESS;
}

AcResult
destroyDevice(Device device)
{
    hipSetDevice(device->id);
    printf("Destroying device %d (%p)\n", device->id, device);

    // Memory
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        hipFree(device->vba.in[i]);
        hipFree(device->vba.out[i]);
    }
    hipFree(device->reduce_scratchpad);
    hipFree(device->reduce_result);

#if PACKED_DATA_TRANSFERS
// Free data required for packed tranfers here (hipFree)
#endif

    // Concurrency
    for (int i = 0; i < NUM_STREAM_TYPES; ++i) {
        hipStreamDestroy(device->streams[i]);
    }

    // Destroy Device
    free(device);
    return AC_SUCCESS;
}

AcResult
boundcondStep(const Device device, const StreamType stream_type, const int3& start, const int3& end)
{
    hipSetDevice(device->id);
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        periodic_boundconds(device->streams[stream_type], start, end, device->vba.in[i]);
    }
    return AC_SUCCESS;
}

AcResult
reduceScal(const Device device, const StreamType stream_type, const ReductionType rtype,
           const VertexBufferHandle vtxbuf_handle, AcReal* result)
{
    hipSetDevice(device->id);

    const int3 start = (int3){device->local_config.int_params[AC_nx_min],
                              device->local_config.int_params[AC_ny_min],
                              device->local_config.int_params[AC_nz_min]};

    const int3 end = (int3){device->local_config.int_params[AC_nx_max],
                            device->local_config.int_params[AC_ny_max],
                            device->local_config.int_params[AC_nz_max]};

    *result = reduce_scal(device->streams[stream_type], rtype, start, end,
                          device->vba.in[vtxbuf_handle], device->reduce_scratchpad,
                          device->reduce_result);
    return AC_SUCCESS;
}

AcResult
reduceVec(const Device device, const StreamType stream_type, const ReductionType rtype,
          const VertexBufferHandle vtxbuf0, const VertexBufferHandle vtxbuf1,
          const VertexBufferHandle vtxbuf2, AcReal* result)
{
    hipSetDevice(device->id);

    const int3 start = (int3){device->local_config.int_params[AC_nx_min],
                              device->local_config.int_params[AC_ny_min],
                              device->local_config.int_params[AC_nz_min]};

    const int3 end = (int3){device->local_config.int_params[AC_nx_max],
                            device->local_config.int_params[AC_ny_max],
                            device->local_config.int_params[AC_nz_max]};

    *result = reduce_vec(device->streams[stream_type], rtype, start, end, device->vba.in[vtxbuf0],
                         device->vba.in[vtxbuf1], device->vba.in[vtxbuf2],
                         device->reduce_scratchpad, device->reduce_result);
    return AC_SUCCESS;
}

AcResult
rkStep(const Device device, const StreamType stream_type, const int step_number, const int3& start,
       const int3& end, const AcReal dt)
{
    hipSetDevice(device->id);

    // const dim3 tpb(32, 1, 4);
    const dim3 tpb = rk3_tpb;

    const int3 n = end - start;
    const dim3 bpg((unsigned int)ceil(n.x / AcReal(tpb.x)), //
                   (unsigned int)ceil(n.y / AcReal(tpb.y)), //
                   (unsigned int)ceil(n.z / AcReal(tpb.z)));

    if (step_number == 0)
        solve<0><<<bpg, tpb, 0, device->streams[stream_type]>>>(start, end, device->vba, dt);
    else if (step_number == 1)
        solve<1><<<bpg, tpb, 0, device->streams[stream_type]>>>(start, end, device->vba, dt);
    else
        solve<2><<<bpg, tpb, 0, device->streams[stream_type]>>>(start, end, device->vba, dt);

    ERRCHK_CUDA_KERNEL();

    return AC_SUCCESS;
}

AcResult
synchronize(const Device device, const StreamType stream_type)
{
    hipSetDevice(device->id);
    if (stream_type == STREAM_ALL) {
        hipDeviceSynchronize();
    }
    else {
        hipStreamSynchronize(device->streams[stream_type]);
    }
    return AC_SUCCESS;
}

static AcResult
loadWithOffset(const Device device, const StreamType stream_type, const AcReal* src,
               const size_t bytes, AcReal* dst)
{
    hipSetDevice(device->id);
    ERRCHK_CUDA(
        hipMemcpyAsync(dst, src, bytes, hipMemcpyHostToDevice, device->streams[stream_type]));
    return AC_SUCCESS;
}

static AcResult
storeWithOffset(const Device device, const StreamType stream_type, const AcReal* src,
                const size_t bytes, AcReal* dst)
{
    hipSetDevice(device->id);
    ERRCHK_CUDA(
        hipMemcpyAsync(dst, src, bytes, hipMemcpyDeviceToHost, device->streams[stream_type]));
    return AC_SUCCESS;
}

AcResult
copyMeshToDevice(const Device device, const StreamType stream_type, const AcMesh& host_mesh,
                 const int3& src, const int3& dst, const int num_vertices)
{
    const size_t src_idx = acVertexBufferIdx(src.x, src.y, src.z, host_mesh.info);
    const size_t dst_idx = acVertexBufferIdx(dst.x, dst.y, dst.z, device->local_config);
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        loadWithOffset(device, stream_type, &host_mesh.vertex_buffer[i][src_idx],
                       num_vertices * sizeof(AcReal), &device->vba.in[i][dst_idx]);
    }
    return AC_SUCCESS;
}

AcResult
copyMeshToHost(const Device device, const StreamType stream_type, const int3& src, const int3& dst,
               const int num_vertices, AcMesh* host_mesh)
{
    const size_t src_idx = acVertexBufferIdx(src.x, src.y, src.z, device->local_config);
    const size_t dst_idx = acVertexBufferIdx(dst.x, dst.y, dst.z, host_mesh->info);
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        storeWithOffset(device, stream_type, &device->vba.in[i][src_idx],
                        num_vertices * sizeof(AcReal), &host_mesh->vertex_buffer[i][dst_idx]);
    }
    return AC_SUCCESS;
}

AcResult
copyMeshDeviceToDevice(const Device src_device, const StreamType stream_type, const int3& src,
                       Device dst_device, const int3& dst, const int num_vertices)
{
    hipSetDevice(src_device->id);
    const size_t src_idx = acVertexBufferIdx(src.x, src.y, src.z, src_device->local_config);
    const size_t dst_idx = acVertexBufferIdx(dst.x, dst.y, dst.z, dst_device->local_config);

    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        ERRCHK_CUDA(hipMemcpyPeerAsync(&dst_device->vba.in[i][dst_idx], dst_device->id,
                                        &src_device->vba.in[i][src_idx], src_device->id,
                                        sizeof(src_device->vba.in[i][0]) * num_vertices,
                                        src_device->streams[stream_type]));
    }
    return AC_SUCCESS;
}

AcResult
swapBuffers(const Device device)
{
    hipSetDevice(device->id);
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        AcReal* tmp        = device->vba.in[i];
        device->vba.in[i]  = device->vba.out[i];
        device->vba.out[i] = tmp;
    }
    return AC_SUCCESS;
}

AcResult
loadDeviceConstant(const Device device, const StreamType stream_type, const AcIntParam param,
                   const int value)
{
    hipSetDevice(device->id);
    // CUDA 10 apparently creates only a single name for a device constant (d_mesh_info here)
    // and something like d_mesh_info.real_params[] cannot be directly accessed.
    // Therefore we have to obfuscate the code a bit and compute the offset address before
    // invoking hipMemcpyToSymbol.
    const size_t offset = (size_t)&d_mesh_info.int_params[param] - (size_t)&d_mesh_info;
    ERRCHK_CUDA_ALWAYS(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset,
                                               hipMemcpyHostToDevice,
                                               device->streams[stream_type]));
    return AC_SUCCESS;
}

AcResult
loadDeviceConstant(const Device device, const StreamType stream_type, const AcRealParam param,
                   const AcReal value)
{
    hipSetDevice(device->id);
    const size_t offset = (size_t)&d_mesh_info.real_params[param] - (size_t)&d_mesh_info;
    ERRCHK_CUDA_ALWAYS(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset,
                                               hipMemcpyHostToDevice,
                                               device->streams[stream_type]));
    return AC_SUCCESS;
}

AcResult
loadGlobalGrid(const Device device, const Grid grid)
{
    hipSetDevice(device->id);
    ERRCHK_CUDA_ALWAYS(
        hipMemcpyToSymbol(HIP_SYMBOL(globalGrid), &grid, sizeof(grid), 0, hipMemcpyHostToDevice));
    return AC_SUCCESS;
}

AcResult
autoOptimize(const Device device)
{
    hipSetDevice(device->id);

    // RK3
    const int3 start = (int3){NGHOST, NGHOST, NGHOST};
    const int3 end   = start + (int3){device->local_config.int_params[AC_nx], //
                                    device->local_config.int_params[AC_ny], //
                                    device->local_config.int_params[AC_nz]};

    dim3 best_dims(0, 0, 0);
    float best_time          = INFINITY;
    const int num_iterations = 10;

    for (int z = 1; z <= MAX_THREADS_PER_BLOCK; ++z) {
        for (int y = 1; y <= MAX_THREADS_PER_BLOCK; ++y) {
            for (int x = WARP_SIZE; x <= MAX_THREADS_PER_BLOCK; x += WARP_SIZE) {

                if (x > end.x - start.x || y > end.y - start.y || z > end.z - start.z)
                    break;
                if (x * y * z > MAX_THREADS_PER_BLOCK)
                    break;

                if (x * y * z * REGISTERS_PER_THREAD > MAX_REGISTERS_PER_BLOCK)
                    break;

                if (((x * y * z) % WARP_SIZE) != 0)
                    continue;

                const dim3 tpb(x, y, z);
                const int3 n = end - start;
                const dim3 bpg((unsigned int)ceil(n.x / AcReal(tpb.x)), //
                               (unsigned int)ceil(n.y / AcReal(tpb.y)), //
                               (unsigned int)ceil(n.z / AcReal(tpb.z)));

                hipDeviceSynchronize();
                if (hipGetLastError() != hipSuccess) // resets the error if any
                    continue;

                // printf("(%d, %d, %d)\n", x, y, z);

                hipEvent_t tstart, tstop;
                hipEventCreate(&tstart);
                hipEventCreate(&tstop);

                hipEventRecord(tstart); // ---------------------------------------- Timing start

                for (int i = 0; i < num_iterations; ++i)
                    solve<2><<<bpg, tpb>>>(start, end, device->vba, FLT_EPSILON);

                hipEventRecord(tstop); // ----------------------------------------- Timing end
                hipEventSynchronize(tstop);
                float milliseconds = 0;
                hipEventElapsedTime(&milliseconds, tstart, tstop);

                ERRCHK_CUDA_KERNEL_ALWAYS();
                if (milliseconds < best_time) {
                    best_time = milliseconds;
                    best_dims = tpb;
                }
            }
        }
    }
#if VERBOSE_PRINTING
    printf(
        "Auto-optimization done. The best threadblock dimensions for rkStep: (%d, %d, %d) %f ms\n",
        best_dims.x, best_dims.y, best_dims.z, double(best_time) / num_iterations);
#endif
    /*
    FILE* fp = fopen("../config/rk3_tbdims.cuh", "w");
    ERRCHK(fp);
    fprintf(fp, "%d, %d, %d\n", best_dims.x, best_dims.y, best_dims.z);
    fclose(fp);
    */

    rk3_tpb = best_dims;
    return AC_SUCCESS;
}

#if PACKED_DATA_TRANSFERS
// Functions for calling packed data transfers
#endif
