#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2019, Johannes Pekkilae, Miikka Vaeisalae.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/

/**
 * @file
 * \brief Brief info.
 *
 * Detailed info.
 *
 */
#include "astaroth_device.h"

#include "errchk.h"

// Device info
#define REGISTERS_PER_THREAD (255)
#define MAX_REGISTERS_PER_BLOCK (65536)
#define MAX_THREADS_PER_BLOCK (1024)
#define WARP_SIZE (32)

typedef struct {
    AcReal* in[NUM_VTXBUF_HANDLES];
    AcReal* out[NUM_VTXBUF_HANDLES];

    AcReal* profiles[NUM_SCALARARRAY_HANDLES];
} VertexBufferArray;

struct device_s {
    int id;
    AcMeshInfo local_config;

    // Concurrency
    hipStream_t streams[NUM_STREAMS];

    // Memory
    VertexBufferArray vba;
    AcReal* reduce_scratchpad;
    AcReal* reduce_result;

#if PACKED_DATA_TRANSFERS
// Declare memory for buffers needed for packed data transfers here
// AcReal* data_packing_buffer;
#endif
};

__constant__ AcMeshInfo d_mesh_info;
static int __device__ __forceinline__
DCONST(const AcIntParam param)
{
    return d_mesh_info.int_params[param];
}
static int3 __device__ __forceinline__
DCONST(const AcInt3Param param)
{
    return d_mesh_info.int3_params[param];
}
static AcReal __device__ __forceinline__
DCONST(const AcRealParam param)
{
    return d_mesh_info.real_params[param];
}
static AcReal3 __device__ __forceinline__
DCONST(const AcReal3Param param)
{
    return d_mesh_info.real3_params[param];
}
static __device__ constexpr VertexBufferHandle
DCONST(const VertexBufferHandle handle)
{
    return handle;
}
#define DEVICE_VTXBUF_IDX(i, j, k) ((i) + (j)*DCONST(AC_mx) + (k)*DCONST(AC_mxy))
#define DEVICE_1D_COMPDOMAIN_IDX(i, j, k) ((i) + (j)*DCONST(AC_nx) + (k)*DCONST(AC_nxy))
#define globalGridN (d_mesh_info.int3_params[AC_global_grid_n])
//#define globalMeshM // Placeholder
//#define localMeshN // Placeholder
//#define localMeshM // Placeholder
//#define localMeshN_min // Placeholder
//#define globalMeshN_min // Placeholder
#define d_multigpu_offset (d_mesh_info.int3_params[AC_multigpu_offset])
//#define d_multinode_offset (d_mesh_info.int3_params[AC_multinode_offset]) // Placeholder
//#include <thrust/complex.h>
// using namespace thrust;
#include <hip/hip_complex.h>
#if AC_DOUBLE_PRECISION == 1
typedef hipDoubleComplex acComplex;
#define acComplex(x, y) make_hipDoubleComplex(x, y)
#else
typedef hipFloatComplex acComplex;
#define acComplex(x, y) make_hipFloatComplex(x, y)
#endif
static __device__ inline acComplex
exp(const acComplex& val)
{
    return acComplex(exp(val.x) * cos(val.y), exp(val.x) * sin(val.y));
}
static __device__ inline acComplex operator*(const AcReal& a, const acComplex& b)
{
    return (acComplex){a * b.x, a * b.y};
}

static __device__ inline acComplex operator*(const acComplex& b, const AcReal& a)
{
    return (acComplex){a * b.x, a * b.y};
}

static __device__ inline acComplex operator*(const acComplex& a, const acComplex& b)
{
    return (acComplex){a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x};
}
//#include <complex>

#include "kernels/boundconds.cuh"
#include "kernels/integration.cuh"
#include "kernels/reductions.cuh"

static dim3 rk3_tpb(32, 1, 4);

#if PACKED_DATA_TRANSFERS // Defined in device.cuh
// #include "kernels/pack_unpack.cuh"
#endif

static __global__ void
dummy_kernel(void)
{
    DCONST((AcIntParam)0);
    DCONST((AcInt3Param)0);
    DCONST((AcRealParam)0);
    DCONST((AcReal3Param)0);
    acComplex a = exp(AcReal(1) * acComplex(1, 1) * AcReal(1));
    a* a;
}

AcResult
acDeviceCreate(const int id, const AcMeshInfo device_config, Device* device_handle)
{
    hipSetDevice(id);
    // hipDeviceReset(); // Would be good for safety, but messes stuff up if we want to emulate
    // multiple devices with a single GPU

    // Create Device
    struct device_s* device = (struct device_s*)malloc(sizeof(*device));
    ERRCHK_ALWAYS(device);

    device->id           = id;
    device->local_config = device_config;
    acDevicePrintInfo(device);

    // Check that the code was compiled for the proper GPU architecture
    printf("Trying to run a dummy kernel. If this fails, make sure that your\n"
           "device supports the CUDA architecture you are compiling for.\n"
           "Running dummy kernel... ");
    fflush(stdout);
    dummy_kernel<<<1, 1>>>();
    ERRCHK_CUDA_KERNEL_ALWAYS();
    printf("Success!\n");

    // Concurrency
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamCreateWithPriority(&device->streams[i], hipStreamNonBlocking, 0);
    }

    // Memory
    // VBA in/out
    const size_t vba_size_bytes = acVertexBufferSizeBytes(device_config);
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        ERRCHK_CUDA_ALWAYS(hipMallocManaged(&device->vba.in[i], vba_size_bytes));
        ERRCHK_CUDA_ALWAYS(hipMallocManaged(&device->vba.out[i], vba_size_bytes));
    }
    // VBA Profiles
    const size_t profile_size_bytes = sizeof(AcReal) * max(device_config.int_params[AC_mx],
                                                           max(device_config.int_params[AC_my],
                                                               device_config.int_params[AC_mz]));
    for (int i = 0; i < NUM_SCALARARRAY_HANDLES; ++i) {
        ERRCHK_CUDA_ALWAYS(hipMallocManaged(&device->vba.profiles[i], profile_size_bytes));
    }

    // Reductions
    ERRCHK_CUDA_ALWAYS(hipMallocManaged(&device->reduce_scratchpad,
                                         acVertexBufferCompdomainSizeBytes(device_config)));
    ERRCHK_CUDA_ALWAYS(hipMallocManaged(&device->reduce_result, sizeof(AcReal)));

#if PACKED_DATA_TRANSFERS
// Allocate data required for packed transfers here (hipMalloc)
#endif

    // Device constants
    acDeviceLoadMeshInfo(device, STREAM_DEFAULT, device_config);

    printf("Created device %d (%p)\n", device->id, device);
    *device_handle = device;

    // Autoptimize
    if (id == 0) {
        acDeviceAutoOptimize(device);
    }

    return AC_SUCCESS;
}

AcResult
acDeviceDestroy(Device device)
{
    hipSetDevice(device->id);
    printf("Destroying device %d (%p)\n", device->id, device);

    // Memory
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        hipFree(device->vba.in[i]);
        hipFree(device->vba.out[i]);
    }
    for (int i = 0; i < NUM_SCALARARRAY_HANDLES; ++i) {
        hipFree(device->vba.profiles[i]);
    }

    hipFree(device->reduce_scratchpad);
    hipFree(device->reduce_result);

#if PACKED_DATA_TRANSFERS
// Free data required for packed tranfers here (hipFree)
#endif

    // Concurrency
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamDestroy(device->streams[i]);
    }

    // Destroy Device
    free(device);
    return AC_SUCCESS;
}

AcResult
acDevicePrintInfo(const Device device)
{
    const int device_id = device->id;

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device_id);
    printf("--------------------------------------------------\n");
    printf("Device Number: %d\n", device_id);
    const size_t bus_id_max_len = 128;
    char bus_id[bus_id_max_len];
    hipDeviceGetPCIBusId(bus_id, bus_id_max_len, device_id);
    printf("  PCI bus ID: %s\n", bus_id);
    printf("    Device name: %s\n", props.name);
    printf("    Compute capability: %d.%d\n", props.major, props.minor);

    // Compute
    printf("  Compute\n");
    printf("    Clock rate (GHz): %g\n", props.clockRate / 1e6); // KHz -> GHz
    printf("    Stream processors: %d\n", props.multiProcessorCount);
    printf("    SP to DP flops performance ratio: %d:1\n", props.singleToDoublePrecisionPerfRatio);
    printf(
        "    Compute mode: %d\n",
        (int)props
            .computeMode); // https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1g7eb25f5413a962faad0956d92bae10d0
    // Memory
    printf("  Global memory\n");
    printf("    Memory Clock Rate (MHz): %d\n", props.memoryClockRate / (1000));
    printf("    Memory Bus Width (bits): %d\n", props.memoryBusWidth);
    printf("    Peak Memory Bandwidth (GiB/s): %f\n",
           2 * (props.memoryClockRate * 1e3) * props.memoryBusWidth / (8. * 1024. * 1024. * 1024.));
    printf("    ECC enabled: %d\n", props.ECCEnabled);

    // Memory usage
    size_t free_bytes, total_bytes;
    hipMemGetInfo(&free_bytes, &total_bytes);
    const size_t used_bytes = total_bytes - free_bytes;
    printf("    Total global mem: %.2f GiB\n", props.totalGlobalMem / (1024.0 * 1024 * 1024));
    printf("    Gmem used (GiB): %.2f\n", used_bytes / (1024.0 * 1024 * 1024));
    printf("    Gmem memory free (GiB): %.2f\n", free_bytes / (1024.0 * 1024 * 1024));
    printf("    Gmem memory total (GiB): %.2f\n", total_bytes / (1024.0 * 1024 * 1024));
    printf("  Caches\n");
    printf("    Local L1 cache supported: %d\n", props.localL1CacheSupported);
    printf("    Global L1 cache supported: %d\n", props.globalL1CacheSupported);
    printf("    L2 size: %d KiB\n", props.l2CacheSize / (1024));
    // MV: props.totalConstMem and props.sharedMemPerBlock cause assembler error
    // MV: while compiling in TIARA gp cluster. Therefore commeted out.
    //!!    printf("    Total const mem: %ld KiB\n", props.totalConstMem / (1024));
    //!!    printf("    Shared mem per block: %ld KiB\n", props.sharedMemPerBlock / (1024));
    printf("  Other\n");
    printf("    Warp size: %d\n", props.warpSize);
    // printf("    Single to double perf. ratio: %dx\n",
    // props.singleToDoublePrecisionPerfRatio); //Not supported with older CUDA
    // versions
    printf("    Stream priorities supported: %d\n", props.streamPrioritiesSupported);
    printf("--------------------------------------------------\n");

    return AC_SUCCESS;
}

AcResult
acDeviceAutoOptimize(const Device device)
{
    hipSetDevice(device->id);

    // RK3
    const int3 start = (int3){NGHOST, NGHOST, NGHOST};
    const int3 end   = start + (int3){device->local_config.int_params[AC_nx], //
                                    device->local_config.int_params[AC_ny], //
                                    device->local_config.int_params[AC_nz]};

    dim3 best_dims(0, 0, 0);
    float best_time          = INFINITY;
    const int num_iterations = 10;

    for (int z = 1; z <= MAX_THREADS_PER_BLOCK; ++z) {
        for (int y = 1; y <= MAX_THREADS_PER_BLOCK; ++y) {
            for (int x = WARP_SIZE; x <= MAX_THREADS_PER_BLOCK; x += WARP_SIZE) {

                if (x > end.x - start.x || y > end.y - start.y || z > end.z - start.z)
                    break;
                if (x * y * z > MAX_THREADS_PER_BLOCK)
                    break;

                if (x * y * z * REGISTERS_PER_THREAD > MAX_REGISTERS_PER_BLOCK)
                    break;

                if (((x * y * z) % WARP_SIZE) != 0)
                    continue;

                const dim3 tpb(x, y, z);
                const int3 n = end - start;
                const dim3 bpg((unsigned int)ceil(n.x / AcReal(tpb.x)), //
                               (unsigned int)ceil(n.y / AcReal(tpb.y)), //
                               (unsigned int)ceil(n.z / AcReal(tpb.z)));

                hipDeviceSynchronize();
                if (hipGetLastError() != hipSuccess) // resets the error if any
                    continue;

                // printf("(%d, %d, %d)\n", x, y, z);

                hipEvent_t tstart, tstop;
                hipEventCreate(&tstart);
                hipEventCreate(&tstop);

                // #ifdef AC_dt
                acDeviceLoadScalarUniform(device, STREAM_DEFAULT, AC_dt, FLT_EPSILON);
                /*#else
                                ERROR("FATAL ERROR: acDeviceAutoOptimize() or
                acDeviceIntegrateSubstep() was " "called, but AC_dt was not defined. Either define
                it or call the generated " "device function acDeviceKernel_<kernel name> which does
                not require the " "timestep to be defined.\n"); #endif*/

                hipEventRecord(tstart); // ---------------------------------------- Timing start
                for (int i = 0; i < num_iterations; ++i)
                    solve<2><<<bpg, tpb>>>(start, end, device->vba);

                hipEventRecord(tstop); // ----------------------------------------- Timing end
                hipEventSynchronize(tstop);
                float milliseconds = 0;
                hipEventElapsedTime(&milliseconds, tstart, tstop);

                ERRCHK_CUDA_KERNEL_ALWAYS();
                if (milliseconds < best_time) {
                    best_time = milliseconds;
                    best_dims = tpb;
                }
            }
        }
    }
#if VERBOSE_PRINTING
    printf(
        "Auto-optimization done. The best threadblock dimensions for rkStep: (%d, %d, %d) %f ms\n",
        best_dims.x, best_dims.y, best_dims.z, double(best_time) / num_iterations);
#endif
    /*
    FILE* fp = fopen("../config/rk3_tbdims.cuh", "w");
    ERRCHK(fp);
    fprintf(fp, "%d, %d, %d\n", best_dims.x, best_dims.y, best_dims.z);
    fclose(fp);
    */

    rk3_tpb = best_dims;
    return AC_SUCCESS;
}

AcResult
acDeviceSynchronizeStream(const Device device, const Stream stream)
{
    hipSetDevice(device->id);
    if (stream == STREAM_ALL) {
        hipDeviceSynchronize();
    }
    else {
        hipStreamSynchronize(device->streams[stream]);
    }
    return AC_SUCCESS;
}

AcResult
acDeviceSwapBuffers(const Device device)
{
    hipSetDevice(device->id);
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        AcReal* tmp        = device->vba.in[i];
        device->vba.in[i]  = device->vba.out[i];
        device->vba.out[i] = tmp;
    }
    return AC_SUCCESS;
}

AcResult
acDeviceLoadScalarUniform(const Device device, const Stream stream, const AcRealParam param,
                          const AcReal value)
{
    hipSetDevice(device->id);
    const size_t offset = (size_t)&d_mesh_info.real_params[param] - (size_t)&d_mesh_info;
    ERRCHK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset,
                                        hipMemcpyHostToDevice, device->streams[stream]));
    return AC_SUCCESS;
}

AcResult
acDeviceLoadVectorUniform(const Device device, const Stream stream, const AcReal3Param param,
                          const AcReal3 value)
{
    hipSetDevice(device->id);
    const size_t offset = (size_t)&d_mesh_info.real3_params[param] - (size_t)&d_mesh_info;
    ERRCHK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset,
                                        hipMemcpyHostToDevice, device->streams[stream]));
    return AC_SUCCESS;
}

AcResult
acDeviceLoadIntUniform(const Device device, const Stream stream, const AcIntParam param,
                       const int value)
{
    hipSetDevice(device->id);
    const size_t offset = (size_t)&d_mesh_info.int_params[param] - (size_t)&d_mesh_info;
    ERRCHK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset,
                                        hipMemcpyHostToDevice, device->streams[stream]));
    return AC_SUCCESS;
}

AcResult
acDeviceLoadInt3Uniform(const Device device, const Stream stream, const AcInt3Param param,
                        const int3 value)
{
    hipSetDevice(device->id);
    const size_t offset = (size_t)&d_mesh_info.int3_params[param] - (size_t)&d_mesh_info;
    ERRCHK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset,
                                        hipMemcpyHostToDevice, device->streams[stream]));
    return AC_SUCCESS;
}

AcResult
acDeviceLoadScalarArray(const Device device, const Stream stream, const ScalarArrayHandle handle,
                        const size_t start, const AcReal* data, const size_t num)
{
    hipSetDevice(device->id);

    ERRCHK((int)(start + num) <= max(device->local_config.int_params[AC_mx],
                                     max(device->local_config.int_params[AC_my],
                                         device->local_config.int_params[AC_mz])));

    ERRCHK_CUDA(hipMemcpyAsync(&device->vba.profiles[handle][start], data, sizeof(data[0]) * num,
                                hipMemcpyHostToDevice, device->streams[stream]));
    return AC_SUCCESS;
}

AcResult
acDeviceLoadMeshInfo(const Device device, const Stream stream, const AcMeshInfo device_config)
{
    hipSetDevice(device->id);

    ERRCHK_ALWAYS(device_config.int_params[AC_nx] == device->local_config.int_params[AC_nx]);
    ERRCHK_ALWAYS(device_config.int_params[AC_ny] == device->local_config.int_params[AC_ny]);
    ERRCHK_ALWAYS(device_config.int_params[AC_nz] == device->local_config.int_params[AC_nz]);
    ERRCHK_ALWAYS(device_config.int_params[AC_multigpu_offset] ==
                  device->local_config.int_params[AC_multigpu_offset]);

    ERRCHK_CUDA_ALWAYS(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mesh_info), &device_config, sizeof(device_config),
                                               0, hipMemcpyHostToDevice, device->streams[stream]));
    return AC_SUCCESS;
}

AcResult
acDeviceLoadVertexBufferWithOffset(const Device device, const Stream stream, const AcMesh host_mesh,
                                   const VertexBufferHandle vtxbuf_handle, const int3 src,
                                   const int3 dst, const int num_vertices)
{
    hipSetDevice(device->id);
    const size_t src_idx = acVertexBufferIdx(src.x, src.y, src.z, host_mesh.info);
    const size_t dst_idx = acVertexBufferIdx(dst.x, dst.y, dst.z, device->local_config);

    const AcReal* src_ptr = &host_mesh.vertex_buffer[vtxbuf_handle][src_idx];
    AcReal* dst_ptr       = &device->vba.in[vtxbuf_handle][dst_idx];
    const size_t bytes    = num_vertices * sizeof(src_ptr[0]);

    ERRCHK_CUDA(                                                                                  //
        hipMemcpyAsync(dst_ptr, src_ptr, bytes, hipMemcpyHostToDevice, device->streams[stream]) //
    );

    return AC_SUCCESS;
}

AcResult
acDeviceLoadMeshWithOffset(const Device device, const Stream stream, const AcMesh host_mesh,
                           const int3 src, const int3 dst, const int num_vertices)
{
    WARNING("This function is deprecated");
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        acDeviceLoadVertexBufferWithOffset(device, stream, host_mesh, (VertexBufferHandle)i, src,
                                           dst, num_vertices);
    }
    return AC_SUCCESS;
}

AcResult
acDeviceLoadVertexBuffer(const Device device, const Stream stream, const AcMesh host_mesh,
                         const VertexBufferHandle vtxbuf_handle)
{
    const int3 src            = (int3){0, 0, 0};
    const int3 dst            = src;
    const size_t num_vertices = acVertexBufferSize(device->local_config);
    acDeviceLoadVertexBufferWithOffset(device, stream, host_mesh, vtxbuf_handle, src, dst,
                                       num_vertices);

    return AC_SUCCESS;
}

AcResult
acDeviceLoadMesh(const Device device, const Stream stream, const AcMesh host_mesh)
{
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        acDeviceLoadVertexBuffer(device, stream, host_mesh, (VertexBufferHandle)i);
    }

    return AC_SUCCESS;
}

AcResult
acDeviceStoreVertexBufferWithOffset(const Device device, const Stream stream,
                                    const VertexBufferHandle vtxbuf_handle, const int3 src,
                                    const int3 dst, const int num_vertices, AcMesh* host_mesh)
{
    hipSetDevice(device->id);
    const size_t src_idx = acVertexBufferIdx(src.x, src.y, src.z, device->local_config);
    const size_t dst_idx = acVertexBufferIdx(dst.x, dst.y, dst.z, host_mesh->info);

    const AcReal* src_ptr = &device->vba.in[vtxbuf_handle][src_idx];
    AcReal* dst_ptr       = &host_mesh->vertex_buffer[vtxbuf_handle][dst_idx];
    const size_t bytes    = num_vertices * sizeof(src_ptr[0]);

    ERRCHK_CUDA(                                                                                  //
        hipMemcpyAsync(dst_ptr, src_ptr, bytes, hipMemcpyDeviceToHost, device->streams[stream]) //
    );

    return AC_SUCCESS;
}

AcResult
acDeviceStoreMeshWithOffset(const Device device, const Stream stream, const int3 src,
                            const int3 dst, const int num_vertices, AcMesh* host_mesh)
{
    WARNING("This function is deprecated");
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        acDeviceStoreVertexBufferWithOffset(device, stream, (VertexBufferHandle)i, src, dst,
                                            num_vertices, host_mesh);
    }

    return AC_SUCCESS;
}

AcResult
acDeviceStoreVertexBuffer(const Device device, const Stream stream,
                          const VertexBufferHandle vtxbuf_handle, AcMesh* host_mesh)
{
    int3 src                  = (int3){0, 0, 0};
    int3 dst                  = src;
    const size_t num_vertices = acVertexBufferSize(device->local_config);

    acDeviceStoreVertexBufferWithOffset(device, stream, vtxbuf_handle, src, dst, num_vertices,
                                        host_mesh);

    return AC_SUCCESS;
}

AcResult
acDeviceStoreMesh(const Device device, const Stream stream, AcMesh* host_mesh)
{
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        acDeviceStoreVertexBuffer(device, stream, (VertexBufferHandle)i, host_mesh);
    }

    return AC_SUCCESS;
}

AcResult
acDeviceTransferVertexBufferWithOffset(const Device src_device, const Stream stream,
                                       const VertexBufferHandle vtxbuf_handle, const int3 src,
                                       const int3 dst, const int num_vertices, Device dst_device)
{
    hipSetDevice(src_device->id);
    const size_t src_idx = acVertexBufferIdx(src.x, src.y, src.z, src_device->local_config);
    const size_t dst_idx = acVertexBufferIdx(dst.x, dst.y, dst.z, dst_device->local_config);

    const AcReal* src_ptr = &src_device->vba.in[vtxbuf_handle][src_idx];
    AcReal* dst_ptr       = &dst_device->vba.in[vtxbuf_handle][dst_idx];
    const size_t bytes    = num_vertices * sizeof(src_ptr[0]);

    ERRCHK_CUDA(hipMemcpyPeerAsync(dst_ptr, dst_device->id, src_ptr, src_device->id, bytes,
                                    src_device->streams[stream]));
    return AC_SUCCESS;
}

AcResult
acDeviceTransferMeshWithOffset(const Device src_device, const Stream stream, const int3 src,
                               const int3 dst, const int num_vertices, Device dst_device)
{
    WARNING("This function is deprecated");
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        acDeviceTransferVertexBufferWithOffset(src_device, stream, (VertexBufferHandle)i, src, dst,
                                               num_vertices, dst_device);
    }
    return AC_SUCCESS;
}

AcResult
acDeviceTransferVertexBuffer(const Device src_device, const Stream stream,
                             const VertexBufferHandle vtxbuf_handle, Device dst_device)
{
    int3 src                  = (int3){0, 0, 0};
    int3 dst                  = src;
    const size_t num_vertices = acVertexBufferSize(src_device->local_config);

    acDeviceTransferVertexBufferWithOffset(src_device, stream, vtxbuf_handle, src, dst,
                                           num_vertices, dst_device);
    return AC_SUCCESS;
}

AcResult
acDeviceTransferMesh(const Device src_device, const Stream stream, Device dst_device)
{
    WARNING("This function is deprecated");
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        acDeviceTransferVertexBuffer(src_device, stream, (VertexBufferHandle)i, dst_device);
    }
    return AC_SUCCESS;
}

AcResult
acDeviceIntegrateSubstep(const Device device, const Stream stream, const int step_number,
                         const int3 start, const int3 end, const AcReal dt)
{
    hipSetDevice(device->id);

    const dim3 tpb = rk3_tpb;

    const int3 n = end - start;
    const dim3 bpg((unsigned int)ceil(n.x / AcReal(tpb.x)), //
                   (unsigned int)ceil(n.y / AcReal(tpb.y)), //
                   (unsigned int)ceil(n.z / AcReal(tpb.z)));

    //#ifdef AC_dt
    acDeviceLoadScalarUniform(device, stream, AC_dt, dt);
    /*#else
        (void)dt;
        ERROR("FATAL ERROR: acDeviceAutoOptimize() or acDeviceIntegrateSubstep() was "
              "called, but AC_dt was not defined. Either define it or call the generated "
              "device function acDeviceKernel_<kernel name> which does not require the "
              "timestep to be defined.\n");
    #endif*/
    if (step_number == 0)
        solve<0><<<bpg, tpb, 0, device->streams[stream]>>>(start, end, device->vba);
    else if (step_number == 1)
        solve<1><<<bpg, tpb, 0, device->streams[stream]>>>(start, end, device->vba);
    else
        solve<2><<<bpg, tpb, 0, device->streams[stream]>>>(start, end, device->vba);

    ERRCHK_CUDA_KERNEL();

    return AC_SUCCESS;
}

AcResult
acDevicePeriodicBoundcondStep(const Device device, const Stream stream_type,
                              const VertexBufferHandle vtxbuf_handle, const int3 start,
                              const int3 end)
{
    hipSetDevice(device->id);
    const hipStream_t stream = device->streams[stream_type];

    const dim3 tpb(8, 2, 8);
    const dim3 bpg((unsigned int)ceil((end.x - start.x) / (float)tpb.x),
                   (unsigned int)ceil((end.y - start.y) / (float)tpb.y),
                   (unsigned int)ceil((end.z - start.z) / (float)tpb.z));

    kernel_periodic_boundconds<<<bpg, tpb, 0, stream>>>(start, end, device->vba.in[vtxbuf_handle]);
    ERRCHK_CUDA_KERNEL();

    return AC_SUCCESS;
}

AcResult
acDevicePeriodicBoundconds(const Device device, const Stream stream, const int3 start,
                           const int3 end)
{
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        acDevicePeriodicBoundcondStep(device, stream, (VertexBufferHandle)i, start, end);
    }
    return AC_SUCCESS;
}

AcResult
acDeviceReduceScal(const Device device, const Stream stream, const ReductionType rtype,
                   const VertexBufferHandle vtxbuf_handle, AcReal* result)
{
    hipSetDevice(device->id);

    const int3 start = (int3){device->local_config.int_params[AC_nx_min],
                              device->local_config.int_params[AC_ny_min],
                              device->local_config.int_params[AC_nz_min]};

    const int3 end = (int3){device->local_config.int_params[AC_nx_max],
                            device->local_config.int_params[AC_ny_max],
                            device->local_config.int_params[AC_nz_max]};

    *result = reduce_scal(device->streams[stream], rtype, start, end, device->vba.in[vtxbuf_handle],
                          device->reduce_scratchpad, device->reduce_result);
    return AC_SUCCESS;
}

AcResult
acDeviceReduceVec(const Device device, const Stream stream, const ReductionType rtype,
                  const VertexBufferHandle vtxbuf0, const VertexBufferHandle vtxbuf1,
                  const VertexBufferHandle vtxbuf2, AcReal* result)
{
    hipSetDevice(device->id);

    const int3 start = (int3){device->local_config.int_params[AC_nx_min],
                              device->local_config.int_params[AC_ny_min],
                              device->local_config.int_params[AC_nz_min]};

    const int3 end = (int3){device->local_config.int_params[AC_nx_max],
                            device->local_config.int_params[AC_ny_max],
                            device->local_config.int_params[AC_nz_max]};

    *result = reduce_vec(device->streams[stream], rtype, start, end, device->vba.in[vtxbuf0],
                         device->vba.in[vtxbuf1], device->vba.in[vtxbuf2],
                         device->reduce_scratchpad, device->reduce_result);
    return AC_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
// MPI tests
////////////////////////////////////////////////////////////////////////////////////////////////////
#if AC_MPI_ENABLED == 1
/**
    Running: mpirun -np <num processes> <executable>
*/
#include <mpi.h>

static void
acDeviceDistributeMeshMPI(const AcMesh src, AcMesh* dst)
{
    MPI_Barrier(MPI_COMM_WORLD);
    printf("Distributing mesh...\n");

    MPI_Datatype datatype = MPI_FLOAT;
    if (sizeof(AcReal) == 8)
        datatype = MPI_DOUBLE;

    int pid, num_processes;
    MPI_Comm_rank(MPI_COMM_WORLD, &pid);
    MPI_Comm_size(MPI_COMM_WORLD, &num_processes);

    const size_t count = acVertexBufferSize(dst->info);
    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {

        if (pid == 0) {
            // Communicate to self
            assert(dst);
            memcpy(&dst->vertex_buffer[i][0], //
                   &src.vertex_buffer[i][0],  //
                   count * sizeof(src.vertex_buffer[i][0]));

            // Communicate to others
            for (int j = 1; j < num_processes; ++j) {
                const size_t src_idx = acVertexBufferIdx(
                    0, 0, j * src.info.int_params[AC_nz] / num_processes, src.info);

                MPI_Send(&src.vertex_buffer[i][src_idx], count, datatype, j, 0, MPI_COMM_WORLD);
            }
        }
        else {
            assert(dst);

            // Recv
            const size_t dst_idx = 0;
            MPI_Status status;
            MPI_Recv(&dst->vertex_buffer[i][dst_idx], count, datatype, 0, 0, MPI_COMM_WORLD,
                     &status);
        }
    }
}

static void
acDeviceGatherMeshMPI(const AcMesh src, AcMesh* dst)
{
    MPI_Barrier(MPI_COMM_WORLD);
    printf("Gathering mesh...\n");
    MPI_Datatype datatype = MPI_FLOAT;
    if (sizeof(AcReal) == 8)
        datatype = MPI_DOUBLE;

    int pid, num_processes;
    MPI_Comm_rank(MPI_COMM_WORLD, &pid);
    MPI_Comm_size(MPI_COMM_WORLD, &num_processes);

    size_t count = acVertexBufferSize(src.info);

    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        // Communicate to self
        if (pid == 0) {
            assert(dst);
            memcpy(&dst->vertex_buffer[i][0], //
                   &src.vertex_buffer[i][0],  //
                   count * sizeof(src.vertex_buffer[i][0]));

            for (int j = 1; j < num_processes; ++j) {
                // Recv
                const size_t dst_idx = acVertexBufferIdx(
                    0, 0, j * dst->info.int_params[AC_nz] / num_processes, dst->info);

                assert(dst_idx + count <= acVertexBufferSize(dst->info));
                MPI_Status status;
                MPI_Recv(&dst->vertex_buffer[i][dst_idx], count, datatype, j, 0, MPI_COMM_WORLD,
                         &status);
            }
        }
        else {
            // Send
            const size_t src_idx = 0;

            assert(src_idx + count <= acVertexBufferSize(src.info));
            MPI_Send(&src.vertex_buffer[i][src_idx], count, datatype, 0, 0, MPI_COMM_WORLD);
        }
    }
}

/** NOTE: Assumes 1 process per GPU */
static AcResult
acDeviceCommunicateHalosMPI(const Device device)
{
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Datatype datatype = MPI_FLOAT;
    if (sizeof(AcReal) == 8)
        datatype = MPI_DOUBLE;

    int pid, num_processes;
    MPI_Comm_rank(MPI_COMM_WORLD, &pid);
    MPI_Comm_size(MPI_COMM_WORLD, &num_processes);

    const size_t count = device->local_config.int_params[AC_mx] *
                         device->local_config.int_params[AC_my] * NGHOST;

    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        { // Front
            // ...|ooooxxx|... -> xxx|ooooooo|...
            const size_t src_idx = acVertexBufferIdx(0, 0, device->local_config.int_params[AC_nz],
                                                     device->local_config);
            const size_t dst_idx = acVertexBufferIdx(0, 0, 0, device->local_config);
            const int send_pid   = (pid + 1) % num_processes;
            const int recv_pid   = (pid + num_processes - 1) % num_processes;

            MPI_Request request;
            MPI_Isend(&device->vba.in[i][src_idx], count, datatype, send_pid, i, MPI_COMM_WORLD,
                      &request);
            fflush(stdout);

            MPI_Status status;
            MPI_Recv(&device->vba.in[i][dst_idx], count, datatype, recv_pid, i, MPI_COMM_WORLD,
                     &status);

            MPI_Wait(&request, &status);
        }
        { // Back
            // ...|ooooooo|xxx <- ...|xxxoooo|...
            const size_t src_idx = acVertexBufferIdx(0, 0, NGHOST, device->local_config);
            const size_t dst_idx = acVertexBufferIdx(
                0, 0, NGHOST + device->local_config.int_params[AC_nz], device->local_config);
            const int send_pid = (pid + num_processes - 1) % num_processes;
            const int recv_pid = (pid + 1) % num_processes;

            MPI_Request request;
            MPI_Isend(&device->vba.in[i][src_idx], count, datatype, send_pid,
                      NUM_VTXBUF_HANDLES + i, MPI_COMM_WORLD, &request);

            MPI_Status status;
            MPI_Recv(&device->vba.in[i][dst_idx], count, datatype, recv_pid, NUM_VTXBUF_HANDLES + i,
                     MPI_COMM_WORLD, &status);

            MPI_Wait(&request, &status);
        }
    }
    return AC_SUCCESS;
}

static void
acHostCommunicateHalosMPI(AcMesh* submesh)
{
    MPI_Barrier(MPI_COMM_WORLD);
    printf("Communicating bounds...\n");
    MPI_Datatype datatype = MPI_FLOAT;
    if (sizeof(AcReal) == 8)
        datatype = MPI_DOUBLE;

    int pid, num_processes;
    MPI_Comm_rank(MPI_COMM_WORLD, &pid);
    MPI_Comm_size(MPI_COMM_WORLD, &num_processes);

    const size_t count = submesh->info.int_params[AC_mx] * submesh->info.int_params[AC_my] * NGHOST;

    for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
        { // Front
            // ...|ooooxxx|... -> xxx|ooooooo|...
            const size_t src_idx = acVertexBufferIdx(0, 0, submesh->info.int_params[AC_nz],
                                                     submesh->info);
            const size_t dst_idx = acVertexBufferIdx(0, 0, 0, submesh->info);
            const int send_pid   = (pid + 1) % num_processes;
            const int recv_pid   = (pid + num_processes - 1) % num_processes;

            MPI_Request request;
            MPI_Isend(&submesh->vertex_buffer[i][src_idx], count, datatype, send_pid, i,
                      MPI_COMM_WORLD, &request);
            fflush(stdout);

            MPI_Status status;
            MPI_Recv(&submesh->vertex_buffer[i][dst_idx], count, datatype, recv_pid, i,
                     MPI_COMM_WORLD, &status);

            MPI_Wait(&request, &status);
        }
        { // Back
            // ...|ooooooo|xxx <- ...|xxxoooo|...
            const size_t src_idx = acVertexBufferIdx(0, 0, NGHOST, submesh->info);
            const size_t dst_idx = acVertexBufferIdx(0, 0, NGHOST + submesh->info.int_params[AC_nz],
                                                     submesh->info);
            const int send_pid   = (pid + num_processes - 1) % num_processes;
            const int recv_pid   = (pid + 1) % num_processes;

            MPI_Request request;
            MPI_Isend(&submesh->vertex_buffer[i][src_idx], count, datatype, send_pid,
                      NUM_VTXBUF_HANDLES + i, MPI_COMM_WORLD, &request);

            MPI_Status status;
            MPI_Recv(&submesh->vertex_buffer[i][dst_idx], count, datatype, recv_pid,
                     NUM_VTXBUF_HANDLES + i, MPI_COMM_WORLD, &status);

            MPI_Wait(&request, &status);
        }
    }
}

// From Astaroth Utils
#include "src/utils/config_loader.h"
#include "src/utils/memory.h"
#include "src/utils/timer_hires.h"
#include "src/utils/verification.h"
// --smpiargs="-gpu"
AcResult
acDeviceRunMPITest(void)
{
    int num_processes, pid;
    MPI_Init(NULL, NULL);
    MPI_Comm_size(MPI_COMM_WORLD, &num_processes);
    MPI_Comm_rank(MPI_COMM_WORLD, &pid);

    char processor_name[MPI_MAX_PROCESSOR_NAME];
    int name_len;
    MPI_Get_processor_name(processor_name, &name_len);
    printf("Processor %s. Process %d of %d.\n", processor_name, pid, num_processes);

    // Check MPI support
    //// Borrowing start (from OpenMPI examples)
#if defined(MPIX_CUDA_AWARE_SUPPORT) && MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library has CUDA-aware support.\n", MPIX_CUDA_AWARE_SUPPORT);
#elif defined(MPIX_CUDA_AWARE_SUPPORT) && !MPIX_CUDA_AWARE_SUPPORT
    printf("This MPI library does not have CUDA-aware support.\n");
#else
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */

    printf("Run time check:\n");
#if defined(MPIX_CUDA_AWARE_SUPPORT)
    if (1 == MPIX_Query_cuda_support()) {
        printf("This MPI library has CUDA-aware support.\n");
    }
    else {
        printf("This MPI library does not have CUDA-aware support.\n");
    }
#else  /* !defined(MPIX_CUDA_AWARE_SUPPORT) */
    printf("This MPI library cannot determine if there is CUDA-aware support.\n");
#endif /* MPIX_CUDA_AWARE_SUPPORT */
       //////// Borrowing end

    // Create model and candidate meshes
    AcMeshInfo info;
    acLoadConfig(AC_DEFAULT_CONFIG, &info);

    const int nn           = 256;
    info.int_params[AC_nx] = info.int_params[AC_ny] = info.int_params[AC_nz] = nn;
    acUpdateConfig(&info);

    AcMesh model, candidate;

    // Master CPU
    if (pid == 0) {
        acMeshCreate(info, &model);
        acMeshCreate(info, &candidate);

        acMeshRandomize(&model);
        acMeshApplyPeriodicBounds(&model);
    }

    assert(info.int_params[AC_nz] % num_processes == 0);

    // Create submesh info
    AcMeshInfo submesh_info                    = info;
    const int submesh_nz                       = info.int_params[AC_nz] / num_processes;
    submesh_info.int_params[AC_nz]             = submesh_nz;
    submesh_info.int3_params[AC_global_grid_n] = (int3){
        info.int_params[AC_nx],
        info.int_params[AC_ny],
        info.int_params[AC_nz],
    };
    submesh_info.int3_params[AC_multigpu_offset] = (int3){0, 0, pid * submesh_nz};
    acUpdateConfig(&submesh_info);

    // Helper dims
    const int3 subgrid_m = (int3){
        submesh_info.int_params[AC_mx],
        submesh_info.int_params[AC_my],
        submesh_info.int_params[AC_mz],
    };

    // Create submesh
    AcMesh submesh;
    acMeshCreate(submesh_info, &submesh);

    acDeviceDistributeMeshMPI(model, &submesh);

    ////////////////////////////////////////////////////////////////////////////////////////////////
    Device device;
    acDeviceCreate(0, submesh_info, &device);
    acDeviceLoadMesh(device, STREAM_DEFAULT, submesh);

    ////////////////////////////// Timer start
    const int num_iters = 100;
    Timer total_time;
    timer_reset(&total_time);
    for (int i = 0; i < num_iters; ++i) {
        ///// Communication start
        {
            const int3 start = (int3){0, 0, NGHOST};
            const int3 end   = (int3){subgrid_m.x, subgrid_m.y, subgrid_m.z - NGHOST};
            acDevicePeriodicBoundconds(device, STREAM_DEFAULT, start, end);
        }
#if 1 // GPU-GPU if CUDA-aware MPI, otherwise managed CPU-GPU-GPU-CPU
        acDeviceSynchronizeStream(device, STREAM_DEFAULT);
        MPI_Barrier(MPI_COMM_WORLD);
        acDeviceCommunicateHalosMPI(
            device); // Includes periodic bounds at first and last ghost zone
        MPI_Barrier(MPI_COMM_WORLD);
#else // Explicit GPU-CPU-CPU-GPU
        acDeviceStoreMesh(device, STREAM_DEFAULT, &submesh);
        acHostCommunicateHalosMPI(&submesh);
        acDeviceLoadMesh(device, STREAM_DEFAULT, submesh);
#endif
        ///// Communication end
    }
    if (pid == 0) {
        const double ms_elapsed = timer_diff_nsec(total_time) / 1e6;
        printf("vertices: %d^3, iterations: %d\n", nn, num_iters);
        printf("Total time: %f ms\n", ms_elapsed);
    }
    ////////////////////////////// Timer end

    acDeviceStoreMesh(device, STREAM_DEFAULT, &submesh);
    acDeviceDestroy(device);
    ////////////////////////////////////////////////////////////////////////////////////////////////
    acDeviceGatherMeshMPI(submesh, &candidate);

    acMeshDestroy(&submesh);

    // Master CPU
    if (pid == 0) {
        acVerifyMesh(model, candidate);
        acMeshDestroy(&model);
        acMeshDestroy(&candidate);
    }

    MPI_Finalize();
    return AC_FAILURE;
}
#else
AcResult
acDeviceRunMPITest(void)
{
    WARNING("MPI was not enabled but acDeviceRunMPITest() was called");
    return AC_FAILURE;
}
#endif

#if PACKED_DATA_TRANSFERS
// Functions for calling packed data transfers
#endif
