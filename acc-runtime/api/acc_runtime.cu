#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2021, Johannes Pekkila, Miikka Vaisala.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/
#define AC_INSIDE_AC_LIBRARY 

#include "acc_runtime.h"
#include "../acc/string_vec.h"
typedef void (*Kernel)(const int3, const int3, VertexBufferArray vba);
#define AcReal3(x,y,z)   (AcReal3){x,y,z}
#define AcComplex(x,y)   (AcComplex){x,y}

static AcBool3 dimension_inactive{};
#include <math.h> 
#include <vector> // tbconfig

#include "errchk.h"
#include "math_utils.h"
#include <unordered_map>
#include <utility>
#include <sys/stat.h>

#if AC_USE_HIP
#include <hip/hip_runtime.h> // Needed in files that include kernels
#include <rocprim/rocprim.hpp>
#endif

#include "user_kernel_declarations.h"
#include "kernel_reduce_info.h"


#define USE_COMPRESSIBLE_MEMORY (0)

//TP: unfortunately cannot use color output since it might not be supported in each env
const bool useColor = false;

#define GREEN "\033[1;32m"
#define YELLOW "\033[1;33m"
#define RESET "\033[0m"

#define COLORIZE(symbol, color) (useColor ? color symbol RESET : symbol)


#include "acc/implementation.h"

static dim3 last_tpb = (dim3){0, 0, 0};
struct Int3Hash {
    std::size_t operator()(const int3& v) const {
        return std::hash<int>()(v.x) ^ std::hash<int>()(v.y) << 1 ^ std::hash<int>()(v.z) << 2;
    }
};
std::array<std::unordered_map<int3,int,Int3Hash>,NUM_KERNELS> reduce_offsets;
int kernel_running_reduce_offsets[NUM_KERNELS];

static int grid_pid = 0;
[[maybe_unused]] static int nprocs   = 0;
static AcMeasurementGatherFunc gather_func  = NULL;

#if AC_MPI_ENABLED
AcResult
acInitializeRuntimeMPI(const int _grid_pid,const int _nprocs, const AcMeasurementGatherFunc mpi_gather_func)
{
	grid_pid = _grid_pid;
	nprocs   = _nprocs;
	gather_func = mpi_gather_func;
	return AC_SUCCESS;
}
#endif

Volume
acKernelLaunchGetLastTPB(void)
{
  return to_volume(last_tpb);
}
int
acGetKernelReduceScratchPadSize(const AcKernel kernel)
{
	return kernel_running_reduce_offsets[(int)kernel];
}
int
acGetKernelReduceScratchPadMinSize()
{
	int res = 0; 
	for(int i = 0; i < NUM_KERNELS; ++i)
		res = (res < kernel_running_reduce_offsets[i]) ? kernel_running_reduce_offsets[i] : res;
	return res;
}
Volume
get_bpg(Volume dims, const Volume tpb)
{
  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING:             // Fallthrough
  case EXPLICIT_CACHING:             // Fallthrough
  case EXPLICIT_CACHING_3D_BLOCKING: // Fallthrough
  case EXPLICIT_CACHING_4D_BLOCKING: // Fallthrough
  case EXPLICIT_PINGPONG_txw:        // Fallthrough
  case EXPLICIT_PINGPONG_txy:        // Fallthrough
  case EXPLICIT_PINGPONG_txyblocked: // Fallthrough
  case EXPLICIT_PINGPONG_txyz:       // Fallthrough
  case EXPLICIT_ROLLING_PINGPONG: {
    return (Volume){
        (size_t)ceil(1. * dims.x / tpb.x),
        (size_t)ceil(1. * dims.y / tpb.y),
        (size_t)ceil(1. * dims.z / tpb.z),
    };
  }
  default: {
    ERROR("Invalid IMPLEMENTATION in get_bpg");
    return (Volume){0, 0, 0};
  }
  }
}

size_t
get_warp_size()
{
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  return props.warpSize;
}


bool
is_valid_configuration(const Volume dims, const Volume tpb, const AcKernel)
{
  const size_t warp_size = get_warp_size();
  const size_t xmax      = (size_t)(warp_size * ceil(1. * dims.x / warp_size));
  const size_t ymax      = (size_t)(warp_size * ceil(1. * dims.y / warp_size));
  const size_t zmax      = (size_t)(warp_size * ceil(1. * dims.z / warp_size));
  const bool too_large   = (tpb.x > xmax) || (tpb.y > ymax) || (tpb.z > zmax);

  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING: {

	    if (too_large)
	      return false;

	    return true;
	  }
	  case EXPLICIT_CACHING_4D_BLOCKING: // Fallthrough
	    if (tpb.z > 1) return false;
	    [[fallthrough]];
	  case EXPLICIT_CACHING: // Fallthrough
	  case EXPLICIT_CACHING_3D_BLOCKING: {

	    // For some reason does not work without this
	    // Probably because of break vs continue when fetching (some threads
	    // quit too early if the dims are not divisible)
	    return !(dims.x % tpb.x) && !(dims.y % tpb.y) && !(dims.z % tpb.z);
	  }
	  case EXPLICIT_PINGPONG_txw: {
	    return (tpb.y == 1) && (tpb.z == 1);
	  }
	  case EXPLICIT_PINGPONG_txy: {
	    return (tpb.z == 1);
	  }
	  case EXPLICIT_PINGPONG_txyblocked: {
	    return (tpb.z == 1);
	  }
	  case EXPLICIT_PINGPONG_txyz: {
	    return true;
	  }
	  case EXPLICIT_ROLLING_PINGPONG: {
	    // OK for every other rolling pingpong implementation
	    // return true;

	    // Required only when unrolling smem loads
	    // Ensures two unrolls is enough to fill the smem buffer
	    return (2 * tpb.x >= STENCIL_WIDTH - 1 + tpb.x) &&
		   (2 * tpb.y >= STENCIL_HEIGHT - 1 + tpb.y);
	  }
	  default: {
	    ERROR("Invalid IMPLEMENTATION in is_valid_configuration");
    return false;
  }
  }
}

size_t
get_smem(const Volume tpb, const size_t stencil_order,
         const size_t bytes_per_elem)
{
  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING: {
    return 0;
  }
  case EXPLICIT_CACHING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) * tpb.z *
           bytes_per_elem;
  }
  case EXPLICIT_CACHING_3D_BLOCKING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) *
           (tpb.z + stencil_order) * bytes_per_elem;
  }
  case EXPLICIT_CACHING_4D_BLOCKING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) * tpb.z *
           (NUM_FIELDS)*bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txw: {
    return 2 * (tpb.x + stencil_order) * NUM_FIELDS * bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txy: {
    return 2 * (tpb.x + stencil_order) * (tpb.y + stencil_order) *
           bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txyblocked: {
    const size_t block_size = 7;
    return 2 * (tpb.x + stencil_order) * (tpb.y + stencil_order) * block_size *
           bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txyz: {
    return 2 * (tpb.x + stencil_order) * (tpb.y + stencil_order) *
           (tpb.z + stencil_order) * bytes_per_elem;
  }
  case EXPLICIT_ROLLING_PINGPONG: {
    // tpbxy slices with halos
    // tpbz depth + 1 rolling cache slab
    return EXPLICIT_ROLLING_PINGPONG_BLOCKSIZE * (tpb.x + stencil_order) *
           (tpb.y + stencil_order) * (tpb.z + 1) * bytes_per_elem;
  }
  default: {
    ERROR("Invalid IMPLEMENTATION in get_smem");
    return (size_t)-1;
  }
  }
}

/*
// Device info (TODO GENERIC)
// Use the maximum available reg count per thread
#define REGISTERS_PER_THREAD (255)
#define MAX_REGISTERS_PER_BLOCK (65536)
#if AC_DOUBLE_PRECISION
#define MAX_THREADS_PER_BLOCK                                                  \
  (MAX_REGISTERS_PER_BLOCK / REGISTERS_PER_THREAD / 2)
#else
#define MAX_THREADS_PER_BLOCK (MAX_REGISTERS_PER_BLOCK / REGISTERS_PER_THREAD)
#endif
*/

__device__ __constant__ AcMeshInfoScalars d_mesh_info;
#include "dconst_arrays_decl.h"
//TP: We do this ugly macro because I want to keep the generated headers the same if we are compiling cpu analysis and for the actual gpu comp
#define DECLARE_GMEM_ARRAY(DATATYPE, DEFINE_NAME, ARR_NAME) __device__ __constant__ DATATYPE* AC_INTERNAL_gmem_##DEFINE_NAME##_arrays_##ARR_NAME 
#define DECLARE_CONST_DIMS_GMEM_ARRAY(DATATYPE, DEFINE_NAME, ARR_NAME, LEN) __device__ DATATYPE AC_INTERNAL_gmem_##DEFINE_NAME##_arrays_##ARR_NAME[LEN]
#include "gmem_arrays_decl.h"



//The macros above generate d arrays like these:

// Astaroth 2.0 backwards compatibility START
#define d_multigpu_offset (d_mesh_info.int3_params[AC_multigpu_offset])


#define DEVICE_INLINE __device__ __forceinline__
#include "dconst_decl.h"



#include "get_address.h"
#include "load_dconst_arrays.h"
#include "store_dconst_arrays.h"


#define DEVICE_VTXBUF_IDX(i, j, k)                                             \
  ((i) + (j)*VAL(AC_mlocal).x + (k)*VAL(AC_mlocal_products).xy)

__device__ int
LOCAL_COMPDOMAIN_IDX(const int3 coord)
{
  return (coord.x) + (coord.y) * VAL(AC_nlocal).x + (coord.z) * VAL(AC_nlocal_products).xy;
}

#define print printf                          // TODO is this a good idea?
// passes an array into a device function and then calls len (need to modify
// the compiler to always pass arrays to functions as references before
// re-enabling)

#include "random.cuh"

#define suppress_unused_warning(X) (void)X
#define longlong long long
#define size(arr) (int)(sizeof(arr) / sizeof(arr[0])) // Leads to bugs if the user
#include "user_kernels.h"
#undef size
#undef longlong


typedef struct {
  AcKernel kernel;
  int3 dims;
  dim3 tpb;
} TBConfig;

static std::vector<TBConfig> tbconfigs;


static TBConfig getOptimalTBConfig(const AcKernel kernel, const int3 dims, VertexBufferArray vba);

static __global__ void
flush_kernel(AcReal* arr, const size_t n, const AcReal value)
{
  const size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n)
    arr[idx] = value;
}
static __global__ void
flush_kernel_int(int* arr, const size_t n, const int value)
{
  const size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n)
    arr[idx] = value;
}
template <typename T>
T TO_CORRECT_ORDER(const T vol)
{
	return vol;
}
size_t TO_CORRECT_ORDER(const size_t size)
{
	return size;
}
#define KERNEL_LAUNCH(func,bgp,tpb,...) func<<<TO_CORRECT_ORDER(bpg),TO_CORRECT_ORDER(tpb),__VA_ARGS__>>>
AcResult
acKernelFlush(const hipStream_t stream, AcReal* arr, const size_t n,
              const AcReal value)
{
  ERRCHK_ALWAYS(arr);
  const size_t tpb = 256;
  const size_t bpg = (size_t)(ceil((double)n / tpb));
  KERNEL_LAUNCH(flush_kernel,bpg,tpb,0,stream)(arr,n,value);
  ERRCHK_CUDA_KERNEL_ALWAYS();
  return AC_SUCCESS;
}

AcResult
acKernelFlushInt(const hipStream_t stream, int* arr, const size_t n,
              const int value)
{
  const size_t tpb = 256;
  const size_t bpg = (size_t)(ceil((double)n / tpb));
  KERNEL_LAUNCH(flush_kernel_int,bpg,tpb,0,stream)(arr,n,value);
  ERRCHK_CUDA_KERNEL_ALWAYS();
  return AC_SUCCESS;
}

#if USE_COMPRESSIBLE_MEMORY
#include <hip/hip_runtime.h>

#define ERRCHK_CU_ALWAYS(x) ERRCHK_ALWAYS((x) == hipSuccess)

static hipError_t
mallocCompressible(void** addr, const size_t requested_bytes)
{
  hipDevice_t device;
  ERRCHK_ALWAYS(hipCtxGetDevice(&device) == hipSuccess);

  hipMemAllocationProp prop;
  memset(&prop, 0, sizeof(hipMemAllocationProp));
  prop.type                       = hipMemAllocationTypePinned;
  prop.location.type              = hipMemLocationTypeDevice;
  prop.location.id                = device;
  prop.allocFlags.compressionType = CU_MEM_ALLOCATION_COMP_GENERIC;

  size_t granularity;
  ERRCHK_CU_ALWAYS(hipMemGetAllocationGranularity(
      &granularity, &prop, hipMemAllocationGranularityMinimum));

  // Pad to align
  const size_t bytes = ((requested_bytes - 1) / granularity + 1) * granularity;

  hipDeviceptr_t dptr;
  ERRCHK_ALWAYS(hipMemAddressReserve(&dptr, bytes, 0, 0, 0) == hipSuccess);

  hipMemGenericAllocationHandle_t handle;
  ERRCHK_ALWAYS(hipMemCreate(&handle, bytes, &prop, 0) == hipSuccess)

  // Check if hipMemCreate was able to allocate compressible memory.
  hipMemAllocationProp alloc_prop;
  memset(&alloc_prop, 0, sizeof(hipMemAllocationProp));
  hipMemGetAllocationPropertiesFromHandle(&alloc_prop, handle);
  ERRCHK_ALWAYS(alloc_prop.allocFlags.compressionType ==
                CU_MEM_ALLOCATION_COMP_GENERIC);

  ERRCHK_ALWAYS(hipMemMap(dptr, bytes, 0, handle, 0) == hipSuccess);
  ERRCHK_ALWAYS(hipMemRelease(handle) == hipSuccess);

  hipMemAccessDesc accessDescriptor;
  accessDescriptor.location.id   = prop.location.id;
  accessDescriptor.location.type = prop.location.type;
  accessDescriptor.flags         = hipMemAccessFlagsProtReadWrite;

  ERRCHK_ALWAYS(hipMemSetAccess(dptr, bytes, &accessDescriptor, 1) ==
                hipSuccess);

  *addr = (void*)dptr;
  return hipSuccess;
}

static void
freeCompressible(void* ptr, const size_t requested_bytes)
{
  hipDevice_t device;
  ERRCHK_ALWAYS(hipCtxGetDevice(&device) == hipSuccess);

  hipMemAllocationProp prop;
  memset(&prop, 0, sizeof(hipMemAllocationProp));
  prop.type                       = hipMemAllocationTypePinned;
  prop.location.type              = hipMemLocationTypeDevice;
  prop.location.id                = device;
  prop.allocFlags.compressionType = CU_MEM_ALLOCATION_COMP_GENERIC;

  size_t granularity = 0;
  ERRCHK_ALWAYS(hipMemGetAllocationGranularity(
                    &granularity, &prop, hipMemAllocationGranularityMinimum) ==
                hipSuccess);
  const size_t bytes = ((requested_bytes - 1) / granularity + 1) * granularity;

  ERRCHK_ALWAYS(ptr);
  ERRCHK_ALWAYS(hipMemUnmap((hipDeviceptr_t)ptr, bytes) == hipSuccess);
  ERRCHK_ALWAYS(hipMemAddressFree((hipDeviceptr_t)ptr, bytes) == hipSuccess);
}
#endif

AcResult
acPBAReset(const hipStream_t stream, ProfileBufferArray* pba, const size3_t counts)
{
  // Set pba.in data to all-nan and pba.out to 0
  for (int i = 0; i < NUM_PROFILES; ++i) {
    acKernelFlush(stream, pba->in[i],  prof_count(Profile(i),counts), (AcReal)0);
    acKernelFlush(stream, pba->out[i], prof_count(Profile(i),counts), (AcReal)0);
  }
  return AC_SUCCESS;
}
void
device_malloc(void** dst, const int bytes)
{
 #if USE_COMPRESSIBLE_MEMORY 
    ERRCHK_CUDA_ALWAYS(mallocCompressible(dst, bytes));
 #else
    ERRCHK_CUDA_ALWAYS(hipMalloc(dst, bytes));
  #endif
  ERRCHK_ALWAYS(dst != NULL);
}
void
device_malloc(AcReal** dst, const int bytes)
{
	device_malloc((void**)dst,bytes);
}

template <typename T>
void
device_free(T** dst, const int bytes)
{
#if USE_COMPRESSIBLE_MEMORY
  freeCompressible(*dst, bytes);
#else
  hipFree(*dst);
  //used to silence unused warning
  (void)bytes;
#endif
  *dst = NULL;
}

ProfileBufferArray
acPBACreate(const size3_t counts)
{
  ProfileBufferArray pba{};
  pba.count = counts.z;
  for (int i = 0; i < NUM_PROFILES; ++i) {
    const size_t bytes = prof_size(Profile(i),counts)*sizeof(AcReal);
    device_malloc(&pba.in[i],  bytes);
    device_malloc(&pba.out[i], bytes);
  }

  acPBAReset(0, &pba, counts);
  hipDeviceSynchronize();
  return pba;
}

void
acPBADestroy(ProfileBufferArray* pba, const size3_t counts)
{
  for (int i = 0; i < NUM_PROFILES; ++i) {
    const size_t bytes = prof_size(Profile(i),counts)*sizeof(AcReal);
    device_free(&pba->in[i],  bytes);
    device_free(&pba->out[i], bytes);
    pba->in[i]  = NULL;
    pba->out[i] = NULL;
  }
  pba->count = 0;
}

AcResult
acVBAReset(const hipStream_t stream, VertexBufferArray* vba)
{
  const size_t count = vba->bytes / sizeof(vba->in[0][0]);

  for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
    ERRCHK_ALWAYS(vba->in[i]);
    ERRCHK_ALWAYS(vba->out[i]);
    acKernelFlush(stream, vba->in[i], count, (AcReal)0);
    acKernelFlush(stream, vba->out[i], count, (AcReal)0);
  }
  memset(&vba->kernel_input_params,0,sizeof(acKernelInputParams));
  // Note: should be moved out when refactoring VBA to KernelParameterArray
  acPBAReset(stream, &vba->profiles, (size3_t){vba->mx,vba->my,vba->mz});
  return AC_SUCCESS;
}


template <typename T>
void
device_malloc(T** dst, const int bytes)
{
 #if USE_COMPRESSIBLE_MEMORY 
    ERRCHK_CUDA_ALWAYS(mallocCompressible((void**)dst, bytes));
 #else
    ERRCHK_CUDA_ALWAYS(hipMalloc((void**)dst, bytes));
  #endif
}

#include "memcpy_to_gmem_arrays.h"

#include "memcpy_from_gmem_arrays.h"

template <typename P>
struct allocate_arrays
{
	void operator()(const AcMeshInfoParams& config) 
	{
		for(P array : get_params<P>())
		{
			if (config[array] != nullptr && !is_dconst(array) && is_alive(array) && !has_const_dims(array))
			{

#if AC_VERBOSE
				fprintf(stderr,"Allocating %s|%d\n",get_name(array),get_array_length(array,config.scalars));
				fflush(stderr);
#endif
				auto d_mem_ptr = get_empty_pointer(array);
			        device_malloc(((void**)&d_mem_ptr), sizeof(config[array][0])*get_array_length(array,config.scalars));
				memcpy_to_gmem_array(array,d_mem_ptr);
			}
		}
	}
};

size3_t
buffer_dims(const AcMeshInfoParams config)
{
	auto mm = config[AC_mlocal];
	return (size3_t){as_size_t(mm.x),as_size_t(mm.y),as_size_t(mm.z)};
}

VertexBufferArray
acVBACreate(const AcMeshInfoParams config)
{
  //TP: !HACK!
  //TP: Get active dimensions at the time VBA is created, works for now but should be moved somewhere else
  dimension_inactive = config[AC_dimension_inactive];
  const size3_t counts = buffer_dims(config);
  VertexBufferArray vba;
  size_t count = counts.x*counts.y*counts.z;
  size_t bytes = sizeof(vba.in[0][0]) * count;
  vba.bytes          = bytes;
  vba.mx             = counts.x;
  vba.my             = counts.y;
  vba.mz             = counts.z;
#if AC_ADJACENT_VERTEX_BUFFERS
  const size_t allbytes = bytes*NUM_VTXBUF_HANDLES;
  AcReal *allbuf_in, *allbuf_out;

  ERRCHK_CUDA_ALWAYS(hipMalloc((void**)&allbuf_in, allbytes));
  ERRCHK_CUDA_ALWAYS(hipMalloc((void**)&allbuf_out, allbytes));

  acKernelFlush(STREAM_DEFAULT,allbuf_in, count*NUM_VTXBUF_HANDLES, (AcReal)0.0);
  ERRCHK_CUDA_ALWAYS(hipMemset((void*)allbuf_out, 0, allbytes));

  vba.in[0]=allbuf_in; vba.out[0]=allbuf_out;
printf("i,vbas[0]= %p %p \n",vba.in[0],vba.out[0]);
  for (size_t i = 1; i < NUM_VTXBUF_HANDLES; ++i) {
    vba.in [i]=vba.in [i-1]+count;
    vba.out[i]=vba.out[i-1]+count;
printf("i,vbas[i]= %zu %p %p\n",i,vba.in[i],vba.out[i]);
  }
#else
  for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
    device_malloc((void**) &vba.in[i],bytes);
    //Auxiliary fields need only a single copy so out can point to in
    if (vtxbuf_is_auxiliary[i])
    {
      vba.out[i] = vba.in[i];
    }else{
      device_malloc((void**) &vba.out[i],bytes);
    }
  }
#endif
  //Allocate workbuffers
  for (int i = 0; i < NUM_WORK_BUFFERS; ++i)
    device_malloc((void**)&vba.w[i],bytes);


  AcArrayTypes::run<allocate_arrays>(config);

  // Note: should be moved out when refactoring VBA to KernelParameterArray
  vba.profiles = acPBACreate(counts);

  acVBAReset(0, &vba);
  hipDeviceSynchronize();
  return vba;
}

template <typename P>
struct update_arrays
{
	void operator()(const AcMeshInfoParams& config)
	{
		for(P array : get_params<P>())
		{
			if (is_dconst(array) || !is_alive(array) || has_const_dims(array)) continue;
			auto config_array = config[array];
			auto gmem_array   = get_empty_pointer(array);
			memcpy_from_gmem_array(array,gmem_array);
			size_t bytes = sizeof(config_array[0])*get_array_length(array,config.scalars);
			if (config_array == nullptr && gmem_array != nullptr) 
				device_free(&gmem_array,bytes);
			else if (config_array != nullptr && gmem_array  == nullptr) 
				device_malloc(&gmem_array,bytes);
			memcpy_to_gmem_array(array,gmem_array);
		}
	}
};
void
acUpdateArrays(const AcMeshInfoParams config)
{
  AcArrayTypes::run<update_arrays>(config);
}

template <typename P>
struct free_arrays
{
	void operator()(const AcMeshInfoParams& config)
	{
		for(P array: get_params<P>())
		{
			auto config_array = config[array];
			if (config_array == nullptr || is_dconst(array) || !is_alive(array) || has_const_dims(array)) continue;
			auto gmem_array = get_empty_pointer(array);
			memcpy_from_gmem_array(array,gmem_array);
			device_free(&gmem_array, get_array_length(array,config.scalars));
			memcpy_to_gmem_array(array,gmem_array);
		}
	}
};

void
acVBADestroy(VertexBufferArray* vba, const AcMeshInfoParams config)
{
  for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) { 
    //TP: if dead then not allocated and thus nothing to free
    device_free(&(vba->in[i]), vba->bytes);
    if (vtxbuf_is_auxiliary[i])
      vba->out[i] = NULL;
    else
      device_free(&(vba->out[i]), vba->bytes);
  }
  //Free workbuffers 
  for (int i = 0; i < NUM_WORK_BUFFERS; ++i) 
    device_free(&(vba->w[i]), vba->bytes);

  //Free arrays
  AcArrayTypes::run<free_arrays>(config);
  // Note: should be moved out when refactoring VBA to KernelParameterArray
  acPBADestroy(&vba->profiles,(size3_t){vba->mx,vba->my,vba->mz});
  vba->bytes = 0;
  vba->mx    = 0;
  vba->my    = 0;
  vba->mz    = 0;
}



int
get_num_of_reduce_output(const dim3 bpg, const dim3 tpb)
{
	const size_t warp_size = get_warp_size();
	const int num_of_warps_per_block = (tpb.x*tpb.y*tpb.z + warp_size-1)/warp_size;
	const int num_of_blocks = bpg.x*bpg.y*bpg.z;
	return num_of_warps_per_block*num_of_blocks;
}

AcResult
acLaunchKernel(AcKernel kernel, const hipStream_t stream, const int3 start,
               const int3 end, VertexBufferArray vba)
{
  const int3 n = end - start;

  const TBConfig tbconf = getOptimalTBConfig(kernel, n, vba);
  const dim3 tpb        = tbconf.tpb;
  const int3 dims       = tbconf.dims;
  const dim3 bpg        = to_dim3(get_bpg(to_volume(dims), to_volume(tpb)));

  const size_t smem = get_smem(to_volume(tpb), STENCIL_ORDER, sizeof(AcReal));
  if (reduce_offsets[kernel].find(start) == reduce_offsets[kernel].end())
  {
  	reduce_offsets[kernel][start] = kernel_running_reduce_offsets[kernel];
  	kernel_running_reduce_offsets[kernel] += get_num_of_reduce_output(bpg,tpb);
  }

  vba.reduce_offset = reduce_offsets[kernel][start];
  // hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferL1);
  KERNEL_LAUNCH(kernels[kernel],bpg,tpb,smem,stream)(start,end,vba);
  ERRCHK_CUDA_KERNEL();

  last_tpb = tpb; // Note: a bit hacky way to get the tpb
  return AC_SUCCESS;
}

AcResult
acBenchmarkKernel(AcKernel kernel, const int3 start, const int3 end,
                  VertexBufferArray vba)
{
  const int3 n = end - start;

  const TBConfig tbconf = getOptimalTBConfig(kernel, n, vba);
  const dim3 tpb        = tbconf.tpb;
  const int3 dims       = tbconf.dims;
  const dim3 bpg        = to_dim3(get_bpg(to_volume(dims), to_volume(tpb)));
  const size_t smem = get_smem(to_volume(tpb), STENCIL_ORDER, sizeof(AcReal));

  // Timer create
  hipEvent_t tstart, tstop;
  hipEventCreate(&tstart);
  hipEventCreate(&tstop);

  // Warmup
  hipEventRecord(tstart);
  KERNEL_LAUNCH(kernels[kernel],bpg, tpb, smem)(start, end, vba);
  hipEventRecord(tstop);
  hipEventSynchronize(tstop);
  ERRCHK_CUDA_KERNEL();
  hipDeviceSynchronize();

  // Benchmark
  hipEventRecord(tstart); // Timing start
  KERNEL_LAUNCH(kernels[kernel],bpg,tpb,smem)(start, end, vba);
  hipEventRecord(tstop); // Timing stop
  hipEventSynchronize(tstop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, tstart, tstop);
  printf("Kernel %s time elapsed: %g ms\n", kernel_names[kernel],(double)milliseconds);

  // Timer destroy
  hipEventDestroy(tstart);
  hipEventDestroy(tstop);

  last_tpb = tpb; // Note: a bit hacky way to get the tpb
  return AC_SUCCESS;
}


AcResult
acLoadStencil(const Stencil stencil, const hipStream_t /* stream */,
              const AcReal data[STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
  ERRCHK_ALWAYS(stencil < NUM_STENCILS);

  // Note important hipDeviceSynchronize below
  //
  // Constant memory allocated for stencils is shared among kernel
  // invocations, therefore a race condition is possible when updating
  // the coefficients. To avoid this, all kernels that can access
  // the coefficients must be completed before starting async copy to
  // constant memory
  hipDeviceSynchronize();

  const size_t bytes = sizeof(data[0][0][0]) * STENCIL_DEPTH * STENCIL_HEIGHT *
                       STENCIL_WIDTH;
  const hipError_t retval = hipMemcpyToSymbol(HIP_SYMBOL(
      stencils), data, bytes, stencil * bytes, hipMemcpyHostToDevice);

  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
};

AcResult
acLoadStencils(const hipStream_t stream,
               const AcReal data[NUM_STENCILS][STENCIL_DEPTH][STENCIL_HEIGHT]
                                [STENCIL_WIDTH])
{
  int retval = 0;
  for (size_t i = 0; i < NUM_STENCILS; ++i)
    retval |= acLoadStencil((Stencil)i, stream, data[i]);
  return (AcResult)retval;
}

AcResult
acStoreStencil(const Stencil stencil, const hipStream_t /* stream */,
               AcReal data[STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
  ERRCHK_ALWAYS(stencil < NUM_STENCILS);

  // Ensure all acLoadUniform calls have completed before continuing
  hipDeviceSynchronize();

  const size_t bytes = sizeof(data[0][0][0]) * STENCIL_DEPTH * STENCIL_HEIGHT *
                       STENCIL_WIDTH;
  const hipError_t retval = hipMemcpyFromSymbol(
      data, HIP_SYMBOL(stencils), bytes, stencil * bytes, hipMemcpyDeviceToHost);

  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
};


template <typename P, typename V>
static AcResult
acLoadUniform(const P param, const V value)
{
	if constexpr (std::is_same<P,AcReal>::value)
	{
  		if (isnan(value)) {
  		  fprintf(stderr,
  		          "WARNING: Passed an invalid value %g to device constant %s. "
  		          "Skipping.\n",
  		          (double)value, realparam_names[param]);
  		  return AC_FAILURE;
  		}
	}
	else if constexpr (std::is_same<P,AcReal3>::value)
	{
  		if (isnan(value.x) || isnan(value.y) || isnan(value.z)) {
  		  fprintf(stderr,
  		          "WARNING: Passed an invalid value (%g, %g, %g) to device constant "
  		          "%s. Skipping.\n",
  		          (double)value.x, (double)value.y, (double)value.z,
  		          real3param_names[param]);
  		  return AC_FAILURE;
  		}
	}
  	ERRCHK_ALWAYS(param < get_num_params<P>());
  	hipDeviceSynchronize(); /* See note in acLoadStencil */

  	const size_t offset =  get_address(param) - (size_t)&d_mesh_info;
  	const hipError_t retval = hipMemcpyToSymbol(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset, hipMemcpyHostToDevice);
  	return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
}



template <typename P, typename V>
static AcResult
acLoadArrayUniform(const P array, const V* values, const size_t length)
{
#if AC_VERBOSE
	fprintf(stderr,"Loading %s\n",get_name(array));
	fflush(stderr);
#endif
	hipDeviceSynchronize();
	ERRCHK_ALWAYS(values  != nullptr);
	const size_t bytes = length*sizeof(values[0]);
	if (!is_dconst(array))
	{
		if (!is_alive(array)) return AC_NOT_ALLOCATED;
		if (has_const_dims(array))
		{
			memcpy_to_const_dims_gmem_array(array,values);
			return AC_SUCCESS;
		}
		auto dst_ptr = get_empty_pointer(array);
		memcpy_from_gmem_array(array,dst_ptr);
		ERRCHK_ALWAYS(dst_ptr != nullptr);
		if (dst_ptr == nullptr)
		{
			fprintf(stderr,"FATAL AC ERROR from acLoadArrayUniform\n");
			exit(EXIT_FAILURE);
		}
#if AC_VERBOSE
		fprintf(stderr,"Calling (cuda/hip)memcpy %s|%ld\n",get_name(array),length);
		fflush(stderr);
#endif
		ERRCHK_CUDA_ALWAYS(hipMemcpy(dst_ptr,values,bytes,hipMemcpyHostToDevice));
	}
	else 
		ERRCHK_CUDA_ALWAYS(load_array(values, bytes, array));
#if AC_VERBOSE
	fprintf(stderr,"Loaded %s\n",get_name(array));
	fflush(stderr);
#endif
	return AC_SUCCESS;
}

template <typename P, typename V>
AcResult
acStoreUniform(const P param, V* value)
{
	ERRCHK_ALWAYS(param < get_num_params<P>());
	hipDeviceSynchronize();
  	const size_t offset =  get_address(param) - (size_t)&d_mesh_info;
	const hipError_t retval = hipMemcpyFromSymbol(value, HIP_SYMBOL(d_mesh_info), sizeof(V), offset, hipMemcpyDeviceToHost);
	return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
}

template <typename P, typename V>
AcResult
acStoreArrayUniform(const P array, V* values, const size_t length)
{
	ERRCHK_ALWAYS(values  != nullptr);
	const size_t bytes = length*sizeof(values[0]);
	if (!is_dconst(array))
	{
		if (!is_alive(array)) return AC_NOT_ALLOCATED;
		if (has_const_dims(array))
		{
			memcpy_from_gmem_array(array,values);
			return AC_SUCCESS;
		}
		auto src_ptr = get_empty_pointer(array);
		memcpy_from_gmem_array(array,src_ptr);
		ERRCHK_ALWAYS(src_ptr != nullptr);
		ERRCHK_CUDA_ALWAYS(hipMemcpy(values, src_ptr, bytes, hipMemcpyDeviceToHost));
	}
	else
		ERRCHK_CUDA_ALWAYS(store_array(values, bytes, array));
	return AC_SUCCESS;
}

#include "load_and_store_uniform_funcs.h"


//TP: best would be to use carriage return to have a single line that simple keeps growing but that seems not to be always supported in SLURM environments. 
// Or at least requires actions from the user
void printProgressBar(FILE* stream, const int progress) {
    int barWidth = 50;
    fprintf(stream,"[");  // Start a new line
    int pos = barWidth * progress / 100;

    for (int i = 0; i < barWidth; ++i) {
        if (i < pos) {
            fprintf(stream,COLORIZE("=",GREEN));  
        } else if (i == pos) {
            fprintf(stream,COLORIZE(">",YELLOW));  
        } else {
            fprintf(stream," ");
        }
    }
    fprintf(stream,"] %d %%", progress);
}
void
logAutotuningStatus(const size_t counter, const size_t num_samples, const AcKernel kernel)
{
    const size_t percent_of_num_samples = num_samples/100;
    for (size_t progress = 0; progress <= 100; ++progress)
    {
	      if (counter == percent_of_num_samples*progress  && grid_pid == 0 && (progress % 10 == 0))
	      {
    			fprintf(stderr,"\nAutotuning %s ",kernel_names[kernel]);
    			printProgressBar(stderr,progress);
			if (progress == 100) fprintf(stderr,"\n");
			fflush(stderr);
	      }
    }
}

static AcAutotuneMeasurement
gather_best_measurement(const AcAutotuneMeasurement local_best)
{
#if AC_MPI_ENABLED
	return gather_func(local_best);
#else
        return local_best;
#endif
}

void
make_vtxbuf_input_params_safe(VertexBufferArray& vba, const AcKernel kernel)
{
  //TP: have to set reduce offset zero since it might not be
  vba.reduce_offset = 0;
#include "safe_vtxbuf_input_params.h"
}

static TBConfig
autotune(const AcKernel kernel, const int3 dims, VertexBufferArray vba)
{
  make_vtxbuf_input_params_safe(vba,kernel);
  // printf("Autotuning kernel '%s' (%p), block (%d, %d, %d), implementation "
  //        "(%d):\n",
  //        kernel_names[id], kernel, dims.x, dims.y, dims.z, IMPLEMENTATION);
  // fflush(stdout);

#if 0
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  size_t size = min(int(prop.l2CacheSize * 0.75), prop.persistingL2CacheMaxSize);
  hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, size);
  // set-aside 3/4 of L2 cache for persisting accesses or the max allowed
#endif

  TBConfig c = {
      .kernel = kernel,
      .dims   = dims,
      .tpb    = (dim3){0, 0, 0},
  };

  const int3 ghosts = (int3){
	  dimension_inactive.x ? 0 : NGHOST,
	  dimension_inactive.y ? 0 : NGHOST,
	  dimension_inactive.z ? 0 : NGHOST
  };
  const int3 start = ghosts;
  const int3 end = start + dims;


  //TP: since autotuning should be quite fast when the dim is not NGHOST only log for actually 3d portions
  const bool builtin_kernel = strlen(kernel_names[kernel]) > 2 && kernel_names[kernel][0] == 'A' && kernel_names[kernel][1] == 'C';
  const bool large_launch = (dims.x > ghosts.x && dims.y > ghosts.y && dims.z > ghosts.z);
  const bool log = !builtin_kernel && large_launch;

  dim3 best_tpb(0, 0, 0);
  float best_time     = INFINITY;
  const int num_iters = 2;

  // Get device hardware information
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  const int max_threads_per_block = MAX_THREADS_PER_BLOCK
                                        ? min(props.maxThreadsPerBlock,
                                              MAX_THREADS_PER_BLOCK)
                                        : props.maxThreadsPerBlock;
  const size_t max_smem           = props.sharedMemPerBlock;

  // Old heuristic
  // for (int z = 1; z <= max_threads_per_block; ++z) {
  //   for (int y = 1; y <= max_threads_per_block; ++y) {
  //     for (int x = max(y, z); x <= max_threads_per_block; ++x) {

  // New: require that tpb.x is a multiple of the minimum transaction or L2
  // cache line size
  const int minimum_transaction_size_in_elems = 32 / sizeof(AcReal);
  // New: restrict tpb.x to be at most dims.x since launching threads that are known to be oob feels simply wasteful
  const int x_increment = min(
		  			minimum_transaction_size_in_elems,
		  			dims.x
		            );

  std::vector<int3> samples{};
  for (int z = 1; z <= min(max_threads_per_block,dims.z); ++z) {
    for (int y = 1; y <= min(max_threads_per_block,dims.y); ++y) {
      for (int x = x_increment;
           x <= min(max_threads_per_block,dims.x); x += x_increment) {


        if (x * y * z > max_threads_per_block)
          break;
        const dim3 tpb(x, y, z);
        const size_t smem = get_smem(to_volume(tpb), STENCIL_ORDER,
                                     sizeof(AcReal));

        if (smem > max_smem)
          continue;

        if ((x * y * z) % props.warpSize && (x*y*z) >props.warpSize)
          continue;

        if (!is_valid_configuration(to_volume(dims), to_volume(tpb),kernel))
          continue;
	//TP: should be emplace back but on my laptop the CUDA compiler gives a cryptic error message that I do not care to debug
        samples.push_back((int3){x,y,z});
      }
    }
  }
  size_t counter  = 0;
  size_t start_samples{};
  size_t end_samples{};
  if(large_launch && AC_MPI_ENABLED)
  {
  	const size_t portion = (size_t)ceil((1.0*samples.size())/nprocs);
  	start_samples = portion*grid_pid;
  	end_samples   = min(samples.size(), portion*(grid_pid+1));
  }
  else
  {
  	start_samples = 0;
  	end_samples   = samples.size();
  }
  const size_t n_samples = end_samples-start_samples;
  const Kernel func = kernels[kernel];
  for(size_t sample  = start_samples; sample < end_samples; ++sample)
  {
        if (log) logAutotuningStatus(counter,n_samples,kernel);
        ++counter;
        auto x = samples[sample].x;
        auto y = samples[sample].y;
        auto z = samples[sample].z;
        const dim3 tpb(x, y, z);
        const dim3 bpg    = to_dim3(
                                get_bpg(to_volume(dims),
                                to_volume(tpb))
                                );
        const size_t smem = get_smem(to_volume(tpb), STENCIL_ORDER,
                                     sizeof(AcReal));

        hipEvent_t tstart, tstop;
        hipEventCreate(&tstart);
        hipEventCreate(&tstop);

        KERNEL_LAUNCH(func,bpg, tpb, smem)(start, end, vba); // Dryrun
        hipDeviceSynchronize();
        hipEventRecord(tstart); // Timing start
        for (int i = 0; i < num_iters; ++i)
          KERNEL_LAUNCH(func,bpg, tpb, smem)(start, end, vba); // Dryrun
        hipEventRecord(tstop); // Timing stop
        hipEventSynchronize(tstop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, tstart, tstop);

        hipEventDestroy(tstart);
        hipEventDestroy(tstop);

        // Discard failed runs (attempt to clear the error to hipSuccess)
        if (hipGetLastError() != hipSuccess) {
	  //TP: reset autotune results
          FILE* fp = fopen(autotune_csv_path,"w");
	  fclose(fp);
          // Exit in case of unrecoverable error that needs a device reset
          ERRCHK_CUDA_KERNEL_ALWAYS();
          ERRCHK_CUDA_ALWAYS(hipGetLastError());
          continue;
        }

        if (milliseconds < best_time) {
          best_time = milliseconds;
          best_tpb  = tpb;
        }

        // printf("Auto-optimizing... Current tpb: (%d, %d, %d), time %f ms\n",
        //        tpb.x, tpb.y, tpb.z, (double)milliseconds / num_iters);
        // fflush(stdout);
  }
  const AcAutotuneMeasurement best_measurement = 
	  large_launch ? gather_best_measurement({best_time,best_tpb}) : (AcAutotuneMeasurement){best_time,best_tpb};
  c.tpb = best_measurement.tpb;
  best_time = best_measurement.time;
  if(grid_pid == 0)
  {
        FILE* fp = fopen(autotune_csv_path, "a");
        ERRCHK_ALWAYS(fp);
#if IMPLEMENTATION == SMEM_HIGH_OCCUPANCY_CT_CONST_TB
        fprintf(fp, "%d, (%d, %d, %d), (%d, %d, %d), %g\n", IMPLEMENTATION, nx, ny,
                nz, best_tpb.x, best_tpb.y, best_tpb.z,
                (double)best_time / num_iters);
#else
        fprintf(fp, "%d, %d, %d, %d, %d, %d, %d, %d, %g, %s\n", IMPLEMENTATION, kernel, dims.x,
                dims.y, dims.z, best_tpb.x, best_tpb.y, best_tpb.z,
                (double)best_time / num_iters, kernel_names[kernel]);
#endif
        fclose(fp);
  }
  if (c.tpb.x * c.tpb.y * c.tpb.z <= 0) {
    fprintf(stderr,
            "Fatal error: failed to find valid thread block dimensions for (%d,%d,%d) launch.\n"
            ,dims.x,dims.y,dims.z);
  }
  ERRCHK_ALWAYS(c.tpb.x * c.tpb.y * c.tpb.z > 0);
  return c;
}

static bool
file_exists(const char* filename)
{
  struct stat   buffer;
  return (stat (filename, &buffer) == 0);
}
static int3
read_optim_tpb(const AcKernel kernel, const int3 dims)
{
  if(!file_exists(autotune_csv_path)) return {-1,-1,-1};
  const char* filename = autotune_csv_path;
  FILE *file = fopen ( filename, "r" );
  int3 res = {-1,-1,-1};
  double best_time     = (double)INFINITY;
  string_vec entries[1000];
  memset(entries,0,sizeof(string_vec)*1000);
  const int n_entries = get_csv_entries(entries,file);
  for(int i = 0; i < n_entries; ++i)
  {
	  string_vec entry = entries[i];
	  if(entry.size > 1)
      	  {
      	     int kernel_index  = atoi(entry.data[1]);
      	     int3 read_dims = {atoi(entry.data[2]), atoi(entry.data[3]), atoi(entry.data[4])};
      	     int3 tpb = {atoi(entry.data[5]), atoi(entry.data[6]), atoi(entry.data[7])};
      	     double time = atof(entry.data[8]);
      	     if(time < best_time && kernel_index == kernel && read_dims == dims)
      	     {
      	    	 best_time = time;
      	    	 res       = tpb;
      	     }
      	  }
      	  for(size_t elem = 0; elem < entry.size; ++elem)
      	         free((char*)entry.data[elem]);
      	  free_str_vec(&entry);
  }
  fclose(file);
  return res;
}


static TBConfig
getOptimalTBConfig(const AcKernel kernel, const int3 dims, VertexBufferArray vba)
{
  for (auto c : tbconfigs)
    if (c.kernel == kernel && c.dims == dims)
      return c;

  const int3 read_tpb = read_optim_tpb(kernel,dims);
  TBConfig c  = (read_tpb != (int3){-1,-1,-1})
          ? (TBConfig){kernel,dims,(dim3){(uint32_t)read_tpb.x, (uint32_t)read_tpb.y, (uint32_t)read_tpb.z}}
          : autotune(kernel,dims,vba);
  tbconfigs.push_back(c);
  return c;
}

AcKernel
acGetOptimizedKernel(const AcKernel kernel_enum, const VertexBufferArray vba)
{
	//#include "user_kernel_ifs.h"
	//silence unused warnings
	(void)vba;
	//TP: for now this is no-op in the future in some cases we choose which kernel to call based on the input params
	return kernel_enum;
	//return kernels[(int) kernel_enum];
}
void
acVBASwapBuffer(const Field field, VertexBufferArray* vba)
{
  AcReal* tmp     = vba->in[field];
  vba->in[field]  = vba->out[field];
  vba->out[field] = tmp;
}

void
acVBASwapBuffers(VertexBufferArray* vba)
{
  for (size_t i = 0; i < NUM_FIELDS; ++i)
    acVBASwapBuffer((Field)i, vba);
}

void
acPBASwapBuffer(const Profile profile, VertexBufferArray* vba)
{
  AcReal* tmp                = vba->profiles.in[profile];
  vba->profiles.in[profile]  = vba->profiles.out[profile];
  vba->profiles.out[profile] = tmp;
}

void
acPBASwapBuffers(VertexBufferArray* vba)
{
  for (int i = 0; i < NUM_PROFILES; ++i)
    acPBASwapBuffer((Profile)i, vba);
}

AcResult
acLoadMeshInfo(const AcMeshInfoScalars info, const hipStream_t stream)
{
  for (int i = 0; i < NUM_INT_PARAMS; ++i)
    ERRCHK_ALWAYS(acLoadIntUniform(stream, (AcIntParam)i, info.int_params[i]) ==
                  AC_SUCCESS);

  for (int i = 0; i < NUM_INT3_PARAMS; ++i)
    ERRCHK_ALWAYS(acLoadInt3Uniform(stream, (AcInt3Param)i,
                                    info.int3_params[i]) == AC_SUCCESS);

  for (int i = 0; i < NUM_REAL_PARAMS; ++i)
    ERRCHK_ALWAYS(acLoadRealUniform(stream, (AcRealParam)i,
                                    info.real_params[i]) == AC_SUCCESS);

  for (int i = 0; i < NUM_REAL3_PARAMS; ++i)
    ERRCHK_ALWAYS(acLoadReal3Uniform(stream, (AcReal3Param)i,
                                     info.real3_params[i]) == AC_SUCCESS);

  return AC_SUCCESS;
}

//---------------
// static __host__ __device__ constexpr size_t
// acShapeSize(const AcShape& shape)
size_t
acShapeSize(const AcShape shape)
{
  return shape.x * shape.y * shape.z * shape.w;
}

static __host__ __device__ constexpr bool
acOutOfBounds(const AcIndex& index, const AcShape& shape)
{
  return (index.x >= shape.x) || //
         (index.y >= shape.y) || //
         (index.z >= shape.z) || //
         (index.w >= shape.w);
}

static __host__ __device__ constexpr AcIndex __attribute__((unused))
min(const AcIndex& a, const AcIndex& b)
{
  return (AcIndex){
      a.x < b.x ? a.x : b.x,
      a.y < b.y ? a.y : b.y,
      a.z < b.z ? a.z : b.z,
      a.w < b.w ? a.w : b.w,
  };
}

static __host__ __device__ constexpr AcIndex
operator+(const AcIndex& a, const AcIndex& b)
{
  return (AcIndex){
      a.x + b.x,
      a.y + b.y,
      a.z + b.z,
      a.w + b.w,
  };
}

static __host__ __device__ constexpr AcIndex __attribute__((unused))
operator-(const AcIndex& a, const AcIndex& b) 
{
  return (AcIndex){
      a.x - b.x,
      a.y - b.y,
      a.z - b.z,
      a.w - b.w,
  };
}

static __host__ __device__ constexpr AcIndex
to_spatial(const size_t i, const AcShape& shape)
{
  return (AcIndex){
      .x = i % shape.x,
      .y = (i / shape.x) % shape.y,
      .z = (i / (shape.x * shape.y)) % shape.z,
      .w = i / (shape.x * shape.y * shape.z),
  };
}

static __host__ __device__ constexpr size_t
to_linear(const AcIndex& index, const AcShape& shape)
{
  return index.x +           //
         index.y * shape.x + //
         index.z * shape.x * shape.y + index.w * shape.x * shape.y * shape.z;
}

static __global__ void
reindex(const AcReal* in, const AcIndex in_offset, const AcShape in_shape,
        AcReal* out, const AcIndex out_offset, const AcShape out_shape,
        const AcShape block_shape)
{
  const size_t i    = (size_t)threadIdx.x + blockIdx.x * blockDim.x;
  const AcIndex idx = to_spatial(i, block_shape);

  const AcIndex in_pos  = idx + in_offset;
  const AcIndex out_pos = idx + out_offset;

  if (acOutOfBounds(idx, block_shape) || //
      acOutOfBounds(in_pos, in_shape) || //
      acOutOfBounds(out_pos, out_shape))
    return;

  const size_t in_idx  = to_linear(in_pos, in_shape);
  const size_t out_idx = to_linear(out_pos, out_shape);

  out[out_idx] = in[in_idx];
}

AcResult
acReindex(const hipStream_t stream, //
          const AcReal* in, const AcIndex in_offset, const AcShape in_shape,
          AcReal* out, const AcIndex out_offset, const AcShape out_shape,
          const AcShape block_shape)
{
  const size_t count = acShapeSize(block_shape);
  const size_t tpb   = min(256ul, count);
  const size_t bpg   = (count + tpb - 1) / tpb;

  KERNEL_LAUNCH(reindex,bpg, tpb, 0, stream)(in, in_offset, in_shape, //
                                   out, out_offset, out_shape, block_shape);
  ERRCHK_CUDA_KERNEL();

  return AC_SUCCESS;
}

typedef struct {
  AcReal *x, *y, *z;
} SOAVector;

typedef struct {
  // Input vectors
  SOAVector A[1];
  size_t A_count;
  SOAVector B[4];
  size_t B_count;
  // Note: more efficient with A_count < B_count

  // Output vectors
  SOAVector C[1 * 4];
  // C count = A_count*B_count
} CrossProductArrays;

static __global__ void UNUSED
reindex_cross(const CrossProductArrays arrays, const AcIndex in_offset,
              const AcShape in_shape, const AcIndex out_offset,
              const AcShape out_shape, const AcShape block_shape)
{
  const AcIndex idx = to_spatial((size_t)threadIdx.x + blockIdx.x * blockDim.x
		  , block_shape);

  const AcIndex in_pos  = idx + in_offset;
  const AcIndex out_pos = idx + out_offset;

  if (acOutOfBounds(idx, block_shape) || //
      acOutOfBounds(in_pos, in_shape) || //
      acOutOfBounds(out_pos, out_shape))
    return;

  const size_t in_idx  = to_linear(in_pos, in_shape);
  const size_t out_idx = to_linear(out_pos, out_shape);

  for (size_t j = 0; j < arrays.A_count; ++j) {
    const AcReal3 a = {
        arrays.A[j].x[in_idx],
        arrays.A[j].y[in_idx],
        arrays.A[j].z[in_idx],
    };
    for (size_t i = 0; i < arrays.B_count; ++i) {
      const AcReal3 b = {
          arrays.B[i].x[in_idx],
          arrays.B[i].y[in_idx],
          arrays.B[i].z[in_idx],
      };
      const AcReal3 res                           = AC_cross(a, b);
      arrays.C[i + j * arrays.B_count].x[out_idx] = res.x;
      arrays.C[i + j * arrays.B_count].y[out_idx] = res.y;
      arrays.C[i + j * arrays.B_count].z[out_idx] = res.z;
    }
  }
}

#if 0
__global__ void
map_cross_product(const CrossProductInputs inputs, const AcIndex start,
                  const AcIndex end)
{

  const AcIndex tid = {
      .x = threadIdx.x + blockIdx.x * blockDim.x,
      .y = threadIdx.y + blockIdx.y * blockDim.y,
      .z = threadIdx.z + blockIdx.z * blockDim.z,
      .w = 0,
  };

  const AcIndex in_idx3d = start + tid;
  const size_t in_idx = DEVICE_VTXBUF_IDX(in_idx3d.x, in_idx3d.y, in_idx3d.z);

  const AcShape dims   = end - start;
  const size_t out_idx = tid.x + tid.y * dims.x + tid.z * dims.x * dims.y;

  const bool within_bounds = in_idx3d.x < end.x && in_idx3d.y < end.y &&
                             in_idx3d.z < end.z;
  if (within_bounds) {
    for (size_t i = 0; i < inputs.A_count; ++i) {
      const AcReal3 a = (AcReal3){
          inputs.A[i].x[in_idx],
          inputs.A[i].y[in_idx],
          inputs.A[i].z[in_idx],
      };
      for (size_t j = 0; j < inputs.B_count; ++j) {
        const AcReal3 b = (AcReal3){
            inputs.B[j].x[in_idx],
            inputs.B[j].y[in_idx],
            inputs.B[j].z[in_idx],
        };
        const AcReal3 res            = cross(a, b);
        inputs.outputs[j].x[out_idx] = res.x;
        inputs.outputs[j].y[out_idx] = res.y;
        inputs.outputs[j].z[out_idx] = res.z;
      }
    }
  }
}
#endif

#ifdef AC_TFM_ENABLED
AcResult
acReindexCross(const hipStream_t stream, //
               const VertexBufferArray vba, const AcIndex in_offset,
               const AcShape in_shape, //
               AcReal* out, const AcIndex out_offset, const AcShape out_shape,
               const AcShape block_shape)
{
  const SOAVector uu = {
      .x = vba.in[VTXBUF_UUX],
      .y = vba.in[VTXBUF_UUY],
      .z = vba.in[VTXBUF_UUZ],
  };
  const SOAVector bb11 = {
      .x = vba.in[TF_b11_x],
      .y = vba.in[TF_b11_y],
      .z = vba.in[TF_b11_z],
  };
  const SOAVector bb12 = {
      .x = vba.in[TF_b12_x],
      .y = vba.in[TF_b12_y],
      .z = vba.in[TF_b12_z],
  };
  const SOAVector bb21 = {
      .x = vba.in[TF_b21_x],
      .y = vba.in[TF_b21_y],
      .z = vba.in[TF_b21_z],
  };
  const SOAVector bb22 = {
      .x = vba.in[TF_b22_x],
      .y = vba.in[TF_b22_y],
      .z = vba.in[TF_b22_z],
  };

  const size_t block_offset = out_shape.x * out_shape.y * out_shape.z;
  const SOAVector out_bb11  = {
       .x = &out[3 * block_offset],
       .y = &out[4 * block_offset],
       .z = &out[5 * block_offset],
  };
  const SOAVector out_bb12 = {
      .x = &out[6 * block_offset],
      .y = &out[7 * block_offset],
      .z = &out[8 * block_offset],
  };
  const SOAVector out_bb21 = {
      .x = &out[9 * block_offset],
      .y = &out[10 * block_offset],
      .z = &out[11 * block_offset],
  };
  const SOAVector out_bb22 = {
      .x = &out[12 * block_offset],
      .y = &out[13 * block_offset],
      .z = &out[14 * block_offset],
  };

  const CrossProductArrays arrays = {
      .A       = {uu},
      .A_count = 1,
      .B       = {bb11, bb12, bb21, bb22},
      .B_count = 4,
      .C       = {out_bb11, out_bb12, out_bb21, out_bb22},
  };

  const size_t count = acShapeSize(block_shape);
  const size_t tpb   = min(256ul, count);
  const size_t bpg   = (count + tpb - 1) / tpb;

  KERNEL_LAUNCH(reindex_cross,bpg, tpb, 0, stream)(arrays, in_offset, in_shape,
                                         out_offset, out_shape, block_shape);
  return AC_SUCCESS;
}
#else
AcResult
acReindexCross(const hipStream_t , //
               const VertexBufferArray , const AcIndex ,
               const AcShape , //
               AcReal* , const AcIndex , const AcShape ,
               const AcShape )
{
  ERROR("acReindexCross called but AC_TFM_ENABLED was false");
  return AC_FAILURE;
}
#endif

#if AC_USE_HIP
#include <hipcub/hipcub.hpp>
#define cub hipcub
#else
#include <hipcub/hipcub.hpp>
#endif

AcResult
acSegmentedReduce(const hipStream_t stream, const AcReal* d_in,
                  const size_t count, const size_t num_segments, AcReal* d_out)
{
  size_t* offsets = (size_t*)malloc(sizeof(offsets[0]) * (num_segments + 1));
  ERRCHK_ALWAYS(offsets);
  for (size_t i = 0; i <= num_segments; ++i) {
    offsets[i] = i * (count / num_segments);
  }

  size_t* d_offsets = NULL;
  hipMalloc(&d_offsets, sizeof(d_offsets[0]) * (num_segments + 1));
  ERRCHK_ALWAYS(d_offsets);
  hipMemcpy(d_offsets, offsets, sizeof(d_offsets[0]) * (num_segments + 1),
             hipMemcpyHostToDevice);

  void* d_temp_storage      = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, d_in,
                                  d_out, num_segments, d_offsets, d_offsets + 1,
                                  stream);
  // printf("Temp storage: %zu bytes\n", temp_storage_bytes);
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  ERRCHK_ALWAYS(d_temp_storage);

  hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, d_in,
                            d_out, num_segments, d_offsets, d_offsets + 1,
                            stream);

  hipStreamSynchronize(
      stream); // Note, would not be needed if allocated at initialization
  hipFree(d_temp_storage);
  hipFree(d_offsets);
  free(offsets);
  return AC_SUCCESS;
}
typedef struct
{
	void* data;
	size_t bytes;
} AcDeviceTmpBuffer;
template <typename T>
void
cub_reduce(AcDeviceTmpBuffer& temp_storage, const hipStream_t stream, const T* d_in, const size_t count, T* d_out,  AcReduceOp reduce_op)
{
  switch(reduce_op)
  {
	  case(REDUCE_SUM):
	  	hipcub::DeviceReduce::Sum(temp_storage.data, temp_storage.bytes, d_in, d_out, count,stream);
	  	break;
	  case(REDUCE_MIN):
	  	hipcub::DeviceReduce::Min(temp_storage.data, temp_storage.bytes, d_in, d_out, count,stream);
	  	break;
	  case(REDUCE_MAX):
	  	hipcub::DeviceReduce::Max(temp_storage.data, temp_storage.bytes, d_in, d_out, count,stream);
	  	break;
	default:
		ERRCHK_ALWAYS(reduce_op != NO_REDUCE);
  }
  if (hipGetLastError() != hipSuccess) {
          ERRCHK_CUDA_KERNEL_ALWAYS();
          ERRCHK_CUDA_ALWAYS(hipGetLastError());
  }
}
template <typename T>
AcResult
acReduceBase(const hipStream_t stream, const T* d_in, const size_t count, T* d_out, const AcReduceOp reduce_op)
{
  ERRCHK_ALWAYS(count != 0);
  ERRCHK_ALWAYS(d_in  != NULL);
  ERRCHK_ALWAYS(d_out != NULL);

  AcDeviceTmpBuffer temp_storage{NULL,0};
  cub_reduce(temp_storage,stream,d_in,count,d_out,reduce_op);

  ERRCHK_ALWAYS(temp_storage.bytes != 0);
  ERRCHK_CUDA_ALWAYS(hipMalloc(&temp_storage.data, temp_storage.bytes));
  ERRCHK_ALWAYS(temp_storage.data);

  cub_reduce(temp_storage,stream,d_in,count,d_out,reduce_op);
  hipStreamSynchronize(
    stream); // Note, would not be needed if allocated at initialization
  hipFree(temp_storage.data);
  return AC_SUCCESS;
}

AcResult
acReduce(const hipStream_t stream, const AcReal* d_in, const size_t count, AcReal* d_out, const AcReduceOp reduce_op)
{
	return acReduceBase(stream,d_in,count,d_out,reduce_op);
}


AcResult
acReduceInt(const hipStream_t stream, const int* d_in, const size_t count, int* d_out, const AcReduceOp reduce_op)
{
	return acReduceBase(stream,d_in,count,d_out,reduce_op);
}

static __global__ void
multiply_inplace(const AcReal value, const size_t count, AcReal* array)
{
  const size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < count)
    array[idx] *= value;
}

AcResult
acMultiplyInplace(const AcReal value, const size_t count, AcReal* array)
{
  const size_t tpb = 256;
  const size_t bpg = (count + tpb - 1) / tpb;
  KERNEL_LAUNCH(multiply_inplace,bpg, tpb,0,0)(value, count, array);
  ERRCHK_CUDA_KERNEL();
  return AC_SUCCESS;
}
#define TILE_DIM (32)

void __global__ 
transpose_xyz_to_zyx(const AcReal* src, AcReal* dst)
{
	__shared__ AcReal tile[TILE_DIM][TILE_DIM];
	const dim3 block_offset =
	{
		blockIdx.x*TILE_DIM,
		blockIdx.y,
		blockIdx.z*TILE_DIM
	};

	const dim3 vertexIdx = 
	{
		threadIdx.x + block_offset.x,
		threadIdx.y + block_offset.y,
		threadIdx.z + block_offset.z
	};
	const dim3 out_vertexIdx = 
	{
		threadIdx.x + block_offset.z,
		threadIdx.y + block_offset.y,
		threadIdx.z + block_offset.x
	};
	const bool in_oob  =  (int)vertexIdx.x  >= VAL(AC_mlocal).x    ||  (int)vertexIdx.y >= VAL(AC_mlocal).y     || (int)vertexIdx.z >= VAL(AC_mlocal).z;
	const bool out_oob =  (int)out_vertexIdx.x >= VAL(AC_mlocal).z ||  (int)out_vertexIdx.y >= VAL(AC_mlocal).y || (int)out_vertexIdx.z >= VAL(AC_mlocal).x;



	tile[threadIdx.z][threadIdx.x] = !in_oob ? src[DEVICE_VTXBUF_IDX(vertexIdx.x,vertexIdx.y,vertexIdx.z)] : 0.0;
	__syncthreads();
	if(!out_oob)
		dst[out_vertexIdx.x +VAL(AC_mlocal).z*out_vertexIdx.y + VAL(AC_mlocal_products).yz*out_vertexIdx.z] = tile[threadIdx.x][threadIdx.z];
}
void __global__ 
transpose_xyz_to_zxy(const AcReal* src, AcReal* dst)
{
	__shared__ AcReal tile[TILE_DIM][TILE_DIM];
	const dim3 block_offset =
	{
		blockIdx.x*TILE_DIM,
		blockIdx.y,
		blockIdx.z*TILE_DIM
	};

	const dim3 vertexIdx = 
	{
		threadIdx.x + block_offset.x,
		threadIdx.y + block_offset.y,
		threadIdx.z + block_offset.z
	};
	const dim3 out_vertexIdx = 
	{
		threadIdx.x + block_offset.z,
		threadIdx.y + block_offset.y,
		threadIdx.z + block_offset.x
	};
	const bool in_oob  =  (int)vertexIdx.x  >= VAL(AC_mlocal).x    ||  (int)vertexIdx.y >= VAL(AC_mlocal).y     || (int)vertexIdx.z >= VAL(AC_mlocal).z;
	const bool out_oob =  (int)out_vertexIdx.x >= VAL(AC_mlocal).z ||  (int)out_vertexIdx.y >= VAL(AC_mlocal).y || (int)out_vertexIdx.z >= VAL(AC_mlocal).x;



	tile[threadIdx.z][threadIdx.x] = !in_oob ? src[DEVICE_VTXBUF_IDX(vertexIdx.x,vertexIdx.y,vertexIdx.z)] : 0.0;
	__syncthreads();
	if(!out_oob)
		dst[out_vertexIdx.x +VAL(AC_mlocal).z*out_vertexIdx.z + VAL(AC_mlocal_products).xz*out_vertexIdx.y] = tile[threadIdx.x][threadIdx.z];
}
void __global__ 
transpose_xyz_to_xyz(const AcReal* src, AcReal* dst)
{
	const dim3 block_offset =
	{
		blockIdx.x*TILE_DIM,
		blockIdx.y*TILE_DIM,
		blockIdx.z
	};

	const dim3 vertexIdx = 
	{
		threadIdx.x + block_offset.x,
		threadIdx.y + block_offset.y,
		threadIdx.z + block_offset.z
	};
	const bool oob  =  (int)vertexIdx.x  >= VAL(AC_mlocal).x    ||  (int)vertexIdx.y >= VAL(AC_mlocal).y     || (int)vertexIdx.z >= VAL(AC_mlocal).z;
	if(oob) return;
	dst[DEVICE_VTXBUF_IDX(vertexIdx.x,vertexIdx.y,vertexIdx.z)] = src[DEVICE_VTXBUF_IDX(vertexIdx.x,vertexIdx.y,vertexIdx.z)];
}
void __global__ 
transpose_xyz_to_yxz(const AcReal* src, AcReal* dst)
{
	__shared__ AcReal tile[TILE_DIM][TILE_DIM];
	const dim3 block_offset =
	{
		blockIdx.x*TILE_DIM,
		blockIdx.y*TILE_DIM,
		blockIdx.z
	};

	const dim3 vertexIdx = 
	{
		threadIdx.x + block_offset.x,
		threadIdx.y + block_offset.y,
		threadIdx.z + block_offset.z
	};
	const dim3 out_vertexIdx = 
	{
		threadIdx.x + block_offset.y,
		threadIdx.y + block_offset.x,
		threadIdx.z + block_offset.z
	};
	const bool in_oob  =  (int)vertexIdx.x  >= VAL(AC_mlocal).x    ||  (int)vertexIdx.y >= VAL(AC_mlocal).y     || (int)vertexIdx.z >= VAL(AC_mlocal).z;
	const bool out_oob =  (int)out_vertexIdx.x >= VAL(AC_mlocal).y ||  (int)out_vertexIdx.y >= VAL(AC_mlocal).x || (int)out_vertexIdx.z >= VAL(AC_mlocal).z;



	tile[threadIdx.y][threadIdx.x] = !in_oob ? src[DEVICE_VTXBUF_IDX(vertexIdx.x,vertexIdx.y,vertexIdx.z)] : 0.0;
	__syncthreads();
	if(!out_oob)
		dst[out_vertexIdx.x +VAL(AC_mlocal).y*out_vertexIdx.y + VAL(AC_mlocal_products).xy*out_vertexIdx.z] = tile[threadIdx.x][threadIdx.y];
}
void __global__ 
transpose_xyz_to_yzx(const AcReal* src, AcReal* dst)
{
	__shared__ AcReal tile[TILE_DIM][TILE_DIM];
	const dim3 block_offset =
	{
		blockIdx.x*TILE_DIM,
		blockIdx.y*TILE_DIM,
		blockIdx.z
	};

	const dim3 vertexIdx = 
	{
		threadIdx.x + block_offset.x,
		threadIdx.y + block_offset.y,
		threadIdx.z + block_offset.z
	};
	const dim3 out_vertexIdx = 
	{
		threadIdx.x + block_offset.y,
		threadIdx.y + block_offset.x,
		threadIdx.z + block_offset.z
	};
	const bool in_oob  =  (int)vertexIdx.x  >= VAL(AC_mlocal).x    ||  (int)vertexIdx.y >= VAL(AC_mlocal).y     || (int)vertexIdx.z >= VAL(AC_mlocal).z;
	const bool out_oob =  (int)out_vertexIdx.x >= VAL(AC_mlocal).y ||  (int)out_vertexIdx.y >= VAL(AC_mlocal).x || (int)out_vertexIdx.z >= VAL(AC_mlocal).z;



	tile[threadIdx.y][threadIdx.x] = !in_oob ? src[DEVICE_VTXBUF_IDX(vertexIdx.x,vertexIdx.y,vertexIdx.z)] : 0.0;
	__syncthreads();
	if(!out_oob)
		dst[out_vertexIdx.x +VAL(AC_mlocal).y*out_vertexIdx.z + VAL(AC_mlocal_products).yz*out_vertexIdx.y] = tile[threadIdx.x][threadIdx.y];
}
void __global__ 
transpose_xyz_to_xzy(const AcReal* src, AcReal* dst)
{
	const dim3 in_block_offset =
	{
		blockIdx.x*blockDim.x,
		blockIdx.y*blockDim.y,
		blockIdx.z*blockDim.z
	};

	const dim3 vertexIdx = 
	{
		threadIdx.x + in_block_offset.x,
		threadIdx.y + in_block_offset.y,
		threadIdx.z + in_block_offset.z
	};

	const bool oob  =  (int)vertexIdx.x  >= VAL(AC_mlocal).x    ||  (int)vertexIdx.y >= VAL(AC_mlocal).y     || (int)vertexIdx.z >= VAL(AC_mlocal).z;
	if(oob) return;
	dst[vertexIdx.x + VAL(AC_mlocal).x*vertexIdx.z + VAL(AC_mlocal_products).xz*vertexIdx.y] 
		= src[DEVICE_VTXBUF_IDX(vertexIdx.x, vertexIdx.y, vertexIdx.z)];
}
static AcResult
acTransposeXYZ_ZYX(const AcReal* src, AcReal* dst, const int3 dims, const hipStream_t stream)
{
	const dim3 tpb = {32,1,32};

	const dim3 bpg = to_dim3(get_bpg(to_volume(dims),to_volume(tpb)));
  	KERNEL_LAUNCH(transpose_xyz_to_zyx,bpg, tpb, 0, stream)(src,dst);
	return AC_SUCCESS;
}
static AcResult
acTransposeXYZ_ZXY(const AcReal* src, AcReal* dst, const int3 dims, const hipStream_t stream)
{
	const dim3 tpb = {32,1,32};

	const dim3 bpg = to_dim3(get_bpg(to_volume(dims),to_volume(tpb)));
  	KERNEL_LAUNCH(transpose_xyz_to_zxy,bpg, tpb, 0, stream)(src,dst);
	return AC_SUCCESS;
}
static AcResult
acTransposeXYZ_YXZ(const AcReal* src, AcReal* dst, const int3 dims, const hipStream_t stream)
{
	const dim3 tpb = {32,32,1};

	const dim3 bpg = to_dim3(get_bpg(to_volume(dims),to_volume(tpb)));
  	KERNEL_LAUNCH(transpose_xyz_to_yxz,bpg, tpb, 0, stream)(src,dst);
	return AC_SUCCESS;
}
static AcResult
acTransposeXYZ_YZX(const AcReal* src, AcReal* dst, const int3 dims, const hipStream_t stream)
{
	const dim3 tpb = {32,32,1};

	const dim3 bpg = to_dim3(get_bpg(to_volume(dims),to_volume(tpb)));
  	KERNEL_LAUNCH(transpose_xyz_to_yzx,bpg, tpb, 0, stream)(src,dst);
	return AC_SUCCESS;
}
static AcResult
acTransposeXYZ_XZY(const AcReal* src, AcReal* dst, const int3 dims, const hipStream_t stream)
{
	const dim3 tpb = {32,32,1};
	const dim3 bpg = to_dim3(get_bpg(to_volume(dims),to_volume(tpb)));
  	KERNEL_LAUNCH(transpose_xyz_to_xzy,bpg, tpb, 0, stream)(src,dst);
	return AC_SUCCESS;
}
static AcResult
acTransposeXYZ_XYZ(const AcReal* src, AcReal* dst, const int3 dims, const hipStream_t stream)
{
	const dim3 tpb = {32,32,1};
	const dim3 bpg = to_dim3(get_bpg(to_volume(dims),to_volume(tpb)));
  	KERNEL_LAUNCH(transpose_xyz_to_xyz,bpg, tpb, 0, stream)(src,dst);
	return AC_SUCCESS;
}
AcResult
acTranspose(const AcMeshOrder order, const AcReal* src, AcReal* dst, const int3 dims, const hipStream_t stream)
{
	switch(order)
	{
		case(XYZ):
			return acTransposeXYZ_XYZ(src,dst,dims,stream);
		case (XZY):
			return acTransposeXYZ_XZY(src,dst,dims,stream);
		case (YXZ):
			return acTransposeXYZ_YXZ(src,dst,dims,stream);
		case (YZX):
			return acTransposeXYZ_YZX(src,dst,dims,stream);
		case(ZXY):
			return acTransposeXYZ_ZXY(src,dst,dims,stream);
		case(ZYX):
			return acTransposeXYZ_ZYX(src,dst,dims,stream);
	}
	return AC_SUCCESS;
}
#include "load_ac_kernel_params.h"
