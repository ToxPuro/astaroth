#include "hip/hip_runtime.h"
#include "acc_runtime.h"

#include "math_utils.h"

__device__ AcMeshInfo d_mesh_info;

// Astaroth 2.0 backwards compatibility START
#define d_multigpu_offset (d_mesh_info.int3_params[AC_multigpu_offset])

int __device__ __forceinline__
DCONST(const AcIntParam param)
{
  return d_mesh_info.int_params[param];
}
int3 __device__ __forceinline__
DCONST(const AcInt3Param param)
{
  return d_mesh_info.int3_params[param];
}
AcReal __device__ __forceinline__
DCONST(const AcRealParam param)
{
  return d_mesh_info.real_params[param];
}
AcReal3 __device__ __forceinline__
DCONST(const AcReal3Param param)
{
  return d_mesh_info.real3_params[param];
}

#define DEVICE_VTXBUF_IDX(i, j, k)                                             \
  ((i) + (j)*DCONST(AC_mx) + (k)*DCONST(AC_mxy))

__device__ constexpr int
IDX(const int i)
{
  return i;
}

__device__ __forceinline__ int
IDX(const int i, const int j, const int k)
{
  return DEVICE_VTXBUF_IDX(i, j, k);
}

__device__ __forceinline__ int
IDX(const int3 idx)
{
  return DEVICE_VTXBUF_IDX(idx.x, idx.y, idx.z);
}
/*
__device__ constexpr inline VertexBufferHandle
DCONST(const VertexBufferHandle handle)
{
  return handle;
}*/
//#define IDX(i, j, k) ((i) + (j)*mm.x + (k)*mm.x * mm.y)
// #define IDX(i, j, k) ((i) + (j)*DCONST(AC_mx) + (k)*DCONST(AC_mx) *
// DCONST(AC_my)) Astaroth 2.0 backwards compatibility END

#define Field3(x, y, z) make_int3((x), (y), (z))
#define real3(i, j, k) ((AcReal3){(i), (j), (k)})
#define print printf // TODO is this a good idea?

#include "user_kernels.h"

AcResult
acLaunchKernel(Kernel func, const hipStream_t stream, const int3 start,
               const int3 end, VertexBufferArray vba)
{
  const int3 n = end - start;

  // const dim3 tpb = getOptimalTBConfig(n, vba).tpb; // TODO
  const dim3 tpb = dim3(32, 4, 1);
  const dim3 bpg((unsigned int)ceil(n.x / AcReal(tpb.x)), //
                 (unsigned int)ceil(n.y / AcReal(tpb.y)), //
                 (unsigned int)ceil(n.z / AcReal(tpb.z)));
  const size_t smem = 0;

  func<<<bpg, tpb, smem, stream>>>(start, end, vba);
  ERRCHK_CUDA_KERNEL();

  return AC_SUCCESS;
}

#define GEN_LOAD_UNIFORM(TYPE)                                                 \
  GEN_LOAD_UNIFORM_DECLARATION(TYPE)                                           \
  {                                                                            \
    hipError_t retval = hipMemcpyToSymbolAsync(HIP_SYMBOL(                              \
        symbol), &value, sizeof(value), 0, hipMemcpyHostToDevice, stream);     \
    return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;                    \
  }

#define GEN_STORE_UNIFORM(TYPE)                                                \
  GEN_STORE_UNIFORM_DECLARATION(TYPE)                                          \
  {                                                                            \
    hipError_t retval = hipMemcpyFromSymbolAsync(                            \
        dst, HIP_SYMBOL(symbol), sizeof(*dst), 0, hipMemcpyDeviceToHost, stream);         \
    return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;                    \
  }

GEN_LOAD_UNIFORM(AcReal)
GEN_LOAD_UNIFORM(AcReal3)
GEN_LOAD_UNIFORM(int)
GEN_LOAD_UNIFORM(int3)

GEN_STORE_UNIFORM(AcReal)
GEN_STORE_UNIFORM(AcReal3)
GEN_STORE_UNIFORM(int)
GEN_STORE_UNIFORM(int3)

/*
int
main(void)
{
  printf("Launching kernel... \n");
  acLaunchKernel(solve, 0, (int3){0, 0, 0}, (int3){1, 0, 0},
                 (VertexBufferArray){0});
  printf("done\n");
  return EXIT_SUCCESS;
}
*/