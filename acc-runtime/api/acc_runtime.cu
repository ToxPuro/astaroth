#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2021, Johannes Pekkila, Miikka Vaisala.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "acc_runtime.h"

#include <vector> // tbconfig

#include "errchk.h"
#include "math_utils.h"

#if AC_USE_HIP
#include <hip/hip_runtime.h> // Needed in files that include kernels
#endif

// Device info (TODO GENERIC)
// Use the maximum available reg count per thread
#define REGISTERS_PER_THREAD (255)
#define MAX_REGISTERS_PER_BLOCK (65536)
#if AC_DOUBLE_PRECISION
#define MAX_THREADS_PER_BLOCK                                                  \
  (MAX_REGISTERS_PER_BLOCK / REGISTERS_PER_THREAD / 2)
#else
#define MAX_THREADS_PER_BLOCK (MAX_REGISTERS_PER_BLOCK / REGISTERS_PER_THREAD)
#endif

__device__ __constant__ AcMeshInfo d_mesh_info;

// Astaroth 2.0 backwards compatibility START
#define d_multigpu_offset (d_mesh_info.int3_params[AC_multigpu_offset])

int __device__ __forceinline__
DCONST(const AcIntParam param)
{
  return d_mesh_info.int_params[param];
}
int3 __device__ __forceinline__
DCONST(const AcInt3Param param)
{
  return d_mesh_info.int3_params[param];
}
AcReal __device__ __forceinline__
DCONST(const AcRealParam param)
{
  return d_mesh_info.real_params[param];
}
AcReal3 __device__ __forceinline__
DCONST(const AcReal3Param param)
{
  return d_mesh_info.real3_params[param];
}

#define DEVICE_VTXBUF_IDX(i, j, k)                                             \
  ((i) + (j)*DCONST(AC_mx) + (k)*DCONST(AC_mxy))

__device__ constexpr int
IDX(const int i)
{
  return i;
}

__device__ __forceinline__ int
IDX(const int i, const int j, const int k)
{
  return DEVICE_VTXBUF_IDX(i, j, k);
}

__device__ __forceinline__ int
IDX(const int3 idx)
{
  return DEVICE_VTXBUF_IDX(idx.x, idx.y, idx.z);
}

#define Field3(x, y, z) make_int3((x), (y), (z))
#define print printf                          // TODO is this a good idea?
#define len(arr) sizeof(arr) / sizeof(arr[0]) // Leads to bugs if the user
// passes an array into a device function and then calls len (need to modify
// the compiler to always pass arrays to functions as references before
// re-enabling)

#include "user_kernels.h"

typedef struct {
  Kernel kernel;
  int3 dims;
  dim3 tpb;
} TBConfig;

static std::vector<TBConfig> tbconfigs;

static TBConfig getOptimalTBConfig(const Kernel kernel, const int3 dims,
                                   VertexBufferArray vba);

AcResult
acLaunchKernel(Kernel kernel, const hipStream_t stream, const int3 start,
               const int3 end, VertexBufferArray vba)
{
  const int3 n = end - start;

  const dim3 tpb = getOptimalTBConfig(kernel, n, vba).tpb;
  const dim3 bpg((unsigned int)ceil(n.x / double(tpb.x)), //
                 (unsigned int)ceil(n.y / double(tpb.y)), //
                 (unsigned int)ceil(n.z / double(tpb.z)));
  const size_t smem = 0;

  kernel<<<bpg, tpb, smem, stream>>>(start, end, vba);
  ERRCHK_CUDA_KERNEL();

  return AC_SUCCESS;
}

AcResult
acLoadStencil(const Stencil stencil, const hipStream_t stream,
              const AcReal data[STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
  ERRCHK_ALWAYS(stencil < NUM_STENCILS);

  const size_t bytes = sizeof(data[0][0][0]) * STENCIL_DEPTH * STENCIL_HEIGHT *
                       STENCIL_WIDTH;
  const hipError_t retval = hipMemcpyToSymbolAsync(HIP_SYMBOL(
      stencils), data, bytes, stencil * bytes, hipMemcpyHostToDevice, stream);

  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
};

AcResult
acStoreStencil(const Stencil stencil, const hipStream_t stream,
               AcReal data[STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
  ERRCHK_ALWAYS(stencil < NUM_STENCILS);

  const size_t bytes = sizeof(data[0][0][0]) * STENCIL_DEPTH * STENCIL_HEIGHT *
                       STENCIL_WIDTH;
  const hipError_t retval = hipMemcpyFromSymbolAsync(
      data, HIP_SYMBOL(stencils), bytes, stencil * bytes, hipMemcpyDeviceToHost, stream);

  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
};

#define GEN_LOAD_UNIFORM(LABEL_UPPER, LABEL_LOWER)                             \
  ERRCHK_ALWAYS(param < NUM_##LABEL_UPPER##_PARAMS);                           \
                                                                               \
  const size_t offset = (size_t)&d_mesh_info.LABEL_LOWER##_params[param] -     \
                        (size_t)&d_mesh_info;                                  \
                                                                               \
  const hipError_t retval = hipMemcpyToSymbolAsync(HIP_SYMBOL(                          \
      d_mesh_info), &value, sizeof(value), offset, hipMemcpyHostToDevice,      \
      stream);                                                                 \
  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;

AcResult
acLoadRealUniform(const hipStream_t stream, const AcRealParam param,
                  const AcReal value)
{
  if (isnan(value)) {
    fprintf(stderr,
            "WARNING: Passed an invalid value %g to device constant %s. "
            "Skipping.\n",
            (double)value, realparam_names[param]);
    return AC_FAILURE;
  }
  GEN_LOAD_UNIFORM(REAL, real);
}

AcResult
acLoadReal3Uniform(const hipStream_t stream, const AcReal3Param param,
                   const AcReal3 value)
{
  if (isnan(value.x) | isnan(value.y) | isnan(value.z)) {
    fprintf(stderr,
            "WARNING: Passed an invalid value (%g, %g, %g) to device constant "
            "%s. Skipping.\n",
            (double)value.x, (double)value.y, (double)value.z,
            real3param_names[param]);
    return AC_FAILURE;
  }
  GEN_LOAD_UNIFORM(REAL3, real3);
}

AcResult
acLoadIntUniform(const hipStream_t stream, const AcIntParam param,
                 const int value)
{
  GEN_LOAD_UNIFORM(INT, int);
}

AcResult
acLoadInt3Uniform(const hipStream_t stream, const AcInt3Param param,
                  const int3 value)
{
  GEN_LOAD_UNIFORM(INT3, int3);
}

#define GEN_STORE_UNIFORM(LABEL_UPPER, LABEL_LOWER)                            \
  ERRCHK_ALWAYS(param < NUM_##LABEL_UPPER##_PARAMS);                           \
                                                                               \
  const size_t offset = (size_t)&d_mesh_info.LABEL_LOWER##_params[param] -     \
                        (size_t)&d_mesh_info;                                  \
                                                                               \
  const hipError_t retval = hipMemcpyFromSymbolAsync(                        \
      value, HIP_SYMBOL(d_mesh_info), sizeof(*value), offset, hipMemcpyDeviceToHost,      \
      stream);                                                                 \
  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;

AcResult
acStoreRealUniform(const hipStream_t stream, const AcRealParam param,
                   AcReal* value)
{
  GEN_STORE_UNIFORM(REAL, real);
}

AcResult
acStoreReal3Uniform(const hipStream_t stream, const AcReal3Param param,
                    AcReal3* value)
{
  GEN_STORE_UNIFORM(REAL3, real3);
}

AcResult
acStoreIntUniform(const hipStream_t stream, const AcIntParam param, int* value)
{
  GEN_STORE_UNIFORM(INT, int);
}

AcResult
acStoreInt3Uniform(const hipStream_t stream, const AcInt3Param param,
                   int3* value)
{
  GEN_STORE_UNIFORM(INT3, int3);
}

static TBConfig
autotune(const Kernel kernel, const int3 dims, VertexBufferArray vba)
{
  printf("Autotuning kernel %p, block (%d, %d, %d)... ", kernel, dims.x, dims.y,
         dims.z);
  fflush(stdout);
  TBConfig c = {
      .kernel = kernel,
      .dims   = dims,
      .tpb    = (dim3){0, 0, 0},
  };

  const int3 start = (int3){
      STENCIL_ORDER / 2,
      STENCIL_ORDER / 2,
      STENCIL_ORDER / 2,
  };
  const int3 end = start + dims;

  dim3 best_tpb(0, 0, 0);
  float best_time     = INFINITY;
  const int num_iters = 2;

  // TODO idea #1:
  // Choose tpb.x s.t. it is at most 'dims.x' rounded upward to the nearest
  // multiple of the warp size
  // xmax = min(MAX_THREADS_PER_BLOCK, (1 + floor((dims.x-1)/ warp_size))

  // TODO idea #2:
  // Break if x*y*z > round_up_to_multiple_of_warp_size(dim.x * dim.y * dim.z)

  for (int z = 1; z <= MAX_THREADS_PER_BLOCK; ++z) {
    for (int y = 1; y <= MAX_THREADS_PER_BLOCK; ++y) {
      for (int x = 1; x <= MAX_THREADS_PER_BLOCK; ++x) {

        if (x * y * z > MAX_THREADS_PER_BLOCK)
          break;

        if (x * y * z * REGISTERS_PER_THREAD > MAX_REGISTERS_PER_BLOCK)
          break;

        const dim3 tpb(x, y, z);
        const dim3 bpg((unsigned int)ceil(dims.x / double(tpb.x)), //
                       (unsigned int)ceil(dims.y / double(tpb.y)), //
                       (unsigned int)ceil(dims.z / double(tpb.z)));

        hipEvent_t tstart, tstop;
        hipEventCreate(&tstart);
        hipEventCreate(&tstop);

        hipDeviceSynchronize();
        hipEventRecord(tstart); // Timing start
        for (int i = 0; i < num_iters; ++i)
          kernel<<<bpg, tpb>>>(start, end, vba);
        hipEventRecord(tstop); // Timing stop
        hipEventSynchronize(tstop);

        if (hipGetLastError() != hipSuccess) // Discard failed runs
          continue;

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, tstart, tstop);

        if (milliseconds < best_time) {
          best_time = milliseconds;
          best_tpb  = tpb;
        }

        // printf("Auto-optimizing... Current tpb: (%d, %d, %d), time %f ms\n",
        //       tpb.x, tpb.y, tpb.z, (double)best_time / num_iters);
        fflush(stdout);
      }
    }
  }
  c.tpb = best_tpb;

  printf("The best tpb: (%d, %d, %d), time %f ms\n", best_tpb.x, best_tpb.y,
         best_tpb.z, (double)best_time / num_iters);

  ERRCHK_ALWAYS(c.tpb.x * c.tpb.y * c.tpb.z > 0);
  return c;
}

static TBConfig
getOptimalTBConfig(const Kernel kernel, const int3 dims, VertexBufferArray vba)
{
  for (auto c : tbconfigs) {
    if (c.kernel == kernel && c.dims == dims)
      return c;
  }
  TBConfig c = autotune(kernel, dims, vba);
  tbconfigs.push_back(c);
  return c;
}
