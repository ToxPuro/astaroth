#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2021, Johannes Pekkila, Miikka Vaisala.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "acc_runtime.h"

#include <vector> // tbconfig

#include "errchk.h"
#include "math_utils.h"

#if AC_USE_HIP
#include <hip/hip_runtime.h> // Needed in files that include kernels
#endif

/*
// Device info (TODO GENERIC)
// Use the maximum available reg count per thread
#define REGISTERS_PER_THREAD (255)
#define MAX_REGISTERS_PER_BLOCK (65536)
#if AC_DOUBLE_PRECISION
#define MAX_THREADS_PER_BLOCK                                                  \
  (MAX_REGISTERS_PER_BLOCK / REGISTERS_PER_THREAD / 2)
#else
#define MAX_THREADS_PER_BLOCK (MAX_REGISTERS_PER_BLOCK / REGISTERS_PER_THREAD)
#endif
*/

__device__ __constant__ AcMeshInfo d_mesh_info;

// Astaroth 2.0 backwards compatibility START
#define d_multigpu_offset (d_mesh_info.int3_params[AC_multigpu_offset])

int __device__ __forceinline__
DCONST(const AcIntParam param)
{
  return d_mesh_info.int_params[param];
}
int3 __device__ __forceinline__
DCONST(const AcInt3Param param)
{
  return d_mesh_info.int3_params[param];
}
AcReal __device__ __forceinline__
DCONST(const AcRealParam param)
{
  return d_mesh_info.real_params[param];
}
AcReal3 __device__ __forceinline__
DCONST(const AcReal3Param param)
{
  return d_mesh_info.real3_params[param];
}

#define DEVICE_VTXBUF_IDX(i, j, k)                                             \
  ((i) + (j)*DCONST(AC_mx) + (k)*DCONST(AC_mxy))

__device__ constexpr int
IDX(const int i)
{
  return i;
}

__device__ __forceinline__ int
IDX(const int i, const int j, const int k)
{
  return DEVICE_VTXBUF_IDX(i, j, k);
}

__device__ __forceinline__ int
IDX(const int3 idx)
{
  return DEVICE_VTXBUF_IDX(idx.x, idx.y, idx.z);
}

#define Field3(x, y, z) make_int3((x), (y), (z))
#define print printf                          // TODO is this a good idea?
#define len(arr) sizeof(arr) / sizeof(arr[0]) // Leads to bugs if the user
// passes an array into a device function and then calls len (need to modify
// the compiler to always pass arrays to functions as references before
// re-enabling)

#include "user_kernels.h"

typedef struct {
  Kernel kernel;
  int3 dims;
  dim3 tpb;
} TBConfig;

static std::vector<TBConfig> tbconfigs;

static TBConfig getOptimalTBConfig(const Kernel kernel, const int3 dims,
                                   VertexBufferArray vba);

static __global__ void
flush_kernel(AcReal* arr, const size_t n)
{
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n)
    arr[idx] = (AcReal)NAN;
}

AcResult
acKernelFlush(AcReal* arr, const size_t n)
{
  const size_t tpb = 256;
  const size_t bpg = (size_t)(ceil((double)n / tpb));
  flush_kernel<<<bpg, tpb>>>(arr, n);
  ERRCHK_CUDA_KERNEL_ALWAYS();
  return AC_SUCCESS;
}

VertexBufferArray
acVBACreate(const size_t count)
{
  VertexBufferArray vba;

  const size_t bytes = sizeof(vba.in[0][0]) * count;

//#define ADJACENT_VERTEX_BUFFERS 1
#if AC_ADJACENT_VERTEX_BUFFERS
  const size_t allbytes = bytes*NUM_VTXBUF_HANDLES;
  AcReal *allbuf_in, *allbuf_out;

  ERRCHK_CUDA_ALWAYS(hipMalloc((void**)&allbuf_in, allbytes));
  ERRCHK_CUDA_ALWAYS(hipMalloc((void**)&allbuf_out, allbytes));
  acKernelFlush(allbuf_in, count*NUM_VTXBUF_HANDLES);
  ERRCHK_CUDA_ALWAYS(hipMemset((void*)allbuf_out, 0, allbytes));

  vba.in[0]=allbuf_in; vba.out[0]=allbuf_out;
printf("i,vbas[0]= 0 %p \n",vba.in[0],vba.out[0]);
  for (size_t i = 1; i < NUM_VTXBUF_HANDLES; ++i) {
    vba.in [i]=vba.in [i-1]+count;
    vba.out[i]=vba.out[i-1]+count;
printf("i,vbas[i]= %d %p \n",i,vba.in[i],vba.out[i]);
  }
#else
  for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
    ERRCHK_CUDA_ALWAYS(hipMalloc((void**)&vba.in[i], bytes));
    ERRCHK_CUDA_ALWAYS(hipMalloc((void**)&vba.out[i], bytes));
printf("i,vbas[i]= %d %p %p\n",i,vba.in[i],vba.out[i]);

    // Set vba.in data to all-nan and vba.out to 0
    acKernelFlush(vba.in[i], count);
    ERRCHK_CUDA_ALWAYS(hipMemset((void*)vba.out[i], 0, bytes));
  }
#endif
  return vba;
}

void
acVBADestroy(VertexBufferArray* vba)
{
  for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
    hipFree(vba->in[i]);
    hipFree(vba->out[i]);
    vba->in[i]  = NULL;
    vba->out[i] = NULL;
  }
}

AcResult
acLaunchKernel(Kernel kernel, const hipStream_t stream, const int3 start,
               const int3 end, VertexBufferArray vba)
{
  const int3 n = end - start;

  const dim3 tpb = getOptimalTBConfig(kernel, n, vba).tpb;
  ERRCHK(tpb.x*tpb.y*tpb.z<=1024);
  const dim3 bpg((unsigned int)ceil(n.x / double(tpb.x)), //
                 (unsigned int)ceil(n.y / double(tpb.y)), //
                 (unsigned int)ceil(n.z / double(tpb.z)));
  const size_t smem = 0;
//printf("before launch tpb,bpg=%d %d %d %d %d %d \n",tpb.x,tpb.y,tpb.z,bpg.x,bpg.y,bpg.z);
//printf("before launch start,end=%d %d %d %d %d %d \n",start.x,start.y,start.z,end.x,end.y,end.z);
  kernel<<<bpg, tpb, smem, stream>>>(start, end, vba);
  ERRCHK_CUDA_KERNEL();

  return AC_SUCCESS;
}

AcResult
acLoadStencil(const Stencil stencil, const hipStream_t stream,
              const AcReal data[STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
  ERRCHK_ALWAYS(stencil < NUM_STENCILS);

  const size_t bytes = sizeof(data[0][0][0]) * STENCIL_DEPTH * STENCIL_HEIGHT *
                       STENCIL_WIDTH;
  const hipError_t retval = hipMemcpyToSymbolAsync(HIP_SYMBOL(
      stencils), data, bytes, stencil * bytes, hipMemcpyHostToDevice, stream);

  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
};

AcResult
acStoreStencil(const Stencil stencil, const hipStream_t stream,
               AcReal data[STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
  ERRCHK_ALWAYS(stencil < NUM_STENCILS);

  const size_t bytes = sizeof(data[0][0][0]) * STENCIL_DEPTH * STENCIL_HEIGHT *
                       STENCIL_WIDTH;
  const hipError_t retval = hipMemcpyFromSymbolAsync(
      data, HIP_SYMBOL(stencils), bytes, stencil * bytes, hipMemcpyDeviceToHost, stream);

  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
};

#define GEN_LOAD_UNIFORM(LABEL_UPPER, LABEL_LOWER)                             \
  ERRCHK_ALWAYS(param < NUM_##LABEL_UPPER##_PARAMS);                           \
                                                                               \
  const size_t offset = (size_t)&d_mesh_info.LABEL_LOWER##_params[param] -     \
                        (size_t)&d_mesh_info;                                  \
                                                                               \
  const hipError_t retval = hipMemcpyToSymbolAsync(HIP_SYMBOL(                          \
      d_mesh_info), &value, sizeof(value), offset, hipMemcpyHostToDevice,      \
      stream);                                                                 \
  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;

AcResult
acLoadRealUniform(const hipStream_t stream, const AcRealParam param,
                  const AcReal value)
{
  if (isnan(value)) {
    fprintf(stderr,
            "WARNING: Passed an invalid value %g to device constant %s. "
            "Skipping.\n",
            (double)value, realparam_names[param]);
    return AC_FAILURE;
  }
  GEN_LOAD_UNIFORM(REAL, real);
}

AcResult
acLoadReal3Uniform(const hipStream_t stream, const AcReal3Param param,
                   const AcReal3 value)
{
  if (isnan(value.x) | isnan(value.y) | isnan(value.z)) {
    fprintf(stderr,
            "WARNING: Passed an invalid value (%g, %g, %g) to device constant "
            "%s. Skipping.\n",
            (double)value.x, (double)value.y, (double)value.z,
            real3param_names[param]);
    return AC_FAILURE;
  }
  GEN_LOAD_UNIFORM(REAL3, real3);
}

AcResult
acLoadIntUniform(const hipStream_t stream, const AcIntParam param,
                 const int value)
{
  GEN_LOAD_UNIFORM(INT, int);
}

AcResult
acLoadInt3Uniform(const hipStream_t stream, const AcInt3Param param,
                  const int3 value)
{
  GEN_LOAD_UNIFORM(INT3, int3);
}

#define GEN_STORE_UNIFORM(LABEL_UPPER, LABEL_LOWER)                            \
  ERRCHK_ALWAYS(param < NUM_##LABEL_UPPER##_PARAMS);                           \
                                                                               \
  const size_t offset = (size_t)&d_mesh_info.LABEL_LOWER##_params[param] -     \
                        (size_t)&d_mesh_info;                                  \
                                                                               \
  const hipError_t retval = hipMemcpyFromSymbolAsync(                        \
      value, HIP_SYMBOL(d_mesh_info), sizeof(*value), offset, hipMemcpyDeviceToHost,      \
      stream);                                                                 \
  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;

AcResult
acStoreRealUniform(const hipStream_t stream, const AcRealParam param,
                   AcReal* value)
{
  GEN_STORE_UNIFORM(REAL, real);
}

AcResult
acStoreReal3Uniform(const hipStream_t stream, const AcReal3Param param,
                    AcReal3* value)
{
  GEN_STORE_UNIFORM(REAL3, real3);
}

AcResult
acStoreIntUniform(const hipStream_t stream, const AcIntParam param, int* value)
{
  GEN_STORE_UNIFORM(INT, int);
}

AcResult
acStoreInt3Uniform(const hipStream_t stream, const AcInt3Param param,
                   int3* value)
{
  GEN_STORE_UNIFORM(INT3, int3);
}

static TBConfig
autotune(const Kernel kernel, const int3 dims, VertexBufferArray vba)
{
  printf("Autotuning kernel %p, block (%d, %d, %d)... ", kernel, dims.x, dims.y,
         dims.z);
  fflush(stdout);
// suppress autotuning for the moment; blocksize seems to be limited to 256
  return (TBConfig){
    .kernel = kernel,
    .dims = dims,
    .tpb = (dim3){64,2,2}
  };

  TBConfig c = {
      .kernel = kernel,
      .dims   = dims,
      .tpb    = (dim3){0, 0, 0},
  };

  const int3 start = (int3){
      STENCIL_ORDER / 2,
      STENCIL_ORDER / 2,
      STENCIL_ORDER / 2,
  };
  const int3 end = start + dims;

  dim3 best_tpb(0, 0, 0);
  float best_time     = INFINITY;
  const int num_iters = 2;

  // Get device hardware information
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  const int warp_size             = props.warpSize;
  const int max_threads_per_block = props.maxThreadsPerBlock;

  for (int z = 1; z <= max_threads_per_block; ++z) {
    for (int y = 1; y <= max_threads_per_block; ++y) {
      for (int x = 1; x <= max_threads_per_block; ++x) {

        if (x * y * z > max_threads_per_block)
          break;

        // if (x * y * z * max_regs_per_thread > max_regs_per_block)
        //  break;

        if ((x * y * z) % warp_size)
          continue;

        // if (max_regs_per_block / (x * y * z) < min_regs_per_thread)
        //   continue;

        // if (x < y || x < z)
        //   continue;

        const dim3 tpb(x, y, z);
        const dim3 bpg((unsigned int)ceil(dims.x / double(tpb.x)), //
                       (unsigned int)ceil(dims.y / double(tpb.y)), //
                       (unsigned int)ceil(dims.z / double(tpb.z)));

        hipEvent_t tstart, tstop;
        hipEventCreate(&tstart);
        hipEventCreate(&tstop);

        hipDeviceSynchronize();
        hipEventRecord(tstart); // Timing start
        for (int i = 0; i < num_iters; ++i)
          kernel<<<bpg, tpb>>>(start, end, vba);
        hipEventRecord(tstop); // Timing stop
        hipEventSynchronize(tstop);

        if (hipGetLastError() != hipSuccess) // Discard failed runs
          continue;

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, tstart, tstop);

        if (milliseconds < best_time) {
          best_time = milliseconds;
          best_tpb  = tpb;
        }

        // printf("Auto-optimizing... Current tpb: (%d, %d, %d), time %f ms\n",
        //       tpb.x, tpb.y, tpb.z, (double)best_time / num_iters);
        fflush(stdout);
      }
    }
  }
  c.tpb = best_tpb;

  printf("The best tpb: (%d, %d, %d), time %f ms\n", best_tpb.x, best_tpb.y,
         best_tpb.z, (double)best_time / num_iters);

  ERRCHK_ALWAYS(c.tpb.x * c.tpb.y * c.tpb.z > 0);
  return c;
}

static TBConfig
getOptimalTBConfig(const Kernel kernel, const int3 dims, VertexBufferArray vba)
{
  for (auto c : tbconfigs) {
    if (c.kernel == kernel && c.dims == dims)
      return c;
  }
  TBConfig c = autotune(kernel, dims, vba);
  tbconfigs.push_back(c);
  return c;
}
