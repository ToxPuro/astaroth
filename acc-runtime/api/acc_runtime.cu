#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2021, Johannes Pekkila, Miikka Vaisala.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "acc_runtime.h"

#include <vector> // tbconfig

#include "errchk.h"
#include "math_utils.h"

#if AC_USE_HIP
#include <hip/hip_runtime.h> // Needed in files that include kernels
#endif

#define USE_COMPRESSIBLE_MEMORY (0)

#include "acc/implementation.h"

static dim3 last_tpb = (dim3){0, 0, 0};

Volume
acKernelLaunchGetLastTPB(void)
{
  return to_volume(last_tpb);
}

Volume
get_bpg(const Volume dims, const Volume tpb)
{
  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING:             // Fallthrough
  case EXPLICIT_CACHING:             // Fallthrough
  case EXPLICIT_CACHING_3D_BLOCKING: // Fallthrough
  case EXPLICIT_CACHING_4D_BLOCKING: // Fallthrough
  case EXPLICIT_PINGPONG_txw:        // Fallthrough
  case EXPLICIT_PINGPONG_txy:        // Fallthrough
  case EXPLICIT_PINGPONG_txyblocked: // Fallthrough
  case EXPLICIT_PINGPONG_txyz:       // Fallthrough
  case EXPLICIT_ROLLING_PINGPONG: {
    return (Volume){
        (size_t)ceil(1. * dims.x / tpb.x),
        (size_t)ceil(1. * dims.y / tpb.y),
        (size_t)ceil(1. * dims.z / tpb.z),
    };
  }
  default: {
    ERROR("Invalid IMPLEMENTATION in get_bpg");
    return (Volume){0, 0, 0};
  }
  }
}

bool
is_valid_configuration(const Volume dims, const Volume tpb)
{
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  const size_t warp_size = props.warpSize;
  const size_t xmax      = (size_t)(warp_size * ceil(1. * dims.x / warp_size));
  const size_t ymax      = (size_t)(warp_size * ceil(1. * dims.y / warp_size));
  const size_t zmax      = (size_t)(warp_size * ceil(1. * dims.z / warp_size));
  const bool too_large   = (tpb.x > xmax) || (tpb.y > ymax) || (tpb.z > zmax);

  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING: {

    if (too_large)
      return false;

    return true;
  }
  case EXPLICIT_CACHING_4D_BLOCKING: // Fallthrough
    if (tpb.z > 1)
      return false;
  case EXPLICIT_CACHING: // Fallthrough
  case EXPLICIT_CACHING_3D_BLOCKING: {

    // For some reason does not work without this
    // Probably because of break vs continue when fetching (some threads
    // quit too early if the dims are not divisible)
    return !(dims.x % tpb.x) && !(dims.y % tpb.y) && !(dims.z % tpb.z);
  }
  case EXPLICIT_PINGPONG_txw: {
    return (tpb.y == 1) && (tpb.z == 1);
  }
  case EXPLICIT_PINGPONG_txy: {
    return (tpb.z == 1);
  }
  case EXPLICIT_PINGPONG_txyblocked: {
    return (tpb.z == 1);
  }
  case EXPLICIT_PINGPONG_txyz: {
    return true;
  }
  case EXPLICIT_ROLLING_PINGPONG: {
    // OK for every other rolling pingpong implementation
    // return true;

    // Required only when unrolling smem loads
    // Ensures two unrolls is enough to fill the smem buffer
    return (2 * tpb.x >= STENCIL_WIDTH - 1 + tpb.x) &&
           (2 * tpb.y >= STENCIL_HEIGHT - 1 + tpb.y);
  }
  default: {
    ERROR("Invalid IMPLEMENTATION in is_valid_configuration");
    return false;
  }
  }
}

size_t
get_smem(const Volume tpb, const size_t stencil_order,
         const size_t bytes_per_elem)
{
  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING: {
    return 0;
  }
  case EXPLICIT_CACHING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) * tpb.z *
           bytes_per_elem;
  }
  case EXPLICIT_CACHING_3D_BLOCKING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) *
           (tpb.z + stencil_order) * bytes_per_elem;
  }
  case EXPLICIT_CACHING_4D_BLOCKING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) * tpb.z *
           (NUM_FIELDS)*bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txw: {
    return 2 * (tpb.x + stencil_order) * NUM_FIELDS * bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txy: {
    return 2 * (tpb.x + stencil_order) * (tpb.y + stencil_order) *
           bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txyblocked: {
    const size_t block_size = 7;
    return 2 * (tpb.x + stencil_order) * (tpb.y + stencil_order) * block_size *
           bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txyz: {
    return 2 * (tpb.x + stencil_order) * (tpb.y + stencil_order) *
           (tpb.z + stencil_order) * bytes_per_elem;
  }
  case EXPLICIT_ROLLING_PINGPONG: {
    // tpbxy slices with halos
    // tpbz depth + 1 rolling cache slab
    return EXPLICIT_ROLLING_PINGPONG_BLOCKSIZE * (tpb.x + stencil_order) *
           (tpb.y + stencil_order) * (tpb.z + 1) * bytes_per_elem;
  }
  default: {
    ERROR("Invalid IMPLEMENTATION in get_smem");
    return (size_t)-1;
  }
  }
}

/*
// Device info (TODO GENERIC)
// Use the maximum available reg count per thread
#define REGISTERS_PER_THREAD (255)
#define MAX_REGISTERS_PER_BLOCK (65536)
#if AC_DOUBLE_PRECISION
#define MAX_THREADS_PER_BLOCK                                                  \
  (MAX_REGISTERS_PER_BLOCK / REGISTERS_PER_THREAD / 2)
#else
#define MAX_THREADS_PER_BLOCK (MAX_REGISTERS_PER_BLOCK / REGISTERS_PER_THREAD)
#endif
*/

__device__ __constant__ AcMeshInfo d_mesh_info;

// Astaroth 2.0 backwards compatibility START
#define d_multigpu_offset (d_mesh_info.int3_params[AC_multigpu_offset])

int __device__ __forceinline__
DCONST(const AcIntParam param)
{
  return d_mesh_info.int_params[param];
}
int3 __device__ __forceinline__
DCONST(const AcInt3Param param)
{
  return d_mesh_info.int3_params[param];
}
AcReal __device__ __forceinline__
DCONST(const AcRealParam param)
{
  return d_mesh_info.real_params[param];
}
AcReal3 __device__ __forceinline__
DCONST(const AcReal3Param param)
{
  return d_mesh_info.real3_params[param];
}

#define DEVICE_VTXBUF_IDX(i, j, k)                                             \
  ((i) + (j) * DCONST(AC_mx) + (k) * DCONST(AC_mxy))

__device__ int
LOCAL_COMPDOMAIN_IDX(const int3 coord)
{
  return (coord.x) + (coord.y) * DCONST(AC_nx) + (coord.z) * DCONST(AC_nxy);
}

__device__ constexpr int
IDX(const int i)
{
  return i;
}

#if 1
__device__ __forceinline__ int
IDX(const int i, const int j, const int k)
{
  return DEVICE_VTXBUF_IDX(i, j, k);
}
#else
constexpr __device__ int
IDX(const uint i, const uint j, const uint k)
{
  /*
  const int precision   = 32; // Bits
  const int dimensions  = 3;
  const int bits = ceil(precision / dimensions);
  */
  const int dimensions = 3;
  const int bits       = 11;

  uint idx = 0;
#pragma unroll
  for (uint bit = 0; bit < bits; ++bit) {
    const uint mask = 0b1 << bit;
    idx |= ((i & mask) << 0) << (dimensions - 1) * bit;
    idx |= ((j & mask) << 1) << (dimensions - 1) * bit;
    idx |= ((k & mask) << 2) << (dimensions - 1) * bit;
  }
  return idx;
}
#endif

// Only used in reductions
__device__ __forceinline__ int
IDX(const int3 idx)
{
  return DEVICE_VTXBUF_IDX(idx.x, idx.y, idx.z);
}

#define Field3(x, y, z) make_int3((x), (y), (z))
#define print printf                          // TODO is this a good idea?
#define len(arr) sizeof(arr) / sizeof(arr[0]) // Leads to bugs if the user
// passes an array into a device function and then calls len (need to modify
// the compiler to always pass arrays to functions as references before
// re-enabling)

#include "random.cuh"

#include "user_kernels.h"

typedef struct {
  Kernel kernel;
  int3 dims;
  dim3 tpb;
} TBConfig;

static std::vector<TBConfig> tbconfigs;

static TBConfig getOptimalTBConfig(const Kernel kernel, const int3 dims,
                                   VertexBufferArray vba);

static __global__ void
flush_kernel(AcReal* arr, const size_t n, const AcReal value)
{
  const size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n)
    arr[idx] = value;
}

AcResult
acKernelFlush(const hipStream_t stream, AcReal* arr, const size_t n,
              const AcReal value)
{
  const size_t tpb = 256;
  const size_t bpg = (size_t)(ceil((double)n / tpb));
  flush_kernel<<<bpg, tpb, 0, stream>>>(arr, n, value);
  ERRCHK_CUDA_KERNEL_ALWAYS();
  return AC_SUCCESS;
}

#if USE_COMPRESSIBLE_MEMORY
#include <hip/hip_runtime.h>

#define ERRCHK_CU_ALWAYS(x) ERRCHK_ALWAYS((x) == hipSuccess)

static hipError_t
mallocCompressible(void** addr, const size_t requested_bytes)
{
  hipDevice_t device;
  ERRCHK_ALWAYS(hipCtxGetDevice(&device) == hipSuccess);

  hipMemAllocationProp prop;
  memset(&prop, 0, sizeof(hipMemAllocationProp));
  prop.type                       = hipMemAllocationTypePinned;
  prop.location.type              = hipMemLocationTypeDevice;
  prop.location.id                = device;
  prop.allocFlags.compressionType = CU_MEM_ALLOCATION_COMP_GENERIC;

  size_t granularity;
  ERRCHK_CU_ALWAYS(hipMemGetAllocationGranularity(
      &granularity, &prop, hipMemAllocationGranularityMinimum));

  // Pad to align
  const size_t bytes = ((requested_bytes - 1) / granularity + 1) * granularity;

  hipDeviceptr_t dptr;
  ERRCHK_ALWAYS(hipMemAddressReserve(&dptr, bytes, 0, 0, 0) == hipSuccess);

  hipMemGenericAllocationHandle_t handle;
  ERRCHK_ALWAYS(hipMemCreate(&handle, bytes, &prop, 0) == hipSuccess)

  // Check if hipMemCreate was able to allocate compressible memory.
  hipMemAllocationProp alloc_prop;
  memset(&alloc_prop, 0, sizeof(hipMemAllocationProp));
  hipMemGetAllocationPropertiesFromHandle(&alloc_prop, handle);
  ERRCHK_ALWAYS(alloc_prop.allocFlags.compressionType ==
                CU_MEM_ALLOCATION_COMP_GENERIC);

  ERRCHK_ALWAYS(hipMemMap(dptr, bytes, 0, handle, 0) == hipSuccess);
  ERRCHK_ALWAYS(hipMemRelease(handle) == hipSuccess);

  hipMemAccessDesc accessDescriptor;
  accessDescriptor.location.id   = prop.location.id;
  accessDescriptor.location.type = prop.location.type;
  accessDescriptor.flags         = hipMemAccessFlagsProtReadWrite;

  ERRCHK_ALWAYS(hipMemSetAccess(dptr, bytes, &accessDescriptor, 1) ==
                hipSuccess);

  *addr = (void*)dptr;
  return hipSuccess;
}

static void
freeCompressible(void* ptr, const size_t requested_bytes)
{
  hipDevice_t device;
  ERRCHK_ALWAYS(hipCtxGetDevice(&device) == hipSuccess);

  hipMemAllocationProp prop;
  memset(&prop, 0, sizeof(hipMemAllocationProp));
  prop.type                       = hipMemAllocationTypePinned;
  prop.location.type              = hipMemLocationTypeDevice;
  prop.location.id                = device;
  prop.allocFlags.compressionType = CU_MEM_ALLOCATION_COMP_GENERIC;

  size_t granularity = 0;
  ERRCHK_ALWAYS(hipMemGetAllocationGranularity(
                    &granularity, &prop, hipMemAllocationGranularityMinimum) ==
                hipSuccess);
  const size_t bytes = ((requested_bytes - 1) / granularity + 1) * granularity;

  ERRCHK_ALWAYS(ptr);
  ERRCHK_ALWAYS(hipMemUnmap((hipDeviceptr_t)ptr, bytes) == hipSuccess);
  ERRCHK_ALWAYS(hipMemAddressFree((hipDeviceptr_t)ptr, bytes) == hipSuccess);
}
#endif

AcResult
acPBAReset(const hipStream_t stream, ProfileBufferArray* pba)
{
  // Set pba.in data to all-nan and pba.out to 0
  for (int i = 0; i < NUM_PROFILES; ++i) {
    acKernelFlush(stream, pba->in[i], pba->count, (AcReal)NAN);
    acKernelFlush(stream, pba->out[i], pba->count, (AcReal)0.0);
  }
  return AC_SUCCESS;
}

ProfileBufferArray
acPBACreate(const size_t count)
{
  ProfileBufferArray pba = {.count = count};

  const size_t bytes = sizeof(pba.in[0][0]) * pba.count * NUM_PROFILES;
  AcReal *in, *out;
#if USE_COMPRESSIBLE_MEMORY
  ERRCHK_CUDA_ALWAYS(mallocCompressible((void**)&in, bytes));
  ERRCHK_CUDA_ALWAYS(mallocCompressible((void**)&out, bytes));
#else
  ERRCHK_CUDA_ALWAYS(hipMalloc((void**)&in, bytes));
  ERRCHK_CUDA_ALWAYS(hipMalloc((void**)&out, bytes));
#endif
  for (int i = 0; i < NUM_PROFILES; ++i) {
    pba.in[i]  = &in[i * pba.count];
    pba.out[i] = &out[i * pba.count];
  }

  acPBAReset(0, &pba);
  hipDeviceSynchronize();
  return pba;
}

void
acPBADestroy(ProfileBufferArray* pba)
{
#if USE_COMPRESSIBLE_MEMORY
  freeCompressible(pba->in[0],
                   sizeof(pba.in[0][0]) * pba->count * NUM_PROFILES);
  freeCompressible(pba->out[0],
                   sizeof(pba.out[0][0]) * pba->count * NUM_PROFILES);
#else
  hipFree(pba->in[0]);
  hipFree(pba->out[0]);
#endif
  for (int i = 0; i < NUM_PROFILES; ++i) {
    pba->in[i]  = NULL;
    pba->out[i] = NULL;
  }
  pba->count = 0;
}

AcResult
acVBAReset(const hipStream_t stream, VertexBufferArray* vba)
{
  const size_t count = vba->bytes / sizeof(vba->in[0][0]);

  // Set vba.in data to all-nan and vba.out to 0
  for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
    acKernelFlush(stream, vba->in[i], count, (AcReal)NAN);
    acKernelFlush(stream, vba->out[i], count, (AcReal)0.0);
  }

  // Note: should be moved out when refactoring VBA to KernelParameterArray
  acPBAReset(stream, &vba->profiles);
  return AC_SUCCESS;
}

VertexBufferArray
acVBACreate(const size_t mx, const size_t my, const size_t mz)
{
  VertexBufferArray vba;

  const size_t bytes = sizeof(vba.in[0][0]) * mx * my * mz;
  vba.bytes          = bytes;
  vba.mx             = mx;
  vba.my             = my;
  vba.mz             = mz;

  for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
#if USE_COMPRESSIBLE_MEMORY
    ERRCHK_CUDA_ALWAYS(mallocCompressible((void**)&vba.in[i], bytes));
    ERRCHK_CUDA_ALWAYS(mallocCompressible((void**)&vba.out[i], bytes));
#else
    ERRCHK_CUDA_ALWAYS(hipMalloc((void**)&vba.in[i], bytes));
    ERRCHK_CUDA_ALWAYS(hipMalloc((void**)&vba.out[i], bytes));
#endif
  }

  // Note: should be moved out when refactoring VBA to KernelParameterArray
  vba.profiles = acPBACreate(mz);

  acVBAReset(0, &vba);
  hipDeviceSynchronize();
  return vba;
}

void
acVBADestroy(VertexBufferArray* vba)
{
  for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
#if USE_COMPRESSIBLE_MEMORY
    freeCompressible(vba->in[i], vba->bytes);
    freeCompressible(vba->out[i], vba->bytes);
#else
    hipFree(vba->in[i]);
    hipFree(vba->out[i]);
#endif
    vba->in[i]  = NULL;
    vba->out[i] = NULL;
  }
  vba->bytes = 0;
  vba->mx    = 0;
  vba->my    = 0;
  vba->mz    = 0;

  // Note: should be moved out when refactoring VBA to KernelParameterArray
  acPBADestroy(&vba->profiles);
}

AcResult
acLaunchKernel(Kernel kernel, const hipStream_t stream, const int3 start,
               const int3 end, VertexBufferArray vba)
{
  const int3 n = end - start;

  const TBConfig tbconf = getOptimalTBConfig(kernel, n, vba);
  const dim3 tpb        = tbconf.tpb;
  const int3 dims       = tbconf.dims;
  const dim3 bpg        = to_dim3(get_bpg(to_volume(dims), to_volume(tpb)));

  const size_t smem = get_smem(to_volume(tpb), STENCIL_ORDER, sizeof(AcReal));

  // hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferL1);
  kernel<<<bpg, tpb, smem, stream>>>(start, end, vba);
  ERRCHK_CUDA_KERNEL();

  last_tpb = tpb; // Note: a bit hacky way to get the tpb
  return AC_SUCCESS;
}

AcResult
acBenchmarkKernel(Kernel kernel, const int3 start, const int3 end,
                  VertexBufferArray vba)
{
  const int3 n = end - start;

  const TBConfig tbconf = getOptimalTBConfig(kernel, n, vba);
  const dim3 tpb        = tbconf.tpb;
  const int3 dims       = tbconf.dims;
  const dim3 bpg        = to_dim3(get_bpg(to_volume(dims), to_volume(tpb)));
  const size_t smem = get_smem(to_volume(tpb), STENCIL_ORDER, sizeof(AcReal));

  // Timer create
  hipEvent_t tstart, tstop;
  hipEventCreate(&tstart);
  hipEventCreate(&tstop);

  // Warmup
  hipEventRecord(tstart);
  kernel<<<bpg, tpb, smem>>>(start, end, vba);
  hipEventRecord(tstop);
  hipEventSynchronize(tstop);
  ERRCHK_CUDA_KERNEL();
  hipDeviceSynchronize();

  // Benchmark
  hipEventRecord(tstart); // Timing start
  kernel<<<bpg, tpb, smem>>>(start, end, vba);
  hipEventRecord(tstop); // Timing stop
  hipEventSynchronize(tstop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, tstart, tstop);

  size_t kernel_id = NUM_KERNELS;
  for (size_t i = 0; i < NUM_KERNELS; ++i) {
    if (kernels[i] == kernel) {
      kernel_id = i;
    }
  }
  ERRCHK_ALWAYS(kernel_id < NUM_KERNELS);
  printf("Kernel %s time elapsed: %g ms\n", kernel_names[kernel_id],
         static_cast<double>(milliseconds));

  // Timer destroy
  hipEventDestroy(tstart);
  hipEventDestroy(tstop);

  last_tpb = tpb; // Note: a bit hacky way to get the tpb
  return AC_SUCCESS;
}

AcResult
acLoadStencil(const Stencil stencil, const hipStream_t /* stream */,
              const AcReal data[STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
  ERRCHK_ALWAYS(stencil < NUM_STENCILS);

  // Note important hipDeviceSynchronize below
  //
  // Constant memory allocated for stencils is shared among kernel
  // invocations, therefore a race condition is possible when updating
  // the coefficients. To avoid this, all kernels that can access
  // the coefficients must be completed before starting async copy to
  // constant memory
  hipDeviceSynchronize();

  const size_t bytes = sizeof(data[0][0][0]) * STENCIL_DEPTH * STENCIL_HEIGHT *
                       STENCIL_WIDTH;
  const hipError_t retval = hipMemcpyToSymbol(HIP_SYMBOL(
      stencils), data, bytes, stencil * bytes, hipMemcpyHostToDevice);

  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
};

AcResult
acLoadStencils(const hipStream_t stream,
               const AcReal data[NUM_STENCILS][STENCIL_DEPTH][STENCIL_HEIGHT]
                                [STENCIL_WIDTH])
{
  int retval = 0;
  for (size_t i = 0; i < NUM_STENCILS; ++i)
    retval |= acLoadStencil((Stencil)i, stream, data[i]);
  return (AcResult)retval;
}

AcResult
acStoreStencil(const Stencil stencil, const hipStream_t /* stream */,
               AcReal data[STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
  ERRCHK_ALWAYS(stencil < NUM_STENCILS);

  // Ensure all acLoadUniform calls have completed before continuing
  hipDeviceSynchronize();

  const size_t bytes = sizeof(data[0][0][0]) * STENCIL_DEPTH * STENCIL_HEIGHT *
                       STENCIL_WIDTH;
  const hipError_t retval = hipMemcpyFromSymbol(
      data, HIP_SYMBOL(stencils), bytes, stencil * bytes, hipMemcpyDeviceToHost);

  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
};

#define GEN_LOAD_UNIFORM(LABEL_UPPER, LABEL_LOWER)                             \
  ERRCHK_ALWAYS(param < NUM_##LABEL_UPPER##_PARAMS);                           \
  hipDeviceSynchronize(); /* See note in acLoadStencil */                     \
                                                                               \
  const size_t offset = (size_t) &                                             \
                        d_mesh_info.LABEL_LOWER##_params[param] - (size_t) &   \
                        d_mesh_info;                                           \
                                                                               \
  const hipError_t retval = hipMemcpyToSymbol(HIP_SYMBOL(                               \
      d_mesh_info), &value, sizeof(value), offset, hipMemcpyHostToDevice);     \
  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;

AcResult
acLoadRealUniform(const hipStream_t /* stream */, const AcRealParam param,
                  const AcReal value)
{
  if (isnan(value)) {
    fprintf(stderr,
            "WARNING: Passed an invalid value %g to device constant %s. "
            "Skipping.\n",
            (double)value, realparam_names[param]);
    return AC_FAILURE;
  }
  GEN_LOAD_UNIFORM(REAL, real);
}

AcResult
acLoadReal3Uniform(const hipStream_t /* stream */, const AcReal3Param param,
                   const AcReal3 value)
{
  if (isnan(value.x) || isnan(value.y) || isnan(value.z)) {
    fprintf(stderr,
            "WARNING: Passed an invalid value (%g, %g, %g) to device constant "
            "%s. Skipping.\n",
            (double)value.x, (double)value.y, (double)value.z,
            real3param_names[param]);
    return AC_FAILURE;
  }
  GEN_LOAD_UNIFORM(REAL3, real3);
}

AcResult
acLoadIntUniform(const hipStream_t /* stream */, const AcIntParam param,
                 const int value)
{
  GEN_LOAD_UNIFORM(INT, int);
}

AcResult
acLoadInt3Uniform(const hipStream_t /* stream */, const AcInt3Param param,
                  const int3 value)
{
  GEN_LOAD_UNIFORM(INT3, int3);
}

#define GEN_STORE_UNIFORM(LABEL_UPPER, LABEL_LOWER)                            \
  ERRCHK_ALWAYS(param < NUM_##LABEL_UPPER##_PARAMS);                           \
  hipDeviceSynchronize(); /* See notes in GEN_LOAD_UNIFORM */                 \
                                                                               \
  const size_t offset = (size_t) &                                             \
                        d_mesh_info.LABEL_LOWER##_params[param] - (size_t) &   \
                        d_mesh_info;                                           \
                                                                               \
  const hipError_t retval = hipMemcpyFromSymbol(                             \
      value, HIP_SYMBOL(d_mesh_info), sizeof(*value), offset, hipMemcpyDeviceToHost);     \
  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;

AcResult
acStoreRealUniform(const hipStream_t /* stream */, const AcRealParam param,
                   AcReal* value)
{
  GEN_STORE_UNIFORM(REAL, real);
}

AcResult
acStoreReal3Uniform(const hipStream_t /* stream */, const AcReal3Param param,
                    AcReal3* value)
{
  GEN_STORE_UNIFORM(REAL3, real3);
}

AcResult
acStoreIntUniform(const hipStream_t /* stream */, const AcIntParam param,
                  int* value)
{
  GEN_STORE_UNIFORM(INT, int);
}

AcResult
acStoreInt3Uniform(const hipStream_t /* stream */, const AcInt3Param param,
                   int3* value)
{
  GEN_STORE_UNIFORM(INT3, int3);
}

static TBConfig
autotune(const Kernel kernel, const int3 dims, VertexBufferArray vba)
{
  size_t id = (size_t)-1;
  for (size_t i = 0; i < NUM_KERNELS; ++i) {
    if (kernels[i] == kernel) {
      id = i;
      break;
    }
  }
  ERRCHK_ALWAYS(id < NUM_KERNELS);
  // printf("Autotuning kernel '%s' (%p), block (%d, %d, %d), implementation "
  //        "(%d):\n",
  //        kernel_names[id], kernel, dims.x, dims.y, dims.z, IMPLEMENTATION);
  // fflush(stdout);

#if 0
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  size_t size = min(int(prop.l2CacheSize * 0.75), prop.persistingL2CacheMaxSize);
  hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, size);
  // set-aside 3/4 of L2 cache for persisting accesses or the max allowed
#endif

  TBConfig c = {
      .kernel = kernel,
      .dims   = dims,
      .tpb    = (dim3){0, 0, 0},
  };

  const int3 start = (int3){
      STENCIL_ORDER / 2,
      STENCIL_ORDER / 2,
      STENCIL_ORDER / 2,
  };
  const int3 end = start + dims;

  dim3 best_tpb(0, 0, 0);
  float best_time     = INFINITY;
  const int num_iters = 2;

  // Get device hardware information
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  const int max_threads_per_block = MAX_THREADS_PER_BLOCK
                                        ? min(props.maxThreadsPerBlock,
                                              MAX_THREADS_PER_BLOCK)
                                        : props.maxThreadsPerBlock;
  const size_t max_smem           = props.sharedMemPerBlock;

  // Old heuristic
  // for (int z = 1; z <= max_threads_per_block; ++z) {
  //   for (int y = 1; y <= max_threads_per_block; ++y) {
  //     for (int x = max(y, z); x <= max_threads_per_block; ++x) {

  // New: require that tpb.x is a multiple of the minimum transaction or L2
  // cache line size
  for (int z = 1; z <= max_threads_per_block; ++z) {
    for (int y = 1; y <= max_threads_per_block; ++y) {
      // 64 bytes on NVIDIA but the minimum L1 cache transaction is 32
      const int minimum_transaction_size_in_elems = 32 / sizeof(AcReal);
      for (int x = minimum_transaction_size_in_elems;
           x <= max_threads_per_block; x += minimum_transaction_size_in_elems) {

        if (x * y * z > max_threads_per_block)
          break;

        // if (x * y * z * max_regs_per_thread > max_regs_per_block)
        //  break;

        // if (max_regs_per_block / (x * y * z) < min_regs_per_thread)
        //   continue;

        // if (x < y || x < z)
        //   continue;

        const dim3 tpb(x, y, z);
        const dim3 bpg    = to_dim3(get_bpg(to_volume(dims), to_volume(tpb)));
        const size_t smem = get_smem(to_volume(tpb), STENCIL_ORDER,
                                     sizeof(AcReal));

        if (smem > max_smem)
          continue;

        if ((x * y * z) % props.warpSize)
          continue;

        if (!is_valid_configuration(to_volume(dims), to_volume(tpb)))
          continue;

        // #if VECTORIZED_LOADS
        //         const size_t window = tpb.x + STENCIL_ORDER;

        //         // Vectorization criterion
        //         if (window % veclen) // Window not divisible into vectorized
        //         blocks
        //           continue;

        //         if (dims.x % tpb.x)
        //           continue;

        //           // May be too strict
        //           // if (dims.x % tpb.x || dims.y % tpb.y || dims.z % tpb.z)
        //           //   continue;
        // #endif
        // #if 0 // Disabled for now (waiting for cleanup)
        // #if USE_SMEM
        //         const size_t max_smem = 128 * 1024;
        //         if (smem > max_smem)
        //           continue;

        // #if VECTORIZED_LOADS
        //         const size_t window = tpb.x + STENCIL_ORDER;

        //         // Vectorization criterion
        //         if (window % veclen) // Window not divisible into vectorized
        //         blocks
        //           continue;

        //         if (dims.x % tpb.x || dims.y % tpb.y || dims.z % tpb.z)
        //           continue;
        // #endif

        //           //  Padding criterion
        //           //  TODO (cannot be checked here)
        // #else
        //         if ((x * y * z) % warp_size)
        //           continue;
        // #endif
        // #endif

        // printf("%d, %d, %d: %lu\n", tpb.x, tpb.y, tpb.z, smem);

        hipEvent_t tstart, tstop;
        hipEventCreate(&tstart);
        hipEventCreate(&tstop);

        kernel<<<bpg, tpb, smem>>>(start, end, vba); // Dryrun
        hipDeviceSynchronize();
        hipEventRecord(tstart); // Timing start
        for (int i = 0; i < num_iters; ++i)
          kernel<<<bpg, tpb, smem>>>(start, end, vba);
        hipEventRecord(tstop); // Timing stop
        hipEventSynchronize(tstop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, tstart, tstop);

        hipEventDestroy(tstart);
        hipEventDestroy(tstop);

        // Discard failed runs (attempt to clear the error to hipSuccess)
        if (hipGetLastError() != hipSuccess) {
          // Exit in case of unrecoverable error that needs a device reset
          ERRCHK_CUDA_KERNEL_ALWAYS();
          ERRCHK_CUDA_ALWAYS(hipGetLastError());
          continue;
        }

        if (milliseconds < best_time) {
          best_time = milliseconds;
          best_tpb  = tpb;
        }

        // printf("Auto-optimizing... Current tpb: (%d, %d, %d), time %f ms\n",
        //        tpb.x, tpb.y, tpb.z, (double)milliseconds / num_iters);
        // fflush(stdout);
      }
    }
  }
  c.tpb = best_tpb;

  // printf("\tThe best tpb: (%d, %d, %d), time %f ms\n", best_tpb.x,
  // best_tpb.y,
  //        best_tpb.z, (double)best_time / num_iters);

  FILE* fp = fopen("autotune.csv", "a");
  ERRCHK_ALWAYS(fp);
#if IMPLEMENTATION == SMEM_HIGH_OCCUPANCY_CT_CONST_TB
  fprintf(fp, "%d, (%d, %d, %d), (%d, %d, %d), %g\n", IMPLEMENTATION, nx, ny,
          nz, best_tpb.x, best_tpb.y, best_tpb.z,
          (double)best_time / num_iters);
#else
  fprintf(fp, "%d, %d, %d, %d, %d, %d, %d, %g\n", IMPLEMENTATION, dims.x,
          dims.y, dims.z, best_tpb.x, best_tpb.y, best_tpb.z,
          (double)best_time / num_iters);
#endif
  fclose(fp);

  if (c.tpb.x * c.tpb.y * c.tpb.z <= 0) {
    fprintf(stderr,
            "Fatal error: failed to find valid thread block dimensions.\n");
  }
  ERRCHK_ALWAYS(c.tpb.x * c.tpb.y * c.tpb.z > 0);
  return c;
}

static TBConfig
getOptimalTBConfig(const Kernel kernel, const int3 dims, VertexBufferArray vba)
{
  for (auto c : tbconfigs) {
    if (c.kernel == kernel && c.dims == dims)
      return c;
  }
  TBConfig c = autotune(kernel, dims, vba);
  tbconfigs.push_back(c);
  return c;
}

void
acVBASwapBuffer(const Field field, VertexBufferArray* vba)
{
  AcReal* tmp     = vba->in[field];
  vba->in[field]  = vba->out[field];
  vba->out[field] = tmp;
}

void
acVBASwapBuffers(VertexBufferArray* vba)
{
  for (size_t i = 0; i < NUM_FIELDS; ++i)
    acVBASwapBuffer((Field)i, vba);
}

void
acPBASwapBuffer(const Profile profile, VertexBufferArray* vba)
{
  AcReal* tmp                = vba->profiles.in[profile];
  vba->profiles.in[profile]  = vba->profiles.out[profile];
  vba->profiles.out[profile] = tmp;
}

void
acPBASwapBuffers(VertexBufferArray* vba)
{
  for (int i = 0; i < NUM_PROFILES; ++i)
    acPBASwapBuffer((Profile)i, vba);
}

AcResult
acLoadMeshInfo(const AcMeshInfo info, const hipStream_t stream)
{
  for (int i = 0; i < NUM_INT_PARAMS; ++i)
    ERRCHK_ALWAYS(acLoadIntUniform(stream, (AcIntParam)i, info.int_params[i]) ==
                  AC_SUCCESS);

  for (int i = 0; i < NUM_INT3_PARAMS; ++i)
    ERRCHK_ALWAYS(acLoadInt3Uniform(stream, (AcInt3Param)i,
                                    info.int3_params[i]) == AC_SUCCESS);

  for (int i = 0; i < NUM_REAL_PARAMS; ++i)
    ERRCHK_ALWAYS(acLoadRealUniform(stream, (AcRealParam)i,
                                    info.real_params[i]) == AC_SUCCESS);

  for (int i = 0; i < NUM_REAL3_PARAMS; ++i)
    ERRCHK_ALWAYS(acLoadReal3Uniform(stream, (AcReal3Param)i,
                                     info.real3_params[i]) == AC_SUCCESS);

  return AC_SUCCESS;
}

//---------------
// static __host__ __device__ constexpr size_t
// acShapeSize(const AcShape& shape)
size_t
acShapeSize(const AcShape shape)
{
  return shape.x * shape.y * shape.z * shape.w;
}

static __host__ __device__ constexpr bool
acOutOfBounds(const AcIndex& index, const AcShape& shape)
{
  return (index.x >= shape.x) || //
         (index.y >= shape.y) || //
         (index.z >= shape.z) || //
         (index.w >= shape.w);
}

static __host__ __device__ constexpr AcIndex
min(const AcIndex& a, const AcIndex& b)
{
  return (AcIndex){
      a.x < b.x ? a.x : b.x,
      a.y < b.y ? a.y : b.y,
      a.z < b.z ? a.z : b.z,
      a.w < b.w ? a.w : b.w,
  };
}

static __host__ __device__ constexpr AcIndex
operator+(const AcIndex& a, const AcIndex& b)
{
  return (AcIndex){
      a.x + b.x,
      a.y + b.y,
      a.z + b.z,
      a.w + b.w,
  };
}

static __host__ __device__ constexpr AcIndex
operator-(const AcIndex& a, const AcIndex& b)
{
  return (AcIndex){
      a.x - b.x,
      a.y - b.y,
      a.z - b.z,
      a.w - b.w,
  };
}

static __host__ __device__ constexpr AcIndex
to_spatial(const size_t i, const AcShape& shape)
{
  return (AcIndex){
      .x = i % shape.x,
      .y = (i / shape.x) % shape.y,
      .z = (i / (shape.x * shape.y)) % shape.z,
      .w = i / (shape.x * shape.y * shape.z),
  };
}

static __host__ __device__ constexpr size_t
to_linear(const AcIndex& index, const AcShape& shape)
{
  return index.x +           //
         index.y * shape.x + //
         index.z * shape.x * shape.y + index.w * shape.x * shape.y * shape.z;
}

static __global__ void
reindex(const AcReal* in, const AcIndex in_offset, const AcShape in_shape,
        AcReal* out, const AcIndex out_offset, const AcShape out_shape,
        const AcShape block_shape)
{
  const size_t i    = (size_t)threadIdx.x + blockIdx.x * blockDim.x;
  const AcIndex idx = to_spatial(i, block_shape);

  const AcIndex in_pos  = idx + in_offset;
  const AcIndex out_pos = idx + out_offset;

  if (acOutOfBounds(idx, block_shape) || //
      acOutOfBounds(in_pos, in_shape) || //
      acOutOfBounds(out_pos, out_shape))
    return;

  const size_t in_idx  = to_linear(in_pos, in_shape);
  const size_t out_idx = to_linear(out_pos, out_shape);

  out[out_idx] = in[in_idx];
}

AcResult
acReindex(const hipStream_t stream, //
          const AcReal* in, const AcIndex in_offset, const AcShape in_shape,
          AcReal* out, const AcIndex out_offset, const AcShape out_shape,
          const AcShape block_shape)
{
  const size_t count = acShapeSize(block_shape);
  const size_t tpb   = min(256ul, count);
  const size_t bpg   = (count + tpb - 1) / tpb;

  reindex<<<bpg, tpb, 0, stream>>>(in, in_offset, in_shape, //
                                   out, out_offset, out_shape, block_shape);
  ERRCHK_CUDA_KERNEL();

  return AC_SUCCESS;
}

typedef struct {
  AcReal *x, *y, *z;
} SOAVector;

typedef struct {
  // Input vectors
  SOAVector A[1];
  size_t A_count;
  SOAVector B[4];
  size_t B_count;
  // Note: more efficient with A_count < B_count

  // Output vectors
  SOAVector C[1 * 4];
  // C count = A_count*B_count
} CrossProductArrays;

static __global__ void
reindex_cross(const CrossProductArrays arrays, const AcIndex in_offset,
              const AcShape in_shape, const AcIndex out_offset,
              const AcShape out_shape, const AcShape block_shape)
{
  const AcIndex idx = to_spatial(static_cast<size_t>(threadIdx.x) + blockIdx.x * blockDim.x, block_shape);

  const AcIndex in_pos  = idx + in_offset;
  const AcIndex out_pos = idx + out_offset;

  if (acOutOfBounds(idx, block_shape) || //
      acOutOfBounds(in_pos, in_shape) || //
      acOutOfBounds(out_pos, out_shape))
    return;

  const size_t in_idx  = to_linear(in_pos, in_shape);
  const size_t out_idx = to_linear(out_pos, out_shape);

  for (size_t j = 0; j < arrays.A_count; ++j) {
    const AcReal3 a = {
        arrays.A[j].x[in_idx],
        arrays.A[j].y[in_idx],
        arrays.A[j].z[in_idx],
    };
    for (size_t i = 0; i < arrays.B_count; ++i) {
      const AcReal3 b = {
          arrays.B[i].x[in_idx],
          arrays.B[i].y[in_idx],
          arrays.B[i].z[in_idx],
      };
      const AcReal3 res                           = cross(a, b);
      arrays.C[i + j * arrays.B_count].x[out_idx] = res.x;
      arrays.C[i + j * arrays.B_count].y[out_idx] = res.y;
      arrays.C[i + j * arrays.B_count].z[out_idx] = res.z;
    }
  }
}

#if 0
__global__ void
map_cross_product(const CrossProductInputs inputs, const AcIndex start,
                  const AcIndex end)
{

  const AcIndex tid = {
      .x = threadIdx.x + blockIdx.x * blockDim.x,
      .y = threadIdx.y + blockIdx.y * blockDim.y,
      .z = threadIdx.z + blockIdx.z * blockDim.z,
      .w = 0,
  };

  const AcIndex in_idx3d = start + tid;
  const size_t in_idx = DEVICE_VTXBUF_IDX(in_idx3d.x, in_idx3d.y, in_idx3d.z);

  const AcShape dims   = end - start;
  const size_t out_idx = tid.x + tid.y * dims.x + tid.z * dims.x * dims.y;

  const bool within_bounds = in_idx3d.x < end.x && in_idx3d.y < end.y &&
                             in_idx3d.z < end.z;
  if (within_bounds) {
    for (size_t i = 0; i < inputs.A_count; ++i) {
      const AcReal3 a = (AcReal3){
          inputs.A[i].x[in_idx],
          inputs.A[i].y[in_idx],
          inputs.A[i].z[in_idx],
      };
      for (size_t j = 0; j < inputs.B_count; ++j) {
        const AcReal3 b = (AcReal3){
            inputs.B[j].x[in_idx],
            inputs.B[j].y[in_idx],
            inputs.B[j].z[in_idx],
        };
        const AcReal3 res            = cross(a, b);
        inputs.outputs[j].x[out_idx] = res.x;
        inputs.outputs[j].y[out_idx] = res.y;
        inputs.outputs[j].z[out_idx] = res.z;
      }
    }
  }
}
#endif

AcResult
acReindexCross(const hipStream_t stream, //
               const VertexBufferArray vba, const AcIndex in_offset,
               const AcShape in_shape, //
               AcReal* out, const AcIndex out_offset, const AcShape out_shape,
               const AcShape block_shape)
{
#if 0 // ifdef AC_TFM_ENABLED
  const SOAVector uu = {
      .x = vba.in[VTXBUF_UUX],
      .y = vba.in[VTXBUF_UUY],
      .z = vba.in[VTXBUF_UUZ],
  };
  const SOAVector bb11 = {
      .x = vba.in[TF_b11_x],
      .y = vba.in[TF_b11_y],
      .z = vba.in[TF_b11_z],
  };
  const SOAVector bb12 = {
      .x = vba.in[TF_b12_x],
      .y = vba.in[TF_b12_y],
      .z = vba.in[TF_b12_z],
  };
  const SOAVector bb21 = {
      .x = vba.in[TF_b21_x],
      .y = vba.in[TF_b21_y],
      .z = vba.in[TF_b21_z],
  };
  const SOAVector bb22 = {
      .x = vba.in[TF_b22_x],
      .y = vba.in[TF_b22_y],
      .z = vba.in[TF_b22_z],
  };

  const size_t block_offset = out_shape.x * out_shape.y * out_shape.z;
  const SOAVector out_bb11  = {
       .x = &out[3 * block_offset],
       .y = &out[4 * block_offset],
       .z = &out[5 * block_offset],
  };
  const SOAVector out_bb12 = {
      .x = &out[6 * block_offset],
      .y = &out[7 * block_offset],
      .z = &out[8 * block_offset],
  };
  const SOAVector out_bb21 = {
      .x = &out[9 * block_offset],
      .y = &out[10 * block_offset],
      .z = &out[11 * block_offset],
  };
  const SOAVector out_bb22 = {
      .x = &out[12 * block_offset],
      .y = &out[13 * block_offset],
      .z = &out[14 * block_offset],
  };

  const CrossProductArrays arrays = {
      .A       = {uu},
      .A_count = 1,
      .B       = {bb11, bb12, bb21, bb22},
      .B_count = 4,
      .C       = {out_bb11, out_bb12, out_bb21, out_bb22},
  };

  const size_t count = acShapeSize(block_shape);
  const size_t tpb   = min(256ul, count);
  const size_t bpg   = (count + tpb - 1) / tpb;

  reindex_cross<<<bpg, tpb, 0, stream>>>(arrays, in_offset, in_shape,
                                         out_offset, out_shape, block_shape);
  return AC_SUCCESS;
#else
  ERROR("acReindexCross called but AC_TFM_ENABLED was false");
  (void)stream; // Unused
  (void)vba; // Unused
  (void)in_offset; // Unused
  (void)in_shape; // Unused
  (void)out; // Unused
  (void)out_offset; // Unused
  (void)out_shape; // Unused
  (void)block_shape; // Unused
  (void)reindex_cross; // Unused
  return AC_FAILURE;
#endif
}

#if AC_USE_HIP
#include <hipcub/hipcub.hpp>
#define cub hipcub
#else
#include <hipcub/hipcub.hpp>
#endif

AcResult
acSegmentedReduce(const hipStream_t stream, const AcReal* d_in,
                  const size_t count, const size_t num_segments, AcReal* d_out)
{
  size_t* offsets = (size_t*)malloc(sizeof(offsets[0]) * (num_segments + 1));
  ERRCHK_ALWAYS(offsets);
  for (size_t i = 0; i <= num_segments; ++i) {
    offsets[i] = i * (count / num_segments);
    // printf("Offset %zu: %zu\n", i, offsets[i]);
  }

  size_t* d_offsets = NULL;
  hipMalloc(&d_offsets, sizeof(d_offsets[0]) * (num_segments + 1));
  ERRCHK_ALWAYS(d_offsets);
  hipMemcpy(d_offsets, offsets, sizeof(d_offsets[0]) * (num_segments + 1),
             hipMemcpyHostToDevice);

  void* d_temp_storage      = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, d_in,
                                  d_out, num_segments, d_offsets, d_offsets + 1,
                                  stream);
  // printf("Temp storage: %zu bytes\n", temp_storage_bytes);
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  ERRCHK_ALWAYS(d_temp_storage);

  hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, d_in,
                                  d_out, num_segments, d_offsets, d_offsets + 1,
                                  stream);

  hipStreamSynchronize(
      stream); // Note, would not be needed if allocated at initialization
  hipFree(d_temp_storage);
  hipFree(d_offsets);
  free(offsets);
  return AC_SUCCESS;
}

static __global__ void
multiply_inplace(const AcReal value, const size_t count, AcReal* array)
{
  const size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < count)
    array[idx] *= value;
}

AcResult
acMultiplyInplace(const AcReal value, const size_t count, AcReal* array)
{
  const size_t tpb = 256;
  const size_t bpg = (count + tpb - 1) / tpb;
  multiply_inplace<<<bpg, tpb>>>(value, count, array);
  ERRCHK_CUDA_KERNEL();
  ERRCHK_CUDA(hipDeviceSynchronize()); // NOTE: explicit sync here for safety
  return AC_SUCCESS;
}

int
acVerifyMeshInfo(const AcMeshInfo info)
{
  int retval = 0;
  for (size_t i = 0; i < NUM_INT_PARAMS; ++i) {
    if (info.int_params[i] == INT_MIN) {
      retval = -1;
      fprintf(stderr, "--- Warning: [%s] uninitialized ---\n",
              intparam_names[i]);
    }
  }
  for (size_t i = 0; i < NUM_INT3_PARAMS; ++i) {
    if (info.int3_params[i].x == INT_MIN || info.int3_params[i].y == INT_MIN ||
        info.int3_params[i].z == INT_MIN) {
      retval = -1;
      fprintf(stderr, "--- Warning: [%s] uninitialized ---\n",
              int3param_names[i]);
    }
  }
  for (size_t i = 0; i < NUM_REAL_PARAMS; ++i) {
    if (isnan(info.real_params[i])) {
      retval = -1;
      fprintf(stderr, "--- Warning: [%s] uninitialized ---\n",
              realparam_names[i]);
    }
  }
  for (int i = 0; i < NUM_REAL3_PARAMS; ++i) {
    if (isnan(info.real3_params[i].x) || isnan(info.real3_params[i].y) ||
        isnan(info.real3_params[i].z)) {
      retval = -1;
      fprintf(stderr, "--- Warning: [%s] uninitialized ---\n",
              real3param_names[i]);
    }
  }
  return retval;
}
