#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2021, Johannes Pekkila, Miikka Vaisala.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "acc_runtime.h"

#include <vector> // tbconfig

#include "errchk.h"
#include "math_utils.h"
#include <functional>

#if AC_USE_HIP
#include <hip/hip_runtime.h> // Needed in files that include kernels
#endif

#define USE_COMPRESSIBLE_MEMORY (0)

#include "acc/implementation.h"

static dim3 last_tpb = (dim3){0, 0, 0};

KernelLambda
kernel_to_kernel_lambda(const Kernel kernel)
{
  kernel_lambda k_l = [kernel](const dim3 bpg,  const dim3 tpb, const size_t smem, const hipStream_t stream, const int3 start, const int3 end, VertexBufferArray vba_in)
                      {kernel<<<bpg, tpb, smem, stream>>>(start,end,vba_in);};
  return {k_l, reinterpret_cast<void*>(kernel)};
};


#define GEN_BIND_SINGLE(TYPE)                                                  \
  KernelLambda bind_single_param(void (*kernel)(const int3 start, const int3 end, VertexBufferArray vba, TYPE input_param), TYPE input_param) \
  { \
  return (KernelLambda){[kernel, input_param](const dim3 bpg, const dim3 tpb, const size_t smem, const hipStream_t stream, const int3 start, const int3 end, VertexBufferArray vba){kernel<<<bpg,tpb,smem,stream>>>(start,end,vba,input_param);}, reinterpret_cast<void*>(kernel)}; \
  } 

GEN_BIND_SINGLE(int)
GEN_BIND_SINGLE(AcReal)
GEN_BIND_SINGLE(AcReal*)
GEN_BIND_SINGLE(int*)
GEN_BIND_SINGLE(bool)
GEN_BIND_SINGLE(bool*)

template <typename T, typename F>
KernelLambda
bind_two_params(void (*kernel)(const int3 start, const int3 end, VertexBufferArray vba, T input_param, F second_input_param), T input_param, F second_input_param)
{
  return (KernelLambda){[kernel, input_param, second_input_param](const dim3 bpg, const dim3 tpb, const size_t smem, const hipStream_t stream, const int3 start, const int3 end, VertexBufferArray vba){kernel<<<bpg,tpb,smem,stream>>>(start,end,vba,input_param, second_input_param);}, reinterpret_cast<void*>(kernel)};
}
template <typename T, typename F, typename H>
KernelLambda
bind_three_params(void (*kernel)(const int3 start, const int3 end, VertexBufferArray vba, T input_param, F second_input_param, H third_input_param), T input_param, F second_input_param, H third_input_param)
{
  return (KernelLambda){[kernel, input_param, second_input_param, third_input_param](const dim3 bpg, const dim3 tpb, const size_t smem, const hipStream_t stream, const int3 start, const int3 end, VertexBufferArray vba){kernel<<<bpg,tpb,smem,stream>>>(start,end,vba,input_param, second_input_param, third_input_param);}, reinterpret_cast<void*>(kernel)};
}
Volume
acKernelLaunchGetLastTPB(void)
{
  return to_volume(last_tpb);
}

Volume
get_bpg(const Volume dims, const Volume tpb)
{
  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING:             // Fallthrough
  case EXPLICIT_CACHING:             // Fallthrough
  case EXPLICIT_CACHING_3D_BLOCKING: // Fallthrough
  case EXPLICIT_CACHING_4D_BLOCKING: // Fallthrough
  case EXPLICIT_PINGPONG_txw:        // Fallthrough
  case EXPLICIT_PINGPONG_txy:        // Fallthrough
  case EXPLICIT_PINGPONG_txyblocked: // Fallthrough
  case EXPLICIT_PINGPONG_txyz:       // Fallthrough
  case EXPLICIT_ROLLING_PINGPONG: {
    return (Volume){
        (size_t)ceil(1. * dims.x / tpb.x),
        (size_t)ceil(1. * dims.y / tpb.y),
        (size_t)ceil(1. * dims.z / tpb.z),
    };
  }
  default: {
    ERROR("Invalid IMPLEMENTATION in get_bpg");
    return (Volume){0, 0, 0};
  }
  }
}

bool
is_valid_configuration(const Volume dims, const Volume tpb)
{
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  const size_t warp_size = props.warpSize;
  const size_t xmax      = (size_t)(warp_size * ceil(1. * dims.x / warp_size));
  const size_t ymax      = (size_t)(warp_size * ceil(1. * dims.y / warp_size));
  const size_t zmax      = (size_t)(warp_size * ceil(1. * dims.z / warp_size));
  const bool too_large   = (tpb.x > xmax) || (tpb.y > ymax) || (tpb.z > zmax);

  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING: {

    if (too_large)
      return false;

    return true;
  }
  case EXPLICIT_CACHING_4D_BLOCKING: // Fallthrough
    if (tpb.z > 1)
      return false;
  case EXPLICIT_CACHING: // Fallthrough
  case EXPLICIT_CACHING_3D_BLOCKING: {

    // For some reason does not work without this
    // Probably because of break vs continue when fetching (some threads
    // quit too early if the dims are not divisible)
    return !(dims.x % tpb.x) && !(dims.y % tpb.y) && !(dims.z % tpb.z);
  }
  case EXPLICIT_PINGPONG_txw: {
    return (tpb.y == 1) && (tpb.z == 1);
  }
  case EXPLICIT_PINGPONG_txy: {
    return (tpb.z == 1);
  }
  case EXPLICIT_PINGPONG_txyblocked: {
    return (tpb.z == 1);
  }
  case EXPLICIT_PINGPONG_txyz: {
    return true;
  }
  case EXPLICIT_ROLLING_PINGPONG: {
    // OK for every other rolling pingpong implementation
    // return true;

    // Required only when unrolling smem loads
    // Ensures two unrolls is enough to fill the smem buffer
    return (2 * tpb.x >= STENCIL_WIDTH - 1 + tpb.x) &&
           (2 * tpb.y >= STENCIL_HEIGHT - 1 + tpb.y);
  }
  default: {
    ERROR("Invalid IMPLEMENTATION in is_valid_configuration");
    return false;
  }
  }
}

size_t
get_smem(const Volume tpb, const size_t stencil_order,
         const size_t bytes_per_elem)
{
  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING: {
    return 0;
  }
  case EXPLICIT_CACHING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) * tpb.z *
           bytes_per_elem;
  }
  case EXPLICIT_CACHING_3D_BLOCKING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) *
           (tpb.z + stencil_order) * bytes_per_elem;
  }
  case EXPLICIT_CACHING_4D_BLOCKING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) * tpb.z *
           (NUM_FIELDS)*bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txw: {
    return 2 * (tpb.x + stencil_order) * NUM_FIELDS * bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txy: {
    return 2 * (tpb.x + stencil_order) * (tpb.y + stencil_order) *
           bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txyblocked: {
    const size_t block_size = 7;
    return 2 * (tpb.x + stencil_order) * (tpb.y + stencil_order) * block_size *
           bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txyz: {
    return 2 * (tpb.x + stencil_order) * (tpb.y + stencil_order) *
           (tpb.z + stencil_order) * bytes_per_elem;
  }
  case EXPLICIT_ROLLING_PINGPONG: {
    // tpbxy slices with halos
    // tpbz depth + 1 rolling cache slab
    return EXPLICIT_ROLLING_PINGPONG_BLOCKSIZE * (tpb.x + stencil_order) *
           (tpb.y + stencil_order) * (tpb.z + 1) * bytes_per_elem;
  }
  default: {
    ERROR("Invalid IMPLEMENTATION in get_smem");
    return (size_t)-1;
  }
  }
}

/*
// Device info (TODO GENERIC)
// Use the maximum available reg count per thread
#define REGISTERS_PER_THREAD (255)
#define MAX_REGISTERS_PER_BLOCK (65536)
#if AC_DOUBLE_PRECISION
#define MAX_THREADS_PER_BLOCK                                                  \
  (MAX_REGISTERS_PER_BLOCK / REGISTERS_PER_THREAD / 2)
#else
#define MAX_THREADS_PER_BLOCK (MAX_REGISTERS_PER_BLOCK / REGISTERS_PER_THREAD)
#endif
*/

__device__ __constant__ AcMeshInfo d_mesh_info;

// Astaroth 2.0 backwards compatibility START
#define d_multigpu_offset (d_mesh_info.int3_params[AC_multigpu_offset])

int __device__ __forceinline__
DCONST(const AcIntParam param)
{
  return d_mesh_info.int_params[param];
}
int3 __device__ __forceinline__
DCONST(const AcInt3Param param)
{
  return d_mesh_info.int3_params[param];
}
AcReal __device__ __forceinline__
DCONST(const AcRealParam param)
{
  return d_mesh_info.real_params[param];
}
AcReal3 __device__ __forceinline__
DCONST(const AcReal3Param param)
{
  return d_mesh_info.real3_params[param];
}

#define DEVICE_VTXBUF_IDX(i, j, k)                                             \
  ((i) + (j)*DCONST(AC_mx) + (k)*DCONST(AC_mxy))

__device__ int
LOCAL_COMPDOMAIN_IDX(const int3 coord)
{
  return (coord.x) + (coord.y) * DCONST(AC_nx) + (coord.z) * DCONST(AC_nxy);
}

__device__ constexpr int
IDX(const int i)
{
  return i;
}

#if 1
__device__ __forceinline__ int
IDX(const int i, const int j, const int k)
{
  return DEVICE_VTXBUF_IDX(i, j, k);
}
#else
constexpr __device__ int
IDX(const uint i, const uint j, const uint k)
{
  /*
  const int precision   = 32; // Bits
  const int dimensions  = 3;
  const int bits = ceil(precision / dimensions);
  */
  const int dimensions = 3;
  const int bits       = 11;

  uint idx = 0;
#pragma unroll
  for (uint bit = 0; bit < bits; ++bit) {
    const uint mask = 0b1 << bit;
    idx |= ((i & mask) << 0) << (dimensions - 1) * bit;
    idx |= ((j & mask) << 1) << (dimensions - 1) * bit;
    idx |= ((k & mask) << 2) << (dimensions - 1) * bit;
  }
  return idx;
}
#endif

// Only used in reductions
__device__ __forceinline__ int
IDX(const int3 idx)
{
  return DEVICE_VTXBUF_IDX(idx.x, idx.y, idx.z);
}

#define Field3(x, y, z) make_int3((x), (y), (z))
#define print printf                          // TODO is this a good idea?
#define len(arr) sizeof(arr) / sizeof(arr[0]) // Leads to bugs if the user
// passes an array into a device function and then calls len (need to modify
// the compiler to always pass arrays to functions as references before
// re-enabling)

#include "random.cuh"

#include "user_kernels.h"

typedef struct {
  void* kernel;
  int3 dims;
  dim3 tpb;
} TBConfig;

static std::vector<TBConfig> tbconfigs;

static TBConfig getOptimalTBConfig(const Kernel kernel, const int3 dims,
                                   VertexBufferArray vba);

static TBConfig getOptimalTBConfig(const KernelLambda lambda, const int3 dims, VertexBufferArray vba);

static __global__ void
flush_kernel(AcReal* arr, const size_t n, const AcReal value)
{
  const size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n)
    arr[idx] = value;
}

AcResult
acKernelFlush(const hipStream_t stream, AcReal* arr, const size_t n,
              const AcReal value)
{
  const size_t tpb = 256;
  const size_t bpg = (size_t)(ceil((double)n / tpb));
  flush_kernel<<<bpg, tpb, 0, stream>>>(arr, n, value);
  ERRCHK_CUDA_KERNEL_ALWAYS();
  return AC_SUCCESS;
}

#if USE_COMPRESSIBLE_MEMORY
#include <hip/hip_runtime.h>

#define ERRCHK_CU_ALWAYS(x) ERRCHK_ALWAYS((x) == hipSuccess)

static hipError_t
mallocCompressible(void** addr, const size_t requested_bytes)
{
  hipDevice_t device;
  ERRCHK_ALWAYS(hipCtxGetDevice(&device) == hipSuccess);

  hipMemAllocationProp prop;
  memset(&prop, 0, sizeof(hipMemAllocationProp));
  prop.type                       = hipMemAllocationTypePinned;
  prop.location.type              = hipMemLocationTypeDevice;
  prop.location.id                = device;
  prop.allocFlags.compressionType = CU_MEM_ALLOCATION_COMP_GENERIC;

  size_t granularity;
  ERRCHK_CU_ALWAYS(hipMemGetAllocationGranularity(
      &granularity, &prop, hipMemAllocationGranularityMinimum));

  // Pad to align
  const size_t bytes = ((requested_bytes - 1) / granularity + 1) * granularity;

  hipDeviceptr_t dptr;
  ERRCHK_ALWAYS(hipMemAddressReserve(&dptr, bytes, 0, 0, 0) == hipSuccess);

  hipMemGenericAllocationHandle_t handle;
  ERRCHK_ALWAYS(hipMemCreate(&handle, bytes, &prop, 0) == hipSuccess)

  // Check if hipMemCreate was able to allocate compressible memory.
  hipMemAllocationProp alloc_prop;
  memset(&alloc_prop, 0, sizeof(hipMemAllocationProp));
  hipMemGetAllocationPropertiesFromHandle(&alloc_prop, handle);
  ERRCHK_ALWAYS(alloc_prop.allocFlags.compressionType ==
                CU_MEM_ALLOCATION_COMP_GENERIC);

  ERRCHK_ALWAYS(hipMemMap(dptr, bytes, 0, handle, 0) == hipSuccess);
  ERRCHK_ALWAYS(hipMemRelease(handle) == hipSuccess);

  hipMemAccessDesc accessDescriptor;
  accessDescriptor.location.id   = prop.location.id;
  accessDescriptor.location.type = prop.location.type;
  accessDescriptor.flags         = hipMemAccessFlagsProtReadWrite;

  ERRCHK_ALWAYS(hipMemSetAccess(dptr, bytes, &accessDescriptor, 1) ==
                hipSuccess);

  *addr = (void*)dptr;
  return hipSuccess;
}

static void
freeCompressible(void* ptr, const size_t requested_bytes)
{
  hipDevice_t device;
  ERRCHK_ALWAYS(hipCtxGetDevice(&device) == hipSuccess);

  hipMemAllocationProp prop;
  memset(&prop, 0, sizeof(hipMemAllocationProp));
  prop.type                       = hipMemAllocationTypePinned;
  prop.location.type              = hipMemLocationTypeDevice;
  prop.location.id                = device;
  prop.allocFlags.compressionType = CU_MEM_ALLOCATION_COMP_GENERIC;

  size_t granularity = 0;
  ERRCHK_ALWAYS(hipMemGetAllocationGranularity(
                    &granularity, &prop, hipMemAllocationGranularityMinimum) ==
                hipSuccess);
  const size_t bytes = ((requested_bytes - 1) / granularity + 1) * granularity;

  ERRCHK_ALWAYS(ptr);
  ERRCHK_ALWAYS(hipMemUnmap((hipDeviceptr_t)ptr, bytes) == hipSuccess);
  ERRCHK_ALWAYS(hipMemAddressFree((hipDeviceptr_t)ptr, bytes) == hipSuccess);
}
#endif

AcResult
acVBAReset(const hipStream_t stream, VertexBufferArray* vba)
{
  const size_t count = vba->bytes / sizeof(vba->in[0][0]);

  // Set vba.in data to all-nan and vba.out to 0
  for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
    if(vtxbuf_is_auxiliary[i])
    {
      acKernelFlush(stream, vba->in[i],count, (AcReal)0.0);
    } else{
      acKernelFlush(stream, vba->in[i],count, (AcReal)NAN);
      acKernelFlush(stream, vba->out[i],count, (AcReal)0.0);
    }
  }
  return AC_SUCCESS;
}

void
device_malloc(void** dst, const int bytes)
{
 #if USE_COMPRESSIBLE_MEMORY 
    ERRCHK_CUDA_ALWAYS(mallocCompressible(dst, bytes));
 #else
    ERRCHK_CUDA_ALWAYS(hipMalloc(dst, bytes));
  #endif
}

VertexBufferArray
acVBACreate(const AcMeshInfo config)
{
  //can't use acVertexBufferDims because of linking issues
  const int3 counts = (int3){
        (config.int_params[AC_mx]),
        (config.int_params[AC_my]),
        (config.int_params[AC_mz])
  };


  VertexBufferArray vba;
  size_t count = counts.x*counts.y*counts.z;
  const size_t bytes = sizeof(vba.in[0][0]) * count;
  vba.bytes          = bytes;

  for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
    //Allocate auxilary fields
    //They need only a single copy so out can point to in
    if(vtxbuf_is_auxiliary[i])
    {
      device_malloc((void**) &vba.in[i],bytes);
      vba.out[i] = vba.in[i];
    }else{
      device_malloc((void**) &vba.in[i],bytes);
      device_malloc((void**) &vba.out[i],bytes);
    }
  }
  //Allocate profiles
  for(int i= 0; i < NUM_PROFILES; ++i){
    //if the user loads in a nullptr for the profile it won't be allocated and set to null (the user will be warned at acGridInit)
    if(config.profiles[i] != nullptr)
    {
      const size_t profile_bytes = sizeof(vba.in[0][0]) * config.int_params[profile_lengths[i]];
      device_malloc((void**)&vba.profiles[i],profile_bytes);
    }else{
      vba.profiles[i] = nullptr;
    }
  }
  //Allocate workbuffers
  for (int i = 0; i < NUM_WORK_BUFFERS; ++i)
    device_malloc((void**)&vba.w[i],bytes);
  //Allocate arrays
  for (int i = 0; i < NUM_REAL_ARRAYS; ++i)
    device_malloc((void**)&vba.arrays[i],sizeof(vba.in[0][0])*config.int_params[real_array_lengths[i]]);
  acVBAReset(0, &vba);
  hipDeviceSynchronize();
  return vba;
}

void
device_free(AcReal** dst, const int bytes)
{
#if USE_COMPRESSIBLE_MEMORY
  freeCompressible(*dst, bytes);
#else
  hipFree(*dst);
#endif
  *dst = NULL;
}
void
acVBADestroy(VertexBufferArray* vba, const AcMeshInfo config)
{
  for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
    device_free(&(vba->in[i]), vba->bytes);
    if(vtxbuf_is_auxiliary[i])
      vba->out[i] = NULL;
    else
      device_free(&(vba->out[i]), vba->bytes);
  }
  //Free workbuffers 
  for (int i = 0; i < NUM_WORK_BUFFERS; ++i) 
    device_free(&(vba->w[i]), vba->bytes);
  //Free profiles
  for(int i=0;i<NUM_PROFILES; ++i)
    //Nothing to free if nullptr, don't know if a nullptr would break compressed memory free so this is safest
    if(config.profiles[i] != nullptr){
      device_free(&(vba->profiles[i]),config.int_params[profile_lengths[i]]);
    }
  //Free arrays
  for(int i=0;i<NUM_REAL_ARRAYS; ++i)
    device_free(&(vba->arrays[i]), config.int_params[real_array_lengths[i]]);
  vba->bytes = 0;
}

AcResult
acLaunchKernel(KernelLambda kernel, const hipStream_t stream, const int3 start, const int3 end, VertexBufferArray vba)
{
  const int3 n = end - start;
  const TBConfig tbconf = getOptimalTBConfig(kernel, n, vba);
  const dim3 tpb        = tbconf.tpb;
  const int3 dims       = tbconf.dims;
  const dim3 bpg        = to_dim3(get_bpg(to_volume(dims), to_volume(tpb)));
  const size_t smem = get_smem(to_volume(tpb), STENCIL_ORDER, sizeof(AcReal));

  kernel.lambda(bpg,tpb,smem,stream,start,end,vba);
  ERRCHK_CUDA_KERNEL();

  last_tpb = tpb; // Note: a bit hacky way to get the tpb
  return AC_SUCCESS;
}

AcResult
acLaunchKernel(Kernel kernel, const hipStream_t stream, const int3 start,
               const int3 end, VertexBufferArray vba)
{
  return acLaunchKernel(kernel_to_kernel_lambda(kernel), stream, start, end, vba);
}


AcResult
acBenchmarkKernel(KernelLambda lambda, const int3 start, const int3 end,
                  VertexBufferArray vba)
{
  const int3 n = end - start;

  const TBConfig tbconf = getOptimalTBConfig(lambda, n, vba);
  const dim3 tpb        = tbconf.tpb;
  const int3 dims       = tbconf.dims;
  const dim3 bpg        = to_dim3(get_bpg(to_volume(dims), to_volume(tpb)));
  const size_t smem = get_smem(to_volume(tpb), STENCIL_ORDER, sizeof(AcReal));

  // Timer create
  hipEvent_t tstart, tstop;
  hipEventCreate(&tstart);
  hipEventCreate(&tstop);

  // Warmup
  hipEventRecord(tstart);
  lambda.lambda(bpg, tpb, smem, 0, start, end, vba);
  hipEventRecord(tstop);
  hipEventSynchronize(tstop);
  ERRCHK_CUDA_KERNEL();
  hipDeviceSynchronize();

  // Benchmark
  hipEventRecord(tstart); // Timing start
  lambda.lambda(bpg, tpb, smem, 0, start, end, vba);
  hipEventRecord(tstop); // Timing stop
  hipEventSynchronize(tstop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, tstart, tstop);

  size_t kernel_id = NUM_KERNELS;
  for (size_t i = 0; i < NUM_KERNELS; ++i) {
    if ((void*)kernels[i] == lambda.kernel) {
      kernel_id = i;
    }
  }
  ERRCHK_ALWAYS(kernel_id < NUM_KERNELS);
  printf("Kernel %s time elapsed: %g ms\n", kernel_names[kernel_id],
         milliseconds);

  // Timer destroy
  hipEventDestroy(tstart);
  hipEventDestroy(tstop);

  last_tpb = tpb; // Note: a bit hacky way to get the tpb
  return AC_SUCCESS;
}

AcResult
acBenchmarkKernel(Kernel kernel, const int3 start, const int3 end,
                  VertexBufferArray vba)
{
  return acBenchmarkKernel(kernel_to_kernel_lambda(kernel), start, end, vba);
}

AcResult
acLoadStencil(const Stencil stencil, const hipStream_t /* stream */,
              const AcReal data[STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
  ERRCHK_ALWAYS(stencil < NUM_STENCILS);

  // Note important hipDeviceSynchronize below
  //
  // Constant memory allocated for stencils is shared among kernel
  // invocations, therefore a race condition is possible when updating
  // the coefficients. To avoid this, all kernels that can access
  // the coefficients must be completed before starting async copy to
  // constant memory
  hipDeviceSynchronize();

  const size_t bytes = sizeof(data[0][0][0]) * STENCIL_DEPTH * STENCIL_HEIGHT *
                       STENCIL_WIDTH;
  const hipError_t retval = hipMemcpyToSymbol(HIP_SYMBOL(
      stencils), data, bytes, stencil * bytes, hipMemcpyHostToDevice);

  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
};

AcResult
acStoreStencil(const Stencil stencil, const hipStream_t /* stream */,
               AcReal data[STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
  ERRCHK_ALWAYS(stencil < NUM_STENCILS);

  // Ensure all acLoadUniform calls have completed before continuing
  hipDeviceSynchronize();

  const size_t bytes = sizeof(data[0][0][0]) * STENCIL_DEPTH * STENCIL_HEIGHT *
                       STENCIL_WIDTH;
  const hipError_t retval = hipMemcpyFromSymbol(
      data, HIP_SYMBOL(stencils), bytes, stencil * bytes, hipMemcpyDeviceToHost);

  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
};

#define GEN_LOAD_UNIFORM(LABEL_UPPER, LABEL_LOWER)                             \
  ERRCHK_ALWAYS(param < NUM_##LABEL_UPPER##_PARAMS);                           \
  hipDeviceSynchronize(); /* See note in acLoadStencil */                     \
                                                                               \
  const size_t offset = (size_t)&d_mesh_info.LABEL_LOWER##_params[param] -     \
                        (size_t)&d_mesh_info;                                  \
                                                                               \
  const hipError_t retval = hipMemcpyToSymbol(HIP_SYMBOL(                               \
      d_mesh_info), &value, sizeof(value), offset, hipMemcpyHostToDevice);     \
  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;

AcResult
acLoadRealUniform(const hipStream_t /* stream */, const AcRealParam param,
                  const AcReal value)
{
  if (isnan(value)) {
    fprintf(stderr,
            "WARNING: Passed an invalid value %g to device constant %s. "
            "Skipping.\n",
            (double)value, realparam_names[param]);
    return AC_FAILURE;
  }
  GEN_LOAD_UNIFORM(REAL, real);
}

AcResult
acLoadReal3Uniform(const hipStream_t /* stream */, const AcReal3Param param,
                   const AcReal3 value)
{
  if (isnan(value.x) || isnan(value.y) || isnan(value.z)) {
    fprintf(stderr,
            "WARNING: Passed an invalid value (%g, %g, %g) to device constant "
            "%s. Skipping.\n",
            (double)value.x, (double)value.y, (double)value.z,
            real3param_names[param]);
    return AC_FAILURE;
  }
  GEN_LOAD_UNIFORM(REAL3, real3);
}

AcResult
acLoadIntUniform(const hipStream_t /* stream */, const AcIntParam param,
                 const int value)
{
  GEN_LOAD_UNIFORM(INT, int);
}

AcResult
acLoadInt3Uniform(const hipStream_t /* stream */, const AcInt3Param param,
                  const int3 value)
{
  GEN_LOAD_UNIFORM(INT3, int3);
}

#define GEN_STORE_UNIFORM(LABEL_UPPER, LABEL_LOWER)                            \
  ERRCHK_ALWAYS(param < NUM_##LABEL_UPPER##_PARAMS);                           \
  hipDeviceSynchronize(); /* See notes in GEN_LOAD_UNIFORM */                 \
                                                                               \
  const size_t offset = (size_t)&d_mesh_info.LABEL_LOWER##_params[param] -     \
                        (size_t)&d_mesh_info;                                  \
                                                                               \
  const hipError_t retval = hipMemcpyFromSymbol(                             \
      value, HIP_SYMBOL(d_mesh_info), sizeof(*value), offset, hipMemcpyDeviceToHost);     \
  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;

AcResult
acStoreRealUniform(const hipStream_t /* stream */, const AcRealParam param,
                   AcReal* value)
{
  GEN_STORE_UNIFORM(REAL, real);
}

AcResult
acStoreReal3Uniform(const hipStream_t /* stream */, const AcReal3Param param,
                    AcReal3* value)
{
  GEN_STORE_UNIFORM(REAL3, real3);
}

AcResult
acStoreIntUniform(const hipStream_t /* stream */, const AcIntParam param,
                  int* value)
{
  GEN_STORE_UNIFORM(INT, int);
}

AcResult
acStoreInt3Uniform(const hipStream_t /* stream */, const AcInt3Param param,
                   int3* value)
{
  GEN_STORE_UNIFORM(INT3, int3);
}

static TBConfig
autotune(const KernelLambda lambda, const int3 dims, VertexBufferArray vba)
{
#if 0
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  size_t size = min(int(prop.l2CacheSize * 0.75), prop.persistingL2CacheMaxSize);
  hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, size);
  // set-aside 3/4 of L2 cache for persisting accesses or the max allowed
#endif

#if 0
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  size_t size = min(int(prop.l2CacheSize * 0.75), prop.persistingL2CacheMaxSize);
  hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, size);
  // set-aside 3/4 of L2 cache for persisting accesses or the max allowed
#endif

  TBConfig c = {
      .kernel = lambda.kernel,
      .dims   = dims,
      .tpb    = (dim3){0, 0, 0},
  };

  const int3 start = (int3){
      STENCIL_ORDER / 2,
      STENCIL_ORDER / 2,
      STENCIL_ORDER / 2,
  };
  const int3 end = start + dims;

  dim3 best_tpb(0, 0, 0);
  float best_time     = INFINITY;
  const int num_iters = 2;

  // Get device hardware information
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  const int max_threads_per_block = MAX_THREADS_PER_BLOCK
                                        ? min(props.maxThreadsPerBlock,
                                              MAX_THREADS_PER_BLOCK)
                                        : props.maxThreadsPerBlock;
  const size_t max_smem           = props.sharedMemPerBlock;

  // Old heuristic
  // for (int z = 1; z <= max_threads_per_block; ++z) {
  //   for (int y = 1; y <= max_threads_per_block; ++y) {
  //     for (int x = max(y, z); x <= max_threads_per_block; ++x) {

  // New: require that tpb.x is a multiple of the minimum transaction or L2
  // cache line size
  for (int z = 1; z <= max_threads_per_block; ++z) {
    for (int y = 1; y <= max_threads_per_block; ++y) {
      // 64 bytes on NVIDIA but the minimum L1 cache transaction is 32
      const int minimum_transaction_size_in_elems = 32 / sizeof(AcReal);
      for (int x = minimum_transaction_size_in_elems;
           x <= max_threads_per_block; x += minimum_transaction_size_in_elems) {

        if (x * y * z > max_threads_per_block)
          break;

        // if (x * y * z * max_regs_per_thread > max_regs_per_block)
        //  break;

        // if (max_regs_per_block / (x * y * z) < min_regs_per_thread)
        //   continue;

        // if (x < y || x < z)
        //   continue;

        const dim3 tpb(x, y, z);
        const dim3 bpg    = to_dim3(get_bpg(to_volume(dims), to_volume(tpb)));
        const size_t smem = get_smem(to_volume(tpb), STENCIL_ORDER,
                                     sizeof(AcReal));

        if (smem > max_smem)
          continue;

        if ((x * y * z) % props.warpSize)
          continue;

        if (!is_valid_configuration(to_volume(dims), to_volume(tpb)))
          continue;

	// #if VECTORIZED_LOADS
        //         const size_t window = tpb.x + STENCIL_ORDER;

        //         // Vectorization criterion
        //         if (window % veclen) // Window not divisible into vectorized
        //         blocks
        //           continue;

        //         if (dims.x % tpb.x)
        //           continue;

        //           // May be too strict
        //           // if (dims.x % tpb.x || dims.y % tpb.y || dims.z % tpb.z)
        //           //   continue;
        // #endif
        // #if 0 // Disabled for now (waiting for cleanup)
        // #if USE_SMEM
        //         const size_t max_smem = 128 * 1024;
        //         if (smem > max_smem)
        //           continue;

        // #if VECTORIZED_LOADS
        //         const size_t window = tpb.x + STENCIL_ORDER;

        //         // Vectorization criterion
        //         if (window % veclen) // Window not divisible into vectorized
        //         blocks
        //           continue;

        //         if (dims.x % tpb.x || dims.y % tpb.y || dims.z % tpb.z)
        //           continue;
        // #endif

        //           //  Padding criterion
        //           //  TODO (cannot be checked here)
        // #else
        //         if ((x * y * z) % warp_size)
        //           continue;
        // #endif
        // #endif

        // printf("%d, %d, %d: %lu\n", tpb.x, tpb.y, tpb.z, smem);

        hipEvent_t tstart, tstop;
        hipEventCreate(&tstart);
        hipEventCreate(&tstop);

        lambda.lambda(bpg, tpb, smem, 0, start, end, vba);
        hipDeviceSynchronize();
        hipEventRecord(tstart); // Timing start
        for (int i = 0; i < num_iters; ++i)
          lambda.lambda(bpg, tpb, smem, 0, start, end, vba);
        hipEventRecord(tstop); // Timing stop
        hipEventSynchronize(tstop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, tstart, tstop);

        hipEventDestroy(tstart);
        hipEventDestroy(tstop);

        // Discard failed runs (attempt to clear the error to hipSuccess)
        if (hipGetLastError() != hipSuccess) {
          // Exit in case of unrecoverable error that needs a device reset
          ERRCHK_CUDA_KERNEL_ALWAYS();
          ERRCHK_CUDA_ALWAYS(hipGetLastError());
          continue;
        }

        if (milliseconds < best_time) {
          best_time = milliseconds;
          best_tpb  = tpb;
        }

        // printf("Auto-optimizing... Current tpb: (%d, %d, %d), time %f ms\n",
        //        tpb.x, tpb.y, tpb.z, (double)milliseconds / num_iters);
        // fflush(stdout);
      }
    }
  }
  c.tpb = best_tpb;

  // printf("\tThe best tpb: (%d, %d, %d), time %f ms\n", best_tpb.x,
  // best_tpb.y,
  //        best_tpb.z, (double)best_time / num_iters);

  FILE* fp = fopen("autotune.csv", "a");
  ERRCHK_ALWAYS(fp);
#if IMPLEMENTATION == SMEM_HIGH_OCCUPANCY_CT_CONST_TB
  fprintf(fp, "%d, (%d, %d, %d), (%d, %d, %d), %g\n", IMPLEMENTATION, nx, ny,
          nz, best_tpb.x, best_tpb.y, best_tpb.z,
          (double)best_time / num_iters);
#else
  fprintf(fp, "%d, %d, %d, %d, %d, %d, %d, %g\n", IMPLEMENTATION, dims.x,
          dims.y, dims.z, best_tpb.x, best_tpb.y, best_tpb.z,
          (double)best_time / num_iters);
#endif
  fclose(fp);

  if (c.tpb.x * c.tpb.y * c.tpb.z <= 0) {
    fprintf(stderr,
            "Fatal error: failed to find valid thread block dimensions.\n");
  }
  ERRCHK_ALWAYS(c.tpb.x * c.tpb.y * c.tpb.z > 0);
  return c;

}

static TBConfig
getOptimalTBConfig(const Kernel kernel, const int3 dims, VertexBufferArray vba)
{
  return getOptimalTBConfig(kernel_to_kernel_lambda(kernel), dims, vba);
}

static TBConfig
getOptimalTBConfig(const KernelLambda lambda, const int3 dims, VertexBufferArray vba)
{
  for (auto c : tbconfigs) {
    if (c.kernel == lambda.kernel && c.dims == dims)
      return c;
  }
  TBConfig c = autotune(lambda, dims, vba);
  tbconfigs.push_back(c);
  return c;
}
