#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2021, Johannes Pekkila, Miikka Vaisala.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "acc_runtime.h"

#include <math.h> 
#include <vector> // tbconfig

#include "errchk.h"
#include "math_utils.h"
#include <unordered_map>
#include <utility>

#if AC_USE_HIP
#include <hip/hip_runtime.h> // Needed in files that include kernels
#include <rocprim/rocprim.hpp>
#endif

#define USE_COMPRESSIBLE_MEMORY (0)

#include "acc/implementation.h"
#include "user_constants.h"

static dim3 last_tpb = (dim3){0, 0, 0};

//the int key in the nested map corresponds to the starting vertexIdx linearized
std::unordered_map<Kernel,std::unordered_map<int,int>> reduce_offsets;
int kernel_running_reduce_offsets[NUM_KERNELS];

Volume
acKernelLaunchGetLastTPB(void)
{
  return to_volume(last_tpb);
}
int
acGetKernelReduceScratchPadSize(const AcKernel kernel)
{
	return kernel_running_reduce_offsets[(int)kernel];
}
int
acGetKernelReduceScratchPadMinSize()
{
	int res = 0; 
	for(int i = 0; i < NUM_KERNELS; ++i)
		res = (res < kernel_running_reduce_offsets[i]) ? kernel_running_reduce_offsets[i] : res;
	return res;
}
Volume
get_bpg(const Volume dims, const Volume tpb)
{
  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING:             // Fallthrough
  case EXPLICIT_CACHING:             // Fallthrough
  case EXPLICIT_CACHING_3D_BLOCKING: // Fallthrough
  case EXPLICIT_CACHING_4D_BLOCKING: // Fallthrough
  case EXPLICIT_PINGPONG_txw:        // Fallthrough
  case EXPLICIT_PINGPONG_txy:        // Fallthrough
  case EXPLICIT_PINGPONG_txyblocked: // Fallthrough
  case EXPLICIT_PINGPONG_txyz:       // Fallthrough
  case EXPLICIT_ROLLING_PINGPONG: {
    return (Volume){
        (size_t)ceil(1. * dims.x / tpb.x),
        (size_t)ceil(1. * dims.y / tpb.y),
        (size_t)ceil(1. * dims.z / tpb.z),
    };
  }
  default: {
    ERROR("Invalid IMPLEMENTATION in get_bpg");
    return (Volume){0, 0, 0};
  }
  }
}

bool
is_valid_configuration(const Volume dims, const Volume tpb)
{
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  const size_t warp_size = props.warpSize;
  const size_t xmax      = (size_t)(warp_size * ceil(1. * dims.x / warp_size));
  const size_t ymax      = (size_t)(warp_size * ceil(1. * dims.y / warp_size));
  const size_t zmax      = (size_t)(warp_size * ceil(1. * dims.z / warp_size));
  const bool too_large   = (tpb.x > xmax) || (tpb.y > ymax) || (tpb.z > zmax);

  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING: {

    if (too_large)
      return false;

    return true;
  }
  case EXPLICIT_CACHING_4D_BLOCKING: // Fallthrough
    if (tpb.z > 1)
      return false;
  case EXPLICIT_CACHING: // Fallthrough
  case EXPLICIT_CACHING_3D_BLOCKING: {

    // For some reason does not work without this
    // Probably because of break vs continue when fetching (some threads
    // quit too early if the dims are not divisible)
    return !(dims.x % tpb.x) && !(dims.y % tpb.y) && !(dims.z % tpb.z);
  }
  case EXPLICIT_PINGPONG_txw: {
    return (tpb.y == 1) && (tpb.z == 1);
  }
  case EXPLICIT_PINGPONG_txy: {
    return (tpb.z == 1);
  }
  case EXPLICIT_PINGPONG_txyblocked: {
    return (tpb.z == 1);
  }
  case EXPLICIT_PINGPONG_txyz: {
    return true;
  }
  case EXPLICIT_ROLLING_PINGPONG: {
    // OK for every other rolling pingpong implementation
    // return true;

    // Required only when unrolling smem loads
    // Ensures two unrolls is enough to fill the smem buffer
    return (2 * tpb.x >= STENCIL_WIDTH - 1 + tpb.x) &&
           (2 * tpb.y >= STENCIL_HEIGHT - 1 + tpb.y);
  }
  default: {
    ERROR("Invalid IMPLEMENTATION in is_valid_configuration");
    return false;
  }
  }
}

size_t
get_smem(const Volume tpb, const size_t stencil_order,
         const size_t bytes_per_elem)
{
  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING: {
    return 0;
  }
  case EXPLICIT_CACHING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) * tpb.z *
           bytes_per_elem;
  }
  case EXPLICIT_CACHING_3D_BLOCKING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) *
           (tpb.z + stencil_order) * bytes_per_elem;
  }
  case EXPLICIT_CACHING_4D_BLOCKING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) * tpb.z *
           (NUM_FIELDS)*bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txw: {
    return 2 * (tpb.x + stencil_order) * NUM_FIELDS * bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txy: {
    return 2 * (tpb.x + stencil_order) * (tpb.y + stencil_order) *
           bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txyblocked: {
    const size_t block_size = 7;
    return 2 * (tpb.x + stencil_order) * (tpb.y + stencil_order) * block_size *
           bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txyz: {
    return 2 * (tpb.x + stencil_order) * (tpb.y + stencil_order) *
           (tpb.z + stencil_order) * bytes_per_elem;
  }
  case EXPLICIT_ROLLING_PINGPONG: {
    // tpbxy slices with halos
    // tpbz depth + 1 rolling cache slab
    return EXPLICIT_ROLLING_PINGPONG_BLOCKSIZE * (tpb.x + stencil_order) *
           (tpb.y + stencil_order) * (tpb.z + 1) * bytes_per_elem;
  }
  default: {
    ERROR("Invalid IMPLEMENTATION in get_smem");
    return (size_t)-1;
  }
  }
}

/*
// Device info (TODO GENERIC)
// Use the maximum available reg count per thread
#define REGISTERS_PER_THREAD (255)
#define MAX_REGISTERS_PER_BLOCK (65536)
#if AC_DOUBLE_PRECISION
#define MAX_THREADS_PER_BLOCK                                                  \
  (MAX_REGISTERS_PER_BLOCK / REGISTERS_PER_THREAD / 2)
#else
#define MAX_THREADS_PER_BLOCK (MAX_REGISTERS_PER_BLOCK / REGISTERS_PER_THREAD)
#endif
*/

__device__ __constant__ AcMeshInfo d_mesh_info;
//we pad with 1 since zero sized arrays are not allowed with some CUDA compilers

__device__ __constant__ AcReal d_real_arrays[D_REAL_ARRAYS_LEN+1];
__device__ __constant__ int d_int_arrays[D_INT_ARRAYS_LEN+1];

// Astaroth 2.0 backwards compatibility START
#define d_multigpu_offset (d_mesh_info.int3_params[AC_multigpu_offset])

int __device__ __forceinline__
DCONST(const AcIntParam param)
{
  return d_mesh_info.int_params[param];
}
int3 __device__ __forceinline__
DCONST(const AcInt3Param param)
{
  return d_mesh_info.int3_params[param];
}
AcReal __device__ __forceinline__
DCONST(const AcRealParam param)
{
  return d_mesh_info.real_params[param];
}
AcReal3 __device__ __forceinline__
DCONST(const AcReal3Param param)
{
  return d_mesh_info.real3_params[param];
}

#define DEVICE_VTXBUF_IDX(i, j, k)                                             \
  ((i) + (j)*DCONST(AC_mx) + (k)*DCONST(AC_mxy))

__device__ int
LOCAL_COMPDOMAIN_IDX(const int3 coord)
{
  return (coord.x) + (coord.y) * DCONST(AC_nx) + (coord.z) * DCONST(AC_nxy);
}

__device__ constexpr int
IDX(const int i)
{
  return i;
}

#if 1
__device__ __forceinline__ int
IDX(const int i, const int j, const int k)
{
  return DEVICE_VTXBUF_IDX(i, j, k);
}
#else
constexpr __device__ int
IDX(const uint i, const uint j, const uint k)
{
  /*
  const int precision   = 32; // Bits
  const int dimensions  = 3;
  const int bits = ceil(precision / dimensions);
  */
  const int dimensions = 3;
  const int bits       = 11;

  uint idx = 0;
#pragma unroll
  for (uint bit = 0; bit < bits; ++bit) {
    const uint mask = 0b1 << bit;
    idx |= ((i & mask) << 0) << (dimensions - 1) * bit;
    idx |= ((j & mask) << 1) << (dimensions - 1) * bit;
    idx |= ((k & mask) << 2) << (dimensions - 1) * bit;
  }
  return idx;
}
#endif

// Only used in reductions
__device__ __forceinline__ int
IDX(const int3 idx)
{
  return DEVICE_VTXBUF_IDX(idx.x, idx.y, idx.z);
}

//#define Field3(x, y, z) make_int3((x), (y), (z))
constexpr int3
Field3(const int& x, const int& y, const int& z)
{
	return make_int3(x,y,z);
}
template <size_t N>
constexpr __device__ __forceinline__
std::array<int3,N>
Field3(const Field (&x)[N], const Field (&y)[N], const Field (&z)[N])
{
	std::array<int3,N> res{};
	for(size_t i = 0; i < N; ++i)
	{
		res[i] = make_int3(x[i],y[i],z[i]);
	}
	return res;
}
#define print printf                          // TODO is this a good idea?
#define len(arr) sizeof(arr) / sizeof(arr[0]) // Leads to bugs if the user
// passes an array into a device function and then calls len (need to modify
// the compiler to always pass arrays to functions as references before
// re-enabling)

#include "random.cuh"

#include "user_dfuncs.h"
#include "user_kernels.h"

typedef struct {
  Kernel kernel;
  int3 dims;
  dim3 tpb;
} TBConfig;

static std::vector<TBConfig> tbconfigs;


static TBConfig getOptimalTBConfig(const Kernel kernel, const int3 dims, VertexBufferArray vba);

static __global__ void
flush_kernel(AcReal* arr, const size_t n, const AcReal value)
{
  const size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n)
    arr[idx] = value;
}

AcResult
acKernelFlush(const hipStream_t stream, AcReal* arr, const size_t n,
              const AcReal value)
{
  const size_t tpb = 256;
  const size_t bpg = (size_t)(ceil((double)n / tpb));
  flush_kernel<<<bpg, tpb, 0, stream>>>(arr, n, value);
  ERRCHK_CUDA_KERNEL_ALWAYS();
  return AC_SUCCESS;
}

#if USE_COMPRESSIBLE_MEMORY
#include <hip/hip_runtime.h>

#define ERRCHK_CU_ALWAYS(x) ERRCHK_ALWAYS((x) == hipSuccess)

static hipError_t
mallocCompressible(void** addr, const size_t requested_bytes)
{
  hipDevice_t device;
  ERRCHK_ALWAYS(hipCtxGetDevice(&device) == hipSuccess);

  hipMemAllocationProp prop;
  memset(&prop, 0, sizeof(hipMemAllocationProp));
  prop.type                       = hipMemAllocationTypePinned;
  prop.location.type              = hipMemLocationTypeDevice;
  prop.location.id                = device;
  prop.allocFlags.compressionType = CU_MEM_ALLOCATION_COMP_GENERIC;

  size_t granularity;
  ERRCHK_CU_ALWAYS(hipMemGetAllocationGranularity(
      &granularity, &prop, hipMemAllocationGranularityMinimum));

  // Pad to align
  const size_t bytes = ((requested_bytes - 1) / granularity + 1) * granularity;

  hipDeviceptr_t dptr;
  ERRCHK_ALWAYS(hipMemAddressReserve(&dptr, bytes, 0, 0, 0) == hipSuccess);

  hipMemGenericAllocationHandle_t handle;
  ERRCHK_ALWAYS(hipMemCreate(&handle, bytes, &prop, 0) == hipSuccess)

  // Check if hipMemCreate was able to allocate compressible memory.
  hipMemAllocationProp alloc_prop;
  memset(&alloc_prop, 0, sizeof(hipMemAllocationProp));
  hipMemGetAllocationPropertiesFromHandle(&alloc_prop, handle);
  ERRCHK_ALWAYS(alloc_prop.allocFlags.compressionType ==
                CU_MEM_ALLOCATION_COMP_GENERIC);

  ERRCHK_ALWAYS(hipMemMap(dptr, bytes, 0, handle, 0) == hipSuccess);
  ERRCHK_ALWAYS(hipMemRelease(handle) == hipSuccess);

  hipMemAccessDesc accessDescriptor;
  accessDescriptor.location.id   = prop.location.id;
  accessDescriptor.location.type = prop.location.type;
  accessDescriptor.flags         = hipMemAccessFlagsProtReadWrite;

  ERRCHK_ALWAYS(hipMemSetAccess(dptr, bytes, &accessDescriptor, 1) ==
                hipSuccess);

  *addr = (void*)dptr;
  return hipSuccess;
}

static void
freeCompressible(void* ptr, const size_t requested_bytes)
{
  hipDevice_t device;
  ERRCHK_ALWAYS(hipCtxGetDevice(&device) == hipSuccess);

  hipMemAllocationProp prop;
  memset(&prop, 0, sizeof(hipMemAllocationProp));
  prop.type                       = hipMemAllocationTypePinned;
  prop.location.type              = hipMemLocationTypeDevice;
  prop.location.id                = device;
  prop.allocFlags.compressionType = CU_MEM_ALLOCATION_COMP_GENERIC;

  size_t granularity = 0;
  ERRCHK_ALWAYS(hipMemGetAllocationGranularity(
                    &granularity, &prop, hipMemAllocationGranularityMinimum) ==
                hipSuccess);
  const size_t bytes = ((requested_bytes - 1) / granularity + 1) * granularity;

  ERRCHK_ALWAYS(ptr);
  ERRCHK_ALWAYS(hipMemUnmap((hipDeviceptr_t)ptr, bytes) == hipSuccess);
  ERRCHK_ALWAYS(hipMemAddressFree((hipDeviceptr_t)ptr, bytes) == hipSuccess);
}
#endif

AcResult
acVBAReset(const hipStream_t stream, VertexBufferArray* vba)
{
  const size_t count = vba->bytes / sizeof(vba->in[0][0]);

  // Set vba.in data to all-nan and vba.out to 0
  for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
    if (vtxbuf_is_auxiliary[i])
    {
      acKernelFlush(stream, vba->in[i],count, (AcReal)0.0);
    } else{
      acKernelFlush(stream, vba->in[i],count, (AcReal)NAN);
      acKernelFlush(stream, vba->out[i],count, (AcReal)0.0);
    }
  }
  return AC_SUCCESS;
}

void
device_malloc(void** dst, const int bytes)
{
 #if USE_COMPRESSIBLE_MEMORY 
    ERRCHK_CUDA_ALWAYS(mallocCompressible(dst, bytes));
 #else
    ERRCHK_CUDA_ALWAYS(hipMalloc(dst, bytes));
  #endif
}

void
device_free(AcReal** dst, const int bytes)
{
#if USE_COMPRESSIBLE_MEMORY
  freeCompressible(*dst, bytes);
#else
  hipFree(*dst);
  //used to silence unused warning
  (void)bytes;
#endif
  *dst = NULL;
}

void
device_free(int** dst, const int bytes)
{
#if USE_COMPRESSIBLE_MEMORY
  freeCompressible(*dst, bytes);
#else
  hipFree(*dst);
  //used to silence unused warning
  (void)bytes;
#endif
  *dst = NULL;
}


VertexBufferArray
acVBACreate(const AcMeshInfo config)
{
  //can't use acVertexBufferDims because of linking issues
  const int3 counts = (int3){
        (config.int_params[AC_mx]),
        (config.int_params[AC_my]),
        (config.int_params[AC_mz])
  };

  VertexBufferArray vba;
  size_t count = counts.x*counts.y*counts.z;
  size_t bytes = sizeof(vba.in[0][0]) * count;
  vba.bytes          = bytes;
#if AC_ADJACENT_VERTEX_BUFFERS
  const size_t allbytes = bytes*NUM_VTXBUF_HANDLES;
  AcReal *allbuf_in, *allbuf_out;

  ERRCHK_CUDA_ALWAYS(hipMalloc((void**)&allbuf_in, allbytes));
  ERRCHK_CUDA_ALWAYS(hipMalloc((void**)&allbuf_out, allbytes));

  acKernelFlush(STREAM_DEFAULT,allbuf_in, count*NUM_VTXBUF_HANDLES, (AcReal)0.0);
  ERRCHK_CUDA_ALWAYS(hipMemset((void*)allbuf_out, 0, allbytes));

  vba.in[0]=allbuf_in; vba.out[0]=allbuf_out;
printf("i,vbas[0]= %p %p \n",vba.in[0],vba.out[0]);
  for (size_t i = 1; i < NUM_VTXBUF_HANDLES; ++i) {
    vba.in [i]=vba.in [i-1]+count;
    vba.out[i]=vba.out[i-1]+count;
printf("i,vbas[i]= %zu %p %p\n",i,vba.in[i],vba.out[i]);
  }
#else
  for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
    //Allocate auxilary fields
    //They need only a single copy so out can point to in
    if (vtxbuf_is_auxiliary[i])
    {
      device_malloc((void**) &vba.in[i],bytes);
      vba.out[i] = vba.in[i];
    }else{
      device_malloc((void**) &vba.in[i],bytes);
      device_malloc((void**) &vba.out[i],bytes);
    }
  }
#endif
  //Allocate workbuffers
  for (int i = 0; i < NUM_WORK_BUFFERS; ++i)
    device_malloc((void**)&vba.w[i],bytes);

  //Allocate arrays
  for (int i = 0; i < NUM_REAL_ARRAYS; ++i)
    if (config.real_arrays[i] != nullptr)
       device_malloc((void**)&vba.real_arrays[i],sizeof(vba.in[0][0])*config.int_params[real_array_lengths[i]]);
  for (int i = 0; i < NUM_INT_ARRAYS; ++i)
    if (config.int_arrays[i] != nullptr)
       device_malloc((void**)&vba.int_arrays[i],sizeof(int)*config.int_params[int_array_lengths[i]]);

  acVBAReset(0, &vba);
  hipDeviceSynchronize();
  return vba;
}

void
acVBAUpdate(VertexBufferArray* vba, const AcMeshInfo config)
{
  size_t bytes;
  //Allocate/Free arrays
  for (int i = 0; i < NUM_REAL_ARRAYS; ++i){
    bytes = sizeof(vba->in[0][0])*config.int_params[real_array_lengths[i]];
    if (config.real_arrays[i] == nullptr){
      if (vba->real_arrays[i] != nullptr) device_free(&vba->real_arrays[i], bytes);
    }
    else{
      if (vba->real_arrays[i] == nullptr) device_malloc((void**)&vba->real_arrays[i],bytes);
    }
  }
  for (int i = 0; i < NUM_INT_ARRAYS; ++i){
    bytes = sizeof(int)*config.int_params[int_array_lengths[i]];
    if (config.int_arrays[i] == nullptr){
      if (vba->int_arrays[i] != nullptr) device_free(&vba->int_arrays[i],bytes);
    }else{
      if (vba->int_arrays[i] == nullptr) device_malloc((void**)&vba->int_arrays[i],bytes);
    }
  }
}
void
acVBADestroy(VertexBufferArray* vba, const AcMeshInfo config)
{
  for (int i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
    device_free(&(vba->in[i]), vba->bytes);
    if (vtxbuf_is_auxiliary[i])
      vba->out[i] = NULL;
    else
      device_free(&(vba->out[i]), vba->bytes);
  }
  //Free workbuffers 
  for (int i = 0; i < NUM_WORK_BUFFERS; ++i) 
    device_free(&(vba->w[i]), vba->bytes);

  //Free arrays
  for (int i=0;i<NUM_REAL_ARRAYS; ++i)
    if (config.real_arrays[i] != nullptr)
    	device_free(&(vba->real_arrays[i]), config.int_params[real_array_lengths[i]]);
  for (int i=0;i<NUM_INT_ARRAYS; ++i)
    if (config.int_arrays[i] != nullptr)
    	device_free(&(vba->int_arrays[i]), config.int_params[int_array_lengths[i]]);
  vba->bytes = 0;
}
int
get_num_of_reduce_output(const dim3 bpg, const dim3 tpb)
{
#if AC_USE_HIP
	const int warp_size = rocprim::host_warp_size();
#else
	const int warp_size = 32;
#endif
	const int num_of_warps_per_block = (tpb.x*tpb.y*tpb.z + warp_size-1)/warp_size;
	const int num_of_blocks = bpg.x*bpg.y*bpg.z;
	return num_of_warps_per_block*num_of_blocks;
}

int
get_kernel_index(const Kernel kernel)
{
	for(int i = 0; i < NUM_KERNELS; ++i)
		if(kernel == kernels[i]) return i;
	return -1;
}
AcResult
acLaunchKernel(Kernel kernel, const hipStream_t stream, const int3 start,
               const int3 end, VertexBufferArray vba)
{
  const int3 n = end - start;

  const TBConfig tbconf = getOptimalTBConfig(kernel, n, vba);
  const dim3 tpb        = tbconf.tpb;
  const int3 dims       = tbconf.dims;
  const dim3 bpg        = to_dim3(get_bpg(to_volume(dims), to_volume(tpb)));

  const size_t smem = get_smem(to_volume(tpb), STENCIL_ORDER, sizeof(AcReal));
  const int key = start.x + 10000*start.y + 10000*10000*start.z;
  if(reduce_offsets[kernel].find(key) == reduce_offsets[kernel].end())
  {
  	reduce_offsets[kernel][key] = kernel_running_reduce_offsets[get_kernel_index(kernel)];
  	kernel_running_reduce_offsets[get_kernel_index(kernel)] += get_num_of_reduce_output(bpg,tpb);
  }

  vba.reduce_offset = reduce_offsets[kernel][key];
  // hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferL1);
  kernel<<<bpg, tpb, smem, stream>>>(start, end, vba);
  ERRCHK_CUDA_KERNEL();

  last_tpb = tpb; // Note: a bit hacky way to get the tpb
  return AC_SUCCESS;
}

AcResult
acBenchmarkKernel(Kernel kernel, const int3 start, const int3 end,
                  VertexBufferArray vba)
{
  const int3 n = end - start;

  const TBConfig tbconf = getOptimalTBConfig(kernel, n, vba);
  const dim3 tpb        = tbconf.tpb;
  const int3 dims       = tbconf.dims;
  const dim3 bpg        = to_dim3(get_bpg(to_volume(dims), to_volume(tpb)));
  const size_t smem = get_smem(to_volume(tpb), STENCIL_ORDER, sizeof(AcReal));

  // Timer create
  hipEvent_t tstart, tstop;
  hipEventCreate(&tstart);
  hipEventCreate(&tstop);

  // Warmup
  hipEventRecord(tstart);
  kernel<<<bpg, tpb, smem>>>(start, end, vba);
  hipEventRecord(tstop);
  hipEventSynchronize(tstop);
  ERRCHK_CUDA_KERNEL();
  hipDeviceSynchronize();

  // Benchmark
  hipEventRecord(tstart); // Timing start
  kernel<<<bpg, tpb, smem>>>(start, end, vba);
  hipEventRecord(tstop); // Timing stop
  hipEventSynchronize(tstop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, tstart, tstop);

  size_t kernel_id = NUM_KERNELS;
  for (size_t i = 0; i < NUM_KERNELS; ++i) {
    if (kernels[i] == kernel) {
      kernel_id = i;
    }
  }
  ERRCHK_ALWAYS(kernel_id < NUM_KERNELS);
  printf("Kernel %s time elapsed: %g ms\n", kernel_names[kernel_id],
         (double)milliseconds);

  // Timer destroy
  hipEventDestroy(tstart);
  hipEventDestroy(tstop);

  last_tpb = tpb; // Note: a bit hacky way to get the tpb
  return AC_SUCCESS;
}


AcResult
acLoadStencil(const Stencil stencil, const hipStream_t /* stream */,
              const AcReal data[STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
  ERRCHK_ALWAYS(stencil < NUM_STENCILS);

  // Note important hipDeviceSynchronize below
  //
  // Constant memory allocated for stencils is shared among kernel
  // invocations, therefore a race condition is possible when updating
  // the coefficients. To avoid this, all kernels that can access
  // the coefficients must be completed before starting async copy to
  // constant memory
  hipDeviceSynchronize();

  const size_t bytes = sizeof(data[0][0][0]) * STENCIL_DEPTH * STENCIL_HEIGHT *
                       STENCIL_WIDTH;
  const hipError_t retval = hipMemcpyToSymbol(HIP_SYMBOL(
      stencils), data, bytes, stencil * bytes, hipMemcpyHostToDevice);

  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
};

AcResult
acStoreStencil(const Stencil stencil, const hipStream_t /* stream */,
               AcReal data[STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
  ERRCHK_ALWAYS(stencil < NUM_STENCILS);

  // Ensure all acLoadUniform calls have completed before continuing
  hipDeviceSynchronize();

  const size_t bytes = sizeof(data[0][0][0]) * STENCIL_DEPTH * STENCIL_HEIGHT *
                       STENCIL_WIDTH;
  const hipError_t retval = hipMemcpyFromSymbol(
      data, HIP_SYMBOL(stencils), bytes, stencil * bytes, hipMemcpyDeviceToHost);

  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
};

#define GEN_LOAD_UNIFORM(LABEL_UPPER, LABEL_LOWER)                             \
  ERRCHK_ALWAYS(param < NUM_##LABEL_UPPER##_PARAMS);                           \
  hipDeviceSynchronize(); /* See note in acLoadStencil */                     \
                                                                               \
  const size_t offset = (size_t)&d_mesh_info.LABEL_LOWER##_params[param] -     \
                        (size_t)&d_mesh_info;                                  \
                                                                               \
  const hipError_t retval = hipMemcpyToSymbol(HIP_SYMBOL(                               \
      d_mesh_info), &value, sizeof(value), offset, hipMemcpyHostToDevice);     \
  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;

AcResult
acLoadRealUniform(const hipStream_t /* stream */, const AcRealParam param,
                  const AcReal value)
{
  if (isnan(value)) {
    fprintf(stderr,
            "WARNING: Passed an invalid value %g to device constant %s. "
            "Skipping.\n",
            (double)value, realparam_names[param]);
    return AC_FAILURE;
  }
  GEN_LOAD_UNIFORM(REAL, real);
}

AcResult
acLoadRealArrayUniform(const hipStream_t /* stream */, const AcRealArrayParam param,
                  const AcReal* values)
{
  ERRCHK_ALWAYS(real_array_is_dconst[(int)param]);
  const int length  = (int)real_array_lengths[(int)param];
  hipDeviceSynchronize();
  const size_t offset = (size_t)d_real_array_offsets[(int)param]*sizeof(AcReal);
  const hipError_t retval = hipMemcpyToSymbol(HIP_SYMBOL(d_real_arrays), values, sizeof(AcReal)*length, offset, hipMemcpyHostToDevice);
  if (retval != hipSuccess)
        return AC_FAILURE;
  return AC_SUCCESS;
}

AcResult
acLoadIntArrayUniform(const hipStream_t /* stream */, const AcIntArrayParam param,
                  const int* values)
{
  ERRCHK_ALWAYS(int_array_is_dconst[(int)param]);
  const int length  = (int)int_array_lengths[(int)param];
  hipDeviceSynchronize();
  const size_t offset = (size_t)d_int_array_offsets[(int)param]*sizeof(int);
  const hipError_t retval = hipMemcpyToSymbol(HIP_SYMBOL(d_int_arrays), values, sizeof(int)*length, offset, hipMemcpyHostToDevice);
  if (retval != hipSuccess)
        return AC_FAILURE;
  return AC_SUCCESS;
}


AcResult
acLoadReal3Uniform(const hipStream_t /* stream */, const AcReal3Param param,
                   const AcReal3 value)
{
  if (isnan(value.x) || isnan(value.y) || isnan(value.z)) {
    fprintf(stderr,
            "WARNING: Passed an invalid value (%g, %g, %g) to device constant "
            "%s. Skipping.\n",
            (double)value.x, (double)value.y, (double)value.z,
            real3param_names[param]);
    return AC_FAILURE;
  }
  GEN_LOAD_UNIFORM(REAL3, real3);
}

AcResult
acLoadIntUniform(const hipStream_t /* stream */, const AcIntParam param,
                 const int value)
{
  GEN_LOAD_UNIFORM(INT, int);
}

AcResult
acLoadInt3Uniform(const hipStream_t /* stream */, const AcInt3Param param,
                  const int3 value)
{
  GEN_LOAD_UNIFORM(INT3, int3);
}

#define GEN_STORE_UNIFORM(LABEL_UPPER, LABEL_LOWER)                            \
  ERRCHK_ALWAYS(param < NUM_##LABEL_UPPER##_PARAMS);                           \
  hipDeviceSynchronize(); /* See notes in GEN_LOAD_UNIFORM */                 \
                                                                               \
  const size_t offset = (size_t)&d_mesh_info.LABEL_LOWER##_params[param] -     \
                        (size_t)&d_mesh_info;                                  \
                                                                               \
  const hipError_t retval = hipMemcpyFromSymbol(                             \
      value, HIP_SYMBOL(d_mesh_info), sizeof(*value), offset, hipMemcpyDeviceToHost);     \
  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;

AcResult
acStoreRealUniform(const hipStream_t /* stream */, const AcRealParam param,
                   AcReal* value)
{
  GEN_STORE_UNIFORM(REAL, real);
}

AcResult
acStoreReal3Uniform(const hipStream_t /* stream */, const AcReal3Param param,
                    AcReal3* value)
{
  GEN_STORE_UNIFORM(REAL3, real3);
}

AcResult
acStoreIntUniform(const hipStream_t /* stream */, const AcIntParam param,
                  int* value)
{
  GEN_STORE_UNIFORM(INT, int);
}

AcResult
acStoreInt3Uniform(const hipStream_t /* stream */, const AcInt3Param param,
                   int3* value)
{
  GEN_STORE_UNIFORM(INT3, int3);
}
static TBConfig
autotune(const Kernel kernel, const int3 dims, VertexBufferArray vba)
{
  vba.reduce_offset = 0;
  size_t id = (size_t)-1;
  for (size_t i = 0; i < NUM_KERNELS; ++i) {
    if (kernels[i] == kernel) {
      id = i;
      break;
    }
  }
  ERRCHK_ALWAYS(id < NUM_KERNELS);
  // printf("Autotuning kernel '%s' (%p), block (%d, %d, %d), implementation "
  //        "(%d):\n",
  //        kernel_names[id], kernel, dims.x, dims.y, dims.z, IMPLEMENTATION);
  // fflush(stdout);

#if 0
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  size_t size = min(int(prop.l2CacheSize * 0.75), prop.persistingL2CacheMaxSize);
  hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, size);
  // set-aside 3/4 of L2 cache for persisting accesses or the max allowed
#endif

  TBConfig c = {
      .kernel = kernel,
      .dims   = dims,
      .tpb    = (dim3){0, 0, 0},
  };

  const int3 start = (int3){
      STENCIL_ORDER / 2,
      STENCIL_ORDER / 2,
      STENCIL_ORDER / 2,
  };
  const int3 end = start + dims;

  dim3 best_tpb(0, 0, 0);
  float best_time     = INFINITY;
  const int num_iters = 2;

  // Get device hardware information
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, 0);
  const int max_threads_per_block = MAX_THREADS_PER_BLOCK
                                        ? min(props.maxThreadsPerBlock,
                                              MAX_THREADS_PER_BLOCK)
                                        : props.maxThreadsPerBlock;
  const size_t max_smem           = props.sharedMemPerBlock;

  // Old heuristic
  // for (int z = 1; z <= max_threads_per_block; ++z) {
  //   for (int y = 1; y <= max_threads_per_block; ++y) {
  //     for (int x = max(y, z); x <= max_threads_per_block; ++x) {

  // New: require that tpb.x is a multiple of the minimum transaction or L2
  // cache line size
  for (int z = 1; z <= max_threads_per_block; ++z) {
    for (int y = 1; y <= max_threads_per_block; ++y) {
      // 64 bytes on NVIDIA but the minimum L1 cache transaction is 32
      const int minimum_transaction_size_in_elems = 32 / sizeof(AcReal);
      for (int x = minimum_transaction_size_in_elems;
           x <= max_threads_per_block; x += minimum_transaction_size_in_elems) {

        if (x * y * z > max_threads_per_block)
          break;

        // if (x * y * z * max_regs_per_thread > max_regs_per_block)
        //  break;

        // if (max_regs_per_block / (x * y * z) < min_regs_per_thread)
        //   continue;

        // if (x < y || x < z)
        //   continue;

        const dim3 tpb(x, y, z);
        const dim3 bpg    = to_dim3(get_bpg(to_volume(dims), to_volume(tpb)));
        const size_t smem = get_smem(to_volume(tpb), STENCIL_ORDER,
                                     sizeof(AcReal));

        if (smem > max_smem)
          continue;

        if ((x * y * z) % props.warpSize)
          continue;

        if (!is_valid_configuration(to_volume(dims), to_volume(tpb)))
          continue;

        // #if VECTORIZED_LOADS
        //         const size_t window = tpb.x + STENCIL_ORDER;

        //         // Vectorization criterion
        //         if (window % veclen) // Window not divisible into vectorized
        //         blocks
        //           continue;

        //         if (dims.x % tpb.x)
        //           continue;

        //           // May be too strict
        //           // if (dims.x % tpb.x || dims.y % tpb.y || dims.z % tpb.z)
        //           //   continue;
        // #endif
        // #if 0 // Disabled for now (waiting for cleanup)
        // #if USE_SMEM
        //         const size_t max_smem = 128 * 1024;
        //         if (smem > max_smem)
        //           continue;

        // #if VECTORIZED_LOADS
        //         const size_t window = tpb.x + STENCIL_ORDER;

        //         // Vectorization criterion
        //         if (window % veclen) // Window not divisible into vectorized
        //         blocks
        //           continue;

        //         if (dims.x % tpb.x || dims.y % tpb.y || dims.z % tpb.z)
        //           continue;
        // #endif

        //           //  Padding criterion
        //           //  TODO (cannot be checked here)
        // #else
        //         if ((x * y * z) % warp_size)
        //           continue;
        // #endif
        // #endif

        // printf("%d, %d, %d: %lu\n", tpb.x, tpb.y, tpb.z, smem);

        hipEvent_t tstart, tstop;
        hipEventCreate(&tstart);
        hipEventCreate(&tstop);

        kernel<<<bpg, tpb, smem>>>(start, end, vba); // Dryrun
        hipDeviceSynchronize();
        hipEventRecord(tstart); // Timing start
        for (int i = 0; i < num_iters; ++i)
          kernel<<<bpg, tpb, smem>>>(start, end, vba);
        hipEventRecord(tstop); // Timing stop
        hipEventSynchronize(tstop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, tstart, tstop);

        hipEventDestroy(tstart);
        hipEventDestroy(tstop);

        // Discard failed runs (attempt to clear the error to hipSuccess)
        if (hipGetLastError() != hipSuccess) {
          // Exit in case of unrecoverable error that needs a device reset
          ERRCHK_CUDA_KERNEL_ALWAYS();
          ERRCHK_CUDA_ALWAYS(hipGetLastError());
          continue;
        }

        if (milliseconds < best_time) {
          best_time = milliseconds;
          best_tpb  = tpb;
        }

        // printf("Auto-optimizing... Current tpb: (%d, %d, %d), time %f ms\n",
        //        tpb.x, tpb.y, tpb.z, (double)milliseconds / num_iters);
        // fflush(stdout);
      }
    }
  }
  c.tpb = best_tpb;

  // printf("\tThe best tpb: (%d, %d, %d), time %f ms\n", best_tpb.x,
  // best_tpb.y,
  //        best_tpb.z, (double)best_time / num_iters);

  FILE* fp = fopen(autotune_csv_path, "a");
  ERRCHK_ALWAYS(fp);
#if IMPLEMENTATION == SMEM_HIGH_OCCUPANCY_CT_CONST_TB
  fprintf(fp, "%d, (%d, %d, %d), (%d, %d, %d), %g\n", IMPLEMENTATION, nx, ny,
          nz, best_tpb.x, best_tpb.y, best_tpb.z,
          (double)best_time / num_iters);
#else
  fprintf(fp, "%d, %d, %d, %d, %d, %d, %d, %d, %g\n", IMPLEMENTATION, get_kernel_index(kernel), dims.x,
          dims.y, dims.z, best_tpb.x, best_tpb.y, best_tpb.z,
          (double)best_time / num_iters);
#endif
  fclose(fp);

  if (c.tpb.x * c.tpb.y * c.tpb.z <= 0) {
    fprintf(stderr,
            "Fatal error: failed to find valid thread block dimensions.\n");
  }
  ERRCHK_ALWAYS(c.tpb.x * c.tpb.y * c.tpb.z > 0);
  return c;
}
int
get_entries(char** dst, const char* line)
{
      char* line_copy = strdup(line);
      int counter = 0;
      char* token;
      token = strtok(line_copy,",");
      while(token != NULL)
      {
              dst[counter] = strdup(token);
              ++counter;
              token = strtok(NULL,",");
      }
      free(line_copy);
      return counter;
}
static int3
read_optim_tpb(const Kernel kernel, const int3 dims)
{
  const char* filename = autotune_csv_path;
  FILE *file = fopen ( filename, "r" );
  int3 res = {-1,-1,-1};
  const double best_time = pow(10.0,20);
  if (file != NULL) {
    char line [1000];
    while(fgets(line,sizeof line,file)!= NULL) /* read a line from a file */ {
      char* entries[9];
      int num_entries = get_entries(entries,line);
      if(num_entries > 1)
      {
         int kernel_index  = atoi(entries[1]);
         int3 read_dims = {atoi(entries[2]), atoi(entries[3]), atoi(entries[4])};
         int3 tpb = {atoi(entries[5]), atoi(entries[6]), atoi(entries[7])};
         double time = atof(entries[8]);
	 res =  (read_dims == dims && kernel_index == get_kernel_index(kernel) && time < best_time) ? tpb  : res;
      }
  }
    fclose(file);
  }
  else {
    perror(filename); //print the error message on stderr.
  }
  return res;
}


static TBConfig
getOptimalTBConfig(const Kernel kernel, const int3 dims, VertexBufferArray vba)
{
  const int3 read_tpb = read_optim_tpb(kernel,dims); 
  if(read_tpb != (int3){-1,-1,-1})
  {
	  return 
	  {
		  kernel,
		  dims,
		  (dim3){(uint32_t)read_tpb.x, (uint32_t)read_tpb.y, (uint32_t)read_tpb.z}
	  };
  }
  for (auto c : tbconfigs) {
    if (c.kernel == kernel && c.dims == dims)
      return c;
  }
  TBConfig c = autotune(kernel, dims, vba);
  tbconfigs.push_back(c);
  return c;
}
Kernel
GetOptimizedKernel(const AcKernel kernel_enum, const VertexBufferArray vba)
{
	#include "user_kernel_ifs.h"
	//silence unused warnings
	(void)vba;
	return kernels[(int) kernel_enum];
}

