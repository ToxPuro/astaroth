#include "hip/hip_runtime.h"
/*
    Copyright (C) 2014-2021, Johannes Pekkila, Miikka Vaisala.

    This file is part of Astaroth.

    Astaroth is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Astaroth is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Astaroth.  If not, see <http://www.gnu.org/licenses/>.
*/
#define rocprim__warpSize() rocprim::warp_size()
#define rocprim__warpId()   rocprim::warp_id()
#define rocprim__warp_shuffle_down rocprim::warp_shuffle_down
#define rocprim__warp_shuffle rocprim::warp_shuffle

#include "acc_runtime.h"
#include "ac_buffer.h"

#include "user_defines_runtime_lib.h"

#include "../acc/string_vec.h"
typedef void (*Kernel)(const int3, const int3, DeviceVertexBufferArray vba);
#define AcReal3(x,y,z)   (AcReal3){x,y,z}
#define AcComplex(x,y)   (AcComplex){x,y}
static AcBool3 dimension_inactive{};
static int3 raytracing_subblock{};
static int  x_ray_shared_mem_block_size{};
static int  z_ray_shared_mem_block_size{};
static bool sparse_autotuning=false;
static int3    max_tpb_for_reduce_kernels{100,100,100};
#include <math.h> 
#include <vector> // tbconfig

#include "errchk.h"
#include "math_utils.h"
#include <unordered_map>
#include <utility>
#include <sys/stat.h>

#if AC_USE_HIP
#include <hip/hip_runtime.h> // Needed in files that include kernels
#include <rocprim/rocprim.hpp>
#include <hip/hip_cooperative_groups.h>
#else
#include <hip/hip_cooperative_groups.h>
#endif

#include "user_kernel_declarations.h"
#include "kernel_reduce_info.h"


#define USE_COMPRESSIBLE_MEMORY (0)

//TP: unfortunately cannot use color output since it might not be supported in each env
const bool useColor = false;

#define GREEN "\033[1;32m"
#define YELLOW "\033[1;33m"
#define RESET "\033[0m"

#define COLORIZE(symbol, color) (useColor ? color symbol RESET : symbol)


#include "acc/implementation.h"
typedef struct
{
	void* data;
	size_t bytes;
} AcDeviceTmpBuffer;

static dim3 last_tpb = (dim3){0, 0, 0};
struct Int3Hash {
    std::size_t operator()(const int3& v) const {
        return std::hash<int>()(v.x) ^ std::hash<int>()(v.y) << 1 ^ std::hash<int>()(v.z) << 2;
    }
};
std::array<std::unordered_map<int3,int,Int3Hash>,NUM_KERNELS> reduce_offsets;
int kernel_running_reduce_offsets[NUM_KERNELS];

AcAutotuneMeasurement
return_own_measurement(const AcAutotuneMeasurement local_measurement) {return local_measurement;}

static int grid_pid = 0;
[[maybe_unused]] static int nprocs   = 0;
static AcMeasurementGatherFunc gather_func  = return_own_measurement;
#if AC_MPI_ENABLED
AcResult
acInitializeRuntimeMPI(const int _grid_pid,const int _nprocs, const AcMeasurementGatherFunc mpi_gather_func)
{
	grid_pid = _grid_pid;
	nprocs   = _nprocs;
	gather_func = mpi_gather_func;
	return AC_SUCCESS;
}
#endif

Volume
acKernelLaunchGetLastTPB(void)
{
  return to_volume(last_tpb);
}
int
acGetKernelReduceScratchPadSize(const AcKernel kernel)
{
	return kernel_running_reduce_offsets[(int)kernel];
}
int
acGetKernelReduceScratchPadMinSize()
{
	int res = 0; 
	for(int i = 0; i < NUM_KERNELS; ++i)
		res = (res < kernel_running_reduce_offsets[i]) ? kernel_running_reduce_offsets[i] : res;
	return res;
}
#include "../../src/helpers/ceil_div.h"

Volume
get_bpg(Volume dims, const Volume tpb)
{
  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING:             // Fallthrough
  case EXPLICIT_CACHING:             // Fallthrough
  case EXPLICIT_CACHING_3D_BLOCKING: // Fallthrough
  case EXPLICIT_CACHING_4D_BLOCKING: // Fallthrough
  case EXPLICIT_PINGPONG_txw:        // Fallthrough
  case EXPLICIT_PINGPONG_txy:        // Fallthrough
  case EXPLICIT_PINGPONG_txyblocked: // Fallthrough
  case EXPLICIT_PINGPONG_txyz:       // Fallthrough
  case EXPLICIT_ROLLING_PINGPONG: {
    return (Volume){
        as_size_t(ceil_div(dims.x,tpb.x)),
        as_size_t(ceil_div(dims.y,tpb.y)),
        as_size_t(ceil_div(dims.z,tpb.z)),
    };
  }
  default: {
    ERROR("Invalid IMPLEMENTATION in get_bpg");
    return (Volume){0, 0, 0};
  }
  }
}
#include "stencil_accesses.h"
#include "../acc/mem_access_helper_funcs.h"

static bool
is_raytracing_kernel(const AcKernel kernel)
{
	for(int ray = 0; ray < NUM_RAYS; ++ray)
	{
		for(int field = 0; field < NUM_ALL_FIELDS; ++field)
			if(incoming_ray_value_accessed[kernel][field][ray]) return true;
	}
	return false;
}

static int
num_fields_ray_accessed_read_and_written(const AcKernel kernel)
{
	int res = 0;
	for(int field = 0; field < NUM_ALL_FIELDS; ++field)
	{
		if(write_called[kernel][field] || stencils_accessed[kernel][field][0])
		{
			res++;
			continue;
		}
		for(int ray = 0; ray < NUM_RAYS; ++ray)
		{
			if(
			    incoming_ray_value_accessed[kernel][field][ray]
			    || outgoing_ray_value_accessed[kernel][field][ray]
			    )
			{
				res++;
				continue;
			}
		}
	}
	return res;
}

static AcBool3
raytracing_step_direction(const AcKernel kernel)
{
	for(int ray = 0; ray < NUM_RAYS; ++ray)
	{
		for(int field = 0; field < NUM_ALL_FIELDS; ++field)
			if(incoming_ray_value_accessed[kernel][field][ray])
			{
				if(ray_directions[ray].z != 0) return (AcBool3){false,false,true};
				if(ray_directions[ray].y != 0) return (AcBool3){false,true,false};
				if(ray_directions[ray].x != 0) return (AcBool3){true,false,false};
			}
	}
	return (AcBool3){false,false,false};
}
static AcBool3
raytracing_directions(const AcKernel kernel)
{
	AcBool3 res = (AcBool3){false,false,false};
	for(int ray = 0; ray < NUM_RAYS; ++ray)
	{
		for(int field = 0; field < NUM_ALL_FIELDS; ++field)
			if(incoming_ray_value_accessed[kernel][field][ray])
			{
				res.x |= ray_directions[ray].x != 0;
				res.y |= ray_directions[ray].y != 0;
				res.z |= ray_directions[ray].z != 0;
			}
	}
	return res;
}
static int
raytracing_number_of_directions(const AcKernel kernel)
{
	const auto dirs = raytracing_directions(kernel);
	return dirs.x+dirs.y+dirs.z;
}

static bool
is_coop_raytracing_kernel(const AcKernel kernel)
{
	return is_raytracing_kernel(kernel) && (raytracing_number_of_directions(kernel) > 1);
}

Volume
get_bpg(Volume dims, const AcKernel kernel, const int3 block_factors, const Volume tpb)
{
	if(kernel_has_block_loops(kernel)) return get_bpg(ceil_div(dims,block_factors), tpb);
	return get_bpg(dims,tpb);
}

static hipDeviceProp_t
get_device_prop()
{
  hipDeviceProp_t props;
  ERRCHK_CUDA_ALWAYS(hipGetDeviceProperties(&props, 0));
  return props;
}


static int3
get_ghosts()
{
  return (int3){
	  dimension_inactive.x ? 0 : NGHOST,
	  dimension_inactive.y ? 0 : NGHOST,
	  dimension_inactive.z ? 0 : NGHOST
  };
}
template <typename T>
bool
is_large_launch(const T dims)
{
  const int3 ghosts = get_ghosts();
  return ((int)dims.x > ghosts.x && (int)dims.y > ghosts.y && (int)dims.z > ghosts.z);
}


bool
is_valid_configuration(const Volume dims, const Volume tpb, const AcKernel kernel)
{
  const size_t warp_size    = get_device_prop().warpSize;
  const size_t xmax         = (size_t)(warp_size * ceil_div(dims.x,warp_size));
  const size_t ymax         = (size_t)(warp_size * ceil_div(dims.y,warp_size));
  const size_t zmax         = (size_t)(warp_size * ceil_div(dims.z,warp_size));
  const bool too_large      = (tpb.x > xmax) || (tpb.y > ymax) || (tpb.z > zmax);
  const bool not_full_warp  = (tpb.x*tpb.y*tpb.z < warp_size);
  if(is_coop_raytracing_kernel(kernel))
  {
	int maxBlocksPerSM{};
	ERRCHK_CUDA_ALWAYS(hipOccupancyMaxActiveBlocksPerMultiprocessor(
			&maxBlocksPerSM,
			kernels[kernel],
			tpb.x*tpb.y*tpb.z,
			0
	));
  	const auto bpg = get_bpg(dims,to_volume(tpb));
	const int totalMaxBlocks = get_device_prop().multiProcessorCount*maxBlocksPerSM;
	if((int)(bpg.x*bpg.y*bpg.z) > totalMaxBlocks) return false;
  }
  if(raytracing_step_direction(kernel).x)
  {
	if(((int)tpb.y - (x_ray_shared_mem_block_size)) != 0) return false;
  }

  else
  {
  	//TP: in most cases this a reasonable limitation but at least theoretically the shape of the threadblock might be more important
  	//    than warp considerations. So impose this limitation only if the user allows it
  	if (sparse_autotuning && (dims.x >= warp_size && tpb.x % warp_size != 0)) return false;
  }

  if(kernel_reduces_profile(kernel))
  {
	  if(tpb.y > (size_t)max_tpb_for_reduce_kernels.y) return false;
	  if(tpb.z > (size_t)max_tpb_for_reduce_kernels.z) return false;
	  //TP: if we enforce that tpb.x is a multiple of the warp size then 
	  //can easily do warp reduce while doing a reduction whose result is not x-dependent --> major saving in memory and performance increase
	  if(dims.x >= warp_size && tpb.x % warp_size != 0) return false;
  }
//  const bool single_tb      = (tpb.x >= dims.x) && (tpb.y >= dims.y) && (tpb.z >= dims.z);

  //TP: if not utilizing the whole warp invalid, expect if dims are so small that could not utilize a whole warp 
  if(not_full_warp && is_large_launch(dims)) return false;

  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING: {

	    if (too_large)
	      return false;

	    return true;
	  }
	  case EXPLICIT_CACHING_4D_BLOCKING: // Fallthrough
	    if (tpb.z > 1) return false;
	    [[fallthrough]];
	  case EXPLICIT_CACHING: // Fallthrough
	  case EXPLICIT_CACHING_3D_BLOCKING: {

	    // For some reason does not work without this
	    // Probably because of break vs continue when fetching (some threads
	    // quit too early if the dims are not divisible)
	    return !(dims.x % tpb.x) && !(dims.y % tpb.y) && !(dims.z % tpb.z);
	  }
	  case EXPLICIT_PINGPONG_txw: {
	    return (tpb.y == 1) && (tpb.z == 1);
	  }
	  case EXPLICIT_PINGPONG_txy: {
	    return (tpb.z == 1);
	  }
	  case EXPLICIT_PINGPONG_txyblocked: {
	    return (tpb.z == 1);
	  }
	  case EXPLICIT_PINGPONG_txyz: {
	    return true;
	  }
	  case EXPLICIT_ROLLING_PINGPONG: {
	    // OK for every other rolling pingpong implementation
	    // return true;

	    // Required only when unrolling smem loads
	    // Ensures two unrolls is enough to fill the smem buffer
	    return (2 * tpb.x >= STENCIL_WIDTH - 1 + tpb.x) &&
		   (2 * tpb.y >= STENCIL_HEIGHT - 1 + tpb.y);
	  }
	  default: {
	    ERROR("Invalid IMPLEMENTATION in is_valid_configuration");
    return false;
  }
  }
}

const bool SHARED_MEM_Z_RAYS = false;
size_t
get_smem(const AcKernel kernel, const Volume tpb, const size_t stencil_order,
         const size_t bytes_per_elem)
{
  if(is_raytracing_kernel(kernel) && raytracing_step_direction(kernel).x)
  {
	//TP: we pad the y dimension by one to avoid bank conflicts
	return bytes_per_elem*(tpb.y+1)*tpb.z*(x_ray_shared_mem_block_size+2)*num_fields_ray_accessed_read_and_written(kernel);
  }
  if(is_raytracing_kernel(kernel) && raytracing_step_direction(kernel).z && SHARED_MEM_Z_RAYS)
  {
	return bytes_per_elem*(tpb.x+2)*(tpb.y+2)*(z_ray_shared_mem_block_size+2)*num_fields_ray_accessed_read_and_written(kernel);
  }
  switch (IMPLEMENTATION) {
  case IMPLICIT_CACHING: {
    return 0;
  }
  case EXPLICIT_CACHING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) * tpb.z *
           bytes_per_elem;
  }
  case EXPLICIT_CACHING_3D_BLOCKING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) *
           (tpb.z + stencil_order) * bytes_per_elem;
  }
  case EXPLICIT_CACHING_4D_BLOCKING: {
    return (tpb.x + stencil_order) * (tpb.y + stencil_order) * tpb.z *
           (NUM_FIELDS)*bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txw: {
    return 2 * (tpb.x + stencil_order) * NUM_FIELDS * bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txy: {
    return 2 * (tpb.x + stencil_order) * (tpb.y + stencil_order) *
           bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txyblocked: {
    const size_t block_size = 7;
    return 2 * (tpb.x + stencil_order) * (tpb.y + stencil_order) * block_size *
           bytes_per_elem;
  }
  case EXPLICIT_PINGPONG_txyz: {
    return 2 * (tpb.x + stencil_order) * (tpb.y + stencil_order) *
           (tpb.z + stencil_order) * bytes_per_elem;
  }
  case EXPLICIT_ROLLING_PINGPONG: {
    // tpbxy slices with halos
    // tpbz depth + 1 rolling cache slab
    return EXPLICIT_ROLLING_PINGPONG_BLOCKSIZE * (tpb.x + stencil_order) *
           (tpb.y + stencil_order) * (tpb.z + 1) * bytes_per_elem;
  }
  default: {
    ERROR("Invalid IMPLEMENTATION in get_smem");
    return (size_t)-1;
  }
  }
}

/*
// Device info (TODO GENERIC)
// Use the maximum available reg count per thread
#define REGISTERS_PER_THREAD (255)
#define MAX_REGISTERS_PER_BLOCK (65536)
#if AC_DOUBLE_PRECISION
#define MAX_THREADS_PER_BLOCK                                                  \
  (MAX_REGISTERS_PER_BLOCK / REGISTERS_PER_THREAD / 2)
#else
#define MAX_THREADS_PER_BLOCK (MAX_REGISTERS_PER_BLOCK / REGISTERS_PER_THREAD)
#endif
*/

__device__ __constant__ AcMeshInfoScalars d_mesh_info;
#include "dconst_arrays_decl.h"
//TP: We do this ugly macro because I want to keep the generated headers the same if we are compiling cpu analysis and for the actual gpu comp
#define DECLARE_GMEM_ARRAY(DATATYPE, DEFINE_NAME, ARR_NAME) __device__ __constant__ DATATYPE* AC_INTERNAL_gmem_##DEFINE_NAME##_arrays_##ARR_NAME 
#define DECLARE_CONST_DIMS_GMEM_ARRAY(DATATYPE, DEFINE_NAME, ARR_NAME, LEN) static __device__ DATATYPE AC_INTERNAL_gmem_##DEFINE_NAME##_arrays_##ARR_NAME[LEN]
#include "gmem_arrays_decl.h"

AcReal
get_reduce_state_flush_var_real(const AcReduceOp state)
{
	return 
		(state == NO_REDUCE || state == REDUCE_SUM) ? 0.0 :
		(state == REDUCE_MIN) ? AC_REAL_MAX :
		(state == REDUCE_MAX) ? -AC_REAL_MAX :
		0.0;
}

int
get_reduce_state_flush_var_int(const AcReduceOp state)
{
	return 
		(state == NO_REDUCE || state == REDUCE_SUM) ? 0 :
		(state == REDUCE_MIN) ? INT_MAX:
		(state == REDUCE_MAX) ? -INT_MAX:
		0;
}

#if AC_DOUBLE_PRECISION
float
get_reduce_state_flush_var_float(const AcReduceOp state)
{
	return 
		(state == NO_REDUCE || state == REDUCE_SUM) ? 0.0 :
		(state == REDUCE_MIN) ? FLT_MAX :
		(state == REDUCE_MAX) ? -FLT_MAX :
		0.0;
}
#endif

typedef struct {
  AcKernel kernel;
  int3 dims;
  dim3 tpb;
} TBConfig;

static std::vector<TBConfig> tbconfigs;


static TBConfig getOptimalTBConfig(const AcKernel kernel, const int3 start, const int3 end, VertexBufferArray vba);


template <typename T>
T TO_CORRECT_ORDER(const T vol)
{
	return vol;
}
size_t TO_CORRECT_ORDER(const size_t size)
{
	return size;
}


#define KERNEL_LAUNCH(func,bgp,tpb,...) \
	func<<<TO_CORRECT_ORDER(bpg),TO_CORRECT_ORDER(tpb),__VA_ARGS__>>>

#define KERNEL_VBA_LAUNCH(func,bgp,tpb,...) \
	func<<<TO_CORRECT_ORDER(bpg),TO_CORRECT_ORDER(tpb),__VA_ARGS__>>>


__device__ __constant__ AcReal* d_symbol_reduce_scratchpads_real[NUM_REAL_SCRATCHPADS];
static AcReal* d_reduce_scratchpads_real[NUM_REAL_SCRATCHPADS];
static size_t d_reduce_scratchpads_size_real[NUM_REAL_SCRATCHPADS];
__device__ __constant__ AcReal  d_reduce_real_res_symbol[NUM_REAL_SCRATCHPADS];

AcResult
acKernelFlush(const hipStream_t stream, AcReal* arr, const size_t n,
              const AcReal value)
{
	return acKernelFlushReal(stream,arr,n,value);
}

AcResult
acKernelFlush(const hipStream_t stream, int* arr, const size_t n,
              const int value)
{
	return acKernelFlushInt(stream,arr,n,value);
}
AcResult
acKernelFlush(const hipStream_t stream, AcComplex* arr, const size_t n,
              const AcComplex value)
{
	return acKernelFlushComplex(stream,arr,n,value);
}

#if AC_DOUBLE_PRECISION
AcResult
acKernelFlush(const hipStream_t stream, float* arr, const size_t n,
              const float value)
{
	return acKernelFlushFloat(stream,arr,n,value);
}
#endif




#include "reduce_helpers.h"


void
resize_scratchpads_to_fit(const size_t n_elems, VertexBufferArray vba, const AcKernel kernel)
{
	resize_reals_to_fit(n_elems,vba,kernel);
	resize_ints_to_fit(n_elems,vba,kernel);
#if AC_DOUBLE_PRECISION
	resize_floats_to_fit(n_elems,vba,kernel);
#endif
}

size_t
acGetRealScratchpadSize(const size_t i)
{
	return d_reduce_scratchpads_size_real[i];
}

//The macros above generate d arrays like these:

// Astaroth 2.0 backwards compatibility START
#define d_multigpu_offset (d_mesh_info.int3_params[AC_multigpu_offset])


#define DEVICE_INLINE __device__ __forceinline__
#include "dconst_decl.h"
#include "output_value_decl.h"



#include "get_address.h"
#include "load_dconst_arrays.h"
#include "store_dconst_arrays.h"

#define PROFILE_X_Y_OR_Z_INDEX(i,j) \
  ((i) + (j)*VAL(AC_mlocal).x)

#define PROFILE_Y_X_OR_Z_INDEX(i,j) \
  ((i) + (j)*VAL(AC_mlocal).y)

#define PROFILE_Z_X_OR_Y_INDEX(i,j) \
  ((i) + (j)*VAL(AC_mlocal).z)


#define DEVICE_VTXBUF_IDX(i, j, k)                                             \
  ((i) + (j)*VAL(AC_mlocal).x + (k)*VAL(AC_mlocal_products).xy)

#define DEVICE_VARIABLE_VTXBUF_IDX(i, j, k,dims)                                             \
  ((i) + dims.x*((j) + (k)*dims.y))

#define LOCAL_COMPDOMAIN_IDX(coord) \
	((coord.x) + (coord.y) * VAL(AC_nlocal).x + (coord.z) * VAL(AC_nlocal_products).xy)

#define print printf                          // TODO is this a good idea?
// passes an array into a device function and then calls len (need to modify
// the compiler to always pass arrays to functions as references before
// re-enabling)

#include "random.cuh"

#define suppress_unused_warning(X) (void)X
#define longlong long long
#define size(arr) (int)(sizeof(arr) / sizeof(arr[0])) // Leads to bugs if the user
#define error_message(error,message) 
#define fatal_error_message(error,message) 

__device__
AcReal
safe_access(const AcReal* arr, const int dims, const int index, const char* name)
{
	if(arr == NULL)
	{
		printf("Trying to access %s which is NULL!\n",name);
		//TP: assert is not defined on Mahti :(
		//assert(false);
		return 0.0;
	}
	else if(index < 0 || index >= dims)
	{
		printf("Trying to access %s out of bounds!: %d\n",name,index);
		//TP: assert is not defined on Mahti :(
		//assert(false);
		return 0.0;
	}
	return arr[index];
}
__device__ UNUSED
AcReal
safe_access(const AcReal* arr, const int dims, const int index, const AcRealArrayParam param)
{
	return safe_access(arr,dims,index,real_array_names__device__[param]);
}

#include "device_fields_info.h"

static __device__ UNUSED
int3
ac_get_field_halos(const Field& field)
{
	return VAL(vtxbuf_device_halos[field]);
}


#define postprocess_reduce_result(DST,OP)
#include "user_kernels.h"
#undef size
#undef longlong


template<typename T1, typename T2>
AcResult
acLaunchKernelVariadic1d(AcKernel kernel, const hipStream_t stream, const size_t start, const size_t end,T1 param1, T2 param2)
{
  const Volume volume_start = {start,0,0};
  const Volume volume_end   = {end,1,1};
  VertexBufferArray vba{};
  acLoadKernelParams(vba.on_device.kernel_input_params,kernel,param1,param2); 
  return acLaunchKernel(kernel,stream,volume_start,volume_end,vba);
}

template<typename T1, typename T2>
AcResult
acLaunchKernelVariadic1d(AcKernel kernel, const int stream, const int start, const size_t end,T1 param1, T2 param2)
{
  const Volume volume_start = {as_size_t(start),0,0};
  const Volume volume_end   = {end,1,1};
  VertexBufferArray vba{};
  acLoadKernelParams(vba.on_device.kernel_input_params,kernel,param1,param2); 
  return acLaunchKernel(kernel,hipStream_t(stream),volume_start,volume_end,vba);
}

AcResult
acKernelFlushReal(const hipStream_t stream, AcReal* arr, const size_t n,
              const AcReal value)
{
  ERRCHK_ALWAYS(arr || n == 0);
  if(n == 0) return AC_SUCCESS;
  acLaunchKernelVariadic1d(AC_FLUSH_REAL,stream,0,n,arr,value);
  ERRCHK_CUDA_KERNEL_ALWAYS();
  return AC_SUCCESS;
}

AcResult
acKernelFlushComplex(const hipStream_t stream, AcComplex* arr, const size_t n,
              const AcComplex value)
{
  ERRCHK_ALWAYS(arr || n == 0);
  if(n == 0) return AC_SUCCESS;
  acLaunchKernelVariadic1d(AC_FLUSH_COMPLEX,stream,0,n,arr,value);
  ERRCHK_CUDA_KERNEL_ALWAYS();
  return AC_SUCCESS;
}

AcResult
acKernelFlushInt(const hipStream_t stream, int* arr, const size_t n,
              const int value)
{
  ERRCHK_ALWAYS(arr || n == 0);
  if(n == 0) return AC_SUCCESS;
  acLaunchKernelVariadic1d(AC_FLUSH_INT,stream,0,n,arr,value);
  ERRCHK_CUDA_KERNEL_ALWAYS();
  return AC_SUCCESS;
}

AcResult
acKernelFlushFloat(const hipStream_t stream, float* arr, const size_t n,
              const float value)
{
  ERRCHK_ALWAYS(arr || n == 0);
  if(n == 0) return AC_SUCCESS;
  acLaunchKernelVariadic1d(AC_FLUSH_FLOAT,stream,0,n,arr,value);
  ERRCHK_CUDA_KERNEL_ALWAYS();
  return AC_SUCCESS;
}


#include "user_built-in_constants.h"
#include "user_builtin_non_scalar_constants.h"



#if USE_COMPRESSIBLE_MEMORY
#include <hip/hip_runtime.h>

#define ERRCHK_CU_ALWAYS(x) ERRCHK_ALWAYS((x) == hipSuccess)

static hipError_t
mallocCompressible(void** addr, const size_t requested_bytes)
{
  hipDevice_t device;
  ERRCHK_ALWAYS(hipCtxGetDevice(&device) == hipSuccess);

  hipMemAllocationProp prop;
  memset(&prop, 0, sizeof(hipMemAllocationProp));
  prop.type                       = hipMemAllocationTypePinned;
  prop.location.type              = hipMemLocationTypeDevice;
  prop.location.id                = device;
  prop.allocFlags.compressionType = CU_MEM_ALLOCATION_COMP_GENERIC;

  size_t granularity;
  ERRCHK_CU_ALWAYS(hipMemGetAllocationGranularity(
      &granularity, &prop, hipMemAllocationGranularityMinimum));

  // Pad to align
  const size_t bytes = ((requested_bytes - 1) / granularity + 1) * granularity;

  hipDeviceptr_t dptr;
  ERRCHK_ALWAYS(hipMemAddressReserve(&dptr, bytes, 0, 0, 0) == hipSuccess);

  hipMemGenericAllocationHandle_t handle;
  ERRCHK_ALWAYS(hipMemCreate(&handle, bytes, &prop, 0) == hipSuccess)

  // Check if hipMemCreate was able to allocate compressible memory.
  hipMemAllocationProp alloc_prop;
  memset(&alloc_prop, 0, sizeof(hipMemAllocationProp));
  hipMemGetAllocationPropertiesFromHandle(&alloc_prop, handle);
  ERRCHK_ALWAYS(alloc_prop.allocFlags.compressionType ==
                CU_MEM_ALLOCATION_COMP_GENERIC);

  ERRCHK_ALWAYS(hipMemMap(dptr, bytes, 0, handle, 0) == hipSuccess);
  ERRCHK_ALWAYS(hipMemRelease(handle) == hipSuccess);

  hipMemAccessDesc accessDescriptor;
  accessDescriptor.location.id   = prop.location.id;
  accessDescriptor.location.type = prop.location.type;
  accessDescriptor.flags         = hipMemAccessFlagsProtReadWrite;

  ERRCHK_ALWAYS(hipMemSetAccess(dptr, bytes, &accessDescriptor, 1) ==
                hipSuccess);

  *addr = (void*)dptr;
  return hipSuccess;
}

static void
freeCompressible(void* ptr, const size_t requested_bytes)
{
  hipDevice_t device;
  ERRCHK_ALWAYS(hipCtxGetDevice(&device) == hipSuccess);

  hipMemAllocationProp prop;
  memset(&prop, 0, sizeof(hipMemAllocationProp));
  prop.type                       = hipMemAllocationTypePinned;
  prop.location.type              = hipMemLocationTypeDevice;
  prop.location.id                = device;
  prop.allocFlags.compressionType = CU_MEM_ALLOCATION_COMP_GENERIC;

  size_t granularity = 0;
  ERRCHK_ALWAYS(hipMemGetAllocationGranularity(
                    &granularity, &prop, hipMemAllocationGranularityMinimum) ==
                hipSuccess);
  const size_t bytes = ((requested_bytes - 1) / granularity + 1) * granularity;

  ERRCHK_ALWAYS(ptr);
  ERRCHK_ALWAYS(hipMemUnmap((hipDeviceptr_t)ptr, bytes) == hipSuccess);
  ERRCHK_ALWAYS(hipMemAddressFree((hipDeviceptr_t)ptr, bytes) == hipSuccess);
}
#endif

AcResult
acPBAReset(const hipStream_t stream, ProfileBufferArray* pba, const AcMeshDims* dims)
{
  // Set pba.in data to all-nan and pba.out to 0
  for (int i = 0; i < NUM_PROFILES; ++i) {
    acKernelFlush(stream, pba->in[i],  prof_count(Profile(i),dims[i].m1), (AcReal)NAN);
    acKernelFlush(stream, pba->out[i], prof_count(Profile(i),dims[i].m1), (AcReal)0);
  }
  return AC_SUCCESS;
}
size_t
get_amount_of_device_memory_free()
{
	size_t free_mem, total_mem;
	ERRCHK_CUDA_ALWAYS(acMemGetInfo(&free_mem,&total_mem));
	return free_mem;
}
void
device_malloc(void** dst, const size_t bytes)
{
  if(get_amount_of_device_memory_free() < bytes)
  {
	fprintf(stderr,"Tried to allocate %ld bytes but have only %ld bytes of memory left on the device\n", bytes, get_amount_of_device_memory_free());
  	ERRCHK_ALWAYS(get_amount_of_device_memory_free() >= bytes);
  }
 #if USE_COMPRESSIBLE_MEMORY 
    ERRCHK_CUDA_ALWAYS(mallocCompressible(dst, bytes));
 #else
    ERRCHK_CUDA_ALWAYS(acMalloc(dst, bytes));
  #endif
  ERRCHK_ALWAYS(dst != NULL);
}
void
device_malloc(AcReal** dst, const size_t bytes)
{
	device_malloc((void**)dst,bytes);
}

template <typename T>
void
device_free(T** dst, const int bytes)
{
#if USE_COMPRESSIBLE_MEMORY
  freeCompressible(*dst, bytes);
#else
  ERRCHK_CUDA_ALWAYS(acFree(*dst));
  //used to silence unused warning
  (void)bytes;
#endif
  *dst = NULL;
}

size_t
device_resize(void** dst,const size_t old_bytes,const size_t new_bytes)
{
	if(old_bytes >= new_bytes) return old_bytes;
	if(old_bytes) device_free(dst,old_bytes);
	device_malloc(dst,new_bytes);
	return new_bytes;
}


ProfileBufferArray
acPBACreate(const AcMeshDims* dims)
{
  ProfileBufferArray pba{};
  for (int i = 0; i < NUM_PROFILES; ++i) {
    const size_t bytes = prof_size(Profile(i),dims[i].m1)*sizeof(AcReal);
    device_malloc(&pba.in[i],  bytes);
    device_malloc(&pba.out[i], bytes);
    //pba.out[i] = pba.in[i];
  }

  acPBAReset(0, &pba, dims);
  ERRCHK_CUDA_ALWAYS(acDeviceSynchronize());
  return pba;
}

void
acPBADestroy(ProfileBufferArray* pba, const AcMeshDims* dims)
{
  for (int i = 0; i < NUM_PROFILES; ++i) {
    const size_t bytes = prof_size(Profile(i),dims[i].m1)*sizeof(AcReal);
    device_free(&pba->in[i],  bytes);
    device_free(&pba->out[i], bytes);
    pba->in[i]  = NULL;
    pba->out[i] = NULL;
  }
}

AcResult
acVBAReset(const hipStream_t stream, VertexBufferArray* vba)
{

  for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
    ERRCHK_ALWAYS(vba->on_device.in[i]);
    ERRCHK_ALWAYS(vba->on_device.out[i]);
    acKernelFlush(stream, vba->on_device.in[i], vba->counts[i], (AcReal)NAN);
    acKernelFlush(stream, vba->on_device.out[i], vba->counts[i], (AcReal)0.0);
  }

  const AcComplex zero_complex{0.0,0.0};
  for(int field = 0; field < NUM_COMPLEX_FIELDS; ++field)
  {
    size_t n = vba->computational_dims.m1.x*vba->computational_dims.m1.y*vba->computational_dims.m1.z;
    ERRCHK_ALWAYS(vba->on_device.complex_in[field]);
    acKernelFlush(stream, vba->on_device.complex_in[field],n,zero_complex);
  }
  memset(&vba->on_device.kernel_input_params,0,sizeof(acKernelInputParams));
  // Note: should be moved out when refactoring VBA to KernelParameterArray
  acPBAReset(stream, &vba->on_device.profiles, vba->profile_dims);
  return AC_SUCCESS;
}


template <typename T>
void
device_malloc(T** dst, const int bytes)
{
 #if USE_COMPRESSIBLE_MEMORY 
    ERRCHK_CUDA_ALWAYS(mallocCompressible((void**)dst, bytes));
 #else
    ERRCHK_CUDA_ALWAYS(acMalloc((void**)dst, bytes));
  #endif
}

#include "memcpy_to_gmem_arrays.h"

#include "memcpy_from_gmem_arrays.h"

template <typename P>
struct allocate_arrays
{
	void operator()(const AcMeshInfo& config) 
	{
		for(P array : get_params<P>())
		{
			if(config[array] == nullptr && is_accessed(array))
			{
				fprintf(stderr,"Passed %s as NULL but it is accessed kernels!!\n",get_name(array));
				fflush(stderr);
				ERRCHK_ALWAYS(config[array] != nullptr);
			}
			if (config[array] != nullptr && !is_dconst(array) && is_alive(array))
			{

#if AC_VERBOSE
				fprintf(stderr,"Allocating %s|%zu\n",get_name(array),get_array_length(array,config));
				fflush(stderr);
#endif
				auto d_mem_ptr = get_empty_pointer(array);
			        device_malloc(((void**)&d_mem_ptr), sizeof(config[array][0])*get_array_length(array,config));
				memcpy_to_gmem_array(array,d_mem_ptr);
			}
		}
	}
};


#if AC_USE_HIP
#include <hipcub/hipcub.hpp>
#define cub hipcub
#else
#include <hipcub/hipcub.hpp>
#endif

template <typename T>
void
cub_reduce(AcDeviceTmpBuffer& temp_storage, const hipStream_t stream, const T* d_in, const size_t count, T* d_out,  AcReduceOp reduce_op)
{
  switch(reduce_op)
  {
	  case(REDUCE_SUM):
	  	ERRCHK_CUDA(hipcub::DeviceReduce::Sum(temp_storage.data, temp_storage.bytes, d_in, d_out, count,stream));
	  	break;
	  case(REDUCE_MIN):
	  	ERRCHK_CUDA(hipcub::DeviceReduce::Min(temp_storage.data, temp_storage.bytes, d_in, d_out, count,stream));
	  	break;
	  case(REDUCE_MAX):
	  	ERRCHK_CUDA(hipcub::DeviceReduce::Max(temp_storage.data, temp_storage.bytes, d_in, d_out, count,stream));
	  	break;
	default:
		ERRCHK_ALWAYS(reduce_op != NO_REDUCE);
  }
  if (acGetLastError() != hipSuccess) {
          ERRCHK_CUDA_KERNEL_ALWAYS();
          ERRCHK_CUDA_ALWAYS(acGetLastError());
  }
}

size3_t
acGetProfileReduceScratchPadDims(const int profile, const AcMeshDims dims)
{
	const auto type = prof_types[profile];
    	if(type == PROFILE_YZ || type == PROFILE_ZY)
    		return
    		{
    		    	    dims.reduction_tile.x,
    		    	    dims.m1.y,
    		    	    dims.m1.z
    		};
    	if(type == PROFILE_XZ || type == PROFILE_ZX)
		return
    		{
    		    	    dims.m1.x,
    		    	    dims.reduction_tile.y,
    		    	    dims.m1.z
    		};
    	if(type == PROFILE_YX || type == PROFILE_XY)
		return
    		{
    		    	    dims.m1.x,
    		    	    dims.m1.y,
    		    	    dims.reduction_tile.z
    		};
	if(type == PROFILE_X)
	{
		return
		{
			dims.m1.x,
			dims.reduction_tile.y,
			dims.reduction_tile.z
		};
	}
	if(type == PROFILE_Y)
	{
		return
		{
			dims.reduction_tile.x,
			dims.m1.y,
			dims.reduction_tile.z
		};
	}
	if(type == PROFILE_Z)
	{
		return
		{
			dims.reduction_tile.x,
			dims.reduction_tile.y,
			dims.m1.z
		};
	}
	return dims.m1;
}

size_t
get_profile_reduce_scratchpad_size(const int profile, const VertexBufferArray vba)
{
	if(!reduced_profiles[profile]) return 0;
	const auto dims = acGetProfileReduceScratchPadDims(profile,vba.profile_dims[profile]);
	return dims.x*dims.y*dims.z*sizeof(AcReal);
}


void
init_scratchpads(VertexBufferArray* vba)
{
    vba->scratchpad_states = (AcScratchpadStates*)malloc(sizeof(AcScratchpadStates));
    memset(vba->scratchpad_states,0,sizeof(AcScratchpadStates));
    // Reductions
    {
	//TP: this is dangerous since it is not always true for DSL reductions but for now keep it
    	for(int i = 0; i < NUM_REAL_SCRATCHPADS; ++i) {
	    const size_t bytes =  
		    		  (i >= NUM_REAL_OUTPUTS) ? get_profile_reduce_scratchpad_size(i-NUM_REAL_OUTPUTS,*vba) :
				  0;
	    allocate_scratchpad_real(i,bytes,vba->scratchpad_states->reals[i]);
	    if(i < NUM_REAL_OUTPUTS)
	    {
	    	vba->reduce_buffer_real[i].src = &d_reduce_scratchpads_real[i];
	    	vba->reduce_buffer_real[i].cub_tmp = (AcReal**)malloc(sizeof(AcReal*));
	    	*(vba->reduce_buffer_real[i].cub_tmp) = NULL;
	    	vba->reduce_buffer_real[i].cub_tmp_size = (size_t*)malloc(sizeof(size_t));
	    	*(vba->reduce_buffer_real[i].cub_tmp_size) = 0;

	    	vba->reduce_buffer_real[i].buffer_size    = &d_reduce_scratchpads_size_real[i];
    		device_malloc((void**) &vba->reduce_buffer_real[i].res,sizeof(AcReal));
	    }
	    else
	    {
		    const Profile prof = (Profile)(i-NUM_REAL_OUTPUTS);
		    const auto dims = acGetProfileReduceScratchPadDims(prof,vba->profile_dims[prof]);
		    vba->profile_reduce_buffers[prof].src = 
		    {
			    d_reduce_scratchpads_real[i],
			    dims.x*dims.y*dims.z,
			    true,
			    (AcShape) { dims.x,dims.y,dims.z,1}
		    };
		    vba->profile_reduce_buffers[prof].transposed = acBufferCreateTransposed(
				vba->profile_reduce_buffers[prof].src, 
				acGetMeshOrderForProfile(prof_types[prof])
				  );
		    vba->profile_reduce_buffers[prof].mem_order = acGetMeshOrderForProfile(prof_types[prof]);

	    	    vba->profile_reduce_buffers[prof].cub_tmp = (AcReal**)malloc(sizeof(AcReal*));
	    	    *(vba->profile_reduce_buffers[prof].cub_tmp) = NULL;
	    	    vba->profile_reduce_buffers[prof].cub_tmp_size = (size_t*)malloc(sizeof(size_t));
	    	    *(vba->profile_reduce_buffers[prof].cub_tmp_size) = 0;
	    }
    	}
    }
    {
    	for(int i = 0; i < NUM_INT_OUTPUTS; ++i) {
	    const size_t bytes = 0;
	    allocate_scratchpad_int(i,bytes,vba->scratchpad_states->ints[i]);

	    vba->reduce_buffer_int[i].src= &d_reduce_scratchpads_int[i];
	    vba->reduce_buffer_int[i].cub_tmp = (int**)malloc(sizeof(int*));
	    *(vba->reduce_buffer_int[i].cub_tmp) = NULL;
	    vba->reduce_buffer_int[i].cub_tmp_size = (size_t*)malloc(sizeof(size_t));
	    *(vba->reduce_buffer_int[i].cub_tmp_size) = 0;
	    vba->reduce_buffer_int[i].buffer_size    = &d_reduce_scratchpads_size_int[i];
    	    device_malloc((void**) &vba->reduce_buffer_int[i].res,sizeof(int));
    	}

#if AC_DOUBLE_PRECISION
    	for(int i = 0; i < NUM_FLOAT_OUTPUTS; ++i) {
	    const size_t bytes = 0;
	    allocate_scratchpad_float(i,bytes,vba->scratchpad_states->floats[i]);

	    vba->reduce_buffer_float[i].src= &d_reduce_scratchpads_float[i];
	    vba->reduce_buffer_float[i].cub_tmp = (float**)malloc(sizeof(float*));
	    *(vba->reduce_buffer_float[i].cub_tmp) = NULL;
	    vba->reduce_buffer_float[i].cub_tmp_size = (size_t*)malloc(sizeof(size_t));
	    *(vba->reduce_buffer_float[i].cub_tmp_size) = 0;
	    vba->reduce_buffer_float[i].buffer_size    = &d_reduce_scratchpads_size_float[i];
    	    device_malloc((void**) &vba->reduce_buffer_float[i].res,sizeof(float));
    	}
#endif
    }
}
static inline AcMeshDims
acGetMeshDims(const AcMeshInfo info)
{
   #include "user_builtin_non_scalar_constants.h"
   const Volume n0 = to_volume(info[AC_nmin]);
   const Volume n1 = to_volume(info[AC_nlocal_max]);
   const Volume m0 = (Volume){0, 0, 0};
   const Volume m1 = to_volume(info[AC_mlocal]);
   const Volume nn = to_volume(info[AC_nlocal]);
   const Volume reduction_tile = (Volume)
   {
	   as_size_t(info.int3_params[AC_reduction_tile_dimensions].x),
	   as_size_t(info.int3_params[AC_reduction_tile_dimensions].y),
	   as_size_t(info.int3_params[AC_reduction_tile_dimensions].z)
   };

   return (AcMeshDims){
       .n0 = n0,
       .n1 = n1,
       .m0 = m0,
       .m1 = m1,
       .nn = nn,
       .reduction_tile = reduction_tile,
   };
}

static inline AcMeshDims
acGetMeshDims(const AcMeshInfo info, const VertexBufferHandle vtxbuf)
{
   #include "user_builtin_non_scalar_constants.h"
   const Volume n0 = to_volume(info[AC_nmin]);
   const Volume m1 = to_volume(info[vtxbuf_dims[vtxbuf]]);
   const Volume n1 = m1-n0;
   const Volume m0 = (Volume){0, 0, 0};
   const Volume nn = m1-n0*2;
   const Volume reduction_tile = (Volume)
   {
	   as_size_t(info.int3_params[AC_reduction_tile_dimensions].x),
	   as_size_t(info.int3_params[AC_reduction_tile_dimensions].y),
	   as_size_t(info.int3_params[AC_reduction_tile_dimensions].z)
   };

   return (AcMeshDims){
       .n0 = n0,
       .n1 = n1,
       .m0 = m0,
       .m1 = m1,
       .nn = nn,
       .reduction_tile = reduction_tile,
   };
}

AcReal* vba_in_buff = NULL;
AcReal* vba_out_buff = NULL;

VertexBufferArray
acVBACreate(const AcMeshInfo config)
{
  //TP: !HACK!
  //TP: Get active dimensions at the time VBA is created, works for now but should be moved somewhere else
  #include "user_builtin_non_scalar_constants.h"
  dimension_inactive = config[AC_dimension_inactive];
  sparse_autotuning  = config[AC_sparse_autotuning];
  raytracing_subblock = config[AC_raytracing_block_factors];
  x_ray_shared_mem_block_size = config[AC_x_ray_shared_mem_block_size];
  z_ray_shared_mem_block_size = config[AC_z_ray_shared_mem_block_size];

  max_tpb_for_reduce_kernels = config[AC_max_tpb_for_reduce_kernels];
  VertexBufferArray vba;
  vba.on_device.block_factor = config[AC_thread_block_loop_factors];

  vba.computational_dims = acGetMeshDims(config);

  size_t in_bytes  = 0;
  size_t out_bytes = 0;
  for(int i = 0; i  < NUM_FIELDS; ++i)
  {
  	vba.dims[i]    = acGetMeshDims(config,Field(i));
  	size_t count = vba.dims[i].m1.x*vba.dims[i].m1.y*vba.dims[i].m1.z;
  	size_t bytes = sizeof(vba.on_device.in[0][0]) * count;
  	vba.counts[i]         = count;
  	vba.bytes[i]          = bytes;
	in_bytes  += vba.bytes[i];
	if(vtxbuf_is_auxiliary[i]) continue;
	out_bytes += vba.bytes[i];
  }
  for(int p = 0; p < NUM_PROFILES; ++p)
  {
	  vba.profile_dims[p] = acGetMeshDims(config);
  	  vba.profile_counts[p] = vba.profile_dims[p].m1.x*vba.profile_dims[p].m1.y*vba.profile_dims[p].m1.z;
  }
  for(int field = 0; field < NUM_COMPLEX_FIELDS; ++field)
  {
  	size_t count = vba.computational_dims.m1.x*vba.computational_dims.m1.y*vba.computational_dims.m1.z;
	device_malloc(&vba.on_device.complex_in[field],sizeof(AcComplex)*count);
  }

  ERRCHK_ALWAYS(vba_in_buff == NULL);
  ERRCHK_ALWAYS(vba_out_buff == NULL);
  device_malloc((void**)&vba_in_buff,in_bytes);
  device_malloc((void**)&vba_out_buff,out_bytes);

  size_t out_offset = 0;
  size_t in_offset = 0;
  for (size_t i = 0; i < NUM_VTXBUF_HANDLES; ++i) {
    vba.on_device.in[i] = vba_in_buff + in_offset;
    ERRCHK_ALWAYS(vba.on_device.in[i] != NULL);
    in_offset += vba.counts[i];
    //device_malloc((void**) &vba.on_device.out[i],bytes);
    if (vtxbuf_is_auxiliary[i])
    {
      vba.on_device.out[i] = vba.on_device.in[i];
      ERRCHK_ALWAYS(vba.on_device.out[i] != NULL);
    }else{
      vba.on_device.out[i] = (vba_out_buff + out_offset);
      out_offset += vba.counts[i];
      if(vba.on_device.out[i] == NULL)
      {
         fprintf(stderr,"In bytes %zu; Out bytes: %zu\n",in_bytes,out_bytes);	
	 fflush(stderr);
       	 ERRCHK_ALWAYS(vba.on_device.out[i] != NULL);
      }
    }
  }


  AcArrayTypes::run<allocate_arrays>(config);

  // Note: should be moved out when refactoring VBA to KernelParameterArray
  vba.on_device.profiles = acPBACreate(vba.profile_dims);
  init_scratchpads(&vba);

  acVBAReset(0, &vba);
  ERRCHK_CUDA_ALWAYS(acDeviceSynchronize());
  return vba;
}

template <typename P>
struct update_arrays
{
	void operator()(const AcMeshInfo& config)
	{
		for(P array : get_params<P>())
		{
			if (is_dconst(array) || !is_alive(array)) continue;
			auto config_array = config[array];
			auto gmem_array   = get_empty_pointer(array);
			memcpy_from_gmem_array(array,gmem_array);
			size_t bytes = sizeof(config_array[0])*get_array_length(array,config);
			if (config_array == nullptr && gmem_array != nullptr) 
				device_free(&gmem_array,bytes);
			else if (config_array != nullptr && gmem_array  == nullptr) 
				device_malloc(&gmem_array,bytes);
			memcpy_to_gmem_array(array,gmem_array);
		}
	}
};
void
acUpdateArrays(const AcMeshInfo config)
{
  AcArrayTypes::run<update_arrays>(config);
}

template <typename P>
struct free_arrays
{
	void operator()(const AcMeshInfo& config)
	{
		for(P array: get_params<P>())
		{
			auto config_array = config[array];
			if (config_array == nullptr || is_dconst(array) || !is_alive(array)) continue;
			auto gmem_array = get_empty_pointer(array);
			memcpy_from_gmem_array(array,gmem_array);
			device_free(&gmem_array, get_array_length(array,config));
			memcpy_to_gmem_array(array,gmem_array);
		}
	}
};
void
destroy_profiles(VertexBufferArray* vba)
{
    for(int i = 0; i < NUM_PROFILES; ++i)
    {
        //TP: will break if allocated with compressed memory but too lazy to fix now: :(
        device_free((void**)&(vba->profile_reduce_buffers[i].transposed),0);
        free_scratchpad_real(i+NUM_REAL_OUTPUTS);
    }
}
void
destroy_real_scratchpads(VertexBufferArray* vba)
{
    for(int j = 0; j < NUM_REAL_OUTPUTS; ++j)
    {
	free_scratchpad_real(j);
	vba->reduce_buffer_real[j].src = NULL;

        ERRCHK_CUDA_ALWAYS(acFree(*vba->reduce_buffer_real[j].cub_tmp));
        ERRCHK_CUDA_ALWAYS(acFree(vba->reduce_buffer_real[j].res));

	free(vba->reduce_buffer_real[j].cub_tmp);
	free(vba->reduce_buffer_real[j].cub_tmp_size);
    }
}

void
destroy_scratchpads(VertexBufferArray* vba)
{
    destroy_real_scratchpads(vba);

    destroy_profiles(vba);

    for(int j = 0; j < NUM_INT_OUTPUTS; ++j)
    {
	free_scratchpad_int(j);
	vba->reduce_buffer_int[j].src = NULL;

        ERRCHK_CUDA_ALWAYS(acFree(*vba->reduce_buffer_int[j].cub_tmp));
        ERRCHK_CUDA_ALWAYS(acFree(vba->reduce_buffer_int[j].res));

	free(vba->reduce_buffer_int[j].cub_tmp);
	free(vba->reduce_buffer_int[j].cub_tmp_size);
    }
#if AC_DOUBLE_PRECISION
    for(int j = 0; j < NUM_FLOAT_OUTPUTS; ++j)
    {
	free_scratchpad_float(j);
	vba->reduce_buffer_float[j].src = NULL;

        ERRCHK_CUDA_ALWAYS(acFree(*vba->reduce_buffer_float[j].cub_tmp));
        ERRCHK_CUDA_ALWAYS(acFree(vba->reduce_buffer_float[j].res));

	free(vba->reduce_buffer_float[j].cub_tmp);
	free(vba->reduce_buffer_float[j].cub_tmp_size);
    }
#endif
}

void
acVBADestroy(VertexBufferArray* vba, const AcMeshInfo config)
{
  destroy_scratchpads(vba);
  //TP: does not work for compressible memory TODO: fix it if needed
  device_free(&(vba_in_buff), 0);
  device_free(&(vba_out_buff), 0);
  for(int field = 0; field < NUM_COMPLEX_FIELDS; ++field)
  {
  	device_free(&vba->on_device.complex_in[field], 0);
  }

  //Free arrays
  AcArrayTypes::run<free_arrays>(config);
  // Note: should be moved out when refactoring VBA to KernelParameterArray
  acPBADestroy(&vba->on_device.profiles,vba->profile_dims);
  memset(vba->profile_dims,0,NUM_PROFILES*sizeof(vba->profile_dims[0]));
  memset(vba->bytes,0,NUM_ALL_FIELDS*sizeof(size_t));
  memset(vba->dims,0,NUM_ALL_FIELDS*sizeof(vba->dims[0]));
}



int
get_num_of_warps(const dim3 bpg, const dim3 tpb)
{
	const size_t warp_size = get_device_prop().warpSize;
	const int num_of_warps_per_block = (tpb.x*tpb.y*tpb.z + warp_size-1)/warp_size;
	const int num_of_blocks = bpg.x*bpg.y*bpg.z;
	return num_of_warps_per_block*num_of_blocks;
}

int
get_current_device()
{
	int device{};
	ERRCHK_CUDA_ALWAYS(acGetDevice(&device));
	return device;
}

bool
supports_cooperative_launches()
{
	static bool called{};
	static int supportsCoopLaunch{};
	if(called)
	{
		ERRCHK_ALWAYS(supportsCoopLaunch);
		return bool(supportsCoopLaunch);
	}
	hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, get_current_device());
	called = true;
	return bool(supportsCoopLaunch);
}
void
launch_kernel(const AcKernel kernel, const int3 start, const int3 end, VertexBufferArray vba, const dim3 bpg, const dim3 tpb, const size_t smem, const hipStream_t stream)
{
  if(is_coop_raytracing_kernel(kernel) && supports_cooperative_launches())
  {
	void* args[] = {(void*)&start,(void*)&end,(void*)&vba.on_device};
	hipLaunchCooperativeKernel((void*)kernels[kernel],bpg,tpb,args,smem,stream);
  }
  else
  {
  	KERNEL_VBA_LAUNCH(kernels[kernel],bpg,tpb,smem,stream)(start,end,vba.on_device);
  }
}
void
launch_kernel(const AcKernel kernel, const int3 start, const int3 end, VertexBufferArray vba, const dim3 bpg, const dim3 tpb, const size_t smem)
{
	launch_kernel(kernel,start,end,vba,bpg,tpb,smem,0);
}


const Volume 
get_kernel_end(const AcKernel kernel, const Volume start, const Volume end)
{
	if(is_raytracing_kernel(kernel))
	{
		const auto step_direction = raytracing_step_direction(kernel);
		if(step_direction.z) return (Volume){end.x,end.y,start.z+1};
		if(step_direction.y) return (Volume){end.x,start.y+1,end.z};
		if(step_direction.x) return (Volume){start.x+1,end.y,end.z};
	}
	return (Volume){end.x,end.y,end.z};

}
AcResult
acLaunchKernel(AcKernel kernel, const hipStream_t stream, const Volume start_volume,
               const Volume end_volume, VertexBufferArray vba)
{
  const int3 start = to_int3(start_volume);
  const int3 end   = to_int3(get_kernel_end(kernel,start_volume,end_volume));

  const TBConfig tbconf = getOptimalTBConfig(kernel, start, end, vba);
  const dim3 tpb        = tbconf.tpb;
  const int3 dims       = tbconf.dims;
  const dim3 bpg        = to_dim3(get_bpg(to_volume(dims),kernel,vba.on_device.block_factor, to_volume(tpb)));
  const size_t smem = get_smem(kernel,to_volume(tpb), STENCIL_ORDER, sizeof(AcReal));
  if (kernel_calls_reduce[kernel] && reduce_offsets[kernel].find(start) == reduce_offsets[kernel].end())
  {
  	reduce_offsets[kernel][start] = kernel_running_reduce_offsets[kernel];
  	kernel_running_reduce_offsets[kernel] += get_num_of_warps(bpg,tpb);
	resize_scratchpads_to_fit(kernel_running_reduce_offsets[kernel],vba,kernel);
  }

  if(kernel_calls_reduce[kernel]) vba.on_device.reduce_offset = reduce_offsets[kernel][start];
  // hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferL1);
  launch_kernel(kernel,start,end,vba,bpg,tpb,smem,stream);
  ERRCHK_CUDA_KERNEL();

  last_tpb = tpb; // Note: a bit hacky way to get the tpb
  return AC_SUCCESS;
}

AcResult
acBenchmarkKernel(AcKernel kernel, const int3 start, const int3 end,
                  VertexBufferArray vba)
{
  const TBConfig tbconf = getOptimalTBConfig(kernel, start, end, vba);
  const dim3 tpb        = tbconf.tpb;
  const int3 dims       = tbconf.dims;
  const dim3 bpg        = to_dim3(get_bpg(to_volume(dims), to_volume(tpb)));
  const size_t smem = get_smem(kernel,to_volume(tpb), STENCIL_ORDER, sizeof(AcReal));

  // Timer create
  hipEvent_t tstart, tstop;
  ERRCHK_CUDA(hipEventCreate(&tstart));
  ERRCHK_CUDA(hipEventCreate(&tstop));

  // Warmup
  ERRCHK_CUDA(hipEventRecord(tstart));
  KERNEL_LAUNCH(kernels[kernel],bpg, tpb, smem)(start, end, vba.on_device);
  ERRCHK_CUDA(hipEventRecord(tstop));
  ERRCHK_CUDA(hipEventSynchronize(tstop));
  ERRCHK_CUDA_KERNEL();
  ERRCHK_CUDA_ALWAYS(acDeviceSynchronize());

  // Benchmark
  ERRCHK_CUDA(hipEventRecord(tstart)); // Timing start
  KERNEL_LAUNCH(kernels[kernel],bpg,tpb,smem)(start, end, vba.on_device);
  ERRCHK_CUDA(hipEventRecord(tstop)); // Timing stop
  ERRCHK_CUDA(hipEventSynchronize(tstop));
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, tstart, tstop);

  ERRCHK_ALWAYS(kernel < NUM_KERNELS);
  printf("Kernel %s time elapsed: %g ms\n", kernel_names[kernel],
         static_cast<double>(milliseconds));

  // Timer destroy
  ERRCHK_CUDA(hipEventDestroy(tstart));
  ERRCHK_CUDA(hipEventDestroy(tstop));

  last_tpb = tpb; // Note: a bit hacky way to get the tpb
  return AC_SUCCESS;
}


AcResult
acLoadStencil(const Stencil stencil, const hipStream_t /* stream */,
              const AcReal data[STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
  ERRCHK_ALWAYS(stencil < NUM_STENCILS);

  // Note important acDeviceSynchronize below
  //
  // Constant memory allocated for stencils is shared among kernel
  // invocations, therefore a race condition is possible when updating
  // the coefficients. To avoid this, all kernels that can access
  // the coefficients must be completed before starting async copy to
  // constant memory
  ERRCHK_CUDA_ALWAYS(acDeviceSynchronize());

  const size_t bytes = sizeof(data[0][0][0]) * STENCIL_DEPTH * STENCIL_HEIGHT *
                       STENCIL_WIDTH;
  const hipError_t retval = hipMemcpyToSymbol(HIP_SYMBOL(
      stencils), data, bytes, stencil * bytes, hipMemcpyHostToDevice);

  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
};

AcResult
acLoadStencils(const hipStream_t stream,
               const AcReal data[NUM_STENCILS][STENCIL_DEPTH][STENCIL_HEIGHT]
                                [STENCIL_WIDTH])
{
  int retval = 0;
  for (size_t i = 0; i < NUM_STENCILS; ++i)
    retval |= acLoadStencil((Stencil)i, stream, data[i]);
  return (AcResult)retval;
}

AcResult
acStoreStencil(const Stencil stencil, const hipStream_t /* stream */,
               AcReal data[STENCIL_DEPTH][STENCIL_HEIGHT][STENCIL_WIDTH])
{
  ERRCHK_ALWAYS(stencil < NUM_STENCILS);

  // Ensure all acLoadUniform calls have completed before continuing
  ERRCHK_CUDA_ALWAYS(acDeviceSynchronize());

  const size_t bytes = sizeof(data[0][0][0]) * STENCIL_DEPTH * STENCIL_HEIGHT *
                       STENCIL_WIDTH;
  const hipError_t retval = hipMemcpyFromSymbol(
      data, HIP_SYMBOL(stencils), bytes, stencil * bytes, hipMemcpyDeviceToHost);

  return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
};

AcResult
acLoadRealReduceRes(hipStream_t stream, const AcRealOutputParam param, const AcReal* value)
{
  	const size_t offset =   (size_t)(&d_reduce_real_res_symbol[param]) - (size_t)&d_reduce_real_res_symbol;
	ERRCHK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_reduce_real_res_symbol), value, sizeof(value), offset, hipMemcpyHostToDevice, stream));
	return AC_SUCCESS;
}

AcResult
acLoadIntReduceRes(hipStream_t stream, const AcIntOutputParam param, const int* value)
{
  	const size_t offset =   (size_t)(&d_reduce_int_res_symbol[param]) - (size_t)&d_reduce_int_res_symbol;
	ERRCHK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_reduce_int_res_symbol), value, sizeof(value), offset, hipMemcpyHostToDevice, stream));
	return AC_SUCCESS;
}

#if AC_DOUBLE_PRECISION
AcResult
acLoadFloatReduceRes(hipStream_t stream, const AcFloatOutputParam param, const float* value)
{
  	const size_t offset =   (size_t)&d_reduce_float_res_symbol[param]- (size_t)&d_reduce_float_res_symbol;
	ERRCHK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_reduce_float_res_symbol), value, sizeof(value), offset, hipMemcpyHostToDevice, stream));
	return AC_SUCCESS;
}
#endif

template <typename P, typename V>
static AcResult
acLoadUniform(const P param, const V value)
{
	if constexpr (std::is_same<P,AcReal>::value)
	{
  		if (isnan(value)) {
  		  fprintf(stderr,
  		          "WARNING: Passed an invalid value %g to device constant %s. "
  		          "Skipping.\n",
  		          (double)value, realparam_names[param]);
  		  return AC_FAILURE;
  		}
	}
	else if constexpr (std::is_same<P,AcReal3>::value)
	{
  		if (isnan(value.x) || isnan(value.y) || isnan(value.z)) {
  		  fprintf(stderr,
  		          "WARNING: Passed an invalid value (%g, %g, %g) to device constant "
  		          "%s. Skipping.\n",
  		          (double)value.x, (double)value.y, (double)value.z,
  		          real3param_names[param]);
  		  return AC_FAILURE;
  		}
	}
  	ERRCHK_ALWAYS(param < get_num_params<P>());
  	ERRCHK_CUDA_ALWAYS(acDeviceSynchronize()); /* See note in acLoadStencil */

  	const size_t offset =  get_address(param) - (size_t)&d_mesh_info;
  	const hipError_t retval = hipMemcpyToSymbol(HIP_SYMBOL(d_mesh_info), &value, sizeof(value), offset, hipMemcpyHostToDevice);
  	return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
}



template <typename P, typename V>
static AcResult
acLoadArrayUniform(const P array, const V* values, const size_t length)
{
#if AC_VERBOSE
	fprintf(stderr,"Loading %s\n",get_name(array));
	fflush(stderr);
#endif
	ERRCHK_CUDA_ALWAYS(acDeviceSynchronize());
	ERRCHK_ALWAYS(values  != nullptr);
	const size_t bytes = length*sizeof(values[0]);
	if (!is_dconst(array))
	{
		if (!is_alive(array)) return AC_NOT_ALLOCATED;
		auto dst_ptr = get_empty_pointer(array);
		memcpy_from_gmem_array(array,dst_ptr);
		ERRCHK_ALWAYS(dst_ptr != nullptr);
		if (dst_ptr == nullptr)
		{
			fprintf(stderr,"FATAL AC ERROR from acLoadArrayUniform\n");
			exit(EXIT_FAILURE);
		}
#if AC_VERBOSE
		fprintf(stderr,"Calling (cuda/hip)memcpy %s|%ld\n",get_name(array),length);
		fflush(stderr);
#endif
		ERRCHK_CUDA_ALWAYS(hipMemcpy(dst_ptr,values,bytes,hipMemcpyHostToDevice));
	}
	else 
		ERRCHK_CUDA_ALWAYS(load_array(values, bytes, array));
#if AC_VERBOSE
	fprintf(stderr,"Loaded %s\n",get_name(array));
	fflush(stderr);
#endif
	return AC_SUCCESS;
}

template <typename P, typename V>
AcResult
acStoreUniform(const P param, V* value)
{
	ERRCHK_ALWAYS(param < get_num_params<P>());
	ERRCHK_CUDA_ALWAYS(acDeviceSynchronize());
  	const size_t offset =  get_address(param) - (size_t)&d_mesh_info;
	const hipError_t retval = hipMemcpyFromSymbol(value, HIP_SYMBOL(d_mesh_info), sizeof(V), offset, hipMemcpyDeviceToHost);
	return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;
}

template <typename P, typename V>
AcResult
acStoreArrayUniform(const P array, V* values, const size_t length)
{
	ERRCHK_ALWAYS(values  != nullptr);
	const size_t bytes = length*sizeof(values[0]);
	if (!is_dconst(array))
	{
		if (!is_alive(array)) return AC_NOT_ALLOCATED;
		auto src_ptr = get_empty_pointer(array);
		memcpy_from_gmem_array(array,src_ptr);
		ERRCHK_ALWAYS(src_ptr != nullptr);
		ERRCHK_CUDA_ALWAYS(hipMemcpy(values, src_ptr, bytes, hipMemcpyDeviceToHost));
	}
	else
		ERRCHK_CUDA_ALWAYS(store_array(values, bytes, array));
	return AC_SUCCESS;
}

#include "load_and_store_uniform_funcs.h"


//TP: best would be to use carriage return to have a single line that simple keeps growing but that seems not to be always supported in SLURM environments. 
// Or at least requires actions from the user
void printProgressBar(FILE* stream, const int progress) {
    int barWidth = 50;
    fprintf(stream,"[");  // Start a new line
    int pos = barWidth * progress / 100;

    for (int i = 0; i < barWidth; ++i) {
        if (i < pos) {
            fprintf(stream,COLORIZE("=",GREEN));  
        } else if (i == pos) {
            fprintf(stream,COLORIZE(">",YELLOW));  
        } else {
            fprintf(stream," ");
        }
    }
    if(progress == 0)
    	fprintf(stream,"] %d%%  ", progress);
    else if(progress != 100)
    	fprintf(stream,"] %d%% ", progress);
    else
    	fprintf(stream,"] %d%%", progress);
}
void
printAutotuningStatus(const AcKernel kernel, const float best_time, const int progress)
{
   if(grid_pid != 0) return;
   fprintf(stderr,"\nAutotuning %s ",kernel_names[kernel]);
   printProgressBar(stderr,progress);
   if(best_time != INFINITY) fprintf(stderr," %14e",(double)best_time);
   if (progress == 100) fprintf(stderr,"\n");
   fflush(stderr);
}

void
logAutotuningStatus(const size_t counter, const size_t num_samples, const AcKernel kernel, const float best_time)
{
    const AcReal percent_of_num_samples = AcReal(num_samples)/100.0;
    for (size_t progress = 0; progress <= 90; ++progress)
    {
	      if (counter == floor(percent_of_num_samples*progress)  && (progress % 10 == 0))
	      {
		        printAutotuningStatus(kernel,best_time,progress);
	      }
    }
}

static AcAutotuneMeasurement
gather_best_measurement(const AcAutotuneMeasurement local_best)
{
	return gather_func(local_best);
}

void
make_vtxbuf_input_params_safe(VertexBufferArray& vba, const AcKernel kernel)
{
  //TP: have to set reduce offset zero since it might not be
  vba.on_device.reduce_offset = 0;
//#include "safe_vtxbuf_input_params.h"
}
int3
get_kernel_dims(const AcKernel kernel, const int3 start, const int3 end)
{
  return is_coop_raytracing_kernel(kernel) ? ceil_div(end-start,raytracing_subblock) : end-start;
}

static TBConfig
autotune(const AcKernel kernel, const int3 start, const int3 end, VertexBufferArray vba)
{
  const int3 dims = get_kernel_dims(kernel,start,end);
  make_vtxbuf_input_params_safe(vba,kernel);
  // printf("Autotuning kernel '%s' (%p), block (%d, %d, %d), implementation "
  //        "(%d):\n",
  //        kernel_names[id], kernel, dims.x, dims.y, dims.z, IMPLEMENTATION);
  // fflush(stdout);

#if 0
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  size_t size = min(int(prop.l2CacheSize * 0.75), prop.persistingL2CacheMaxSize);
  hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, size);
  // set-aside 3/4 of L2 cache for persisting accesses or the max allowed
#endif

  TBConfig c = {
      .kernel = kernel,
      .dims   = dims,
      .tpb    = (dim3){0, 0, 0},
  };


  //TP: since autotuning should be quite fast when the dim is not NGHOST only log for actually 3d portions
  const bool builtin_kernel = strlen(kernel_names[kernel]) > 2 && kernel_names[kernel][0] == 'A' && kernel_names[kernel][1] == 'C';
  const bool large_launch   = is_large_launch(dims);
  const bool log = is_raytracing_kernel(kernel) || (!builtin_kernel && large_launch);

  AcAutotuneMeasurement best_measurement = {INFINITY,(dim3){0,0,0}};
  const int num_iters = 2;

  // Get device hardware information
  const auto props = get_device_prop();
  const int max_threads_per_block = MAX_THREADS_PER_BLOCK
                                        ? min(props.maxThreadsPerBlock,
                                              MAX_THREADS_PER_BLOCK)
                                        : props.maxThreadsPerBlock;
  const size_t max_smem           = props.sharedMemPerBlock;

  // Old heuristic
  // for (int z = 1; z <= max_threads_per_block; ++z) {
  //   for (int y = 1; y <= max_threads_per_block; ++y) {
  //     for (int x = max(y, z); x <= max_threads_per_block; ++x) {

  // New: require that tpb.x is a multiple of the minimum transaction or L2
  // cache line size
  const int minimum_transaction_size_in_elems = 32 / sizeof(AcReal);
  // New: restrict tpb.x to be at most dims.x since launching threads that are known to be oob feels simply wasteful

  int3 tpb_end = dims; 
  if(is_raytracing_kernel(kernel))
  {
	const auto dir = raytracing_step_direction(kernel);
	if(dir.z) tpb_end.z = 1;
	if(dir.y) tpb_end.y = 1;
	if(dir.x) tpb_end.x = 1;
  }
  //TP: emprically and thinking about it y and z usually cannot be too big (since x is usually quite large) so we can limit then when performing sparse autotuning
  if(sparse_autotuning)
  {
	  if(!raytracing_step_direction(kernel).x)
	  {
	  	tpb_end.y = min(tpb_end.y,32);
	  }
	  tpb_end.z = min(tpb_end.z,32);
  }
  const int x_increment = min(
		  			minimum_transaction_size_in_elems,
					tpb_end.x
		            );


  std::vector<int3> samples{};
  for (int z = 1; z <= min(max_threads_per_block,tpb_end.z); ++z) {
    for (int y = 1; y <= min(max_threads_per_block,tpb_end.y); ++y) {
      for (int x = x_increment;
           x <= min(max_threads_per_block,tpb_end.x); x += x_increment) {


        if (x * y * z > max_threads_per_block)
          break;
        const dim3 tpb(x, y, z);
        const size_t smem = get_smem(kernel,to_volume(tpb), STENCIL_ORDER,
                                     sizeof(AcReal));

        if (smem > max_smem)
          continue;

        //if ((x * y * z) % props.warpSize && (x*y*z) >props.warpSize)
        //  continue;

        if (!is_valid_configuration(to_volume(dims), to_volume(tpb),kernel))
          continue;
	//TP: should be emplace back but on my laptop the CUDA compiler gives a cryptic error message that I do not care to debug
        samples.push_back((int3){x,y,z});
      }
    }
  }
  if(samples.size() == 0)
  {
	fprintf(stderr,"Found no suitable thread blocks for Kernel %s!\n",kernel_names[kernel]);
	fflush(stderr);
  	ERRCHK_ALWAYS(samples.size() > 0);
  }
  size_t counter  = 0;
  size_t start_samples{};
  size_t end_samples{};

  const bool on_halos =
	  (start.x < (int)vba.computational_dims.n0.x) ||
	  (start.y < (int)vba.computational_dims.n0.y) ||
	  (start.z < (int)vba.computational_dims.n0.z) ||
                          
	  (end.x >=  (int)vba.computational_dims.n1.x) ||
	  (end.y >=  (int)vba.computational_dims.n1.y) ||
	  (end.z >=  (int)vba.computational_dims.n1.z);

  const bool parallel_autotuning = !on_halos && AC_MPI_ENABLED;
  if(parallel_autotuning)
  {
  	const size_t portion = ceil_div(samples.size(),nprocs);
  	start_samples = portion*grid_pid;
  	end_samples   = min(samples.size(), portion*(grid_pid+1));
  }
  else
  {
  	start_samples = 0;
  	end_samples   = samples.size();
  }
  const size_t n_samples = end_samples-start_samples;

  //TP: logs the percent 0% which is useful to know the autotuning has started
  if (log) logAutotuningStatus(counter,n_samples,kernel,best_measurement.time / num_iters);
  for(size_t sample  = start_samples; sample < end_samples; ++sample)
  {
        auto x = samples[sample].x;
        auto y = samples[sample].y;
        auto z = samples[sample].z;
        const dim3 tpb(x, y, z);
        const dim3 bpg    = to_dim3(
                                get_bpg(to_volume(dims),kernel,vba.on_device.block_factor,
                                to_volume(tpb)
                                ));
	const int n_warps = get_num_of_warps(bpg,tpb);
	if(kernel_calls_reduce[kernel])
		resize_scratchpads_to_fit(n_warps,vba,kernel);
        const size_t smem = get_smem(kernel,to_volume(tpb), STENCIL_ORDER,
                                     sizeof(AcReal));

        hipEvent_t tstart, tstop;
        ERRCHK_CUDA(hipEventCreate(&tstart));
        ERRCHK_CUDA(hipEventCreate(&tstop));

        launch_kernel(kernel,start,end,vba,bpg,tpb,smem);
        ERRCHK_CUDA_ALWAYS(acDeviceSynchronize());
        ERRCHK_CUDA(hipEventRecord(tstart)); // Timing start
        for (int i = 0; i < num_iters; ++i)
	{
        	launch_kernel(kernel,start,end,vba,bpg,tpb,smem);
	}
        ERRCHK_CUDA(hipEventRecord(tstop)); // Timing stop
        ERRCHK_CUDA(hipEventSynchronize(tstop));

        float milliseconds = 0;
        ERRCHK_CUDA(hipEventElapsedTime(&milliseconds, tstart, tstop));

        ERRCHK_CUDA(hipEventDestroy(tstart));
        ERRCHK_CUDA(hipEventDestroy(tstop));
        ++counter;
        if (log) logAutotuningStatus(counter,n_samples,kernel,best_measurement.time / num_iters);

        // Discard failed runs (attempt to clear the error to hipSuccess)
        const auto err = hipGetLastError();
        //TP: it is fine to simply skip invalid configuration values since it can be because of too large tpb's
        //We simply do not count them for finding the optim config
        if(err == hipErrorInvalidConfiguration) continue;
        if(err == hipErrorLaunchOutOfResources) continue;
        if (err != hipSuccess) {
          //TP: reset autotune results
          fprintf(stderr,"\nFailed while autotuning: %s\nReason: %s\n",kernel_names[kernel],hipGetErrorName(err));
          FILE* fp = fopen(autotune_csv_path,"w");
          fclose(fp);
          ERRCHK_ALWAYS(err == hipSuccess);
        }

        if (milliseconds < best_measurement.time) {
          best_measurement.time = milliseconds;
          best_measurement.tpb = tpb;
        }

        // printf("Auto-optimizing... Current tpb: (%d, %d, %d), time %f ms\n",
        //        tpb.x, tpb.y, tpb.z, (double)milliseconds / num_iters);
        // fflush(stdout);
  }
  best_measurement =  parallel_autotuning ? gather_best_measurement(best_measurement) : best_measurement;
  if(log) printAutotuningStatus(kernel,best_measurement.time/num_iters,100);
  c.tpb = best_measurement.tpb;
  if(grid_pid == 0)
  {
        FILE* fp = fopen(autotune_csv_path, "a");
        ERRCHK_ALWAYS(fp);
#if IMPLEMENTATION == SMEM_HIGH_OCCUPANCY_CT_CONST_TB
        fprintf(fp, "%d, (%d, %d, %d), (%d, %d, %d), %g\n", IMPLEMENTATION, nx, ny,
                nz, best_measurement.tpb.x, best_measurement.tpb.y, best_measurement.tpb.z,
                (double)best_measurement.time / num_iters);
#else
        fprintf(fp, "%d, %d, %d, %d, %d, %d, %d, %d, %g, %s, %d, %d, %d, %d, %d, %d, %d\n", IMPLEMENTATION, kernel, dims.x,
                dims.y, dims.z, best_measurement.tpb.x, best_measurement.tpb.y, best_measurement.tpb.z,
                (double)best_measurement.time / num_iters, kernel_names[kernel],
		vba.on_device.block_factor.x,vba.on_device.block_factor.y,vba.on_device.block_factor.z
		,raytracing_subblock.x
		,raytracing_subblock.y
		,raytracing_subblock.z
		,sparse_autotuning
		);
#endif
        fclose(fp);
	fflush(fp);
  }
  if (c.tpb.x * c.tpb.y * c.tpb.z <= 0) {
    fprintf(stderr,
            "Fatal error: failed to find valid thread block dimensions for (%d,%d,%d) launch of %s.\n"
            ,dims.x,dims.y,dims.z,kernel_names[kernel]);
  }
  ERRCHK_ALWAYS(c.tpb.x * c.tpb.y * c.tpb.z > 0);
  //TP: done to ensure scratchpads are reset after autotuning
  if(vba.scratchpad_states) memset(vba.scratchpad_states,0,sizeof(AcScratchpadStates));
  return c;
}

static bool
file_exists(const char* filename)
{
  struct stat   buffer;
  return (stat (filename, &buffer) == 0);
}

int3
acReadOptimTBConfig(const AcKernel kernel, const int3 dims, const int3 block_factors)
{
  if(!file_exists(autotune_csv_path)) return {-1,-1,-1};
  const char* filename = autotune_csv_path;
  FILE *file = fopen ( filename, "r" );
  int3 res = {-1,-1,-1};
  double best_time     = (double)INFINITY;
  string_vec entries[1000];
  memset(entries,0,sizeof(string_vec)*1000);
  const int n_entries = get_csv_entries(entries,file);
  for(int i = 0; i < n_entries; ++i)
  {
	  string_vec entry = entries[i];
	  if(entry.size == 17)
      	  {
      	     int kernel_index  = atoi(entry.data[1]);
      	     int3 read_dims = {atoi(entry.data[2]), atoi(entry.data[3]), atoi(entry.data[4])};
      	     int3 tpb = {atoi(entry.data[5]), atoi(entry.data[6]), atoi(entry.data[7])};
      	     double time = atof(entry.data[8]);
      	     int3 read_block_factors = {atoi(entry.data[10]), atoi(entry.data[11]), atoi(entry.data[12])};
      	     int3 read_raytracing_factors = {atoi(entry.data[13]), atoi(entry.data[14]), atoi(entry.data[15])};
	     int  was_sparse = atoi(entry.data[16]);
      	     if(time < best_time && kernel_index == kernel && read_dims == dims && read_block_factors == block_factors && read_raytracing_factors == raytracing_subblock && was_sparse == sparse_autotuning)
      	     {
      	    	 best_time = time;
      	    	 res       = tpb;
      	     }
      	  }
      	  for(size_t elem = 0; elem < entry.size; ++elem)
      	         free((char*)entry.data[elem]);
      	  free_str_vec(&entry);
  }
  fclose(file);
  return res;
}


static TBConfig
getOptimalTBConfig(const AcKernel kernel, const int3 start, const int3 end, VertexBufferArray vba)
{
  const int3 dims = get_kernel_dims(kernel,start,end);
  for (auto c : tbconfigs)
    if (c.kernel == kernel && c.dims == dims)
      return c;

  const int3 read_tpb = acReadOptimTBConfig(kernel,dims,vba.on_device.block_factor);
  TBConfig c  = (read_tpb != (int3){-1,-1,-1})
          ? (TBConfig){kernel,dims,(dim3){(uint32_t)read_tpb.x, (uint32_t)read_tpb.y, (uint32_t)read_tpb.z}}
          : autotune(kernel,start,end,vba);
  tbconfigs.push_back(c);
  return c;
}

AcKernel
acGetOptimizedKernel(const AcKernel kernel_enum, const VertexBufferArray vba)
{
	#include "user_kernels_ifs.h"
	//silence unused warnings
	(void)vba;
	return kernel_enum;
	//return kernels[(int) kernel_enum];
}
void
acVBASwapBuffer(const Field field, VertexBufferArray* vba)
{
  AcReal* tmp     = vba->on_device.in[field];
  vba->on_device.in[field]  = vba->on_device.out[field];
  vba->on_device.out[field] = tmp;
}

void
acVBASwapBuffers(VertexBufferArray* vba)
{
  for (size_t i = 0; i < NUM_FIELDS; ++i)
    acVBASwapBuffer((Field)i, vba);
}

void
acPBASwapBuffer(const Profile profile, VertexBufferArray* vba)
{
  AcReal* tmp                = vba->on_device.profiles.in[profile];
  vba->on_device.profiles.in[profile]  = vba->on_device.profiles.out[profile];
  vba->on_device.profiles.out[profile] = tmp;
}

void
acPBASwapBuffers(VertexBufferArray* vba)
{
  for (int i = 0; i < NUM_PROFILES; ++i)
    acPBASwapBuffer((Profile)i, vba);
}

template <typename P>
struct load_all_scalars_uniform
{
	void operator()(const AcMeshInfo& config)
	{
		for(P i : get_params<P>())
			acLoadUniform(0,  i, config[i]);
	}
};

template <typename P>
struct load_all_arrays_uniform
{
	void operator()(const AcMeshInfo& config)
	{
		for(const P array : get_params<P>())
		{
			auto config_array = config[array];
      			if (config_array != nullptr)
				acLoadArrayUniform(array,config_array, get_array_length(array,config));
		}
	}
};

AcResult
acLoadMeshInfo(const AcMeshInfo info, const hipStream_t)
{
  /* See note in acLoadStencil */
  ERRCHK_CUDA(acDeviceSynchronize());
  AcResult retval = AC_SUCCESS;
  AcScalarTypes::run<load_all_scalars_uniform>(info);
  AcArrayTypes::run<load_all_arrays_uniform>(info);
  return retval;
}

//---------------
// static __host__ __device__ constexpr size_t
// acShapeSize(const AcShape& shape)
size_t
acShapeSize(const AcShape shape)
{
  return shape.x * shape.y * shape.z * shape.w;
}

__host__ __device__ constexpr bool
acOutOfBounds(const AcIndex& index, const AcShape& shape)
{
  return (index.x >= shape.x) || //
         (index.y >= shape.y) || //
         (index.z >= shape.z) || //
         (index.w >= shape.w);
}

static __host__ __device__ constexpr AcIndex __attribute__((unused))
min(const AcIndex& a, const AcIndex& b)
{
  return (AcIndex){
      a.x < b.x ? a.x : b.x,
      a.y < b.y ? a.y : b.y,
      a.z < b.z ? a.z : b.z,
      a.w < b.w ? a.w : b.w,
  };
}

__host__ __device__ constexpr AcIndex
operator+(const AcIndex& a, const AcIndex& b)
{
  return (AcIndex){
      a.x + b.x,
      a.y + b.y,
      a.z + b.z,
      a.w + b.w,
  };
}

static __host__ __device__ constexpr AcIndex __attribute__((unused))
operator-(const AcIndex& a, const AcIndex& b) 
{
  return (AcIndex){
      a.x - b.x,
      a.y - b.y,
      a.z - b.z,
      a.w - b.w,
  };
}

__host__ __device__ constexpr AcIndex
to_spatial(const size_t i, const AcShape& shape)
{
  return (AcIndex){
      .x = i % shape.x,
      .y = (i / shape.x) % shape.y,
      .z = (i / (shape.x * shape.y)) % shape.z,
      .w = i / (shape.x * shape.y * shape.z),
  };
}

__host__ __device__ constexpr size_t
to_linear(const AcIndex& index, const AcShape& shape)
{
  return index.x +           //
         index.y * shape.x + //
         index.z * shape.x * shape.y + index.w * shape.x * shape.y * shape.z;
}

static __global__ void
reindex(const AcReal* in, const AcIndex in_offset, const AcShape in_shape,
        AcReal* out, const AcIndex out_offset, const AcShape out_shape,
        const AcShape block_shape)
{
  const size_t i    = (size_t)threadIdx.x + blockIdx.x * blockDim.x;
  const AcIndex idx = to_spatial(i, block_shape);

  const AcIndex in_pos  = idx + in_offset;
  const AcIndex out_pos = idx + out_offset;

  if (acOutOfBounds(idx, block_shape) || //
      acOutOfBounds(in_pos, in_shape) || //
      acOutOfBounds(out_pos, out_shape))
    return;

  const size_t in_idx  = to_linear(in_pos, in_shape);
  const size_t out_idx = to_linear(out_pos, out_shape);

  out[out_idx] = in[in_idx];
}

AcResult
acReindex(const hipStream_t stream, //
          const AcReal* in, const AcIndex in_offset, const AcShape in_shape,
          AcReal* out, const AcIndex out_offset, const AcShape out_shape,
          const AcShape block_shape)
{
  const size_t count = acShapeSize(block_shape);
  const size_t tpb   = min(256ul, count);
  const size_t bpg   = (count + tpb - 1) / tpb;

  KERNEL_LAUNCH(reindex,bpg, tpb, 0, stream)(in, in_offset, in_shape, //
                                   out, out_offset, out_shape, block_shape);
  ERRCHK_CUDA_KERNEL();

  return AC_SUCCESS;
}

typedef struct {
  AcReal *x, *y, *z;
} SOAVector;

typedef struct {
  // Input vectors
  SOAVector A[1];
  size_t A_count;
  SOAVector B[4];
  size_t B_count;
  // Note: more efficient with A_count < B_count

  // Output vectors
  SOAVector C[1 * 4];
  // C count = A_count*B_count
} CrossProductArrays;

static __global__ void UNUSED
reindex_cross(const CrossProductArrays arrays, const AcIndex in_offset,
              const AcShape in_shape, const AcIndex out_offset,
              const AcShape out_shape, const AcShape block_shape)
{
  const AcIndex idx = to_spatial(
      static_cast<size_t>(threadIdx.x) + blockIdx.x * blockDim.x, block_shape);

  const AcIndex in_pos  = idx + in_offset;
  const AcIndex out_pos = idx + out_offset;

  if (acOutOfBounds(idx, block_shape) || //
      acOutOfBounds(in_pos, in_shape) || //
      acOutOfBounds(out_pos, out_shape))
    return;

  const size_t in_idx  = to_linear(in_pos, in_shape);
  const size_t out_idx = to_linear(out_pos, out_shape);

  for (size_t j = 0; j < arrays.A_count; ++j) {
    const AcReal3 a = {
        arrays.A[j].x[in_idx],
        arrays.A[j].y[in_idx],
        arrays.A[j].z[in_idx],
    };
    for (size_t i = 0; i < arrays.B_count; ++i) {
      const AcReal3 b = {
          arrays.B[i].x[in_idx],
          arrays.B[i].y[in_idx],
          arrays.B[i].z[in_idx],
      };
      const AcReal3 res                           = AC_cross(a, b);
      arrays.C[i + j * arrays.B_count].x[out_idx] = res.x;
      arrays.C[i + j * arrays.B_count].y[out_idx] = res.y;
      arrays.C[i + j * arrays.B_count].z[out_idx] = res.z;
    }
  }
}

#if 0
__global__ void
map_cross_product(const CrossProductInputs inputs, const AcIndex start,
                  const AcIndex end)
{

  const AcIndex tid = {
      .x = threadIdx.x + blockIdx.x * blockDim.x,
      .y = threadIdx.y + blockIdx.y * blockDim.y,
      .z = threadIdx.z + blockIdx.z * blockDim.z,
      .w = 0,
  };

  const AcIndex in_idx3d = start + tid;
  const size_t in_idx = DEVICE_VTXBUF_IDX(in_idx3d.x, in_idx3d.y, in_idx3d.z);

  const AcShape dims   = end - start;
  const size_t out_idx = tid.x + tid.y * dims.x + tid.z * dims.x * dims.y;

  const bool within_bounds = in_idx3d.x < end.x && in_idx3d.y < end.y &&
                             in_idx3d.z < end.z;
  if (within_bounds) {
    for (size_t i = 0; i < inputs.A_count; ++i) {
      const AcReal3 a = (AcReal3){
          inputs.A[i].x[in_idx],
          inputs.A[i].y[in_idx],
          inputs.A[i].z[in_idx],
      };
      for (size_t j = 0; j < inputs.B_count; ++j) {
        const AcReal3 b = (AcReal3){
            inputs.B[j].x[in_idx],
            inputs.B[j].y[in_idx],
            inputs.B[j].z[in_idx],
        };
        const AcReal3 res            = cross(a, b);
        inputs.outputs[j].x[out_idx] = res.x;
        inputs.outputs[j].y[out_idx] = res.y;
        inputs.outputs[j].z[out_idx] = res.z;
      }
    }
  }
}
#endif

#ifdef AC_TFM_ENABLED
AcResult
acReindexCross(const hipStream_t stream, //
               const VertexBufferArray vba, const AcIndex in_offset,
               const AcShape in_shape, //
               AcReal* out, const AcIndex out_offset, const AcShape out_shape,
               const AcShape block_shape)
{
  const SOAVector uu = {
      .x = vba.in[VTXBUF_UUX],
      .y = vba.in[VTXBUF_UUY],
      .z = vba.in[VTXBUF_UUZ],
  };
  const SOAVector bb11 = {
      .x = vba.in[TF_b11_x],
      .y = vba.in[TF_b11_y],
      .z = vba.in[TF_b11_z],
  };
  const SOAVector bb12 = {
      .x = vba.in[TF_b12_x],
      .y = vba.in[TF_b12_y],
      .z = vba.in[TF_b12_z],
  };
  const SOAVector bb21 = {
      .x = vba.in[TF_b21_x],
      .y = vba.in[TF_b21_y],
      .z = vba.in[TF_b21_z],
  };
  const SOAVector bb22 = {
      .x = vba.in[TF_b22_x],
      .y = vba.in[TF_b22_y],
      .z = vba.in[TF_b22_z],
  };

  const size_t block_offset = out_shape.x * out_shape.y * out_shape.z;
  const SOAVector out_bb11  = {
       .x = &out[3 * block_offset],
       .y = &out[4 * block_offset],
       .z = &out[5 * block_offset],
  };
  const SOAVector out_bb12 = {
      .x = &out[6 * block_offset],
      .y = &out[7 * block_offset],
      .z = &out[8 * block_offset],
  };
  const SOAVector out_bb21 = {
      .x = &out[9 * block_offset],
      .y = &out[10 * block_offset],
      .z = &out[11 * block_offset],
  };
  const SOAVector out_bb22 = {
      .x = &out[12 * block_offset],
      .y = &out[13 * block_offset],
      .z = &out[14 * block_offset],
  };

  const CrossProductArrays arrays = {
      .A       = {uu},
      .A_count = 1,
      .B       = {bb11, bb12, bb21, bb22},
      .B_count = 4,
      .C       = {out_bb11, out_bb12, out_bb21, out_bb22},
  };

  const size_t count = acShapeSize(block_shape);
  const size_t tpb   = min(256ul, count);
  const size_t bpg   = (count + tpb - 1) / tpb;

  KERNEL_LAUNCH(reindex_cross,bpg, tpb, 0, stream)(arrays, in_offset, in_shape,
                                         out_offset, out_shape, block_shape);
  return AC_SUCCESS;
}
#else
AcResult
acReindexCross(const hipStream_t , //
               const VertexBufferArray , const AcIndex ,
               const AcShape , //
               AcReal* , const AcIndex , const AcShape ,
               const AcShape )
{
  ERROR("acReindexCross called but AC_TFM_ENABLED was false");
  return AC_FAILURE;
}
#endif

typedef struct
{
	size_t x;
	size_t y;
} size_t2;

struct size_t2Hash {
    std::size_t operator()(const size_t2& v) const {
        return std::hash<size_t>()(v.x) ^ std::hash<size_t>()(v.y) << 1;
    }
};

std::unordered_map<size_t2,size_t*,size_t2Hash> segmented_reduce_offsets{};

static HOST_DEVICE_INLINE bool
operator==(const size_t2& a, const size_t2& b)
{
  return a.x == b.x && a.y == b.y;
}

//TP: will return a cached allocation if one is found
size_t*
get_offsets(const size_t count, const size_t num_segments)
{
  const size_t2 key = {count,num_segments};
  if(segmented_reduce_offsets.find(key) != segmented_reduce_offsets.end())
	  return segmented_reduce_offsets[key];

  size_t* offsets = (size_t*)malloc(sizeof(offsets[0]) * (num_segments + 1));
  ERRCHK_ALWAYS(num_segments > 0);
  ERRCHK_ALWAYS(offsets);
  ERRCHK_ALWAYS(count % num_segments == 0);
  for (size_t i = 0; i <= num_segments; ++i) {
    offsets[i] = i * (count / num_segments);
    ERRCHK_ALWAYS(offsets[i] <= count);
  }
  size_t* d_offsets = NULL;
  ERRCHK_CUDA_ALWAYS(hipMalloc(&d_offsets, sizeof(d_offsets[0]) * (num_segments + 1)));
  ERRCHK_ALWAYS(d_offsets);
  ERRCHK_CUDA(hipMemcpy(d_offsets, offsets, sizeof(d_offsets[0]) * (num_segments + 1),hipMemcpyHostToDevice));
  free(offsets);
  segmented_reduce_offsets[key] = d_offsets;
  return d_offsets;
}


AcResult
acSegmentedReduce(const hipStream_t stream, const AcReal* d_in,
                  const size_t count, const size_t num_segments, AcReal* d_out, AcReal** tmp_buffer, size_t* tmp_size)
{

  size_t* d_offsets = get_offsets(count,num_segments);

  void* d_temp_storage      = NULL;
  size_t temp_storage_bytes = 0;
  ERRCHK_CUDA(hipcub::DeviceSegmentedReduce::Sum(d_temp_storage, temp_storage_bytes, d_in,
                                  d_out, num_segments, d_offsets, d_offsets + 1,
                                  stream));

  *tmp_size = device_resize((void**)tmp_buffer,*tmp_size,temp_storage_bytes);
  ERRCHK_CUDA(hipcub::DeviceSegmentedReduce::Sum((void*)(*tmp_buffer), temp_storage_bytes, d_in,
                            d_out, num_segments, d_offsets, d_offsets + 1,
                            stream));
  ERRCHK_CUDA_KERNEL();
  return AC_SUCCESS;
}

template <typename T>
AcResult
acReduceBase(const hipStream_t stream, const AcReduceOp reduce_op, T buffer, const size_t count)
{
  ERRCHK(*(buffer.buffer_size)/sizeof(*(buffer.src)[0]) >= count);
  ERRCHK(buffer.src   != NULL);
  ERRCHK(buffer.src   != NULL);

  AcDeviceTmpBuffer temp_storage{NULL,0};
  cub_reduce(temp_storage,stream,*(buffer.src),count,buffer.res,reduce_op);

  *buffer.cub_tmp_size = device_resize((void**)buffer.cub_tmp,*buffer.cub_tmp_size,temp_storage.bytes);
  temp_storage.data = (void*)(*buffer.cub_tmp);
  cub_reduce(temp_storage,stream,*(buffer.src),count,buffer.res,reduce_op);
  return AC_SUCCESS;
}

AcResult
acReduceReal(const hipStream_t stream, const AcReduceOp op, const AcRealScalarReduceBuffer buffer, const size_t count)
{
	return acReduceBase(stream,op,buffer,count);
}

#if AC_DOUBLE_PRECISION
AcResult
acReduceFloat(const hipStream_t stream, const AcReduceOp op, const AcFloatScalarReduceBuffer buffer, const size_t count)
{
	return acReduceBase(stream,op,buffer,count);
}
#endif

AcResult
acReduceInt(const hipStream_t stream, const AcReduceOp op, const AcIntScalarReduceBuffer buffer, const size_t count)
{
	return acReduceBase(stream,op,buffer,count);
}

AcResult
acComplexToReal(const AcComplex* src, const size_t count, AcReal* dst)
{
  acLaunchKernelVariadic1d(AC_COMPLEX_TO_REAL,0,0,count,(AcComplex*)src,dst);
  ERRCHK_CUDA_KERNEL();
  ERRCHK_CUDA(acDeviceSynchronize()); // NOTE: explicit sync here for safety
  return AC_SUCCESS;
}

AcResult
acRealToComplex(const AcReal* src, const size_t count, AcComplex* dst)
{
  acLaunchKernelVariadic1d(AC_REAL_TO_COMPLEX,0,0,count,(AcReal*)src,dst);
  ERRCHK_CUDA_KERNEL();
  ERRCHK_CUDA(acDeviceSynchronize()); // NOTE: explicit sync here for safety
  return AC_SUCCESS;
}


AcResult
acMultiplyInplaceComplex(const AcReal value, const size_t count, AcComplex* array)
{
  acLaunchKernelVariadic1d(AC_MULTIPLY_INPLACE_COMPLEX,0,0,count,value,array);
  ERRCHK_CUDA_KERNEL();
  ERRCHK_CUDA(acDeviceSynchronize()); // NOTE: explicit sync here for safety
  return AC_SUCCESS;
}

AcResult
acMultiplyInplace(const AcReal value, const size_t count, AcReal* array)
{
  acLaunchKernelVariadic1d(AC_MULTIPLY_INPLACE,0,0,count,value,array);
  ERRCHK_CUDA_KERNEL();
  ERRCHK_CUDA(acDeviceSynchronize()); // NOTE: explicit sync here for safety
  return AC_SUCCESS;
}
#define TILE_DIM (32)

void __global__ 
transpose_xyz_to_zyx(const AcReal* src, AcReal* dst, const Volume dims, const Volume start, const Volume end)
{
	__shared__ AcReal tile[TILE_DIM][TILE_DIM];
	const dim3 block_offset =
	{
		blockIdx.x*TILE_DIM,
		blockIdx.y,
		blockIdx.z*TILE_DIM
	};

	const dim3 vertexIdx = 
	{
		(int)start.x + threadIdx.x + block_offset.x,
		(int)start.y + threadIdx.y + block_offset.y,
		(int)start.z + threadIdx.z + block_offset.z
	};
	const dim3 out_vertexIdx = 
	{
		(int)start.z + threadIdx.x + block_offset.z,
		(int)start.y + threadIdx.y + block_offset.y,
		(int)start.x + threadIdx.z + block_offset.x
	};
	const bool in_oob  =  vertexIdx.x  >= end.x    ||  vertexIdx.y >= end.y     || vertexIdx.z >= end.z;
	const bool out_oob =  out_vertexIdx.x >= end.z ||  out_vertexIdx.y >= end.y || out_vertexIdx.z >= end.x;



	tile[threadIdx.z][threadIdx.x] = !in_oob ? src[vertexIdx.x + dims.x*(vertexIdx.y + dims.y*vertexIdx.z)] : 0.0;
	__syncthreads();
	if(!out_oob)
		dst[out_vertexIdx.x +dims.z*out_vertexIdx.y + dims.z*dims.y*out_vertexIdx.z] = tile[threadIdx.x][threadIdx.z];
}
void __global__ 
transpose_xyz_to_zxy(const AcReal* src, AcReal* dst, const Volume dims, const Volume start, const Volume end)
{
	__shared__ AcReal tile[TILE_DIM][TILE_DIM];
	const dim3 block_offset =
	{
		blockIdx.x*TILE_DIM,
		blockIdx.y,
		blockIdx.z*TILE_DIM
	};

	const dim3 vertexIdx = 
	{
		(int) start.x + threadIdx.x + block_offset.x,
		(int) start.y + threadIdx.y + block_offset.y,
		(int) start.z + threadIdx.z + block_offset.z
	};
	const dim3 out_vertexIdx = 
	{
		(int)start.z + threadIdx.x + block_offset.z,
		(int)start.y + threadIdx.y + block_offset.y,
		(int)start.x + threadIdx.z + block_offset.x
	};
	const bool in_oob  =  vertexIdx.x  >= end.x    ||  vertexIdx.y >= end.y     || vertexIdx.z >= end.z;
	const bool out_oob =  out_vertexIdx.x >= end.z ||  out_vertexIdx.y >= end.y || out_vertexIdx.z >= end.x;



	tile[threadIdx.z][threadIdx.x] = !in_oob ? src[vertexIdx.x + dims.x*(vertexIdx.y + dims.y*vertexIdx.z)] : 0.0;
	__syncthreads();
	if(!out_oob)
		dst[out_vertexIdx.x +dims.z*out_vertexIdx.z + dims.z*dims.x*out_vertexIdx.y] = tile[threadIdx.x][threadIdx.z];
}
void __global__ 
transpose_xyz_to_yxz(const AcReal* src, AcReal* dst, const Volume dims, const Volume start, const Volume end)
{
	__shared__ AcReal tile[TILE_DIM][TILE_DIM];
	const dim3 block_offset =
	{
		blockIdx.x*TILE_DIM,
		blockIdx.y*TILE_DIM,
		blockIdx.z
	};

	const dim3 vertexIdx = 
	{
		(int) start.x + threadIdx.x + block_offset.x,
		(int) start.y + threadIdx.y + block_offset.y,
		(int) start.z + threadIdx.z + block_offset.z
	};
	const dim3 out_vertexIdx = 
	{
		(int)start.y + threadIdx.x + block_offset.y,
		(int)start.x + threadIdx.y + block_offset.x,
		(int)start.z + threadIdx.z + block_offset.z
	};
	const bool in_oob  =  vertexIdx.x  >= end.x    ||  vertexIdx.y >= end.y     || vertexIdx.z >= end.z;
	const bool out_oob =  out_vertexIdx.x >= end.y ||  out_vertexIdx.y >= end.x || out_vertexIdx.z >= end.z;



	tile[threadIdx.y][threadIdx.x] = !in_oob ? src[vertexIdx.x + dims.x*(vertexIdx.y + dims.y*vertexIdx.z)] : 0.0;
	__syncthreads();
	if(!out_oob)
		dst[out_vertexIdx.x +dims.y*out_vertexIdx.y + dims.x*dims.y*out_vertexIdx.z] = tile[threadIdx.x][threadIdx.y];
}
void __global__ 
transpose_xyz_to_yzx(const AcReal* src, AcReal* dst, const Volume dims, const Volume start, const Volume end)
{
	__shared__ AcReal tile[TILE_DIM][TILE_DIM];
	const dim3 block_offset =
	{
		blockIdx.x*TILE_DIM,
		blockIdx.y*TILE_DIM,
		blockIdx.z
	};

	const dim3 vertexIdx = 
	{
		(int)start.x +threadIdx.x + block_offset.x,
		(int)start.y +threadIdx.y + block_offset.y,
		(int)start.z +threadIdx.z + block_offset.z
	};
	const dim3 out_vertexIdx = 
	{
		(int)start.y + threadIdx.x + block_offset.y,
		(int)start.x + threadIdx.y + block_offset.x,
		(int)start.z + threadIdx.z + block_offset.z
	};
	const bool in_oob  =  vertexIdx.x  >= end.x    ||  vertexIdx.y >= end.y     || vertexIdx.z >= end.z;
	const bool out_oob =  out_vertexIdx.x >= end.y ||  out_vertexIdx.y >= end.x || out_vertexIdx.z >= end.z;



	tile[threadIdx.y][threadIdx.x] = !in_oob ? src[vertexIdx.x + dims.x*(vertexIdx.y + dims.y*vertexIdx.z)] : 0.0;
	__syncthreads();
	if(!out_oob)
		dst[out_vertexIdx.x +dims.y*out_vertexIdx.z + dims.y*dims.z*out_vertexIdx.y] = tile[threadIdx.x][threadIdx.y];
}
void __global__ 
transpose_xyz_to_xzy(const AcReal* src, AcReal* dst, const Volume dims, const Volume start, const Volume end)
{
	const dim3 in_block_offset =
	{
		blockIdx.x*blockDim.x,
		blockIdx.y*blockDim.y,
		blockIdx.z*blockDim.z
	};

	const dim3 vertexIdx = 
	{
		(int)start.x + threadIdx.x + in_block_offset.x,
		(int)start.y + threadIdx.y + in_block_offset.y,
		(int)start.z + threadIdx.z + in_block_offset.z
	};

	const bool oob  =  vertexIdx.x  >= end.x    ||  vertexIdx.y >= end.y     || vertexIdx.z >= end.z;
	if(oob) return;
	dst[vertexIdx.x + dims.x*vertexIdx.z + dims.x*dims.z*vertexIdx.y] 
		= src[vertexIdx.x + dims.x*(vertexIdx.y + dims.y*vertexIdx.z)];
}
static AcResult
acTransposeXYZ_ZYX(const AcReal* src, AcReal* dst, const Volume dims, const Volume start, const Volume end, const hipStream_t stream)
{
	const dim3 tpb = {32,1,32};
	const Volume sub_dims = end-start;
	const dim3 bpg = to_dim3(get_bpg(sub_dims,to_volume(tpb)));
  	KERNEL_LAUNCH(transpose_xyz_to_zyx,bpg, tpb, 0, stream)(src,dst,dims,start,end);
	ERRCHK_CUDA_KERNEL();
	return AC_SUCCESS;
}
static AcResult
acTransposeXYZ_ZXY(const AcReal* src, AcReal* dst, const Volume dims, const Volume start, const Volume end, const hipStream_t stream)
{
	const dim3 tpb = {32,1,32};

	const Volume sub_dims = end-start;
	const dim3 bpg = to_dim3(get_bpg(sub_dims,to_volume(tpb)));
  	KERNEL_LAUNCH(transpose_xyz_to_zxy,bpg, tpb, 0, stream)(src,dst,dims,start,end);
	ERRCHK_CUDA_KERNEL();
	return AC_SUCCESS;
}
static AcResult
acTransposeXYZ_YXZ(const AcReal* src, AcReal* dst, const Volume dims, const Volume start, const Volume end, const hipStream_t stream)
{
	const dim3 tpb = {32,32,1};

	const Volume sub_dims = end-start;
	const dim3 bpg = to_dim3(get_bpg(sub_dims,to_volume(tpb)));
  	KERNEL_LAUNCH(transpose_xyz_to_yxz,bpg, tpb, 0, stream)(src,dst,dims,start,end);
	ERRCHK_CUDA_KERNEL();
	return AC_SUCCESS;
}
static AcResult
acTransposeXYZ_YZX(const AcReal* src, AcReal* dst, const Volume dims, const Volume start, const Volume end, const hipStream_t stream)
{
	const dim3 tpb = {32,32,1};

	const Volume sub_dims = end-start;
	const dim3 bpg = to_dim3(get_bpg(sub_dims,to_volume(tpb)));
  	KERNEL_LAUNCH(transpose_xyz_to_yzx,bpg, tpb, 0, stream)(src,dst,dims,start,end);
	ERRCHK_CUDA_KERNEL();
	return AC_SUCCESS;
}
static AcResult
acTransposeXYZ_XZY(const AcReal* src, AcReal* dst, const Volume dims, const Volume start, const Volume end, const hipStream_t stream)
{
	const dim3 tpb = {32,32,1};
	const Volume sub_dims = end-start;
	const dim3 bpg = to_dim3(get_bpg(sub_dims,to_volume(tpb)));
  	KERNEL_LAUNCH(transpose_xyz_to_xzy,bpg, tpb, 0, stream)(src,dst,dims,start,end);
	ERRCHK_CUDA_KERNEL();
	return AC_SUCCESS;
}
static AcResult
acTransposeXYZ_XYZ(const AcReal* src, AcReal* dst, const Volume dims, const Volume start, const Volume end, const hipStream_t stream)
{
	const Volume sub_dims = end-start;
	const size_t bytes = sub_dims.x*sub_dims.y*sub_dims.z*sizeof(AcReal);
	src = &src[start.x + dims.x*start.y + dims.x*dims.y*start.z];
	dst = &dst[start.x + dims.x*start.y + dims.x*dims.y*start.z];
	ERRCHK_CUDA_ALWAYS(hipMemcpyAsync(dst,src,bytes,hipMemcpyDeviceToDevice,stream));
	return AC_SUCCESS;
}
AcResult
acTransposeWithBounds(const AcMeshOrder order, const AcReal* src, AcReal* dst, const Volume dims, const Volume start, const Volume end, const hipStream_t stream)
{
	switch(order)
	{
		case(XYZ):
			return acTransposeXYZ_XYZ(src,dst,dims,start,end,stream);
		case (XZY):
			return acTransposeXYZ_XZY(src,dst,dims,start,end,stream);
		case (YXZ):
			return acTransposeXYZ_YXZ(src,dst,dims,start,end,stream);
		case (YZX):
			return acTransposeXYZ_YZX(src,dst,dims,start,end,stream);
		case(ZXY):
			return acTransposeXYZ_ZXY(src,dst,dims,start,end,stream);
		case(ZYX):
			return acTransposeXYZ_ZYX(src,dst,dims,start,end,stream);
	}
        ERRCHK_CUDA_KERNEL();
	return AC_SUCCESS;
}

AcResult
acTranspose(const AcMeshOrder order, const AcReal* src, AcReal* dst, const Volume dims, const hipStream_t stream)
{
	return acTransposeWithBounds(order,src,dst,dims,(Volume){0,0,0},dims,stream);
}

size_t
get_count(const AcShape shape)
{
	return shape.x*shape.y*shape.z*shape.w;
}
static AcResult
ac_flush_scratchpad(VertexBufferArray vba, const int variable, const AcType type, const AcReduceOp op)
{

	const int n_elems = 
				type == AC_REAL_TYPE ?  NUM_REAL_OUTPUTS :
				type == AC_PROF_TYPE ?  NUM_PROFILES     :
				type == AC_INT_TYPE  ?  NUM_INT_OUTPUTS  :
#if AC_DOUBLE_PRECISION
				type == AC_FLOAT_TYPE  ?  NUM_FLOAT_OUTPUTS  :
#endif
				0;
	ERRCHK_ALWAYS(variable < n_elems);
	const size_t counts = 
			type == AC_INT_TYPE  ? (*vba.reduce_buffer_int[variable].buffer_size)/sizeof(int) :
#if AC_DOUBLE_PRECISION
			type == AC_FLOAT_TYPE  ? (*vba.reduce_buffer_float[variable].buffer_size)/sizeof(float) :
#endif
			type == AC_REAL_TYPE ? (*vba.reduce_buffer_real[variable].buffer_size)/sizeof(AcReal) :
			type == AC_PROF_TYPE ? (get_count(vba.profile_reduce_buffers[variable].src.shape)) :
			0;

	if(type == AC_REAL_TYPE)
	{
		if constexpr (NUM_REAL_OUTPUTS == 0) return AC_FAILURE;
		AcReal* dst = *(vba.reduce_buffer_real[variable].src);
		acKernelFlush(0,dst,counts,get_reduce_state_flush_var_real(op));
	}
	else if(type == AC_PROF_TYPE)
	{
		if constexpr(NUM_PROFILES == 0) return AC_FAILURE;
		AcReal* dst = vba.profile_reduce_buffers[variable].src.data;
		acKernelFlush(0,dst,counts,get_reduce_state_flush_var_real(op));
	}
#if AC_DOUBLE_PRECISION
	else if(type == AC_FLOAT_TYPE)
	{
		if constexpr(NUM_FLOAT_OUTPUTS  == 0) return AC_FAILURE;
		float* dst = *(vba.reduce_buffer_float[variable].src);
		acKernelFlush(0,dst,counts,get_reduce_state_flush_var_float(op));
	}
#endif
	else
	{
		if constexpr (NUM_INT_OUTPUTS == 0) return AC_FAILURE;
		int* dst = *(vba.reduce_buffer_int[variable].src);
		acKernelFlush(0,dst,counts,get_reduce_state_flush_var_int(op));
	}
  	ERRCHK_CUDA_ALWAYS(acDeviceSynchronize());
	return AC_SUCCESS;
}
static AcReduceOp*
get_reduce_buffer_states(const VertexBufferArray vba, const AcType type)
{
	return
#if AC_DOUBLE_PRECISION
			type == AC_FLOAT_TYPE  ? vba.scratchpad_states->floats :
#endif
			type == AC_INT_TYPE    ? vba.scratchpad_states->ints  :
			type == AC_REAL_TYPE   ? vba.scratchpad_states->reals :
			type == AC_PROF_TYPE   ? &vba.scratchpad_states->reals[NUM_REAL_OUTPUTS] :
			NULL;
}
static UNUSED AcReduceOp
get_reduce_buffer_state(const VertexBufferArray vba, const int variable, const AcType type)
{
	return get_reduce_buffer_states(vba,type)[variable];
}
AcResult
acPreprocessScratchPad(VertexBufferArray vba, const int variable, const AcType type,const AcReduceOp op)
{
	AcReduceOp* states = get_reduce_buffer_states(vba,type);
	if(states[variable] == op) return AC_SUCCESS;
	states[variable] = op;
	return ac_flush_scratchpad(vba,variable,type,op);
}

AcMeshOrder
acGetMeshOrderForProfile(const AcProfileType type)
{
    	switch(type)
    	{
    	        case(PROFILE_X):
    	    	    return ZYX;
    	        case(PROFILE_Y):
		    return XZY;
    	        case(PROFILE_Z):
			return XYZ;
    	        case(PROFILE_XY):
			return ZXY;
    	        case(PROFILE_XZ):
			return YXZ;
    	        case(PROFILE_YX):
			return ZYX;
    	        case(PROFILE_YZ):
			return XYZ;
    	        case(PROFILE_ZX):
			return YZX;
    	        case(PROFILE_ZY):
			return XZY;
		case(PROFILE_NONE):
			return XYZ;
    	}
	return XYZ;
};

#include "load_ac_kernel_params.h"

int
acVerifyMeshInfo(const AcMeshInfo info)
{
  int retval = 0;
  for (size_t i = 0; i < NUM_INT_PARAMS; ++i) {
    if (info.int_params[i] == INT_MIN) {
      retval = -1;
      fprintf(stderr, "--- Warning: [%s] uninitialized ---\n",
              intparam_names[i]);
    }
  }
  for (size_t i = 0; i < NUM_INT3_PARAMS; ++i) {
    if (info.int3_params[i].x == INT_MIN || info.int3_params[i].y == INT_MIN ||
        info.int3_params[i].z == INT_MIN) {
      retval = -1;
      fprintf(stderr, "--- Warning: [%s] uninitialized ---\n",
              int3param_names[i]);
    }
  }
  for (size_t i = 0; i < NUM_REAL_PARAMS; ++i) {
    if (isnan(info.real_params[i])) {
      retval = -1;
      fprintf(stderr, "--- Warning: [%s] uninitialized ---\n",
              realparam_names[i]);
    }
  }
  for (int i = 0; i < NUM_REAL3_PARAMS; ++i) {
    if (isnan(info.real3_params[i].x) || isnan(info.real3_params[i].y) ||
        isnan(info.real3_params[i].z)) {
      retval = -1;
      fprintf(stderr, "--- Warning: [%s] uninitialized ---\n",
              real3param_names[i]);
    }
  }
  return retval;
}

const AcKernel*
acGetKernels()
{
	return kernel_enums;
}

AcResult
acRuntimeQuit()
{
	tbconfigs.clear();
	for(int kernel = 0; kernel < NUM_KERNELS; ++kernel)
	{
		reduce_offsets[kernel].clear();
		kernel_running_reduce_offsets[kernel] = 0;
	}
	segmented_reduce_offsets.clear();
	return AC_SUCCESS;
}
#if AC_FFT_ENABLED


#if AC_USE_HIP
#if AC_DOUBLE_PRECISION
#define AC_FFT_PRECISION rocfft_precision_double
#else
#define AC_FFT_PRECISION rocfft_precision_single
#endif

#include <rocfft.h>

rocfft_plan_description 
get_data_layout(const Volume domain_size)
{
    //TP: not sure are the offsets for rocfft in bytes or in number of elements so prefer to do the offseting via pointer arithmetic myself
    size_t offsets[]  = {0,0,0};
    size_t strides[]  = {domain_size.x*domain_size.y,domain_size.x,1};
    size_t distance = domain_size.x*domain_size.y*domain_size.z;
    // Create plan description
    rocfft_plan_description desc = nullptr;
    rocfft_status status = rocfft_plan_description_create(&desc);
    ERRCHK_ALWAYS((status == rocfft_status_success));
    status = rocfft_plan_description_set_data_layout(
        desc,
        rocfft_array_type_complex_interleaved,  // in_array_type
        rocfft_array_type_complex_interleaved,  // out_array_type
	offsets,
	offsets,
	3,
	strides,
	distance,
	3,
	strides,
	distance
        );

    ERRCHK_ALWAYS((status == rocfft_status_success));
    return desc;
}

AcResult
acFFTForwardTransformC2C(const AcComplex* src, const Volume domain_size,
                                const Volume subdomain_size, const Volume starting_point,
                                AcComplex* dst) {
    rocfft_plan_description desc = get_data_layout(domain_size);
    const size_t starting_offset = starting_point.x + domain_size.x*(starting_point.y + domain_size.y*starting_point.z);
    // Create plan
    rocfft_plan plan = nullptr;
    size_t lengths[] = {subdomain_size.z,subdomain_size.y,subdomain_size.x};
    rocfft_status status = rocfft_plan_create(
        &plan,
        rocfft_placement_notinplace,
        rocfft_transform_type_complex_forward,
	AC_FFT_PRECISION,
        3,            // Dimensions
        lengths,      // lengths
        1,            // batch
        desc);        // description
    if (status != rocfft_status_success) return AC_FAILURE;

    // Create execution info
    rocfft_execution_info info = nullptr;
    status = rocfft_execution_info_create(&info);
    if (status != rocfft_status_success) return AC_FAILURE;

    // Execute
    void* in_buffer[] = {const_cast<void*>(reinterpret_cast<const void*>(src+starting_offset))};
    void* out_buffer[] = {reinterpret_cast<void*>(dst+starting_offset)};
    status = rocfft_execute(plan, in_buffer, out_buffer, info);
    if (status != rocfft_status_success) return AC_FAILURE;

    // Cleanup
    rocfft_execution_info_destroy(info);
    rocfft_plan_destroy(plan);
    rocfft_plan_description_destroy(desc);

    // Scaling (just like CUFFT doesn't scale by default)
    size_t complex_domain_size = domain_size.x * domain_size.y * domain_size.z;
    const AcReal scale = 1.0 / (subdomain_size.x * subdomain_size.y * subdomain_size.z);
    acMultiplyInplaceComplex(scale, complex_domain_size, dst);

    return AC_SUCCESS;
}


AcResult
acFFTBackwardTransformC2C(const AcComplex* src,
                                 const Volume domain_size,
                                 const Volume subdomain_size,
                                 const Volume starting_point,
                                 AcComplex* dst) {
    // Create plan description
    rocfft_plan_description desc = get_data_layout(domain_size);
    // Create inverse plan
    rocfft_plan plan = nullptr;
    size_t lengths[] = {subdomain_size.z,subdomain_size.y,subdomain_size.x};
    const size_t starting_offset = starting_point.x + domain_size.x*(starting_point.y + domain_size.y*starting_point.z);
    rocfft_status status = rocfft_plan_create(
        &plan,
        rocfft_placement_notinplace,
        rocfft_transform_type_complex_inverse,
	AC_FFT_PRECISION,
        3,           // Dimensions
        lengths,     // FFT size
        1,           // Batch size
        desc);
    if (status != rocfft_status_success) return AC_FAILURE;

    // Create execution info
    rocfft_execution_info info = nullptr;
    status = rocfft_execution_info_create(&info);
    if (status != rocfft_status_success) return AC_FAILURE;

    void* in_buffer[] = {const_cast<void*>(reinterpret_cast<const void*>(src+starting_offset))};
    void* out_buffer[] = {reinterpret_cast<void*>(dst+starting_offset)};

    status = rocfft_execute(plan, in_buffer, out_buffer, info);
    if (status != rocfft_status_success) return AC_FAILURE;

    // Cleanup
    rocfft_execution_info_destroy(info);
    rocfft_plan_destroy(plan);
    rocfft_plan_description_destroy(desc);

    return AC_SUCCESS;
}
AcResult
acFFTForwardTransformSymmetricR2C(const AcReal*, const Volume, const Volume, const Volume, AcComplex*) {
	return AC_FAILURE;
}

AcResult
acFFTBackwardTransformSymmetricC2R(const AcComplex*,const Volume, const Volume,const Volume, AcReal*) {
	return AC_FAILURE;
}
#else
#include <hipfft/hipfftXt.h>
#include <hip/hip_complex.h>

// cufft API error chekcing
#ifndef CUFFT_CALL
#define CUFFT_CALL( call )                                                                                             \
    {                                                                                                                  \
        auto status = static_cast<hipfftResult>( call );                                                                \
        if ( status != HIPFFT_SUCCESS )                                                                                 \
	    {                                                                                                          \
            fprintf( stderr,                                                                                           \
                     "ERROR: CUFFT call \"%s\" in line %d of file %s failed "                                          \
                     "with "                                                                                           \
                     "code (%d).\n",                                                                                   \
                     #call,                                                                                            \
                     __LINE__,                                                                                         \
                     __FILE__,                                                                                         \
                     status );                                                                                         \
		abort();                                                                                               \
	    }                                                                                                          \
    }
#endif  // CUFFT_CALL


// TODO: if the buffer on GPU would be properly padded:
// https://docs.nvidia.com/cuda/cufft/index.html#data-layout
// we could use in-place transformation and save one buffer allocation
// Padding as mentioned in the link: padded to (n/2 + 1) in the least significant dimension.
AcResult
acFFTForwardTransformSymmetricR2C(const AcReal* buffer, const Volume domain_size, const Volume subdomain_size, const Volume starting_point, AcComplex* transformed_in) {
    buffer = buffer + (starting_point.x + domain_size.x*(starting_point.y + domain_size.y*starting_point.z));
    // Number of elements in each dimension to use
    int dims[] = {(int)subdomain_size.z, (int)subdomain_size.y, (int)subdomain_size.x};
    // NOTE: inembed[0] and onembed[0] are not used directly, but as idist and odist
    // Sizes of input dimension of the buffer used
    int inembed[] = {(int)domain_size.z, (int)domain_size.y, (int)domain_size.x};
    // Sizes of the output dimension of the buffer used
    int onembed[] = {(int)subdomain_size.z, (int)subdomain_size.y, (int)(subdomain_size.x / 2) + 1};
    
    hipfftHandle plan_r2c{};
    CUFFT_CALL(hipfftCreate(&plan_r2c));
    size_t workspace_size;
    CUFFT_CALL(hipfftMakePlanMany(plan_r2c, 3, dims,
        inembed, 1, inembed[0], // in case inembed and onembed not needed could be: nullptr, 1, 0
        onembed, 1, onembed[0], //                                                  nullptr, 1, 0
        HIPFFT_D2Z, 1, &workspace_size));
    
    size_t orig_domain_size = inembed[0] * inembed[1] * inembed[2];
    size_t complex_domain_size = onembed[0] * onembed[1] * onembed[2];    
    
    hipDoubleComplex* transformed = reinterpret_cast<hipDoubleComplex*>(transformed_in);
    // Execute the plan_r2c
    CUFFT_CALL(hipfftXtExec(plan_r2c, (void*)buffer, transformed, HIPFFT_FORWARD));
    CUFFT_CALL(hipfftDestroy(plan_r2c));
    // Scale complex results that inverse FFT results in original values
    const AcReal scale{1.0 / orig_domain_size};
    acMultiplyInplaceComplex(scale, complex_domain_size, transformed_in);
    return AC_SUCCESS;
}

AcResult
acFFTForwardTransformC2C(const AcComplex* buffer, const Volume domain_size, const Volume subdomain_size, const Volume starting_point, AcComplex* transformed_in) {
    const size_t starting_offset = starting_point.x + domain_size.x*(starting_point.y + domain_size.y*starting_point.z);
    buffer = buffer + starting_offset;
    // Number of elements in each dimension to use
    int dims[] = {(int)subdomain_size.z, (int)subdomain_size.y, (int)subdomain_size.x};
    // NOTE: inembed[0] and onembed[0] are not used directly, but as idist and odist
    // Sizes of input dimension of the buffer used
    int inembed[] = {(int)domain_size.z, (int)domain_size.y, (int)domain_size.x};
    // Sizes of the output dimension of the buffer used
    int onembed[] = {(int)domain_size.z, (int)domain_size.y, (int)(domain_size.x)};
    
    hipfftHandle plan_r2c{};
    CUFFT_CALL(hipfftCreate(&plan_r2c));
    size_t workspace_size;
    CUFFT_CALL(hipfftMakePlanMany(plan_r2c, 3, dims,
        inembed, 1, inembed[0], // in case inembed and onembed not needed could be: nullptr, 1, 0
        onembed, 1, onembed[0], //                                                  nullptr, 1, 0
        HIPFFT_Z2Z, 1, &workspace_size));
    
    size_t complex_domain_size = onembed[0] * onembed[1] * onembed[2];    
    
    hipDoubleComplex* transformed = reinterpret_cast<hipDoubleComplex*>(transformed_in + starting_offset);
    // Execute the plan_r2c
    CUFFT_CALL(hipfftXtExec(plan_r2c, (void*)buffer, transformed, HIPFFT_FORWARD));
    CUFFT_CALL(hipfftDestroy(plan_r2c));
    // Scale complex results that inverse FFT results in original values
    const AcReal scale{1.0 / ( dims[0] * dims[1] * dims[2])};
    acMultiplyInplaceComplex(scale, complex_domain_size, transformed_in);
    return AC_SUCCESS;
}




AcResult
acFFTBackwardTransformSymmetricC2R(const AcComplex* transformed_in,const Volume domain_size, const Volume subdomain_size,const Volume starting_point, AcReal* buffer) {
    buffer = buffer + (starting_point.x + domain_size.x*(starting_point.y + domain_size.y*starting_point.z));
    // Number of elements in each dimension to use
    int dims[] = {(int)subdomain_size.z, (int)subdomain_size.y, (int)subdomain_size.x};
    // NOTE: inembed[0] and onembed[0] are not used directly, but as idist and odist
    // Sizes of input dimension of the buffer used
    int inembed[] = {(int)domain_size.z, (int)domain_size.y, (int)domain_size.x};
    // Sizes of the output dimension of the buffer used
    int onembed[] = {(int)subdomain_size.z, (int)subdomain_size.y, (int)(((int)subdomain_size.x) / 2) + 1};
    
    hipfftHandle plan_c2r{};
    CUFFT_CALL(hipfftCreate(&plan_c2r));
    size_t workspace_size;
    CUFFT_CALL(hipfftMakePlanMany(plan_c2r, 3, dims,
        onembed, 1, onembed[0],
        inembed, 1, inembed[0],
        HIPFFT_Z2D, 1, &workspace_size));
    const hipDoubleComplex* transformed = reinterpret_cast<const hipDoubleComplex*>(transformed_in);
    CUFFT_CALL(hipfftXtExec(plan_c2r, (void*)transformed, buffer, HIPFFT_BACKWARD));
    CUFFT_CALL(hipfftDestroy(plan_c2r));
    return AC_SUCCESS;
}

AcResult
acFFTBackwardTransformC2C(const AcComplex* transformed_in,const Volume domain_size, const Volume subdomain_size,const Volume starting_point, AcComplex* buffer) {
    const size_t starting_offset = starting_point.x + domain_size.x*(starting_point.y + domain_size.y*starting_point.z);
    buffer = buffer + starting_offset;
    // Number of elements in each dimension to use
    int dims[] = {(int)subdomain_size.z, (int)subdomain_size.y, (int)subdomain_size.x};
    // NOTE: inembed[0] and onembed[0] are not used directly, but as idist and odist
    // Sizes of input dimension of the buffer used
    int inembed[] = {(int)domain_size.z, (int)domain_size.y, (int)domain_size.x};
    // Sizes of the output dimension of the buffer used
    int onembed[] = {(int)domain_size.z, (int)domain_size.y, (int)(((int)domain_size.x))};
    
    hipfftHandle plan_c2r{};
    CUFFT_CALL(hipfftCreate(&plan_c2r));
    size_t workspace_size;
    CUFFT_CALL(hipfftMakePlanMany(plan_c2r, 3, dims,
        onembed, 1, onembed[0],
        inembed, 1, inembed[0],
        HIPFFT_Z2Z, 1, &workspace_size));
    const hipDoubleComplex* transformed = reinterpret_cast<const hipDoubleComplex*>(transformed_in + starting_offset);
    CUFFT_CALL(hipfftXtExec(plan_c2r, (void*)transformed, buffer, HIPFFT_BACKWARD));
    CUFFT_CALL(hipfftDestroy(plan_c2r));
    return AC_SUCCESS;
}
#endif //AC_USE_HIP
AcResult
acFFTBackwardTransformC2R(const AcComplex* transformed_in,const Volume domain_size, const Volume subdomain_size,const Volume starting_point, AcReal* buffer) {
    const size_t count = domain_size.x*domain_size.y*domain_size.z;
    const size_t bytes = sizeof(AcComplex)*count;
    AcComplex* tmp = NULL;
    device_malloc(&tmp,bytes);
    acFFTBackwardTransformC2C(transformed_in,domain_size,subdomain_size,starting_point,tmp);
    acComplexToReal(tmp,count,buffer);
    device_free(&tmp,0);
    return AC_SUCCESS;
}

AcResult
acFFTForwardTransformR2C(const AcReal* buffer, const Volume domain_size, const Volume subdomain_size, const Volume starting_point, AcComplex* transformed_in) {
    const size_t count = domain_size.x*domain_size.y*domain_size.z;
    const size_t bytes = sizeof(AcComplex)*count;
    AcComplex* tmp = NULL;
    device_malloc(&tmp,bytes);
    acRealToComplex(buffer,count,tmp);
    acFFTForwardTransformC2C(tmp, domain_size,subdomain_size,starting_point,transformed_in);
    device_free(&tmp,0);
    return AC_SUCCESS;
}

#else
AcResult
acFFTForwardTransformSymmetricR2C(const AcReal*, const Volume, const Volume, const Volume, AcComplex*) {
	fprintf(stderr,"FATAL: need to have FFT_ENABLED=ON for acFFTForwardTransform!\n");
	fflush(stderr);
	exit(EXIT_FAILURE);
	return AC_FAILURE;
}
AcResult
acFFTForwardTransformR2C(const AcReal*, const Volume, const Volume, const Volume, AcComplex*) {
	fprintf(stderr,"FATAL: need to have FFT_ENABLED=ON for acFFTForwardTransform!\n");
	fflush(stderr);
	exit(EXIT_FAILURE);
	return AC_FAILURE;
}
AcResult
acFFTBackwardTransformSymmetricC2R(const AcComplex*,const Volume, const Volume,const Volume, AcReal*)
{
	fprintf(stderr,"FATAL: need to have FFT_ENABLED=ON for acFFTBackwardTransform!\n");
	fflush(stderr);
	exit(EXIT_FAILURE);
	return AC_FAILURE;
}
AcResult
acFFTBackwardTransformC2R(const AcComplex*,const Volume, const Volume,const Volume, AcReal*)
{
	fprintf(stderr,"FATAL: need to have FFT_ENABLED=ON for acFFTBackwardTransform!\n");
	fflush(stderr);
	exit(EXIT_FAILURE);
	return AC_FAILURE;
}
#endif

hipError_t
acStreamSynchronize(hipStream_t stream)
{
	return hipStreamSynchronize(stream);
}
hipError_t
acDeviceSynchronize()
{
	return hipDeviceSynchronize();
}
hipError_t
acSetDevice(const int id)
{
	return hipSetDevice(id);
}
hipError_t
acGetDeviceCount(int* dst)
{
	return hipGetDeviceCount(dst);
}
hipError_t
acDeviceSetSharedMemConfig(const hipSharedMemConfig config)
{
	return hipDeviceSetSharedMemConfig(config);
}
hipError_t
acStreamCreateWithPriority(hipStream_t* dst, int option, int priority)
{
	return hipStreamCreateWithPriority(dst,option,priority);
}
hipError_t
acStreamDestroy(hipStream_t stream)
{
	return hipStreamDestroy(stream);
}
hipError_t
acMemcpy(AcReal* dst, const AcReal* src, const size_t bytes, hipMemcpyKind kind)
{
	return hipMemcpy(dst,src,bytes,kind);
}
hipError_t
acMemcpyAsync(AcReal* dst, const AcReal* src, const size_t bytes, hipMemcpyKind kind, const hipStream_t stream)
{
	return hipMemcpyAsync(dst,src,bytes,kind,stream);
}
hipError_t
acMemcpyPeerAsync(AcReal* dst, int dst_id, const AcReal* src, int src_id, const size_t bytes, const hipStream_t stream)
{
	return hipMemcpyPeerAsync(dst,dst_id,src,src_id,bytes,stream);
}
hipError_t
acMemGetInfo(size_t* free_mem, size_t* total_mem)
{
	return hipMemGetInfo(free_mem,total_mem);
}
hipError_t
acStreamQuery(hipStream_t stream)
{
    return hipStreamQuery(stream);
}
const char*
acGetErrorString(hipError_t err)
{
    return hipGetErrorString(err);
}
hipError_t
acDeviceGetStreamPriorityRange(int* leastPriority, int* greatestPriority)
{
	return hipDeviceGetStreamPriorityRange(leastPriority,greatestPriority);
}
hipError_t
acStreamCreateWithPriority(hipStream_t* stream, unsigned int flags, int priority)
{
	return hipStreamCreateWithPriority(stream, flags, priority);
}
hipError_t
acMalloc(void** dst, const size_t bytes)
{
	return hipMalloc(dst,bytes);
}
hipError_t
acFree(void* dst)
{
	return hipFree(dst);
}
hipError_t
acMallocHost(void** dst, const size_t bytes)
{
	return hipHostMalloc(dst,bytes);
}
hipError_t
acGetDevice(int* dst)
{
	return hipGetDevice(dst);
}
hipError_t
acGetLastError()
{
	return hipGetLastError();
}

