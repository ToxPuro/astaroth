#include "hip/hip_runtime.h"
#include "kernels.h"

#include "math_utils.h"

__device__ AcMeshInfo d_mesh_info;
__device__ dim3 mm;
__device__ dim3 multigpu_offset;

#define Field3(x, y, z) make_int3((x), (y), (z))
#define real3(i, j, k) ((AcReal3){(i), (j), (k)})
#define print printf // TODO is this a good idea?

#include "user_kernels.h"

AcResult
acLaunchKernel(Kernel func, const hipStream_t stream, const int3 start,
               const int3 end, VertexBufferArray vba)
{
  const int3 n = end - start;

  // const dim3 tpb = getOptimalTBConfig(n, vba).tpb; // TODO
  const dim3 tpb = dim3(32, 4, 1);
  const dim3 bpg((unsigned int)ceil(n.x / AcReal(tpb.x)), //
                 (unsigned int)ceil(n.y / AcReal(tpb.y)), //
                 (unsigned int)ceil(n.z / AcReal(tpb.z)));
  const size_t smem = 0;

  func<<<bpg, tpb, smem, stream>>>(start, end, vba);
  ERRCHK_CUDA_KERNEL();

  return AC_SUCCESS;
}

#define GEN_LOAD_UNIFORM(TYPE)                                                 \
  GEN_LOAD_UNIFORM_DECLARATION(TYPE)                                           \
  {                                                                            \
    hipError_t retval = hipMemcpyToSymbolAsync(HIP_SYMBOL(                              \
        symbol), &value, sizeof(value), 0, hipMemcpyHostToDevice, stream);     \
    ERRCHK_CUDA(retval);                                                       \
    return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;                    \
  }

#define GEN_STORE_UNIFORM(TYPE)                                                \
  GEN_STORE_UNIFORM_DECLARATION(TYPE)                                          \
  {                                                                            \
    hipError_t retval = hipMemcpyFromSymbolAsync(                            \
        dst, HIP_SYMBOL(symbol), sizeof(*dst), 0, hipMemcpyDeviceToHost, stream);         \
    ERRCHK_CUDA(retval);                                                       \
    return retval == hipSuccess ? AC_SUCCESS : AC_FAILURE;                    \
  }

GEN_LOAD_UNIFORM(AcReal)
GEN_LOAD_UNIFORM(AcReal3)
GEN_LOAD_UNIFORM(int)
GEN_LOAD_UNIFORM(int3)

GEN_STORE_UNIFORM(AcReal)
GEN_STORE_UNIFORM(AcReal3)
GEN_STORE_UNIFORM(int)
GEN_STORE_UNIFORM(int3)

int
main(void)
{
  printf("Launching kernel... \n");
  acLaunchKernel(solve, 0, (int3){0, 0, 0}, (int3){1, 0, 0},
                 (VertexBufferArray){0});
  printf("done\n");
  return EXIT_SUCCESS;
}